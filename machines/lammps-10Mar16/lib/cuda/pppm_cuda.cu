#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include "cuda_precision.h"
//#define FFT_CUFFT
#define MY_PREFIX pppm
#include "cuda_shared.h"
#include "cuda_common.h"
#include "pppm_cuda_cu.h"
#include "hip/hip_runtime.h"
#include <stdio.h>

//#include "crm_cuda_utils.cu"
#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))

__device__ __constant__ FFT_CFLOAT* work1;
__device__ __constant__ FFT_CFLOAT* work2;
__device__ __constant__ FFT_CFLOAT* work3;
__device__ __constant__ PPPM_CFLOAT* greensfn;
__device__ __constant__ PPPM_CFLOAT* gf_b;
__device__ __constant__ PPPM_CFLOAT* fkx;
__device__ __constant__ PPPM_CFLOAT* fky;
__device__ __constant__ PPPM_CFLOAT* fkz;
__device__ __constant__ PPPM_CFLOAT* vg;
__device__ __constant__ int* part2grid;
__device__ __constant__ PPPM_CFLOAT* density_brick;
__device__ __constant__ int* density_brick_int;
__device__ __constant__ PPPM_CFLOAT density_intScale;
__device__ __constant__ PPPM_CFLOAT* vdx_brick;
__device__ __constant__ PPPM_CFLOAT* vdy_brick;
__device__ __constant__ PPPM_CFLOAT* vdz_brick;
__device__ __constant__ PPPM_CFLOAT* density_fft;
__device__ __constant__ ENERGY_CFLOAT* energy;
__device__ __constant__ ENERGY_CFLOAT* virial;
__device__ __constant__ int nxlo_in;
__device__ __constant__ int nxhi_in;
__device__ __constant__ int nxlo_out;
__device__ __constant__ int nxhi_out;
__device__ __constant__ int nylo_in;
__device__ __constant__ int nyhi_in;
__device__ __constant__ int nylo_out;
__device__ __constant__ int nyhi_out;
__device__ __constant__ int nzlo_in;
__device__ __constant__ int nzhi_in;
__device__ __constant__ int nzlo_out;
__device__ __constant__ int nzhi_out;
__device__ __constant__ int nxlo_fft;
__device__ __constant__ int nxhi_fft;
__device__ __constant__ int nylo_fft;
__device__ __constant__ int nyhi_fft;
__device__ __constant__ int nzlo_fft;
__device__ __constant__ int nzhi_fft;
__device__ __constant__ int nx_pppm;
__device__ __constant__ int ny_pppm;
__device__ __constant__ int nz_pppm;
__device__ __constant__ int slabflag;
__device__ __constant__ PPPM_CFLOAT qqrd2e;
__device__ __constant__ int order;
//__device__ __constant__ float3 sublo;
__device__ __constant__ PPPM_CFLOAT* rho_coeff;
__device__ __constant__ int nmax;
__device__ __constant__ int nlocal;
__device__ __constant__ PPPM_CFLOAT* debugdata;
__device__ __constant__ PPPM_CFLOAT delxinv;
__device__ __constant__ PPPM_CFLOAT delyinv;
__device__ __constant__ PPPM_CFLOAT delzinv;
__device__ __constant__ int nlower;
__device__ __constant__ int nupper;
__device__ __constant__ PPPM_CFLOAT shiftone;


#include "pppm_cuda_kernel.cu"
#include "stdio.h"
void pppm_device_init(void* cu_density_brick, void* cu_vdx_brick, void* cu_vdy_brick, void* cu_vdz_brick, void* cu_density_fft, void* cu_energy, void* cu_virial
                      , void* cu_work1, void* cu_work2, void* cu_work3, void* cu_greensfn, void* cu_fkx, void* cu_fky, void* cu_fkz, void* cu_vg
                      , int cu_nxlo_in, int cu_nxhi_in, int cu_nylo_in, int cu_nyhi_in, int cu_nzlo_in, int cu_nzhi_in, int cu_nxlo_out, int cu_nxhi_out, int cu_nylo_out, int cu_nyhi_out, int cu_nzlo_out, int cu_nzhi_out, int cu_nx_pppm, int cu_ny_pppm, int cu_nz_pppm
                      , int cu_nxlo_fft, int cu_nxhi_fft, int cu_nylo_fft, int cu_nyhi_fft, int cu_nzlo_fft, int cu_nzhi_fft, void* cu_gf_b
                      , double cu_qqrd2e, int cu_order, void* cu_rho_coeff, void* cu_debugdata, void* cu_density_brick_int, int cu_slabflag
                     )
{
  CUT_CHECK_ERROR("ERROR-CUDA poisson_init Start");
  hipMemcpyToSymbol(HIP_SYMBOL(density_brick), &cu_density_brick, sizeof(PPPM_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(density_brick_int), &cu_density_brick_int, sizeof(PPPM_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(vdx_brick), &cu_vdx_brick, sizeof(PPPM_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(vdy_brick), &cu_vdy_brick, sizeof(PPPM_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(vdz_brick), &cu_vdz_brick, sizeof(PPPM_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(density_fft), &cu_density_fft, sizeof(PPPM_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(energy), &cu_energy, sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(virial), &cu_virial, sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(nxlo_in), &cu_nxlo_in, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nxhi_in), &cu_nxhi_in, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nxlo_out), &cu_nxlo_out, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nxhi_out), &cu_nxhi_out, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nylo_in), &cu_nylo_in, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nyhi_in), &cu_nyhi_in, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nylo_out), &cu_nylo_out, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nyhi_out), &cu_nyhi_out, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nzlo_in), &cu_nzlo_in, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nzhi_in), &cu_nzhi_in, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nzlo_out), &cu_nzlo_out, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nzhi_out), &cu_nzhi_out, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nxlo_fft), &cu_nxlo_fft, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nxhi_fft), &cu_nxhi_fft, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nylo_fft), &cu_nylo_fft, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nyhi_fft), &cu_nyhi_fft, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nzlo_fft), &cu_nzlo_fft, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nzhi_fft), &cu_nzhi_fft, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(slabflag), &cu_slabflag, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nx_pppm), &cu_nx_pppm, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(ny_pppm), &cu_ny_pppm, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nz_pppm), &cu_nz_pppm, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(work1), &cu_work1, sizeof(FFT_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(work2), &cu_work2, sizeof(FFT_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(work3), &cu_work3, sizeof(FFT_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(greensfn), &cu_greensfn, sizeof(PPPM_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(gf_b), &cu_gf_b, sizeof(PPPM_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(fkx), &cu_fkx, sizeof(PPPM_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(fky), &cu_fky, sizeof(PPPM_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(fkz), &cu_fkz, sizeof(PPPM_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(vg), &cu_vg, sizeof(PPPM_CFLOAT*));

  PPPM_CFLOAT cu_qqrd2e_a = cu_qqrd2e;
  hipMemcpyToSymbol(HIP_SYMBOL(qqrd2e), &cu_qqrd2e_a, sizeof(PPPM_CFLOAT));
  hipMemcpyToSymbol(HIP_SYMBOL(order), &cu_order, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(rho_coeff), &cu_rho_coeff, sizeof(PPPM_CFLOAT*));
  hipMemcpyToSymbol(HIP_SYMBOL(debugdata), &cu_debugdata, sizeof(PPPM_CFLOAT*));

  CUT_CHECK_ERROR("ERROR-CUDA poisson_init");

  /*if(sizeof(CUDA_CFLOAT)==sizeof(float)) printf("PPPMCuda Kernel: Using single precision\n");

  #ifdef PPPM_PRECISION
  if(sizeof(PPPM_CFLOAT)==sizeof(float)) printf("PPPMCuda Kernel: Using single precision for pppm core\n");
  if(sizeof(PPPM_CFLOAT)==sizeof(double)) printf("PPPMCuda Kernel: Using double precision for pppm core\n");
  #endif
  #ifdef ENERGY_PRECISION
  if(sizeof(ENERGY_CFLOAT)==sizeof(float)) printf("PPPMCuda Kernel: Using single precision for energy\n");
  if(sizeof(ENERGY_CFLOAT)==sizeof(double)) printf("PPPMCuda Kernel: Using double precision for energy\n");
  #endif
  #ifdef ENERGY_PRECISION
  if(sizeof(FFT_CFLOAT)==sizeof(float)) printf("PPPMCuda Kernel: Using single precision for fft\n");
  if(sizeof(FFT_CFLOAT)==sizeof(double)) printf("PPPMCuda Kernel: Using double precision for fft\n");
  #endif
  #ifdef X_PRECISION
  if(sizeof(X_CFLOAT)==sizeof(float)) printf("PPPMCuda Kernel: Using single precision for positions\n");
  if(sizeof(X_CFLOAT)==sizeof(double)) printf("PPPMCuda Kernel: Using double precision for positions\n");
  #endif
  #ifdef F_PRECISION
  if(sizeof(F_CFLOAT)==sizeof(float)) printf("PPPMCuda Kernel: Using single precision for forces\n");
  if(sizeof(F_CFLOAT)==sizeof(double)) printf("PPPMCuda Kernel: Using double precision for forces\n");
  #endif*/
}

void pppm_device_init_setup(cuda_shared_data* sdata, PPPM_CFLOAT cu_shiftone, PPPM_CFLOAT cu_delxinv, PPPM_CFLOAT cu_delyinv, PPPM_CFLOAT cu_delzinv, int cu_nlower, int cu_nupper)
{
  hipMemcpyToSymbol(HIP_SYMBOL(delxinv), &cu_delxinv, sizeof(PPPM_CFLOAT));
  hipMemcpyToSymbol(HIP_SYMBOL(delyinv), &cu_delyinv, sizeof(PPPM_CFLOAT));
  hipMemcpyToSymbol(HIP_SYMBOL(delzinv), &cu_delzinv, sizeof(PPPM_CFLOAT));
  hipMemcpyToSymbol(HIP_SYMBOL(shiftone), &cu_shiftone, sizeof(PPPM_CFLOAT));
  hipMemcpyToSymbol(HIP_SYMBOL(nlower), &cu_nlower, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nupper), &cu_nupper, sizeof(int));
  hipMemcpyToSymbol(MY_AP(sublo)   , sdata->domain.sublo, 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(subhi)   , sdata->domain.subhi, 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(boxlo)   , sdata->domain.boxlo, 3 * sizeof(X_CFLOAT));
  CUT_CHECK_ERROR("ERROR-CUDA pppm_init_setup");
}

void pppm_device_update(cuda_shared_data* sdata, void* cu_part2grid, int nlocala, int nmaxa)
{
  hipMemcpyToSymbol(HIP_SYMBOL(part2grid), &cu_part2grid, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(x)   , & sdata->atom.x   .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(f)   , & sdata->atom.f   .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(q)   , & sdata->atom.q   .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(tag)   , & sdata->atom.tag   .dev_data, sizeof(int*));
  //hipMemcpyToSymbol(MY_AP(nlocal)   , & sdata->atom.nlocal   .dev_data, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nlocal   ), &nlocala, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(nmax   ), &nmaxa, sizeof(int));
  CUT_CHECK_ERROR("ERROR-CUDA pppm_device_update");

}

void pppm_update_nlocal(int nlocala)
{
  hipMemcpyToSymbol(HIP_SYMBOL(nlocal   ), &nlocala, sizeof(int));
  CUT_CHECK_ERROR("ERROR-CUDA update_nlocal b");
}


void Cuda_PPPM_Setup_fkxyz_vg(int nx_pppma, int ny_pppma, int nz_pppma, PPPM_CFLOAT unitkx, PPPM_CFLOAT unitky, PPPM_CFLOAT unitkz, PPPM_CFLOAT g_ewald)
{
  dim3 grid;
  dim3 threads;
  grid.x = nz_pppma;
  grid.y = ny_pppma;
  grid.z = 1;
  threads.x = nx_pppma;
  threads.y = 1;
  threads.z = 1;
  setup_fkxyz_vg <<< grid, threads, 0>>>(unitkx, unitky, unitkz, g_ewald);
  hipDeviceSynchronize();

  CUT_CHECK_ERROR("ERROR-CUDA Cuda_PPPM_Setup_fkxyz_vg ");
}

void Cuda_PPPM_setup_greensfn(int nx_pppma, int ny_pppma, int nz_pppma, PPPM_CFLOAT unitkx, PPPM_CFLOAT unitky, PPPM_CFLOAT unitkz, PPPM_CFLOAT g_ewald,
                              int nbx, int nby, int nbz, PPPM_CFLOAT xprd, PPPM_CFLOAT yprd, PPPM_CFLOAT zprd_slab)
{
  dim3 grid;
  dim3 threads;
  grid.x = nz_pppma;
  grid.y = ny_pppma;
  grid.z = 1;
  threads.x = nx_pppma;
  threads.y = 1;
  threads.z = 1;
  setup_greensfn <<< grid, threads, 0>>>(unitkx, unitky, unitkz, g_ewald, nbx, nby, nbz, xprd, yprd, zprd_slab);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("ERROR-CUDA Cuda_PPPM_Setup_greensfn ");
}

void poisson_scale(int nx_pppma, int ny_pppma, int nz_pppma)
{
  dim3 grid;
  dim3 threads;
  grid.x = nz_pppma;
  grid.y = ny_pppma;
  grid.z = 1;
  threads.x = nx_pppma;
  threads.y = 1;
  threads.z = 1;
  poisson_scale_kernel <<< grid, threads, 0>>>();
  CUT_CHECK_ERROR("ERROR-CUDA poisson_scale ");

}

void poisson_xgrad(int nx_pppma, int ny_pppma, int nz_pppma)
{
  dim3 grid;
  dim3 threads;
  grid.x = nz_pppma;
  grid.y = ny_pppma;
  grid.z = 1;
  threads.x = nx_pppma;
  threads.y = 1;
  threads.z = 1;
  poisson_xgrad_kernel <<< grid, threads, 0>>>();
  CUT_CHECK_ERROR("ERROR-CUDA poisson_xgrad ");
}

void poisson_ygrad(int nx_pppma, int ny_pppma, int nz_pppma)
{
  dim3 grid;
  dim3 threads;
  grid.x = nz_pppma;
  grid.y = ny_pppma;
  grid.z = 1;
  threads.x = nx_pppma;
  threads.y = 1;
  threads.z = 1;
  poisson_ygrad_kernel <<< grid, threads, 0>>>();
  CUT_CHECK_ERROR("ERROR-CUDA poisson_ygrad ");
}

void poisson_zgrad(int nx_pppma, int ny_pppma, int nz_pppma)
{
  dim3 grid;
  dim3 threads;
  grid.x = nz_pppma;
  grid.y = ny_pppma;
  grid.z = 1;
  threads.x = nx_pppma;
  threads.y = 1;
  threads.z = 1;
  poisson_zgrad_kernel <<< grid, threads, 0>>>();
  CUT_CHECK_ERROR("ERROR-CUDA poisson_zgrad ");
}

void poisson_vdx_brick(int ihi, int ilo, int jhi, int jlo, int khi, int klo, int nx_pppma, int ny_pppma, int nz_pppma)
{

  dim3 grid;
  dim3 threads;
  grid.x = khi - klo + 1;
  grid.y = jhi - jlo + 1;
  grid.z = 1;
  threads.x = ihi - ilo + 1;
  threads.y = 1;
  threads.z = 1;
  //printf("VDX_BRICK CUDA: %i %i %i\n",grid.x,grid.y,threads.x);
  poisson_vdx_brick_kernel <<< grid, threads, 0>>>(ilo, jlo, klo);
  CUT_CHECK_ERROR("ERROR-CUDA poisson_vdxbrick ");
  hipDeviceSynchronize();
}

void poisson_vdy_brick(int ihi, int ilo, int jhi, int jlo, int khi, int klo, int nx_pppm, int ny_pppm, int nz_pppm)
{
  dim3 grid;
  dim3 threads;
  grid.x = khi - klo + 1;
  grid.y = jhi - jlo + 1;
  grid.z = 1;
  threads.x = ihi - ilo + 1;
  threads.y = 1;
  threads.z = 1;
  poisson_vdy_brick_kernel <<< grid, threads, 0>>>(ilo, jlo, klo);
  CUT_CHECK_ERROR("ERROR-CUDA poisson_vdybrick ");
  hipDeviceSynchronize();
}

void poisson_vdz_brick(int ihi, int ilo, int jhi, int jlo, int khi, int klo, int nx_pppm, int ny_pppm, int nz_pppm)
{
  dim3 grid;
  dim3 threads;
  grid.x = khi - klo + 1;
  grid.y = jhi - jlo + 1;
  grid.z = 1;
  threads.x = ihi - ilo + 1;
  threads.y = 1;
  threads.z = 1;
  poisson_vdz_brick_kernel <<< grid, threads, 0>>>(ilo, jlo, klo);
  CUT_CHECK_ERROR("ERROR-CUDA poisson_vdzbrick ");
  hipDeviceSynchronize();
}


void poisson_energy(int nxlo_fft, int nxhi_fft, int nylo_fft, int nyhi_fft, int nzlo_fft, int nzhi_fft, int vflag)
{
  //printf("VFLAG_GPU: %i\n",vflag);
  CUT_CHECK_ERROR("ERROR-CUDA poisson_energy start ");
  dim3 grid;
  dim3 threads;
  grid.x = nzhi_fft - nzlo_fft + 1;
  grid.y = nyhi_fft - nylo_fft + 1;
  grid.z = 1;
  threads.x = nxhi_fft - nxlo_fft + 1;
  threads.y = 1;
  threads.z = 1;
  poisson_energy_kernel <<< grid, threads, threads.x* sizeof(ENERGY_CFLOAT)>>>(nxlo_fft, nylo_fft, nzlo_fft, vflag);

  hipDeviceSynchronize();
  CUT_CHECK_ERROR("ERROR-CUDA poisson_energy end ");
}

ENERGY_CFLOAT sum_energy(void* cu_virial, void* cu_energy, int nx_pppma, int ny_pppma, int nz_pppma, int vflag, ENERGY_CFLOAT* cpu_virial)
{
  ENERGY_CFLOAT host_energy = 0;
  dim3 grid;
  dim3 threads;

  grid.x = nz_pppma;
  grid.y = 1;
  grid.z = 1;
  threads.x = ny_pppma;
  threads.y = 1;
  threads.z = 1;
  sum_energy_kernel1 <<< grid, threads, ny_pppma* sizeof(ENERGY_CFLOAT)>>>(vflag);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("ERROR-CUDA sumenergy_kernel1 ");

  grid.x = 1;
  grid.y = 1;
  grid.z = 1;
  threads.x = nz_pppma;
  threads.y = 1;
  threads.z = 1;
  sum_energy_kernel2 <<< grid, threads, nz_pppma* sizeof(ENERGY_CFLOAT)>>>(vflag);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("ERROR-CUDA sumenergy_kernel2 ");

  hipMemcpy((void*)(&host_energy), cu_energy, sizeof(ENERGY_CFLOAT), hipMemcpyDeviceToHost);

  if(vflag)
    hipMemcpy((void*) cpu_virial, (void*) cu_virial, 6 * sizeof(ENERGY_CFLOAT), hipMemcpyDeviceToHost);
  CUT_CHECK_ERROR("ERROR-CUDA sumenergy_memcopy");

  return host_energy;
}

void cuda_make_rho(cuda_shared_data* sdata, void* flag, PPPM_CFLOAT* cu_density_intScale, int ihi, int ilo, int jhi, int jlo, int khi, int klo, void* cu_density_brick, void* cu_density_brick_int)
{
  CUT_CHECK_ERROR("cuda_make_rho begin");
  dim3 grid, threads;
  int cpu_flag[3];
  grid.x = (sdata->atom.nlocal + 31) / 32;
  grid.y = 1;
  grid.z = 1;
  threads.x = 32;
  threads.y = 1;
  threads.z = 1;
  int sharedmemsize = (32 + 32 * (sdata->pppm.nupper - sdata->pppm.nlower + 1) + sdata->pppm.order * (sdata->pppm.order / 2 - (1 - sdata->pppm.order) / 2 + 1)) * sizeof(PPPM_CFLOAT);

  do {
    cpu_flag[0] = 0;
    cpu_flag[1] = 0;
    cpu_flag[2] = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(density_intScale), cu_density_intScale, sizeof(PPPM_CFLOAT*));
    CUT_CHECK_ERROR("ERROR-CUDA make_rho pre Z");
    hipMemset(flag, 0, 3 * sizeof(int));
    CUT_CHECK_ERROR("ERROR-CUDA make_rho pre A");
    hipMemset(cu_density_brick, 0, (khi - klo + 1) * (jhi - jlo + 1) * (ihi - ilo + 1)*sizeof(PPPM_CFLOAT));
    CUT_CHECK_ERROR("ERROR-CUDA make_rho pre B");
    hipMemset(cu_density_brick_int, 0, (khi - klo + 1) * (jhi - jlo + 1) * (ihi - ilo + 1)*sizeof(int));
    CUT_CHECK_ERROR("ERROR-CUDA make_rho pre C");
    make_rho_kernel <<< grid, threads, sharedmemsize>>>((int*) flag, 32 / (sdata->pppm.nupper - sdata->pppm.nlower + 1));
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("ERROR-CUDA make_rho A");
    hipMemcpy((void*) &cpu_flag, flag, 3 * sizeof(int), hipMemcpyDeviceToHost);

    if(cpu_flag[0] != 0) {
      (*cu_density_intScale) /= 2;
      MYDBG(printf("PPPM_Cuda::cuda_make_rho: Decrease cu_density_intScale to: %e\n", *cu_density_intScale);)
    }
    if((cpu_flag[0] == 0) && (cpu_flag[1] == 0)) {
      (*cu_density_intScale) *= 2;
      MYDBG(printf("PPPM_Cuda::cuda_make_rho: Increase cu_density_intScale to: %e\n", *cu_density_intScale);)
    }
    /* if((*cu_density_intScale)>0xe0000000)
     {
     	printf("Error Scaling\n");
         cpu_flag[0]=0;
         cpu_flag[1]=1;
     }*/
    CUT_CHECK_ERROR("ERROR-CUDA make_rho B");
  } while((cpu_flag[0] != 0) || (cpu_flag[1] == 0));


  grid.x = khi - klo + 1;
  grid.y = jhi - jlo + 1;
  threads.x = ihi - ilo + 1;
  scale_rho_kernel <<< grid, threads, 0>>>();
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("ERROR-CUDA make_rho_scale");
}


int cuda_particle_map(cuda_shared_data* sdata, void* flag)
{
  dim3 grid, threads;
  int cpu_flag;
  grid.x = (sdata->atom.nlocal + 31) / 32;
  grid.y = 1;
  grid.z = 1;
  threads.x = 32;
  threads.y = 1;
  threads.z = 1;
  CUT_CHECK_ERROR("ERROR-CUDA particla_map ..pre");
  particle_map_kernel <<< grid, threads, 0>>>((int*) flag);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("ERROR-CUDA particla_map a");
  hipMemcpy((void*) &cpu_flag, flag, sizeof(int), hipMemcpyDeviceToHost);
  CUT_CHECK_ERROR("ERROR-CUDA particla_map b");
  return cpu_flag;
}


void cuda_fieldforce(cuda_shared_data* sdata, void* flag)
{
  dim3 grid, threads;
  grid.x = (sdata->atom.nlocal + 31) / 32;
  grid.y = 1;
  grid.z = 1;
  threads.x = 32;
  threads.y = 1;
  threads.z = 1;
  int sharedmemsize = (32 + 3 * 32 * (sdata->pppm.nupper - sdata->pppm.nlower + 1) + sdata->pppm.order * (sdata->pppm.order / 2 - (1 - sdata->pppm.order) / 2 + 1)) * sizeof(PPPM_CFLOAT);
  fieldforce_kernel <<< grid, threads, sharedmemsize>>>
  (sdata->pppm.nupper - sdata->pppm.nlower + 1, 32 / (sdata->pppm.nupper - sdata->pppm.nlower + 1), (int*) flag);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("ERROR-CUDA fieldforce");
}

double cuda_slabcorr_energy(cuda_shared_data* sdata, ENERGY_CFLOAT* buf, ENERGY_CFLOAT* dev_buf)
{
  dim3 grid, threads;
  grid.x = (sdata->atom.nlocal + 31) / 32;
  grid.y = 1;
  grid.z = 1;
  threads.x = 32;
  threads.y = 1;
  threads.z = 1;
  slabcorr_energy_kernel <<< grid, threads, 32* sizeof(ENERGY_CFLOAT)>>>(dev_buf);
  hipDeviceSynchronize();
  hipMemcpy((void*) buf, dev_buf, grid.x* sizeof(ENERGY_CFLOAT), hipMemcpyDeviceToHost);

  double dipole_all = 0.0;

  for(int i = 0; i < grid.x; i++)
    dipole_all += buf[i];

  return dipole_all;
}

void cuda_slabcorr_force(cuda_shared_data* sdata, F_CFLOAT ffact)
{
  dim3 grid, threads;
  grid.x = (sdata->atom.nlocal + 31) / 32;
  grid.y = 1;
  grid.z = 1;
  threads.x = 32;
  threads.y = 1;
  threads.z = 1;
  slabcorr_force_kernel <<< grid, threads>>>(ffact);
  hipDeviceSynchronize();
}

void sum_virial(double* host_virial)
{
}

void pppm_initfftdata(cuda_shared_data* sdata, PPPM_CFLOAT* in, FFT_CFLOAT* out)
{
  int nslow = sdata->pppm.nzhi_in - sdata->pppm.nzlo_in;
  int nmid = sdata->pppm.nyhi_in - sdata->pppm.nylo_in;
  int nfast = sdata->pppm.nxhi_in - sdata->pppm.nxlo_in;
  int nrimz = MAX(sdata->pppm.nzlo_in - sdata->pppm.nzlo_out, sdata->pppm.nzhi_out - sdata->pppm.nzhi_in);
  int nrimy = MAX(sdata->pppm.nylo_in - sdata->pppm.nylo_out, sdata->pppm.nyhi_out - sdata->pppm.nyhi_in);
  int nrimx = MAX(sdata->pppm.nxlo_in - sdata->pppm.nxlo_out, sdata->pppm.nxhi_out - sdata->pppm.nxhi_in);
  dim3 grid;
  grid.x = nslow + 1;
  grid.y = nmid + 1;
  grid.z = 1;
  dim3 threads;
  threads.x = nfast + 1;
  threads.y = 1;
  threads.z = 1;
  hipDeviceSynchronize();
  initfftdata_core_kernel <<< grid, threads, 0>>>(in, out);
  hipDeviceSynchronize();
  grid.x = nrimz;
  grid.y = nmid + 1;
  threads.x = nfast + 1;
  initfftdata_z_kernel <<< grid, threads, 0>>>(in, out);
  hipDeviceSynchronize();
  grid.x = nslow + 1;
  grid.y = nrimy;
  threads.x = nfast + 1;
  initfftdata_y_kernel <<< grid, threads, 0>>>(in, out);
  hipDeviceSynchronize();
  grid.x = nslow + 1;
  grid.y = nmid + 1;
  threads.x = nrimx;
  initfftdata_x_kernel <<< grid, threads, 0>>>(in, out);
  hipDeviceSynchronize();
  grid.x = nrimz;
  grid.y = nrimy;
  threads.x = nfast + 1;
  initfftdata_yz_kernel <<< grid, threads, 0>>>(in, out);
  hipDeviceSynchronize();
  grid.x = nrimz;
  grid.y = nmid + 1;
  threads.x = nrimx;
  initfftdata_xz_kernel <<< grid, threads, 0>>>(in, out);
  hipDeviceSynchronize();
  grid.x = nslow + 1;
  grid.y = nrimy;
  threads.x = nrimx;
  initfftdata_xy_kernel <<< grid, threads, 0>>>(in, out);
  hipDeviceSynchronize();
  grid.x = nrimz;
  grid.y = nrimy;
  threads.x = nrimx;
  initfftdata_xyz_kernel <<< grid, threads, 0>>>(in, out);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("ERROR-CUDA initfftdata_kernel");
}


