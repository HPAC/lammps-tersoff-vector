/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>

#define _lj1 MY_AP(coeff1)
#define _lj2 MY_AP(coeff2)
#define _lj3 MY_AP(coeff3)
#define _lj4 MY_AP(coeff4)
#define _shift MY_AP(coeff5)

#include "pair_lj_expand_cuda_cu.h"
#include "pair_lj_expand_cuda_kernel_nc.cu"
#include <time.h>


void Cuda_PairLJExpandCuda_Init(cuda_shared_data* sdata)
{
  Cuda_Pair_Init_AllStyles(sdata, 5);
}




void Cuda_PairLJExpandCuda(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist, int eflag, int vflag, int eflag_atom, int vflag_atom)
{

  // initialize only on first call
  static  short init = 0;

  if(! init) {
    init = 1;
    Cuda_PairLJExpandCuda_Init(sdata);
  }

  dim3 grid, threads;
  int sharedperproc;

  Cuda_Pair_PreKernel_AllStyles(sdata, sneighlist, eflag, vflag, grid, threads, sharedperproc, false, 256);

  hipStream_t* streams = (hipStream_t*) CudaWrapper_returnStreams();

  if(sdata->pair.use_block_per_atom)
    Pair_Kernel_BpA<PAIR_LJ_EXPAND, COUL_NONE, DATA_NONE>
    <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x, streams[1]>>> (eflag, vflag, eflag_atom, vflag_atom);
  else
    Pair_Kernel_TpA<PAIR_LJ_EXPAND, COUL_NONE, DATA_NONE>
    <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x, streams[1]>>> (eflag, vflag, eflag_atom, vflag_atom);

  Cuda_Pair_PostKernel_AllStyles(sdata, grid, sharedperproc, eflag, vflag);
}

#undef _lj1
#undef _lj2
#undef _lj3
#undef _lj4


