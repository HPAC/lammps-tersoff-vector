/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

//#define CUDA_PRECISION 1
#include "cuda_precision.h"
#include "cuda_common.h"
struct  FFT_DATA {
  FFT_CFLOAT re;
  FFT_CFLOAT im;
};

#include "fft3d_cuda_cu.h"
#include "fft3d_cuda_kernel.cu"
#include <stdio.h>

void initfftdata(double* in, FFT_CFLOAT* out, int nfast, int nmid, int nslow)
{

  dim3 grid;
  grid.x = nslow;
  grid.y = nmid;
  grid.z = 1;
  dim3 threads;
  threads.x = nfast;
  threads.y = 1;
  threads.z = 1;
  hipDeviceSynchronize();
  initfftdata_kernel <<< grid, threads, 0>>>(in, out);
  hipDeviceSynchronize();
  MYDBG(printf("ERROR-CUDA initfftdata_kernel: %s\n", hipGetErrorString(hipGetLastError())));
}


void permute(FFT_DATA* in, FFT_DATA* out, int nfast, int nmid, int nslow)
{

  dim3 grid;
  grid.x = nslow;
  grid.y = nmid;
  grid.z = 1;
  dim3 threads;
  threads.x = nfast * 2;
  threads.y = 1;
  threads.z = 1;
  permute_kernel <<< grid, threads, 0>>>((FFT_CFLOAT*)in, (FFT_CFLOAT*)out);
  hipDeviceSynchronize();
  MYDBG(printf("ERROR-CUDA permute_kernel: %s\n", hipGetErrorString(hipGetLastError())));
}

void permute_scale(FFT_DATA* in, FFT_DATA* out, int nfast, int nmid, int nslow)
{

  dim3 grid;
  grid.x = nslow;
  grid.y = nmid;
  grid.z = 1;
  dim3 threads;
  threads.x = nfast * 2;
  threads.y = 1;
  threads.z = 1;
  permute_kernel <<< grid, threads, 0>>>((FFT_CFLOAT*)in, (FFT_CFLOAT*)out);
  hipDeviceSynchronize();
}
void permute_part(FFT_DATA* in, FFT_DATA* out, int nfast, int nmid, int nslow, int ihi, int ilo, int jhi, int jlo, int khi, int klo)
{

  dim3 grid;
  grid.x = (ihi - ilo + 1);
  grid.y = (jhi - jlo + 1);
  grid.z = 1;
  dim3 threads;
  threads.x = (khi - klo + 1) * 2;
  threads.y = 1;
  threads.z = 1;
  permute_part_kernel <<< grid, threads, 0>>>((FFT_CFLOAT*)in, (FFT_CFLOAT*)out, nfast, nmid, nslow, ihi, ilo, jhi, jlo, khi, klo);
  hipDeviceSynchronize();
}

void FFTsyncthreads()
{
  hipDeviceSynchronize();
}

