#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

__device__ void v_tally(int &vflag_global, int &vflag_atom, int &n, int* list, ENERGY_CFLOAT total, ENERGY_CFLOAT* v)
{
  /*if(vflag_global)
  {
    ENERGY_CFLOAT fraction = n/total;
  ENERGY_CFLOAT* shared = &sharedmem[threadIdx.x];
    *shared += fraction*v[0]; shared+=blockDim.x;
    *shared += fraction*v[1]; shared+=blockDim.x;
    *shared += fraction*v[2]; shared+=blockDim.x;
    *shared += fraction*v[3]; shared+=blockDim.x;
    *shared += fraction*v[4]; shared+=blockDim.x;
    *shared += fraction*v[5];
  }*/
  if(vflag_atom) {
    ENERGY_CFLOAT fraction = ENERGY_F(1.0) / total;

    for(int i = 0; i < n; i++) {
      int m = list[i];
      ENERGY_CFLOAT* myvatom = &_vatom[m];

      *myvatom += fraction * v[0];
      myvatom += _nmax;
      *myvatom += fraction * v[1];
      myvatom += _nmax;
      *myvatom += fraction * v[2];
      myvatom += _nmax;
      *myvatom += fraction * v[3];
      myvatom += _nmax;
      *myvatom += fraction * v[4];
      myvatom += _nmax;
      *myvatom += fraction * v[5];
    }
  }
}

inline __device__ void minimum_image(X_CFLOAT3 &delta)
{
  if(_triclinic == 0) {
    if(_periodicity[0]) {
      delta.x += delta.x < -X_F(0.5) * _prd[0] ? _prd[0] :
                 (delta.x >  X_F(0.5) * _prd[0] ? -_prd[0] : X_F(0.0));
    }

    if(_periodicity[1]) {
      delta.y += delta.y < -X_F(0.5) * _prd[1] ? _prd[1] :
                 (delta.y >  X_F(0.5) * _prd[1] ? -_prd[1] : X_F(0.0));
    }

    if(_periodicity[2]) {
      delta.z += delta.z < -X_F(0.5) * _prd[2] ? _prd[2] :
                 (delta.z >  X_F(0.5) * _prd[2] ? -_prd[2] : X_F(0.0));
    }

  } else {
    if(_periodicity[1]) {
      delta.z += delta.z < -X_F(0.5) * _prd[2] ? _prd[2] :
                 (delta.z >  X_F(0.5) * _prd[2] ? -_prd[2] : X_F(0.0));
      delta.y += delta.z < -X_F(0.5) * _prd[2] ? _h[3] :
                 (delta.z >  X_F(0.5) * _prd[2] ? -_h[3] : X_F(0.0));
      delta.x += delta.z < -X_F(0.5) * _prd[2] ? _h[4] :
                 (delta.z >  X_F(0.5) * _prd[2] ? -_h[4] : X_F(0.0));

    }

    if(_periodicity[1]) {
      delta.y += delta.y < -X_F(0.5) * _prd[1] ? _prd[1] :
                 (delta.y >  X_F(0.5) * _prd[1] ? -_prd[1] : X_F(0.0));
      delta.x += delta.y < -X_F(0.5) * _prd[1] ? _h[5] :
                 (delta.y >  X_F(0.5) * _prd[1] ? -_h[5] : X_F(0.0));

    }

    if(_periodicity[0]) {
      delta.x += delta.x < -X_F(0.5) * _prd[0] ? _prd[0] :
                 (delta.x >  X_F(0.5) * _prd[0] ? -_prd[0] : X_F(0.0));
    }
  }
}

__global__ void FixShakeCuda_UnconstrainedUpdate_Kernel()
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i >= _nlocal) return;

  X_CFLOAT3 my_xshake = {X_F(0.0), X_F(0.0), X_F(0.0)};

  if(_shake_flag[i]) {
    F_CFLOAT* my_f = _f + i;
    V_CFLOAT* my_v = _v + i;
    X_CFLOAT* my_x = _x + i;

    V_CFLOAT 		dtfmsq = _dtfsq;

    if(_rmass_flag) dtfmsq *= V_F(1.0) / _rmass[i];
    else 			dtfmsq *= V_F(1.0) / _mass[_type[i]];

    my_xshake.x =  *my_x + _dtv* *my_v + dtfmsq* *my_f;
    my_f += _nmax;
    my_v += _nmax;
    my_x += _nmax;
    my_xshake.y =  *my_x + _dtv* *my_v + dtfmsq* *my_f;
    my_f += _nmax;
    my_v += _nmax;
    my_x += _nmax;
    my_xshake.z =  *my_x + _dtv* *my_v + dtfmsq* *my_f;
  }

  _xshake[i] = my_xshake;
}




__device__ void FixShakeCuda_Shake2(int &vflag, int &vflag_atom, int &m)
{
  int nlist, list[2];
  ENERGY_CFLOAT v[6];
  X_CFLOAT invmass0, invmass1;

  // local atom IDs and constraint distances

  int i0 = _map_array[_shake_atom[m]];
  int i1 = _map_array[_shake_atom[m + _nmax]];
  X_CFLOAT bond1 = _bond_distance[_shake_type[m]];

  // r01 = distance vec between atoms, with PBC

  X_CFLOAT3 r01;

  X_CFLOAT4 x_i0, x_i1;
  x_i0 = fetchXType(i0);
  x_i1 = fetchXType(i1);

  r01.x = x_i0.x - x_i1.x;
  r01.y = x_i0.y - x_i1.y;
  r01.z = x_i0.z - x_i1.z;
  minimum_image(r01);

  // s01 = distance vec after unconstrained update, with PBC

  X_CFLOAT3 s01;
  X_CFLOAT3 xs_i0 = _xshake[i0];
  X_CFLOAT3 xs_i1 = _xshake[i1];

  s01.x = xs_i0.x - xs_i1.x;
  s01.y = xs_i0.y - xs_i1.y;
  s01.z = xs_i0.z - xs_i1.z;
  minimum_image(s01);

  // scalar distances between atoms

  X_CFLOAT r01sq = r01.x * r01.x + r01.y * r01.y + r01.z * r01.z;
  X_CFLOAT s01sq = s01.x * s01.x + s01.y * s01.y + s01.z * s01.z;

  // a,b,c = coeffs in quadratic equation for lamda

  if(_rmass_flag) {
    invmass0 = X_F(1.0) / _rmass[i0];
    invmass1 = X_F(1.0) / _rmass[i1];
  } else {
    invmass0 = X_F(1.0) / _mass[static_cast <int>(x_i0.w)];
    invmass1 = X_F(1.0) / _mass[static_cast <int>(x_i1.w)];
  }

  X_CFLOAT a = (invmass0 + invmass1) * (invmass0 + invmass1) * r01sq;
  X_CFLOAT b = X_F(2.0) * (invmass0 + invmass1) *
              (s01.x * r01.x + s01.y * r01.y + s01.z * r01.z);
  X_CFLOAT c = s01sq - bond1 * bond1;

  // error check

  X_CFLOAT determ = b * b - X_F(4.0) * a * c;

  if(determ < X_F(0.0)) {
    _flag[0]++;
    determ = X_F(0.0);
  }

  // exact quadratic solution for lamda

  X_CFLOAT lamda, lamda1, lamda2;
  lamda1 = -b + _SQRT_(determ);
  lamda2 = -lamda1 - X_F(2.0) * b;
  lamda1 *= X_F(1.0) / (X_F(2.0) * a);
  lamda2 *= X_F(1.0) / (X_F(2.0) * a);

  lamda = (fabs(lamda1) <= fabs(lamda2)) ? lamda1 : lamda2;

  // update forces if atom is owned by this processor

  lamda *= X_F(1.0) / _dtfsq;


  //attenion: are shake clusters <-> atom unique?
  nlist = 0;

  if(i0 < _nlocal) {
    _f[i0]         += lamda * r01.x;
    _f[i0 + _nmax]   += lamda * r01.y;
    _f[i0 + 2 * _nmax] += lamda * r01.z;
    list[nlist++] = i0;
  }

  if(i1 < _nlocal) {
    _f[i1]         -= lamda * r01.x;
    _f[i1 + _nmax]   -= lamda * r01.y;
    _f[i1 + 2 * _nmax] -= lamda * r01.z;
    list[nlist++] = i1;
  }

  if(vflag || vflag_atom) {
    ENERGY_CFLOAT* shared = &sharedmem[threadIdx.x];
    X_CFLOAT factor = nlist;
    v[0] = lamda * r01.x * r01.x;
    *shared = factor * v[0];
    shared += blockDim.x; //times 2.0 since the reducing function is the same as in force calculations, which adds a factor 0.5
    v[1] = lamda * r01.y * r01.y;
    *shared = factor * v[1];
    shared += blockDim.x;
    v[2] = lamda * r01.z * r01.z;
    *shared = factor * v[2];
    shared += blockDim.x;
    v[3] = lamda * r01.x * r01.y;
    *shared = factor * v[3];
    shared += blockDim.x;
    v[4] = lamda * r01.x * r01.z;
    *shared = factor * v[4];
    shared += blockDim.x;
    v[5] = lamda * r01.y * r01.z;
    *shared = factor * v[5];
    shared += blockDim.x;

    v_tally(vflag, vflag_atom, nlist, list, 2.0, v);
  }
}


__device__ void FixShakeCuda_Shake3(int &vflag, int &vflag_atom, int &m)
{
  int nlist, list[3];
  ENERGY_CFLOAT v[6];
  X_CFLOAT invmass0, invmass1, invmass2;

  // local atom IDs and constraint distances

  int i0 = _map_array[_shake_atom[m]];
  int i1 = _map_array[_shake_atom[m + _nmax]];
  int i2 = _map_array[_shake_atom[m + 2 * _nmax]];
  X_CFLOAT bond1 = _bond_distance[_shake_type[m]];
  X_CFLOAT bond2 = _bond_distance[_shake_type[m + _nmax]];

  // r01 = distance vec between atoms, with PBC

  X_CFLOAT3 r01, r02;

  X_CFLOAT4 x_i0, x_i1, x_i2;
  x_i0 = fetchXType(i0);
  x_i1 = fetchXType(i1);
  x_i2 = fetchXType(i2);

  r01.x = x_i0.x - x_i1.x;
  r01.y = x_i0.y - x_i1.y;
  r01.z = x_i0.z - x_i1.z;
  minimum_image(r01);

  r02.x = x_i0.x - x_i2.x;
  r02.y = x_i0.y - x_i2.y;
  r02.z = x_i0.z - x_i2.z;
  minimum_image(r02);

  // s01 = distance vec after unconstrained update, with PBC

  X_CFLOAT3 s01, s02;
  X_CFLOAT3 xs_i0 = _xshake[i0];
  X_CFLOAT3 xs_i1 = _xshake[i1];
  X_CFLOAT3 xs_i2 = _xshake[i2];

  s01.x = xs_i0.x - xs_i1.x;
  s01.y = xs_i0.y - xs_i1.y;
  s01.z = xs_i0.z - xs_i1.z;
  minimum_image(s01);

  s02.x = xs_i0.x - xs_i2.x;
  s02.y = xs_i0.y - xs_i2.y;
  s02.z = xs_i0.z - xs_i2.z;
  minimum_image(s02);

  // scalar distances between atoms

  X_CFLOAT r01sq = r01.x * r01.x + r01.y * r01.y + r01.z * r01.z;
  X_CFLOAT r02sq = r02.x * r02.x + r02.y * r02.y + r02.z * r02.z;
  X_CFLOAT s01sq = s01.x * s01.x + s01.y * s01.y + s01.z * s01.z;
  X_CFLOAT s02sq = s02.x * s02.x + s02.y * s02.y + s02.z * s02.z;

  // a,b,c = coeffs in quadratic equation for lamda

  if(_rmass_flag) {
    invmass0 = X_F(1.0) / _rmass[i0];
    invmass1 = X_F(1.0) / _rmass[i1];
    invmass2 = X_F(1.0) / _rmass[i2];
  } else {
    invmass0 = X_F(1.0) / _mass[static_cast <int>(x_i0.w)];
    invmass1 = X_F(1.0) / _mass[static_cast <int>(x_i1.w)];
    invmass2 = X_F(1.0) / _mass[static_cast <int>(x_i2.w)];
  }

  X_CFLOAT a11 = X_F(2.0) * (invmass0 + invmass1) *
                (s01.x * r01.x + s01.y * r01.y + s01.z * r01.z);
  X_CFLOAT a12 = X_F(2.0) * invmass0 *
                (s01.x * r02.x + s01.y * r02.y + s01.z * r02.z);
  X_CFLOAT a21 = X_F(2.0) * invmass0 *
                (s02.x * r01.x + s02.y * r01.y + s02.z * r01.z);
  X_CFLOAT a22 = X_F(2.0) * (invmass0 + invmass2) *
                (s02.x * r02.x + s02.y * r02.y + s02.z * r02.z);

  // error check

  X_CFLOAT determ = a11 * a22 - a12 * a21;

  if(determ == X_F(0.0)) _flag[0]++;

  X_CFLOAT determinv = X_F(1.0) / determ;

  X_CFLOAT a11inv = a22 * determinv;
  X_CFLOAT a12inv = -a12 * determinv;
  X_CFLOAT a21inv = -a21 * determinv;
  X_CFLOAT a22inv = a11 * determinv;

  // quadratic correction coeffs

  X_CFLOAT r0102 = (r01.x * r02.x + r01.y * r02.y + r01.z * r02.z);

  X_CFLOAT quad1_0101 = (invmass0 + invmass1) * (invmass0 + invmass1) * r01sq;
  X_CFLOAT quad1_0202 = invmass0 * invmass0 * r02sq;
  X_CFLOAT quad1_0102 = X_F(2.0) * (invmass0 + invmass1) * invmass0 * r0102;

  X_CFLOAT quad2_0202 = (invmass0 + invmass2) * (invmass0 + invmass2) * r02sq;
  X_CFLOAT quad2_0101 = invmass0 * invmass0 * r01sq;
  X_CFLOAT quad2_0102 = X_F(2.0) * (invmass0 + invmass2) * invmass0 * r0102;

  // iterate until converged

  X_CFLOAT lamda01 = X_F(0.0);
  X_CFLOAT lamda02 = X_F(0.0);
  int niter = 0;
  int done = 0;

  X_CFLOAT quad1, quad2, b1, b2, lamda01_new, lamda02_new;

  //maybe all running full loop?
  while(__any(!done) && niter < _max_iter) {
    quad1 = quad1_0101 * lamda01 * lamda01 + quad1_0202 * lamda02 * lamda02 +
            quad1_0102 * lamda01 * lamda02;
    quad2 = quad2_0101 * lamda01 * lamda01 + quad2_0202 * lamda02 * lamda02 +
            quad2_0102 * lamda01 * lamda02;

    b1 = bond1 * bond1 - s01sq - quad1;
    b2 = bond2 * bond2 - s02sq - quad2;

    lamda01_new = a11inv * b1 + a12inv * b2;
    lamda02_new = a21inv * b1 + a22inv * b2;

    done++;
    done = (fabs(lamda01_new - lamda01) > _tolerance) ? 0 : done;
    done = (fabs(lamda02_new - lamda02) > _tolerance) ? 0 : done;


    lamda01 = done < 2 ? lamda01_new : lamda01;
    lamda02 = done < 2 ? lamda02_new : lamda02;
    niter++;
  }

  // update forces if atom is owned by this processor

  lamda01 *= X_F(1.0) / _dtfsq;
  lamda02 *= X_F(1.0) / _dtfsq;


  //attenion: are shake clusters <-> atom unique?
  nlist = 0;

  if(i0 < _nlocal) {
    _f[i0] += lamda01 * r01.x + lamda02 * r02.x;
    _f[i0 + _nmax] += lamda01 * r01.y + lamda02 * r02.y;
    _f[i0 + 2 * _nmax] += lamda01 * r01.z + lamda02 * r02.z;
    list[nlist++] = i0;
  }

  if(i1 < _nlocal) {
    _f[i1] -= lamda01 * r01.x;
    _f[i1 + _nmax] -= lamda01 * r01.y;
    _f[i1 + 2 * _nmax] -= lamda01 * r01.z;
    list[nlist++] = i1;
  }

  if(i2 < _nlocal) {
    _f[i2] -= lamda02 * r02.x;
    _f[i2 + _nmax] -= lamda02 * r02.y;
    _f[i2 + 2 * _nmax] -= lamda02 * r02.z;
    list[nlist++] = i2;
  }

  if(vflag || vflag_atom) {
    ENERGY_CFLOAT* shared = &sharedmem[threadIdx.x];
    X_CFLOAT factor = X_F(2.0) / X_F(3.0) * nlist;
    v[0] = lamda01 * r01.x * r01.x + lamda02 * r02.x * r02.x;
    *shared = factor * v[0];
    shared += blockDim.x; //times 2.0 since the reducing function is the same as in force calculations, which adds a factor 0.5
    v[1] = lamda01 * r01.y * r01.y + lamda02 * r02.y * r02.y;
    *shared = factor * v[1];
    shared += blockDim.x;
    v[2] = lamda01 * r01.z * r01.z + lamda02 * r02.z * r02.z;
    *shared = factor * v[2];
    shared += blockDim.x;
    v[3] = lamda01 * r01.x * r01.y + lamda02 * r02.x * r02.y;
    *shared = factor * v[3];
    shared += blockDim.x;
    v[4] = lamda01 * r01.x * r01.z + lamda02 * r02.x * r02.z;
    *shared = factor * v[4];
    shared += blockDim.x;
    v[5] = lamda01 * r01.y * r01.z + lamda02 * r02.y * r02.z;
    *shared = factor * v[5];
    shared += blockDim.x;

    v_tally(vflag, vflag_atom, nlist, list, 3.0, v);
  }
}

__device__ void FixShakeCuda_Shake4(int &vflag, int &vflag_atom, int &m)
{
  int nlist, list[4];
  ENERGY_CFLOAT v[6];
  X_CFLOAT invmass0, invmass1, invmass2, invmass3;

  // local atom IDs and constraint distances

  int i0 = _map_array[_shake_atom[m]];
  int i1 = _map_array[_shake_atom[m + _nmax]];
  int i2 = _map_array[_shake_atom[m + 2 * _nmax]];
  int i3 = _map_array[_shake_atom[m + 3 * _nmax]];
  X_CFLOAT bond1 = _bond_distance[_shake_type[m]];
  X_CFLOAT bond2 = _bond_distance[_shake_type[m + _nmax]];
  X_CFLOAT bond3 = _bond_distance[_shake_type[m + 2 * _nmax]];

  // r01 = distance vec between atoms, with PBC

  X_CFLOAT3 r01, r02, r03;

  X_CFLOAT4 x_i0, x_i1, x_i2, x_i3;
  x_i0 = fetchXType(i0);
  x_i1 = fetchXType(i1);
  x_i2 = fetchXType(i2);
  x_i3 = fetchXType(i3);

  r01.x = x_i0.x - x_i1.x;
  r01.y = x_i0.y - x_i1.y;
  r01.z = x_i0.z - x_i1.z;
  minimum_image(r01);

  r02.x = x_i0.x - x_i2.x;
  r02.y = x_i0.y - x_i2.y;
  r02.z = x_i0.z - x_i2.z;
  minimum_image(r02);

  r03.x = x_i0.x - x_i3.x;
  r03.y = x_i0.y - x_i3.y;
  r03.z = x_i0.z - x_i3.z;
  minimum_image(r03);

  // s01 = distance vec after unconstrained update, with PBC

  X_CFLOAT3 s01, s02, s03;
  X_CFLOAT3 xs_i0 = _xshake[i0];
  X_CFLOAT3 xs_i1 = _xshake[i1];
  X_CFLOAT3 xs_i2 = _xshake[i2];
  X_CFLOAT3 xs_i3 = _xshake[i3];

  s01.x = xs_i0.x - xs_i1.x;
  s01.y = xs_i0.y - xs_i1.y;
  s01.z = xs_i0.z - xs_i1.z;
  minimum_image(s01);

  s02.x = xs_i0.x - xs_i2.x;
  s02.y = xs_i0.y - xs_i2.y;
  s02.z = xs_i0.z - xs_i2.z;
  minimum_image(s02);

  s03.x = xs_i0.x - xs_i3.x;
  s03.y = xs_i0.y - xs_i3.y;
  s03.z = xs_i0.z - xs_i3.z;
  minimum_image(s03);

  // scalar distances between atoms

  X_CFLOAT r01sq = r01.x * r01.x + r01.y * r01.y + r01.z * r01.z;
  X_CFLOAT r02sq = r02.x * r02.x + r02.y * r02.y + r02.z * r02.z;
  X_CFLOAT r03sq = r03.x * r03.x + r03.y * r03.y + r03.z * r03.z;
  X_CFLOAT s01sq = s01.x * s01.x + s01.y * s01.y + s01.z * s01.z;
  X_CFLOAT s02sq = s02.x * s02.x + s02.y * s02.y + s02.z * s02.z;
  X_CFLOAT s03sq = s03.x * s03.x + s03.y * s03.y + s03.z * s03.z;

  // a,b,c = coeffs in quadratic equation for lamda

  if(_rmass_flag) {
    invmass0 = X_F(1.0) / _rmass[i0];
    invmass1 = X_F(1.0) / _rmass[i1];
    invmass2 = X_F(1.0) / _rmass[i2];
    invmass3 = X_F(1.0) / _rmass[i3];
  } else {
    invmass0 = X_F(1.0) / _mass[static_cast <int>(x_i0.w)];
    invmass1 = X_F(1.0) / _mass[static_cast <int>(x_i1.w)];
    invmass2 = X_F(1.0) / _mass[static_cast <int>(x_i2.w)];
    invmass3 = X_F(1.0) / _mass[static_cast <int>(x_i3.w)];
  }

  X_CFLOAT a11 = X_F(2.0) * (invmass0 + invmass1) *
                (s01.x * r01.x + s01.y * r01.y + s01.z * r01.z);
  X_CFLOAT a12 = X_F(2.0) * invmass0 *
                (s01.x * r02.x + s01.y * r02.y + s01.z * r02.z);
  X_CFLOAT a13 = X_F(2.0) * invmass0 *
                (s01.x * r03.x + s01.y * r03.y + s01.z * r03.z);
  X_CFLOAT a21 = X_F(2.0) * invmass0 *
                (s02.x * r01.x + s02.y * r01.y + s02.z * r01.z);
  X_CFLOAT a22 = X_F(2.0) * (invmass0 + invmass2) *
                (s02.x * r02.x + s02.y * r02.y + s02.z * r02.z);
  X_CFLOAT a23 = X_F(2.0) * (invmass0) *
                (s02.x * r03.x + s02.y * r03.y + s02.z * r03.z);
  X_CFLOAT a31 = X_F(2.0) * (invmass0) *
                (s03.x * r01.x + s03.y * r01.y + s03.z * r01.z);
  X_CFLOAT a32 = X_F(2.0) * (invmass0) *
                (s03.x * r02.x + s03.y * r02.y + s03.z * r02.z);
  X_CFLOAT a33 = X_F(2.0) * (invmass0 + invmass3) *
                (s03.x * r03.x + s03.y * r03.y + s03.z * r03.z);

  // error check

  X_CFLOAT determ = a11 * a22 * a33 + a12 * a23 * a31 + a13 * a21 * a32 -
                   a11 * a23 * a32 - a12 * a21 * a33 - a13 * a22 * a31;

  if(determ == X_F(0.0)) _flag[0]++;

  X_CFLOAT determinv = X_F(1.0) / determ;

  X_CFLOAT a11inv = determinv * (a22 * a33 - a23 * a32);
  X_CFLOAT a12inv = -determinv * (a12 * a33 - a13 * a32);
  X_CFLOAT a13inv = determinv * (a12 * a23 - a13 * a22);
  X_CFLOAT a21inv = -determinv * (a21 * a33 - a23 * a31);
  X_CFLOAT a22inv = determinv * (a11 * a33 - a13 * a31);
  X_CFLOAT a23inv = -determinv * (a11 * a23 - a13 * a21);
  X_CFLOAT a31inv = determinv * (a21 * a32 - a22 * a31);
  X_CFLOAT a32inv = -determinv * (a11 * a32 - a12 * a31);
  X_CFLOAT a33inv = determinv * (a11 * a22 - a12 * a21);

  // quadratic correction coeffs

  X_CFLOAT r0102 = (r01.x * r02.x + r01.y * r02.y + r01.z * r02.z);
  X_CFLOAT r0103 = (r01.x * r03.x + r01.y * r03.y + r01.z * r03.z);
  X_CFLOAT r0203 = (r02.x * r03.x + r02.y * r03.y + r02.z * r03.z);

  X_CFLOAT quad1_0101 = (invmass0 + invmass1) * (invmass0 + invmass1) * r01sq;
  X_CFLOAT quad1_0202 = invmass0 * invmass0 * r02sq;
  X_CFLOAT quad1_0303 = invmass0 * invmass0 * r03sq;
  X_CFLOAT quad1_0102 = X_F(2.0) * (invmass0 + invmass1) * invmass0 * r0102;
  X_CFLOAT quad1_0103 = X_F(2.0) * (invmass0 + invmass1) * invmass0 * r0103;
  X_CFLOAT quad1_0203 = X_F(2.0) * invmass0 * invmass0 * r0203;

  X_CFLOAT quad2_0101 = invmass0 * invmass0 * r01sq;
  X_CFLOAT quad2_0202 = (invmass0 + invmass2) * (invmass0 + invmass2) * r02sq;
  X_CFLOAT quad2_0303 = invmass0 * invmass0 * r03sq;
  X_CFLOAT quad2_0102 = X_F(2.0) * (invmass0 + invmass2) * invmass0 * r0102;
  X_CFLOAT quad2_0103 = X_F(2.0) * invmass0 * invmass0 * r0103;
  X_CFLOAT quad2_0203 = X_F(2.0) * (invmass0 + invmass2) * invmass0 * r0203;

  X_CFLOAT quad3_0101 = invmass0 * invmass0 * r01sq;
  X_CFLOAT quad3_0202 = invmass0 * invmass0 * r02sq;
  X_CFLOAT quad3_0303 = (invmass0 + invmass3) * (invmass0 + invmass3) * r03sq;
  X_CFLOAT quad3_0102 = X_F(2.0) * invmass0 * invmass0 * r0102;
  X_CFLOAT quad3_0103 = X_F(2.0) * (invmass0 + invmass3) * invmass0 * r0103;
  X_CFLOAT quad3_0203 = X_F(2.0) * (invmass0 + invmass3) * invmass0 * r0203;
  // iterate until converged

  X_CFLOAT lamda01 = X_F(0.0);
  X_CFLOAT lamda02 = X_F(0.0);
  X_CFLOAT lamda03 = X_F(0.0);
  int niter = 0;
  int done = 0;

  X_CFLOAT quad1, quad2, quad3, b1, b2, b3, lamda01_new, lamda02_new, lamda03_new;

  //maybe all running full loop?
  while(__any(!done) && niter < _max_iter) {
    quad1 = quad1_0101 * lamda01 * lamda01 +
            quad1_0202 * lamda02 * lamda02 +
            quad1_0303 * lamda03 * lamda03 +
            quad1_0102 * lamda01 * lamda02 +
            quad1_0103 * lamda01 * lamda03 +
            quad1_0203 * lamda02 * lamda03;

    quad2 = quad2_0101 * lamda01 * lamda01 +
            quad2_0202 * lamda02 * lamda02 +
            quad2_0303 * lamda03 * lamda03 +
            quad2_0102 * lamda01 * lamda02 +
            quad2_0103 * lamda01 * lamda03 +
            quad2_0203 * lamda02 * lamda03;

    quad3 = quad3_0101 * lamda01 * lamda01 +
            quad3_0202 * lamda02 * lamda02 +
            quad3_0303 * lamda03 * lamda03 +
            quad3_0102 * lamda01 * lamda02 +
            quad3_0103 * lamda01 * lamda03 +
            quad3_0203 * lamda02 * lamda03;

    b1 = bond1 * bond1 - s01sq - quad1;
    b2 = bond2 * bond2 - s02sq - quad2;
    b3 = bond3 * bond3 - s03sq - quad3;

    lamda01_new = a11inv * b1 + a12inv * b2 + a13inv * b3;
    lamda02_new = a21inv * b1 + a22inv * b2 + a23inv * b3;
    lamda03_new = a31inv * b1 + a32inv * b2 + a33inv * b3;

    done++;
    done = (fabs(lamda01_new - lamda01) > _tolerance) ? 0 : done;
    done = (fabs(lamda02_new - lamda02) > _tolerance) ? 0 : done;
    done = (fabs(lamda03_new - lamda03) > _tolerance) ? 0 : done;

    lamda01 = done < 2 ? lamda01_new : lamda01;
    lamda02 = done < 2 ? lamda02_new : lamda02;
    lamda03 = done < 2 ? lamda03_new : lamda03;
    niter++;
  }

  // update forces if atom is owned by this processor

  lamda01 *= X_F(1.0) / _dtfsq;
  lamda02 *= X_F(1.0) / _dtfsq;
  lamda03 *= X_F(1.0) / _dtfsq;


  //attenion: are shake clusters <-> atom unique?
  nlist = 0;

  if(i0 < _nlocal) {
    _f[i0] 			+= lamda01 * r01.x + lamda02 * r02.x + lamda03 * r03.x;
    _f[i0 + _nmax] 	+= lamda01 * r01.y + lamda02 * r02.y + lamda03 * r03.y;
    _f[i0 + 2 * _nmax] 	+= lamda01 * r01.z + lamda02 * r02.z + lamda03 * r03.z;
    list[nlist++] = i0;
  }

  if(i1 < _nlocal) {
    _f[i1] -= lamda01 * r01.x;
    _f[i1 + _nmax] -= lamda01 * r01.y;
    _f[i1 + 2 * _nmax] -= lamda01 * r01.z;
    list[nlist++] = i1;
  }

  if(i2 < _nlocal) {
    _f[i2] -= lamda02 * r02.x;
    _f[i2 + _nmax] -= lamda02 * r02.y;
    _f[i2 + 2 * _nmax] -= lamda02 * r02.z;
    list[nlist++] = i2;
  }

  if(i3 < _nlocal) {
    _f[i3] -= lamda03 * r03.x;
    _f[i3 + _nmax] -= lamda03 * r03.y;
    _f[i3 + 2 * _nmax] -= lamda03 * r03.z;
    list[nlist++] = i3;
  }

  if(vflag || vflag_atom) {
    ENERGY_CFLOAT* shared = &sharedmem[threadIdx.x];
    X_CFLOAT factor = X_F(2.0) / X_F(4.0) * nlist;
    v[0] = lamda01 * r01.x * r01.x + lamda02 * r02.x * r02.x + lamda03 * r03.x * r03.x;
    *shared = factor * v[0];
    shared += blockDim.x; //times 2.0 since the reducing function is the same as in force calculations, which adds a factor 0.5
    v[1] = lamda01 * r01.y * r01.y + lamda02 * r02.y * r02.y + lamda03 * r03.y * r03.y;
    *shared = factor * v[1];
    shared += blockDim.x;
    v[2] = lamda01 * r01.z * r01.z + lamda02 * r02.z * r02.z + lamda03 * r03.z * r03.z;
    *shared = factor * v[2];
    shared += blockDim.x;
    v[3] = lamda01 * r01.x * r01.y + lamda02 * r02.x * r02.y + lamda03 * r03.x * r03.y;
    *shared = factor * v[3];
    shared += blockDim.x;
    v[4] = lamda01 * r01.x * r01.z + lamda02 * r02.x * r02.z + lamda03 * r03.x * r03.z;
    *shared = factor * v[4];
    shared += blockDim.x;
    v[5] = lamda01 * r01.y * r01.z + lamda02 * r02.y * r02.z + lamda03 * r03.y * r03.z;
    *shared = factor * v[5];
    shared += blockDim.x;

    v_tally(vflag, vflag_atom, nlist, list, 4.0, v);
  }
}

__device__ void FixShakeCuda_Shake3Angle(int &vflag, int &vflag_atom, int &m)
{
  int nlist, list[3];
  ENERGY_CFLOAT v[6];
  X_CFLOAT invmass0, invmass1, invmass2;

  // local atom IDs and constraint distances

  int i0 = _map_array[_shake_atom[m]];
  int i1 = _map_array[_shake_atom[m + _nmax]];
  int i2 = _map_array[_shake_atom[m + 2 * _nmax]];
  X_CFLOAT bond1 = _bond_distance[_shake_type[m]];
  X_CFLOAT bond2 = _bond_distance[_shake_type[m + _nmax]];
  X_CFLOAT bond12 = _angle_distance[_shake_type[m + 2 * _nmax]];

  // r01 = distance vec between atoms, with PBC

  X_CFLOAT3 r01, r02, r12;

  X_CFLOAT4 x_i0, x_i1, x_i2;
  x_i0 = fetchXType(i0);
  x_i1 = fetchXType(i1);
  x_i2 = fetchXType(i2);

  r01.x = x_i0.x - x_i1.x;
  r01.y = x_i0.y - x_i1.y;
  r01.z = x_i0.z - x_i1.z;
  minimum_image(r01);

  r02.x = x_i0.x - x_i2.x;
  r02.y = x_i0.y - x_i2.y;
  r02.z = x_i0.z - x_i2.z;
  minimum_image(r02);

  r12.x = x_i1.x - x_i2.x;
  r12.y = x_i1.y - x_i2.y;
  r12.z = x_i1.z - x_i2.z;
  minimum_image(r12);

  // s01 = distance vec after unconstrained update, with PBC

  X_CFLOAT3 s01, s02, s12;
  X_CFLOAT3 xs_i0 = _xshake[i0];
  X_CFLOAT3 xs_i1 = _xshake[i1];
  X_CFLOAT3 xs_i2 = _xshake[i2];

  s01.x = xs_i0.x - xs_i1.x;
  s01.y = xs_i0.y - xs_i1.y;
  s01.z = xs_i0.z - xs_i1.z;
  minimum_image(s01);

  s02.x = xs_i0.x - xs_i2.x;
  s02.y = xs_i0.y - xs_i2.y;
  s02.z = xs_i0.z - xs_i2.z;
  minimum_image(s02);

  s12.x = xs_i1.x - xs_i2.x;
  s12.y = xs_i1.y - xs_i2.y;
  s12.z = xs_i1.z - xs_i2.z;
  minimum_image(s12);

  // scalar distances between atoms

  X_CFLOAT r01sq = r01.x * r01.x + r01.y * r01.y + r01.z * r01.z;
  X_CFLOAT r02sq = r02.x * r02.x + r02.y * r02.y + r02.z * r02.z;
  X_CFLOAT r12sq = r12.x * r12.x + r12.y * r12.y + r12.z * r12.z;
  X_CFLOAT s01sq = s01.x * s01.x + s01.y * s01.y + s01.z * s01.z;
  X_CFLOAT s02sq = s02.x * s02.x + s02.y * s02.y + s02.z * s02.z;
  X_CFLOAT s12sq = s12.x * s12.x + s12.y * s12.y + s12.z * s12.z;

  // a,b,c = coeffs in quadratic equation for lamda

  if(_rmass_flag) {
    invmass0 = X_F(1.0) / _rmass[i0];
    invmass1 = X_F(1.0) / _rmass[i1];
    invmass2 = X_F(1.0) / _rmass[i2];
  } else {
    invmass0 = X_F(1.0) / _mass[static_cast <int>(x_i0.w)];
    invmass1 = X_F(1.0) / _mass[static_cast <int>(x_i1.w)];
    invmass2 = X_F(1.0) / _mass[static_cast <int>(x_i2.w)];
  }

  X_CFLOAT a11 = X_F(2.0) * (invmass0 + invmass1) *
                (s01.x * r01.x + s01.y * r01.y + s01.z * r01.z);
  X_CFLOAT a12 = X_F(2.0) * invmass0 *
                (s01.x * r02.x + s01.y * r02.y + s01.z * r02.z);
  X_CFLOAT a13 = - X_F(2.0) * invmass1 *
                (s01.x * r12.x + s01.y * r12.y + s01.z * r12.z);
  X_CFLOAT a21 = X_F(2.0) * invmass0 *
                (s02.x * r01.x + s02.y * r01.y + s02.z * r01.z);
  X_CFLOAT a22 = X_F(2.0) * (invmass0 + invmass2) *
                (s02.x * r02.x + s02.y * r02.y + s02.z * r02.z);
  X_CFLOAT a23 = X_F(2.0) * invmass2 *
                (s02.x * r12.x + s02.y * r12.y + s02.z * r12.z);
  X_CFLOAT a31 = - X_F(2.0) * invmass1 *
                (s12.x * r01.x + s12.y * r01.y + s12.z * r01.z);
  X_CFLOAT a32 = X_F(2.0) * invmass2 *
                (s12.x * r02.x + s12.y * r02.y + s12.z * r02.z);
  X_CFLOAT a33 = X_F(2.0) * (invmass1 + invmass2) *
                (s12.x * r12.x + s12.y * r12.y + s12.z * r12.z);

  // inverse of matrix

  X_CFLOAT determ = a11 * a22 * a33 + a12 * a23 * a31 + a13 * a21 * a32 -
                   a11 * a23 * a32 - a12 * a21 * a33 - a13 * a22 * a31;

  if(determ == X_F(0.0)) _flag[0]++;

  X_CFLOAT determinv = X_F(1.0) / determ;

  X_CFLOAT a11inv = determinv * (a22 * a33 - a23 * a32);
  X_CFLOAT a12inv = -determinv * (a12 * a33 - a13 * a32);
  X_CFLOAT a13inv = determinv * (a12 * a23 - a13 * a22);
  X_CFLOAT a21inv = -determinv * (a21 * a33 - a23 * a31);
  X_CFLOAT a22inv = determinv * (a11 * a33 - a13 * a31);
  X_CFLOAT a23inv = -determinv * (a11 * a23 - a13 * a21);
  X_CFLOAT a31inv = determinv * (a21 * a32 - a22 * a31);
  X_CFLOAT a32inv = -determinv * (a11 * a32 - a12 * a31);
  X_CFLOAT a33inv = determinv * (a11 * a22 - a12 * a21);

  // quadratic correction coeffs

  X_CFLOAT r0102 = (r01.x * r02.x + r01.y * r02.y + r01.z * r02.z);
  X_CFLOAT r0112 = (r01.x * r12.x + r01.y * r12.y + r01.z * r12.z);
  X_CFLOAT r0212 = (r02.x * r12.x + r02.y * r12.y + r02.z * r12.z);

  X_CFLOAT quad1_0101 = (invmass0 + invmass1) * (invmass0 + invmass1) * r01sq;
  X_CFLOAT quad1_0202 = invmass0 * invmass0 * r02sq;
  X_CFLOAT quad1_1212 = invmass1 * invmass1 * r12sq;
  X_CFLOAT quad1_0102 = X_F(2.0) * (invmass0 + invmass1) * invmass0 * r0102;
  X_CFLOAT quad1_0112 = - X_F(2.0) * (invmass0 + invmass1) * invmass1 * r0112;
  X_CFLOAT quad1_0212 = - X_F(2.0) * invmass0 * invmass1 * r0212;

  X_CFLOAT quad2_0101 = invmass0 * invmass0 * r01sq;
  X_CFLOAT quad2_0202 = (invmass0 + invmass2) * (invmass0 + invmass2) * r02sq;
  X_CFLOAT quad2_1212 = invmass2 * invmass2 * r12sq;
  X_CFLOAT quad2_0102 = X_F(2.0) * (invmass0 + invmass2) * invmass0 * r0102;
  X_CFLOAT quad2_0112 = X_F(2.0) * invmass0 * invmass2 * r0112;
  X_CFLOAT quad2_0212 = X_F(2.0) * (invmass0 + invmass2) * invmass2 * r0212;

  X_CFLOAT quad3_0101 = invmass1 * invmass1 * r01sq;
  X_CFLOAT quad3_0202 = invmass2 * invmass2 * r02sq;
  X_CFLOAT quad3_1212 = (invmass1 + invmass2) * (invmass1 + invmass2) * r12sq;
  X_CFLOAT quad3_0102 = - X_F(2.0) * invmass1 * invmass2 * r0102;
  X_CFLOAT quad3_0112 = - X_F(2.0) * (invmass1 + invmass2) * invmass1 * r0112;
  X_CFLOAT quad3_0212 = X_F(2.0) * (invmass1 + invmass2) * invmass2 * r0212;
  // iterate until converged

  X_CFLOAT lamda01 = X_F(0.0);
  X_CFLOAT lamda02 = X_F(0.0);
  X_CFLOAT lamda12 = X_F(0.0);
  int niter = 0;
  int done = 0;

  X_CFLOAT quad1, quad2, quad3, b1, b2, b3, lamda01_new, lamda02_new, lamda12_new;

  //maybe all running full loop?
  while(__any(!done) && niter < _max_iter) {
    quad1 = quad1_0101 * lamda01 * lamda01 +
            quad1_0202 * lamda02 * lamda02 +
            quad1_1212 * lamda12 * lamda12 +
            quad1_0102 * lamda01 * lamda02 +
            quad1_0112 * lamda01 * lamda12 +
            quad1_0212 * lamda02 * lamda12;

    quad2 = quad2_0101 * lamda01 * lamda01 +
            quad2_0202 * lamda02 * lamda02 +
            quad2_1212 * lamda12 * lamda12 +
            quad2_0102 * lamda01 * lamda02 +
            quad2_0112 * lamda01 * lamda12 +
            quad2_0212 * lamda02 * lamda12;

    quad3 = quad3_0101 * lamda01 * lamda01 +
            quad3_0202 * lamda02 * lamda02 +
            quad3_1212 * lamda12 * lamda12 +
            quad3_0102 * lamda01 * lamda02 +
            quad3_0112 * lamda01 * lamda12 +
            quad3_0212 * lamda02 * lamda12;

    b1 = bond1 * bond1 - s01sq - quad1;
    b2 = bond2 * bond2 - s02sq - quad2;
    b3 = bond12 * bond12 - s12sq - quad3;

    lamda01_new = a11inv * b1 + a12inv * b2 + a13inv * b3;
    lamda02_new = a21inv * b1 + a22inv * b2 + a23inv * b3;
    lamda12_new = a31inv * b1 + a32inv * b2 + a33inv * b3;

    done++;
    done = (fabs(lamda01_new - lamda01) > _tolerance) ? 0 : done;
    done = (fabs(lamda02_new - lamda02) > _tolerance) ? 0 : done;
    done = (fabs(lamda12_new - lamda12) > _tolerance) ? 0 : done;

    lamda01 = done < 2 ? lamda01_new : lamda01;
    lamda02 = done < 2 ? lamda02_new : lamda02;
    lamda12 = done < 2 ? lamda12_new : lamda12;
    niter++;
  }

  // update forces if atom is owned by this processor

  lamda01 *= X_F(1.0) / _dtfsq;
  lamda02 *= X_F(1.0) / _dtfsq;
  lamda12 *= X_F(1.0) / _dtfsq;


  //attenion: are shake clusters <-> atom unique?
  nlist = 0;

  if(i0 < _nlocal) {
    _f[i0] 			+= lamda01 * r01.x + lamda02 * r02.x;
    _f[i0 + _nmax] 	+= lamda01 * r01.y + lamda02 * r02.y;
    _f[i0 + 2 * _nmax] 	+= lamda01 * r01.z + lamda02 * r02.z;
    list[nlist++] = i0;
  }

  if(i1 < _nlocal) {
    _f[i1] 			-= lamda01 * r01.x - lamda12 * r12.x;
    _f[i1 + _nmax] 	-= lamda01 * r01.y - lamda12 * r12.y;
    _f[i1 + 2 * _nmax] 	-= lamda01 * r01.z - lamda12 * r12.z;
    list[nlist++] = i1;
  }

  if(i2 < _nlocal) {
    _f[i2] 			-= lamda02 * r02.x + lamda12 * r12.x;
    _f[i2 + _nmax] 	-= lamda02 * r02.y + lamda12 * r12.y;
    _f[i2 + 2 * _nmax] 	-= lamda02 * r02.z + lamda12 * r12.z;
    list[nlist++] = i2;
  }

  if(vflag || vflag_atom) {
    ENERGY_CFLOAT* shared = &sharedmem[threadIdx.x];
    X_CFLOAT factor = X_F(2.0) / X_F(3.0) * nlist;
    v[0] = lamda01 * r01.x * r01.x + lamda02 * r02.x * r02.x + lamda12 * r12.x * r12.x;
    *shared = factor * v[0];
    shared += blockDim.x; //times 2.0 since the reducing function is the same as in force calculations, which adds a factor 0.5
    v[1] = lamda01 * r01.y * r01.y + lamda02 * r02.y * r02.y + lamda12 * r12.y * r12.y;
    *shared = factor * v[1];
    shared += blockDim.x;
    v[2] = lamda01 * r01.z * r01.z + lamda02 * r02.z * r02.z + lamda12 * r12.z * r12.z;
    *shared = factor * v[2];
    shared += blockDim.x;
    v[3] = lamda01 * r01.x * r01.y + lamda02 * r02.x * r02.y + lamda12 * r12.x * r12.y;
    *shared = factor * v[3];
    shared += blockDim.x;
    v[4] = lamda01 * r01.x * r01.z + lamda02 * r02.x * r02.z + lamda12 * r12.x * r12.z;
    *shared = factor * v[4];
    shared += blockDim.x;
    v[5] = lamda01 * r01.y * r01.z + lamda02 * r02.y * r02.z + lamda12 * r12.y * r12.z;
    *shared = factor * v[5];
    shared += blockDim.x;

    v_tally(vflag, vflag_atom, nlist, list, 3.0, v);
  }
}

__global__ void FixShakeCuda_Shake_Kernel(int vflag, int vflag_atom, int* list, int nlist)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < nlist) {

    int m = list[i];
    int sflag = _shake_flag[m];

    if(sflag == 2) FixShakeCuda_Shake2(vflag, vflag_atom, m);
    else if(sflag == 3) FixShakeCuda_Shake3(vflag, vflag_atom, m);
    else if(sflag == 4) FixShakeCuda_Shake4(vflag, vflag_atom, m);
    else FixShakeCuda_Shake3Angle(vflag, vflag_atom, m);
  } else {
    ENERGY_CFLOAT* shared = &sharedmem[threadIdx.x];
    *shared = ENERGY_F(0.0);
    shared += blockDim.x;
    *shared = ENERGY_F(0.0);
    shared += blockDim.x;
    *shared = ENERGY_F(0.0);
    shared += blockDim.x;
    *shared = ENERGY_F(0.0);
    shared += blockDim.x;
    *shared = ENERGY_F(0.0);
    shared += blockDim.x;
    *shared = ENERGY_F(0.0);
  }

  if(vflag) {
    __syncthreads();
    int eflag = 0;
    PairVirialCompute_A_Kernel(eflag, vflag);
  }

}

__global__ void FixShakeCuda_PackComm_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_CFLOAT dx, X_CFLOAT dy, X_CFLOAT dz)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];

    if(j > _nmax) _flag[0] = 1;

    X_CFLOAT3 xs = _xshake[j];
    ((X_CFLOAT*) _buffer)[i] = xs.x + dx;
    ((X_CFLOAT*) _buffer)[i + 1 * n] = xs.y + dy;
    ((X_CFLOAT*) _buffer)[i + 2 * n] = xs.z + dz;
  }

}

__global__ void FixShakeCuda_PackComm_Self_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_CFLOAT dx, X_CFLOAT dy, X_CFLOAT dz, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];

    if(j > _nmax) _flag[0] = 1;

    X_CFLOAT3 xs = _xshake[j];
    xs.x += dx;
    xs.y += dy;
    xs.z += dz;
    _xshake[i + first] = xs;
  }

}

__global__ void FixShakeCuda_UnpackComm_Kernel(int n, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < n) {
    X_CFLOAT3 xs;
    xs.x = ((X_CFLOAT*) _buffer)[i];
    xs.y = ((X_CFLOAT*) _buffer)[i + 1 * n];
    xs.z = ((X_CFLOAT*) _buffer)[i + 2 * n];
    _xshake[i + first] = xs;
  }
}

