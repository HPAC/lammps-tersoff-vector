#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

__global__ void initfftdata_kernel(double* in, FFT_CFLOAT* out)
{
  out[2 * (((blockIdx.x * gridDim.y + blockIdx.y)*blockDim.x) + threadIdx.x)] = in[((blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x) + threadIdx.x];
  out[2 * (((blockIdx.x * gridDim.y + blockIdx.y)*blockDim.x) + threadIdx.x) + 1] = 0;
}


__global__ void permute_kernel(FFT_CFLOAT* in, FFT_CFLOAT* out)
{
  out[2 * (((threadIdx.x / 2)*gridDim.x + blockIdx.x)*gridDim.y + blockIdx.y) + threadIdx.x - 2 * (threadIdx.x / 2)] = in[((blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x) + threadIdx.x];
}

__global__ void permute_scale_kernel(FFT_CFLOAT* in, FFT_CFLOAT* out)
{
  out[2 * (((threadIdx.x / 2)*gridDim.x + blockIdx.x)*gridDim.y + blockIdx.y) + threadIdx.x - 2 * (threadIdx.x / 2)] = in[((blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x) + threadIdx.x] * gridDim.x * gridDim.y * blockDim.x * 0.5;
}

__global__ void permute_part_kernel(FFT_CFLOAT* in, FFT_CFLOAT* out, int nfast, int nmid, int nslow, int ihi, int ilo, int jhi, int jlo, int khi, int klo)
{
  {
    out[2 * ((threadIdx.x / 2) * (ihi - ilo + 1) * (jhi - jlo + 1) + (blockIdx.x) * (jhi - jlo + 1) + blockIdx.y - jlo) + threadIdx.x - 2 * (threadIdx.x / 2)] = in[2 * (blockIdx.x + ilo) * nmid * nslow + 2 * (blockIdx.y + jlo) * nmid + threadIdx.x + 2 * klo];
  }
}
