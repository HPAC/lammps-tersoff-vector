#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */


__global__ void Cuda_FixEnforce2dCuda_PostForce_Kernel(int groupbit)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal)
    if(_mask[i] & groupbit) {
      _v[i + 2 * _nmax] = V_F(0.0);
      _f[i + 2 * _nmax] = F_F(0.0);
    }
}
