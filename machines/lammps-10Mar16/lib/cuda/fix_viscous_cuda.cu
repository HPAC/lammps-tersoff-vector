#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX fix_viscous_cuda
#include "cuda_shared.h"
#include "cuda_common.h"
#include "crm_cuda_utils.cu"

#include "fix_viscous_cuda_cu.h"
#include "fix_viscous_cuda_kernel.cu"

void Cuda_FixViscousCuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(v)       , & sdata->atom.x    .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(f)       , & sdata->atom.f    .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(type)    , & sdata->atom.type .dev_data, sizeof(int*));
}

void Cuda_FixViscousCuda_Init(cuda_shared_data* sdata)
{
  Cuda_FixViscousCuda_UpdateNmax(sdata);

}


void Cuda_FixViscousCuda_PostForce(cuda_shared_data* sdata, int groupbit, void* gamma)
{
  if(sdata->atom.update_nmax)
    Cuda_FixViscousCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));


  int3 layout = getgrid(sdata->atom.nlocal, 0);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  Cuda_FixViscousCuda_PostForce_Kernel <<< grid, threads, 0>>> (groupbit, (F_CFLOAT*) gamma);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Cuda_FixViscousCuda_PostForce: Kernel execution failed");

}
