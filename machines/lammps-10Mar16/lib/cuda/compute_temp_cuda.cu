#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX compute_temp_cuda
#include "cuda_shared.h"
#include "cuda_common.h"

#include "crm_cuda_utils.cu"

#include "compute_temp_cuda_cu.h"
#include "compute_temp_cuda_kernel.cu"

void Cuda_ComputeTempCuda_UpdateBuffer(cuda_shared_data* sdata)
{
  int size = (unsigned)((sdata->atom.nlocal + 63) / 64.0) * 6 * sizeof(ENERGY_CFLOAT);

  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_ComputeTempCuda Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
    CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)
  }

  hipMemcpyToSymbol(MY_AP(buffer), & sdata->buffer, sizeof(int*));
}

void Cuda_ComputeTempCuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(mass)    , & sdata->atom.mass .dev_data, sizeof(V_CFLOAT*));

  if(sdata->atom.rmass_flag)
    hipMemcpyToSymbol(MY_AP(rmass)   , & sdata->atom.rmass.dev_data, sizeof(V_CFLOAT*));

  hipMemcpyToSymbol(MY_AP(rmass_flag)   , & sdata->atom.rmass_flag, sizeof(int));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(v)       , & sdata->atom.v    .dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(type)       , & sdata->atom.type    .dev_data, sizeof(int*));
}

void Cuda_ComputeTempCuda_Init(cuda_shared_data* sdata)
{
  Cuda_ComputeTempCuda_UpdateNmax(sdata);
}


void Cuda_ComputeTempCuda_Vector(cuda_shared_data* sdata, int groupbit, ENERGY_CFLOAT* t)
{
  //if(sdata->atom.update_nmax) //is most likely not called every timestep, therefore update of constants is necessary
  Cuda_ComputeTempCuda_UpdateNmax(sdata);
  //if(sdata->atom.update_nlocal)
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  //if(sdata->buffer_new)
  Cuda_ComputeTempCuda_UpdateBuffer(sdata);

  int3 layout = getgrid(sdata->atom.nlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    Cuda_ComputeTempCuda_Vector_Kernel <<< grid, threads, threads.x* 6* sizeof(ENERGY_CFLOAT)>>> (groupbit);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_ComputeTempCuda_Vector: compute_vector Kernel execution failed");

    int oldgrid = grid.x * grid.y;
    grid.x = 6;
    grid.y = 1;
    threads.x = 512;
    Cuda_ComputeTempCuda_Reduce_Kernel <<< grid, threads, threads.x* sizeof(ENERGY_CFLOAT)>>> (oldgrid, t);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_ComputeTempCuda_Vector: reduce_vector Kernel execution failed");
  }
}

void Cuda_ComputeTempCuda_Scalar(cuda_shared_data* sdata, int groupbit, ENERGY_CFLOAT* t)
{
  //if(sdata->atom.update_nmax) //is most likely not called every timestep, therefore update of constants is necessary
  Cuda_ComputeTempCuda_UpdateNmax(sdata);
  //if(sdata->atom.update_nlocal)
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  //if(sdata->buffer_new)
  Cuda_ComputeTempCuda_UpdateBuffer(sdata);
  MYDBG(printf("#CUDA ComputeTempCuda_Scalar: %i\n", sdata->atom.nlocal);)
  int3 layout = getgrid(sdata->atom.nlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    CUT_CHECK_ERROR("Cuda_ComputeTempCuda_Scalar: pre compute_scalar Kernel");
    Cuda_ComputeTempCuda_Scalar_Kernel <<< grid, threads, threads.x* sizeof(ENERGY_CFLOAT)>>> (groupbit);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_ComputeTempCuda_Scalar: compute_scalar Kernel execution failed");

    int oldgrid = grid.x * grid.y;
    grid.x = 1;
    grid.y = 1;
    threads.x = 512;
    Cuda_ComputeTempCuda_Reduce_Kernel <<< grid, threads, threads.x* sizeof(ENERGY_CFLOAT)>>> (oldgrid, t);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_ComputeTempCuda_Scalar: reduce_scalar Kernel execution failed");
  }
}
