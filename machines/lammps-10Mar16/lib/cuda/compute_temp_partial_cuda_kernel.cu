#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

extern __shared__ ENERGY_CFLOAT sharedmem[];


__global__ void Cuda_ComputeTempPartialCuda_Scalar_Kernel(int groupbit, int xflag, int yflag, int zflag)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  sharedmem[threadIdx.x] = 0;

  if(i < _nlocal) {
    if(_rmass_flag) {
      if(_mask[i] & groupbit)
        sharedmem[threadIdx.x] = (_v[i] * _v[i] * xflag + _v[i + _nmax] * _v[i + _nmax] * yflag + _v[i + 2 * _nmax] * _v[i + 2 * _nmax] * zflag) * _rmass[i];
    } else {
      if(_mask[i] & groupbit)
        sharedmem[threadIdx.x] = (_v[i] * _v[i] * xflag + _v[i + _nmax] * _v[i + _nmax] * yflag + _v[i + 2 * _nmax] * _v[i + 2 * _nmax] * zflag) * (_mass[_type[i]]);
    }
  }

  reduceBlock(sharedmem);
  ENERGY_CFLOAT* buffer = (ENERGY_CFLOAT*) _buffer;

  if(threadIdx.x == 0) {
    buffer[blockIdx.x * gridDim.y + blockIdx.y] = sharedmem[0];
  }
}

__global__ void Cuda_ComputeTempPartialCuda_Vector_Kernel(int groupbit, int xflag, int yflag, int zflag)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  sharedmem[threadIdx.x] = 0;
  sharedmem[threadIdx.x + blockDim.x] = 0;
  sharedmem[threadIdx.x + 2 * blockDim.x] = 0;
  sharedmem[threadIdx.x + 3 * blockDim.x] = 0;
  sharedmem[threadIdx.x + 4 * blockDim.x] = 0;
  sharedmem[threadIdx.x + 5 * blockDim.x] = 0;

  if(i < _nlocal)
    if(_mask[i] & groupbit) {
      V_CFLOAT massone;

      if(_rmass_flag) massone = _rmass[i];
      else massone = _mass[_type[i]];

      sharedmem[threadIdx.x] = massone * _v[i] * _v[i] * xflag;
      sharedmem[threadIdx.x + blockDim.x] = massone * _v[i + _nmax] * _v[i + _nmax] * yflag;
      sharedmem[threadIdx.x + 2 * blockDim.x] = massone * _v[i + 2 * _nmax] * _v[i + 2 * _nmax] * zflag;
      sharedmem[threadIdx.x + 3 * blockDim.x] = massone * _v[i] * _v[i + _nmax] * xflag * yflag;
      sharedmem[threadIdx.x + 4 * blockDim.x] = massone * _v[i] * _v[i + 2 * _nmax] * xflag * zflag;
      sharedmem[threadIdx.x + 5 * blockDim.x] = massone * _v[i + _nmax] * _v[i + 2 * _nmax] * yflag * zflag;
    }

  reduceBlock(sharedmem);
  reduceBlock(&sharedmem[blockDim.x]);
  reduceBlock(&sharedmem[2 * blockDim.x]);
  reduceBlock(&sharedmem[3 * blockDim.x]);
  reduceBlock(&sharedmem[4 * blockDim.x]);
  reduceBlock(&sharedmem[5 * blockDim.x]);
  ENERGY_CFLOAT* buffer = (ENERGY_CFLOAT*) _buffer;

  if(threadIdx.x == 0) {
    buffer[blockIdx.x * gridDim.y + blockIdx.y] = sharedmem[0];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + gridDim.x * gridDim.y] = sharedmem[blockDim.x];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + 2 * gridDim.x * gridDim.y] = sharedmem[2 * blockDim.x];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + 3 * gridDim.x * gridDim.y] = sharedmem[3 * blockDim.x];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + 4 * gridDim.x * gridDim.y] = sharedmem[4 * blockDim.x];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + 5 * gridDim.x * gridDim.y] = sharedmem[5 * blockDim.x];
  }
}


__global__ void Cuda_ComputeTempPartialCuda_Reduce_Kernel(int n, ENERGY_CFLOAT* t)
{
  int i = 0;
  sharedmem[threadIdx.x] = 0;
  ENERGY_CFLOAT myforig = 0.0;
  ENERGY_CFLOAT* buf = (ENERGY_CFLOAT*) _buffer;
  buf = &buf[blockIdx.x * n];

  while(i < n) {
    sharedmem[threadIdx.x] = 0;

    if(i + threadIdx.x < n)
      sharedmem[threadIdx.x] = buf[i + threadIdx.x];

    __syncthreads();
    reduceBlock(sharedmem);
    i += blockDim.x;

    if(threadIdx.x == 0)
      myforig += sharedmem[0];
  }

  if(threadIdx.x == 0)
    t[blockIdx.x] = myforig;
}

__global__ void Cuda_ComputeTempPartialCuda_RemoveBiasAll_Kernel(int groupbit, int xflag, int yflag, int zflag, V_CFLOAT* vbiasall)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal)
    if(_mask[i] & groupbit) {
      if(!xflag) {
        vbiasall[i] = _v[i];
        _v[i] = V_F(0.0);
      }

      if(!yflag) {
        vbiasall[i + _nmax] = _v[i + _nmax];
        _v[i + _nmax] = V_F(0.0);
      }

      if(!zflag) {
        vbiasall[i + 2 * _nmax] = _v[i + 2 * _nmax];
        _v[i + 2 * _nmax] = V_F(0.0);
      }
    }
}

__global__ void Cuda_ComputeTempPartialCuda_RestoreBiasAll_Kernel(int groupbit, int xflag, int yflag, int zflag, V_CFLOAT* vbiasall)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal)
    if(_mask[i] & groupbit) {
      if(!xflag) {
        _v[i] += vbiasall[i];
      }

      if(!yflag) {
        _v[i + _nmax] += vbiasall[i + _nmax];
      }

      if(!zflag) {
        _v[i + 2 * _nmax] += vbiasall[i + 2 * _nmax];
      }
    }
}
