#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */
#define RIMLARGER 1.000001
#define RIMSMALLER 0.999999
#define SMALL 1e-5

extern __shared__ int shared[];

template <const unsigned int data_mask>
__global__ void Cuda_AtomVecCuda_PackComm_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_CFLOAT dx, X_CFLOAT dy, X_CFLOAT dz, void* buffer)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];

    if(j > _nmax) _flag[0] = 1;

    int k = 0;

    if(data_mask & X_MASK) {
      ((X_CFLOAT*) buffer)[i + k * n] = _x[j] + dx;
      k++;
      ((X_CFLOAT*) buffer)[i + k * n] = _x[j + _nmax] + dy;
      k++;
      ((X_CFLOAT*) buffer)[i + k * n] = _x[j + 2 * _nmax] + dz;
      k++;
    }

    if(data_mask & V_MASK) {
      ((X_CFLOAT*) buffer)[i + k * n] = _v[j];
      k++;
      ((X_CFLOAT*) buffer)[i + k * n] = _v[j + _nmax];
      k++;
      ((X_CFLOAT*) buffer)[i + k * n] = _v[j + 2 * _nmax];
      k++;
    }

    if(data_mask & OMEGA_MASK) {
      ((X_CFLOAT*) buffer)[i + k * n] = _omega[j];
      k++;
      ((X_CFLOAT*) buffer)[i + k * n] = _omega[j + _nmax];
      k++;
      ((X_CFLOAT*) buffer)[i + k * n] = _omega[j + 2 * _nmax];
      k++;
    }

    if(data_mask & RADIUS_MASK)((X_CFLOAT*) buffer)[i + k * n] = _radius[j];

    k++;

    if(data_mask & RMASS_MASK)((X_CFLOAT*) buffer)[i + k * n] = _rmass[j];

    k++;
  }
}

template <const unsigned int data_mask>
__global__ void Cuda_AtomVecCuda_PackComm_Self_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_CFLOAT dx, X_CFLOAT dy, X_CFLOAT dz, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = i;
    j = list[i];

    if(data_mask & X_MASK) {
      _x[i + first] = _x[j] + dx;
      _x[i + first + _nmax] = _x[j + _nmax] + dy;
      _x[i + first + 2 * _nmax] = _x[j + 2 * _nmax] + dz;
    }

    if(data_mask & V_MASK) {
      _v[i + first] = _v[j];
      _v[i + first + _nmax] = _v[j + _nmax];
      _v[i + first + 2 * _nmax] = _v[j + 2 * _nmax];
    }

    if(data_mask & OMEGA_MASK) {
      _omega[i + first] = _omega[j];
      _omega[i + first + _nmax] = _omega[j + _nmax];
      _omega[i + first + 2 * _nmax] = _omega[j + 2 * _nmax];
    }

    if(data_mask & RADIUS_MASK) _radius[i + first] = _radius[j];

    if(data_mask & RMASS_MASK) _rmass[i + first] = _rmass[j];
  }
}


template <const unsigned int data_mask>
__global__ void Cuda_AtomVecCuda_UnpackComm_Kernel(int n, int first, void* buffer)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < n) {
    int k = 0;

    if(data_mask & X_MASK) {
      _x[i + first] = ((X_CFLOAT*) buffer)[i + k * n];
      k++;
      _x[i + first + _nmax] = ((X_CFLOAT*) buffer)[i + k * n];
      k++;
      _x[i + first + 2 * _nmax] = ((X_CFLOAT*) buffer)[i + k * n];
      k++;
    }

    if(data_mask & V_MASK) {
      _v[i + first] = ((X_CFLOAT*) buffer)[i + k * n];
      k++;
      _v[i + first + _nmax] = ((X_CFLOAT*) buffer)[i + k * n];
      k++;
      _v[i + first + 2 * _nmax] = ((X_CFLOAT*) buffer)[i + k * n];
      k++;
    }

    if(data_mask & OMEGA_MASK) {
      _omega[i + first] = ((X_CFLOAT*) buffer)[i + k * n];
      k++;
      _omega[i + first + _nmax] = ((X_CFLOAT*) buffer)[i + k * n];
      k++;
      _omega[i + first + 2 * _nmax] = ((X_CFLOAT*) buffer)[i + k * n];
      k++;
    }

    if(data_mask & RADIUS_MASK) _radius[i + first] = ((X_CFLOAT*) buffer)[i + k * n];

    k++;

    if(data_mask & RMASS_MASK) _rmass[i + first] = ((X_CFLOAT*) buffer)[i + k * n];

    k++;
  }
}


__global__ void Cuda_AtomVecCuda_PackExchangeList_Kernel(int n, int dim)
{
  double* buf = (double*) _buffer;
  buf = &buf[1];

  //X_CFLOAT lo=slablo[iswap];
  //X_CFLOAT hi=slabhi[iswap];

  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  bool add = false;

  if(i < _nlocal) {
    double xdim_tmp = static_cast <double>(_x[i + dim * _nmax]);

    if(xdim_tmp < _sublo[dim] || xdim_tmp >= _subhi[dim]) {
      add = true;
    }
  }

  shared[threadIdx.x] = add ? 1 : 0;
  __syncthreads();
  int nsend = 0;

  if(threadIdx.x == 0) {
    for(int k = 0; k < blockDim.x; k++) {
      if(shared[k]) {
        nsend++;
        shared[k] = nsend;
      }
    }

    shared[blockDim.x] = atomicAdd((int*) _buffer, nsend);
  }

  __syncthreads();

  nsend = shared[blockDim.x] + shared[threadIdx.x] - 1;

  if(add && nsend + 1 < n)
    buf[nsend] = i;
}

template <const unsigned int data_mask>
__global__ void Cuda_AtomVecCuda_PackExchange_Kernel(int nsend, int* copylist)
{
  double* buf = (double*) _buffer;
  int k = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(k >= nsend) return;

  buf = &buf[1 + k];

  int i = static_cast <int>(buf[0]);
  int j = copylist[k];

  int m = 1;

  if(data_mask & X_MASK) {
    buf[(m++)*nsend] = static_cast <double>(_x[i]);
    buf[(m++)*nsend] = static_cast <double>(_x[i + _nmax]);
    buf[(m++)*nsend] = static_cast <double>(_x[i + 2 * _nmax]);
  }

  if(data_mask & V_MASK) {
    buf[(m++)*nsend] = _v[i];
    buf[(m++)*nsend] = _v[i + _nmax];
    buf[(m++)*nsend] = _v[i + 2 * _nmax];
  }

  if(data_mask & TAG_MASK) 		buf[(m++)*nsend] = _tag[i];

  if(data_mask & TYPE_MASK) 	buf[(m++)*nsend] = _type[i];

  if(data_mask & MASK_MASK) 	buf[(m++)*nsend] = _mask[i];

  if(data_mask & IMAGE_MASK) 	buf[(m++)*nsend] = _image[i];

  if(data_mask & Q_MASK) 		buf[(m++)*nsend] = _q[i];

  if(data_mask & MOLECULE_MASK) buf[(m++)*nsend] = _molecule[i];

  if(data_mask & RADIUS_MASK) 	buf[(m++)*nsend] = _radius[i];

  if(data_mask & DENSITY_MASK) 	buf[(m++)*nsend] = _density[i];

  if(data_mask & RMASS_MASK) 	buf[(m++)*nsend] = _rmass[i];

  if(data_mask & OMEGA_MASK) {
    buf[(m++)*nsend] = _omega[i];
    buf[(m++)*nsend] = _omega[i + _nmax];
    buf[(m++)*nsend] = _omega[i + 2 * _nmax];
  }

  /*  if(data_mask & NSPECIAL_MASK)
    {
    	buf[(m++)*nsend] = _nspecial[i];
    	buf[(m++)*nsend] = _nspecial[i+_nmax];
    	buf[(m++)*nsend] = _nspecial[i+2* _nmax];
    }*/

  if(i >= _nlocal) return;

  if(data_mask & X_MASK) {
    _x[i] = _x[j];
    _x[i + _nmax] = _x[j + _nmax];
    _x[i + 2 * _nmax] = _x[j + 2 * _nmax];
  }

  if(data_mask & V_MASK) {
    _v[i] = _v[j];
    _v[i + _nmax] = _v[j + _nmax];
    _v[i + 2 * _nmax] = _v[j + 2 * _nmax];
  }

  if(data_mask & TAG_MASK)		_tag[i] 	= _tag[j];

  if(data_mask & TYPE_MASK)		_type[i] 	= _type[j];

  if(data_mask & MASK_MASK)		_mask[i] 	= _mask[j];

  if(data_mask & IMAGE_MASK)	_image[i] 	= _image[j];

  if(data_mask & Q_MASK) 		_q[i] 		= _q[j];

  if(data_mask & MOLECULE_MASK) _molecule[i] = _molecule[j];

  if(data_mask & RADIUS_MASK) 	_radius[i] 	= _radius[j];

  if(data_mask & DENSITY_MASK) 	_density[i] = _density[j];

  if(data_mask & RMASS_MASK) 	_rmass[i] 	= _rmass[j];

  if(data_mask & OMEGA_MASK) {
    _omega[i] = _omega[j];
    _omega[i + _nmax] = _omega[j + _nmax];
    _omega[i + 2 * _nmax] = _omega[j + 2 * _nmax];
  }

  /* if(data_mask & NSPECIAL_MASK)
  {
  _nspecial[i] = _nspecial[j];
  _nspecial[i+_nmax] = _nspecial[j+_nmax];
  _nspecial[i+2* _nmax] = _nspecial[j+2* _nmax];
  }*/
}

template <const unsigned int data_mask>
__global__ void Cuda_AtomVecCuda_UnpackExchange_Kernel(int dim, int nsend, int* copylist)
{
  double* buf = (double*) _buffer;
  int k = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(k >= nsend) return;

  buf = &buf[1 + k];
  int i = -1;
  double xdim_tmp = buf[(1 + dim) * nsend];

  if(xdim_tmp >= _sublo[dim] - SMALL && xdim_tmp < _subhi[dim] + SMALL) {
    i = atomicAdd(_flag, 1) + _nlocal;

    int m = 1;

    if(data_mask & X_MASK) {
      _x[i] = buf[(m++) * nsend];
      _x[i + _nmax] = buf[(m++) * nsend];
      _x[i + 2 * _nmax] = buf[(m++) * nsend];
    }

    if(data_mask & V_MASK) {
      _v[i] = buf[(m++) * nsend];
      _v[i + _nmax] = buf[(m++) * nsend];
      _v[i + 2 * _nmax] = buf[(m++) * nsend];
    }

    if(data_mask & TAG_MASK) 	_tag[i] = buf[(m++) * nsend];

    if(data_mask & TYPE_MASK) 	_type[i] = buf[(m++) * nsend];

    if(data_mask & MASK_MASK) 	_mask[i] = buf[(m++) * nsend];

    if(data_mask & IMAGE_MASK) _image[i] = buf[(m++) * nsend];

    if(data_mask & Q_MASK) _q[i] = buf[(m++) * nsend];

    if(data_mask & MOLECULE_MASK) _molecule[i] = buf[(m++) * nsend];

    if(data_mask & RADIUS_MASK) _radius[i] = buf[(m++) * nsend];

    if(data_mask & DENSITY_MASK) _density[i] = buf[(m++) * nsend];

    if(data_mask & RMASS_MASK) _rmass[i] = buf[(m++) * nsend];

    if(data_mask & OMEGA_MASK) {
      _omega[i] = buf[(m++) * nsend];
      _omega[i + _nmax] = buf[(m++) * nsend];
      _omega[i + 2 * _nmax] = buf[(m++) * nsend];
    }

    /*  if(data_mask & NSPECIAL_MASK)
      {
       _nspecial[i] = buf[(m++)*nsend];
       _nspecial[i+_nmax] = buf[(m++)*nsend];
       _nspecial[i+2*_nmax] = buf[(m++)*nsend];
      }*/
  }

  copylist[k] = i;
}

template <const unsigned int data_mask>
__global__ void Cuda_AtomVecCuda_PackBorder_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_CFLOAT dx, X_CFLOAT dy, X_CFLOAT dz)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];
    int m = 0;

    if(data_mask & X_MASK) {
      ((X_CFLOAT*) _buffer)[i + (m++)*n] = _x[j] + dx;
      ((X_CFLOAT*) _buffer)[i + (m++)*n] = _x[j + _nmax] + dy;
      ((X_CFLOAT*) _buffer)[i + (m++)*n] = _x[j + 2 * _nmax] + dz;
    }

    if(data_mask & V_MASK) {
      ((X_CFLOAT*) _buffer)[i + (m++)*n] = _v[j];
      ((X_CFLOAT*) _buffer)[i + (m++)*n] = _v[j + _nmax];
      ((X_CFLOAT*) _buffer)[i + (m++)*n] = _v[j + 2 * _nmax];
    }

    if(data_mask & TAG_MASK)((X_CFLOAT*) _buffer)[i + (m++)*n] = _tag[j];

    if(data_mask & TYPE_MASK)((X_CFLOAT*) _buffer)[i + (m++)*n] = _type[j];

    if(data_mask & MASK_MASK)((X_CFLOAT*) _buffer)[i + (m++)*n] = _mask[j];

    if(data_mask & Q_MASK)((X_CFLOAT*) _buffer)[i + (m++)*n] = _q[j];

    if(data_mask & MOLECULE_MASK)((X_CFLOAT*) _buffer)[i + (m++)*n] = _molecule[j];

    if(data_mask & RADIUS_MASK)((X_CFLOAT*) _buffer)[i + (m++)*n] = _radius[i];

    if(data_mask & DENSITY_MASK)((X_CFLOAT*) _buffer)[i + (m++)*n] = _density[i];

    if(data_mask & RMASS_MASK)((X_CFLOAT*) _buffer)[i + (m++)*n] = _rmass[i];

    if(data_mask & OMEGA_MASK) {
      ((X_CFLOAT*) _buffer)[i + (m++)*n] = _omega[i];
      ((X_CFLOAT*) _buffer)[i + (m++)*n] = _omega[i + _nmax];
      ((X_CFLOAT*) _buffer)[i + (m++)*n] = _omega[i + 2 * _nmax];
    }
  }
}



template <const unsigned int data_mask>
__global__ void Cuda_AtomVecCuda_PackBorder_Self_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_CFLOAT dx, X_CFLOAT dy, X_CFLOAT dz, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];

    if(data_mask & X_MASK) {
      _x[i + first] = _x[j] + dx;
      _x[i + first + _nmax] = _x[j + _nmax] + dy;
      _x[i + first + 2 * _nmax] = _x[j + 2 * _nmax] + dz;
    }

    if(data_mask & V_MASK) {
      _v[i + first] = _v[j];
      _v[i + first + _nmax] = _v[j + _nmax];
      _v[i + first + 2 * _nmax] =  _v[j + 2 * _nmax];
    }

    if(data_mask & TAG_MASK) _tag[i + first] = _tag[j];

    if(data_mask & TYPE_MASK) _type[i + first] = _type[j];

    if(data_mask & MASK_MASK) _mask[i + first] = _mask[j];

    if(data_mask & Q_MASK) _q[i + first] = _q[j];

    if(data_mask & MOLECULE_MASK) _molecule[i + first] = _molecule[j];

    if(data_mask & RADIUS_MASK) _radius[i + first] = _radius[j];

    if(data_mask & DENSITY_MASK) _density[i + first] = _density[j];

    if(data_mask & RMASS_MASK) _rmass[i + first] = _rmass[j];

    if(data_mask & OMEGA_MASK) {
      _omega[i + first] = _omega[j];
      _omega[i + first + _nmax] = _omega[j + _nmax];
      _omega[i + first + 2 * _nmax] =  _omega[j + 2 * _nmax];
    }
  }
}

template <const unsigned int data_mask>
__global__ void Cuda_AtomVecCuda_UnpackBorder_Kernel(int n, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < n) {
    if(i + first < _nmax) {
      int m = 0;

      if(data_mask & X_MASK) {
        _x[i + first] = ((X_CFLOAT*) _buffer)[i + (m++) * n];
        _x[i + first + _nmax] = ((X_CFLOAT*) _buffer)[i + (m++) * n];
        _x[i + first + 2 * _nmax] = ((X_CFLOAT*) _buffer)[i + (m++) * n];
      }

      if(data_mask & V_MASK) {
        _v[i + first] = ((X_CFLOAT*) _buffer)[i + (m++) * n];
        _v[i + first + _nmax] = ((X_CFLOAT*) _buffer)[i + (m++) * n];
        _v[i + first + 2 * _nmax] = ((X_CFLOAT*) _buffer)[i + (m++) * n];
      }

      if(data_mask & TAG_MASK) _tag[i + first] = static_cast<int>(((X_CFLOAT*) _buffer)[i + (m++) * n]);

      if(data_mask & TYPE_MASK) _type[i + first] = static_cast<int>(((X_CFLOAT*) _buffer)[i + (m++) * n]);

      if(data_mask & MASK_MASK) _mask[i + first] = static_cast<int>(((X_CFLOAT*) _buffer)[i + (m++) * n]);

      if(data_mask & Q_MASK) _q[i + first] = ((X_CFLOAT*) _buffer)[i + (m++) * n];

      if(data_mask & MOLECULE_MASK) _molecule[i + first] = static_cast<int>(((X_CFLOAT*) _buffer)[i + (m++) * n]);

      if(data_mask & RADIUS_MASK) _radius[i + first] = ((X_CFLOAT*) _buffer)[i + (m++) * n];

      if(data_mask & DENSITY_MASK) _density[i + first] = ((X_CFLOAT*) _buffer)[i + (m++) * n];

      if(data_mask & RMASS_MASK) _rmass[i + first] = ((X_CFLOAT*) _buffer)[i + (m++) * n];

      if(data_mask & OMEGA_MASK) {
        _omega[i + first] = ((X_CFLOAT*) _buffer)[i + (m++) * n];
        _omega[i + first + _nmax] = ((X_CFLOAT*) _buffer)[i + (m++) * n];
        _omega[i + first + 2 * _nmax] = ((X_CFLOAT*) _buffer)[i + (m++) * n];
      }
    } else {
      _flag[0] = 1;
    }
  }
}


