#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX fix_temp_rescale_limit_cuda
#include "cuda_shared.h"
#include "cuda_common.h"
#include "crm_cuda_utils.cu"

#include "fix_temp_rescale_limit_cuda_cu.h"
#include "fix_temp_rescale_limit_cuda_kernel.cu"


void Cuda_FixTempRescaleLimitCuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(v)       , & sdata->atom.v    .dev_data, sizeof(X_CFLOAT*));
}

void Cuda_FixTempRescaleLimitCuda_Init(cuda_shared_data* sdata)
{
  Cuda_FixTempRescaleLimitCuda_UpdateNmax(sdata);

}


void Cuda_FixTempRescaleLimitCuda_EndOfStep(cuda_shared_data* sdata, int groupbit, double afactor, double limit)
{
  V_CFLOAT factor = afactor;
  //if(sdata->atom.update_nmax) //fix temp rescale is usually not called every timestep so it might miss an update step
  Cuda_FixTempRescaleLimitCuda_UpdateNmax(sdata);
  //if(sdata->atom.update_nlocal)
  //hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int)      );

  int3 layout = getgrid(sdata->atom.nlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  Cuda_FixTempRescaleLimitCuda_EndOfStep_Kernel <<< grid, threads, 0>>> (groupbit, factor, limit);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Cuda_FixTempRescaleLimitCuda_PostForce: fix add_force post_force compute Kernel execution failed");
}
