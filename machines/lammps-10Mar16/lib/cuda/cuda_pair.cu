#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

enum PAIR_FORCES {PAIR_NONE, PAIR_BORN, PAIR_BUCK, PAIR_CG_CMM, PAIR_LJ_CHARMM, PAIR_LJ_CLASS2, PAIR_LJ_CUT, PAIR_LJ_EXPAND, PAIR_LJ_GROMACS, PAIR_LJ_SMOOTH, PAIR_LJ96_CUT, PAIR_MORSE, PAIR_MORSE_R6};
enum COUL_FORCES {COUL_NONE, COUL_CHARMM, COUL_CHARMM_IMPLICIT, COUL_CUT, COUL_LONG, COUL_DEBYE, COUL_GROMACS, COUL_SPECIAL};
#define DATA_NONE 0
#define DATA_V 1
#define DATA_TAG 2
#define DATA_RMASS 4
#define DATA_MASS 8
#define DATA_TORQUE 16
#define DATA_OMEGA 32
#define DATA_RADIUS 64
#define DATA_DENSITY 128
#define DATA_MASK 256
#define DATA_V_RADIUS 512
#define DATA_OMEGA_RMASS 1024

#define NEIGHMASK 0x3FFFFFFF

#define MY_PREFIX cuda_pair
#define IncludeCommonNeigh
#include "cuda_shared.h"
#include "cuda_common.h"
#include "cuda_wrapper_cu.h"
#include "crm_cuda_utils.cu"

//constants used by multiple forces

//general
#define _cutsq MY_AP(cutsq)
#define _offset MY_AP(offset)
#define _special_lj MY_AP(special_lj)
#define _special_coul MY_AP(special_coul)
#define _cutsq_global MY_AP(cutsq_global)
#define _collect_forces_later MY_AP(collect_forces_later)

__device__ __constant__ X_CFLOAT _cutsq[CUDA_MAX_TYPES2];
__device__ __constant__ ENERGY_CFLOAT _offset[CUDA_MAX_TYPES2];
__device__ __constant__ F_CFLOAT _special_lj[4];
__device__ __constant__ F_CFLOAT _special_coul[4];
__device__ __constant__ X_CFLOAT _cutsq_global;
__device__ __constant__ int _collect_forces_later;

__device__ __constant__ F_CFLOAT MY_AP(coeff1)[CUDA_MAX_TYPES2]; //pair force coefficients in case ntypes < CUDA_MAX_TYPES (coeffs fit into constant space)
__device__ __constant__ F_CFLOAT MY_AP(coeff2)[CUDA_MAX_TYPES2];
__device__ __constant__ F_CFLOAT MY_AP(coeff3)[CUDA_MAX_TYPES2];
__device__ __constant__ F_CFLOAT MY_AP(coeff4)[CUDA_MAX_TYPES2];
__device__ __constant__ F_CFLOAT MY_AP(coeff5)[CUDA_MAX_TYPES2];


__device__ __constant__ F_CFLOAT* MY_AP(coeff1_gm); //pair force coefficients in case ntypes > CUDA_MAX_TYPES (coeffs do not fit into constant space)
__device__ __constant__ F_CFLOAT* MY_AP(coeff2_gm);
__device__ __constant__ F_CFLOAT* MY_AP(coeff3_gm);
__device__ __constant__ F_CFLOAT* MY_AP(coeff4_gm);
__device__ __constant__ F_CFLOAT* MY_AP(coeff5_gm);
__device__ __constant__ F_CFLOAT* MY_AP(coeff6_gm);
__device__ __constant__ F_CFLOAT* MY_AP(coeff7_gm);
__device__ __constant__ F_CFLOAT* MY_AP(coeff8_gm);
__device__ __constant__ F_CFLOAT* MY_AP(coeff9_gm);
__device__ __constant__ F_CFLOAT* MY_AP(coeff10_gm);

#define _coeff1_gm_tex         MY_AP(coeff1_gm_tex)
#if F_PRECISION == 1
texture<float> _coeff1_gm_tex;
#else
texture<int2, 1> _coeff1_gm_tex;
#endif

#define _coeff2_gm_tex         MY_AP(coeff2_gm_tex)
#if F_PRECISION == 1
texture<float> _coeff2_gm_tex;
#else
texture<int2, 1> _coeff2_gm_tex;
#endif

#define _coeff3_gm_tex         MY_AP(coeff3_gm_tex)
#if F_PRECISION == 1
texture<float> _coeff3_gm_tex;
#else
texture<int2, 1> _coeff3_gm_tex;
#endif

#define _coeff4_gm_tex         MY_AP(coeff4_gm_tex)
#if F_PRECISION == 1
texture<float> _coeff4_gm_tex;
#else
texture<int2, 1> _coeff4_gm_tex;
#endif

#define _coeff5_gm_tex         MY_AP(coeff5_gm_tex)
#if F_PRECISION == 1
texture<float> _coeff5_gm_tex;
#else
texture<int2, 1> _coeff5_gm_tex;
#endif

#define _coeff6_gm_tex         MY_AP(coeff6_gm_tex)
#if F_PRECISION == 1
texture<float> _coeff6_gm_tex;
#else
texture<int2, 1> _coeff6_gm_tex;
#endif

#define _coeff7_gm_tex         MY_AP(coeff7_gm_tex)
#if F_PRECISION == 1
texture<float> _coeff7_gm_tex;
#else
texture<int2, 1> _coeff7_gm_tex;
#endif

#define _coeff8_gm_tex         MY_AP(coeff8_gm_tex)
#if F_PRECISION == 1
texture<float> _coeff8_gm_tex;
#else
texture<int2, 1> _coeff8_gm_tex;
#endif

#define _coeff9_gm_tex         MY_AP(coeff9_gm_tex)
#if F_PRECISION == 1
texture<float> _coeff9_gm_tex;
#else
texture<int2, 1> _coeff9_gm_tex;
#endif

#define _coeff10_gm_tex         MY_AP(coeff10_gm_tex)
#if F_PRECISION == 1
texture<float> _coeff10_gm_tex;
#else
texture<int2, 1> _coeff10_gm_tex;
#endif

//if more than 5 coefficients are needed for a pair potential add them here


//coulomb
#define _cut_coulsq MY_AP(cut_coulsq)
#define _cut_coulsq_global MY_AP(cut_coulsq_global)
#define _g_ewald MY_AP(g_ewald)
#define _qqrd2e MY_AP(qqrd2e)
#define _kappa MY_AP(kappa)
__device__ __constant__ X_CFLOAT _cut_coulsq[CUDA_MAX_TYPES2];
__device__ __constant__ X_CFLOAT _cut_coulsq_global;
__device__ __constant__ F_CFLOAT _g_ewald;
__device__ __constant__ F_CFLOAT _qqrd2e;
__device__ __constant__ F_CFLOAT _kappa;

//inner cutoff
#define _cut_innersq MY_AP(cut_innersq)
#define _cut_innersq_global MY_AP(cut_innersq_global)
__device__ __constant__ X_CFLOAT _cut_innersq[CUDA_MAX_TYPES2];
__device__ __constant__ X_CFLOAT _cut_innersq_global;


template <const PAIR_FORCES pair_type, const COUL_FORCES coul_type, const unsigned int extended_data>
__global__ void Pair_Kernel_TpA(int eflag, int vflag, int eflag_atom, int vflag_atom);

template <const PAIR_FORCES pair_type, const COUL_FORCES coul_type, const unsigned int extended_data>
__global__ void Pair_Kernel_BpA(int eflag, int vflag, int eflag_atom, int vflag_atom);

template <const PAIR_FORCES pair_type, const COUL_FORCES coul_type, const unsigned int extended_data>
__global__ void Pair_Kernel_TpA_opt(int eflag, int vflag, int eflag_atom, int vflag_atom, int comm_phase);

template <const PAIR_FORCES pair_type, const COUL_FORCES coul_type, const unsigned int extended_data>
__global__ void Pair_Kernel_BpA_opt(int eflag, int vflag, int eflag_atom, int vflag_atom, int comm_phase);

#include <stdio.h>
#include "cuda_pair_cu.h"
#include "cuda_pair_virial_kernel_nc.cu"

//Functions which are shared by pair styles

//Update Buffersize
void Cuda_UpdateBuffer(cuda_shared_data* sdata, int size)
{
  CUT_CHECK_ERROR("Cuda_Pair_UpdateBuffer_AllStyles: before updateBuffer failed");

  if(sdata->buffersize < size) {
    MYDBG(printf("Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
    CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)
  }

  hipMemcpyToSymbol(MY_AP(buffer), & sdata->buffer, sizeof(int*));
  CUT_CHECK_ERROR("Cuda_Pair_UpdateBuffer_AllStyles failed");
}

void Cuda_Pair_UpdateNeighbor_AllStyles(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist)
{
  //Neighbor
  hipMemcpyToSymbol(MY_AP(neighbor_maxlocal)  , & sneighlist->firstneigh.dim[0]  , sizeof(unsigned));
  hipMemcpyToSymbol(MY_AP(firstneigh)     , & sneighlist->firstneigh.dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(ilist)          , & sneighlist->ilist     .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(inum)           , & sneighlist->inum               , sizeof(int));
  hipMemcpyToSymbol(MY_AP(numneigh)       , & sneighlist->numneigh  .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(neighbors)      , & sneighlist->neighbors  .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(maxneighbors)       , & sneighlist->maxneighbors     , sizeof(int));
  hipMemcpyToSymbol(MY_AP(overlap_comm)       , & sdata->overlap_comm, sizeof(int));

  if(sdata->overlap_comm) {
    hipMemcpyToSymbol(MY_AP(numneigh_border)  , & sneighlist->numneigh_border .dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(numneigh_inner)   , & sneighlist->numneigh_inner  .dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(neighbors_border) , & sneighlist->neighbors_border.dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(neighbors_inner)  , & sneighlist->neighbors_inner .dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(ilist_border)     , & sneighlist->ilist_border    .dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(inum_border)      , & sneighlist->inum_border     .dev_data, sizeof(int*));
  }

}
//Update constants after nmax change which are generally needed by all pair styles
void Cuda_Pair_UpdateNmax_AllStyles(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist)
{
  CUT_CHECK_ERROR("Cuda_Pair_UpdateNmax_AllStyles: Begin");

  //System
  hipMemcpyToSymbol(MY_AP(nlocal)    			, & sdata->atom.nlocal             , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nall)      			, & sdata->atom.nall               , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)      			, & sdata->atom.nmax               , sizeof(int));

  //Atom
  hipMemcpyToSymbol(MY_AP(x)         			, & sdata->atom.x         .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(x_type)         	, & sdata->atom.x_type    .dev_data, sizeof(X_CFLOAT4*));
  hipMemcpyToSymbol(MY_AP(f)         			, & sdata->atom.f         .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(type)      			, & sdata->atom.type      .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(q)         			, & sdata->atom.q         .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(tag)      			, & sdata->atom.tag       .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(eatom)     			, & sdata->atom.eatom     .dev_data, sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(vatom)     			, & sdata->atom.vatom     .dev_data, sizeof(ENERGY_CFLOAT*));


  //Other
  hipMemcpyToSymbol(MY_AP(debugdata)      , & sdata->debugdata      , sizeof(int*));
  CUT_CHECK_ERROR("Cuda_Pair_UpdateNmax_AllStyles: End");
}

//Initialisation of GPU Constants which rarely change
void Cuda_Pair_Init_AllStyles(cuda_shared_data* sdata, int ncoeff, bool need_q = false, bool use_global_params = false, bool need_innercut = false, bool need_cut = true)
{
  unsigned cuda_ntypes = sdata->atom.ntypes + 1;
  unsigned cuda_ntypes2 = cuda_ntypes * cuda_ntypes;
  unsigned n = sizeof(F_CFLOAT) * cuda_ntypes2;
  unsigned nx = sizeof(X_CFLOAT) * cuda_ntypes2;

  //check if enough constant memory is available
  if((cuda_ntypes2 > CUDA_MAX_TYPES2) && !use_global_params)
    printf("# CUDA: Cuda_Pair_Init: you need %u types. this is more than %u "
           "(assumed at compile time). re-compile with -DCUDA_MAX_TYPES_PLUS_ONE=32 "
           "or ajust this in cuda_common.h\n", cuda_ntypes, CUDA_MAX_TYPES_PLUS_ONE - 1);

  if((cuda_ntypes2 > CUDA_MAX_TYPES2) && !use_global_params)
    exit(0);

  //type conversion of cutoffs and parameters
  if(need_cut) {
    X_CFLOAT cutsq[cuda_ntypes2];

    for(int i = 1; i <= sdata->atom.ntypes; ++i) {
      for(int j = 1; j <= sdata->atom.ntypes; ++j) {
        cutsq[i * cuda_ntypes + j] = (X_CFLOAT)(sdata->pair.cut_global * sdata->pair.cut_global);
      }
    }

    int cutsqdiffer = 0;
    X_CFLOAT cutsq_global;
    cutsq_global = (X_CFLOAT)(sdata->pair.cut_global * sdata->pair.cut_global);

    if(sdata->pair.cut) {
      for(int i = 1; i <= sdata->atom.ntypes; ++i) {
        for(int j = i; j <= sdata->atom.ntypes; ++j) {
          if(sdata->pair.cut[i][j] > 1e-6) {
            cutsq[i * cuda_ntypes + j] = (X_CFLOAT)(sdata->pair.cut[i][j] * sdata->pair.cut[i][j]);
            cutsq[j * cuda_ntypes + i] = (X_CFLOAT)(sdata->pair.cut[i][j] * sdata->pair.cut[i][j]);
          }

          if(i == 1 && j == 1) cutsq_global = cutsq[i * cuda_ntypes + j];

          if((cutsq_global - cutsq[i * cuda_ntypes + j]) * (cutsq_global - cutsq[i * cuda_ntypes + j]) > 1e-6)
            cutsqdiffer++;
        }
      }
    }

    if(sdata->pair.cutsq) {
      for(int i = 1; i <= sdata->atom.ntypes; ++i) {
        for(int j = i; j <= sdata->atom.ntypes; ++j) {
          if(sdata->pair.cut[i][j] > 1e-6) {
            cutsq[i * cuda_ntypes + j] = (X_CFLOAT)(sdata->pair.cutsq[i][j]);
            cutsq[j * cuda_ntypes + i] = (X_CFLOAT)(sdata->pair.cutsq[i][j]);
          }

          if(i == 1 && j == 1) cutsq_global = cutsq[i * cuda_ntypes + j];

          if((cutsq_global - cutsq[i * cuda_ntypes + j]) * (cutsq_global - cutsq[i * cuda_ntypes + j]) > 1e-6)
            cutsqdiffer++;
        }
      }
    }

    //printf("CUTSQGLOB: %i %e\n",cutsqdiffer,cutsq_global);
    if(cutsqdiffer) {

      cutsq_global = -1.0;
      hipMemcpyToSymbol(MY_AP(cutsq)      	, cutsq                    		, nx);
    }

    hipMemcpyToSymbol(MY_AP(cutsq_global)	, &cutsq_global  				, sizeof(X_CFLOAT));
  }

  if(need_innercut) {
    X_CFLOAT cut_innersq[cuda_ntypes2];

    for(int i = 1; i <= sdata->atom.ntypes; ++i) {
      for(int j = 1; j <= sdata->atom.ntypes; ++j) {
        cut_innersq[i * cuda_ntypes + j] = (X_CFLOAT)(sdata->pair.cut_inner_global * sdata->pair.cut_inner_global);
      }
    }

    int cutsqdiffer = 0;
    X_CFLOAT cut_innersq_global;
    cut_innersq_global = (X_CFLOAT)(sdata->pair.cut_inner_global * sdata->pair.cut_inner_global);

    if(sdata->pair.cut_inner) {
      for(int i = 1; i <= sdata->atom.ntypes; ++i) {
        for(int j = i; j <= sdata->atom.ntypes; ++j) {
          if(sdata->pair.cut_inner[i][j] > 1e-6) {
            cut_innersq[i * cuda_ntypes + j] = (X_CFLOAT)(sdata->pair.cut_inner[i][j] * sdata->pair.cut_inner[i][j]);
            cut_innersq[j * cuda_ntypes + i] = (X_CFLOAT)(sdata->pair.cut_inner[i][j] * sdata->pair.cut_inner[i][j]);
          }

          if(i == 1 && j == 1) cut_innersq_global = cut_innersq[i * cuda_ntypes + j];

          if((cut_innersq_global - cut_innersq[i * cuda_ntypes + j]) * (cut_innersq_global - cut_innersq[i * cuda_ntypes + j]) > 1e-6)
            cutsqdiffer++;
        }
      }
    }

    if(cutsqdiffer) {
      cut_innersq_global = -1.0;
      hipMemcpyToSymbol(MY_AP(cut_innersq)      	, cut_innersq                    		, nx);
    }

    hipMemcpyToSymbol(MY_AP(cut_innersq_global)	, &cut_innersq_global  				, sizeof(X_CFLOAT));
  }

  if(need_q) {
    X_CFLOAT cut_coulsq[cuda_ntypes2];

    for(int i = 1; i <= sdata->atom.ntypes; ++i) {
      for(int j = 1; j <= sdata->atom.ntypes; ++j) {
        cut_coulsq[i * cuda_ntypes + j] = (X_CFLOAT)(sdata->pair.cut_coul_global * sdata->pair.cut_coul_global);
      }
    }

    int cutsqdiffer = 0;
    X_CFLOAT cut_coulsq_global;
    cut_coulsq_global = (X_CFLOAT)(sdata->pair.cut_coul_global * sdata->pair.cut_coul_global);

    if(sdata->pair.cut_coulsq_global > cut_coulsq_global)  cut_coulsq_global = (X_CFLOAT) sdata->pair.cut_coulsq_global;

    if(sdata->pair.cut_coul) {
      for(int i = 1; i <= sdata->atom.ntypes; ++i) {
        for(int j = i; j <= sdata->atom.ntypes; ++j) {
          if(sdata->pair.cut_coul[i][j] > 1e-6) {
            cut_coulsq[i * cuda_ntypes + j] = (X_CFLOAT)(sdata->pair.cut_coul[i][j] * sdata->pair.cut_coul[i][j]);
            cut_coulsq[j * cuda_ntypes + i] = (X_CFLOAT)(sdata->pair.cut_coul[i][j] * sdata->pair.cut_coul[i][j]);
          }

          if(i == 1 && j == 1) cut_coulsq_global = cut_coulsq[i * cuda_ntypes + j];

          if((cut_coulsq_global - cut_coulsq[i * cuda_ntypes + j]) * (cut_coulsq_global - cut_coulsq[i * cuda_ntypes + j]) > 1e-6)
            cutsqdiffer++;
        }
      }
    }

    if(cutsqdiffer) {
      cut_coulsq_global = -1.0;
      hipMemcpyToSymbol(MY_AP(cut_coulsq)      	, cut_coulsq                    		, nx);
    }

    hipMemcpyToSymbol(MY_AP(cut_coulsq_global), &cut_coulsq_global  					, sizeof(X_CFLOAT));
  }

  CUT_CHECK_ERROR("Cuda_Pair: init pre Coeff failed");

  if(ncoeff > 0) {
    F_CFLOAT coeff1[cuda_ntypes2];

    for(int i = 1; i <= sdata->atom.ntypes; ++i) {
      for(int j = 1; j <= sdata->atom.ntypes; ++j) {
        coeff1[i * cuda_ntypes + j] = (F_CFLOAT) sdata->pair.coeff1[i][j];
      }
    }

    if(use_global_params) {
      hipMemcpyToSymbol(MY_AP(coeff1_gm)  , &sdata->pair.coeff1_gm.dev_data   , sizeof(F_CFLOAT*));
      hipMemcpy((sdata->pair.coeff1_gm.dev_data), coeff1, n, hipMemcpyHostToDevice);

      _coeff1_gm_tex.normalized = false;                      // access with normalized texture coordinates
      _coeff1_gm_tex.filterMode = hipFilterModePoint;        // Point mode, so no
      _coeff1_gm_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
      const textureReference* coeff1_gm_texture_ptr = &MY_AP(coeff1_gm_tex);
      CUT_CHECK_ERROR("Cuda_Pair: init Coeff0 a failed");

#if F_PRECISION == 1
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<float>();
      CUT_CHECK_ERROR("Cuda_Pair: init Coeff0 b failed");
      hipBindTexture(0, coeff1_gm_texture_ptr, sdata->pair.coeff1_gm.dev_data, &channelDescXType, sdata->atom.nmax * sizeof(F_CFLOAT));
      CUT_CHECK_ERROR("Cuda_Pair: init Coeff0 c failed");
#else
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<int2>();
      CUT_CHECK_ERROR("Cuda_Pair: init Coeff0 b-d failed");
      hipBindTexture(0, coeff1_gm_texture_ptr, sdata->pair.coeff1_gm.dev_data, &channelDescXType, sdata->atom.nmax * 2 * sizeof(int2));
      CUT_CHECK_ERROR("Cuda_Pair: init Coeff0 c-d failed");
#endif

    } else
      hipMemcpyToSymbol(MY_AP(coeff1), coeff1 , n);
  }

  CUT_CHECK_ERROR("Cuda_Pair: init Coeff0 failed");

  if(ncoeff > 1) {
    F_CFLOAT coeff2[cuda_ntypes2];

    for(int i = 1; i <= sdata->atom.ntypes; ++i) {
      for(int j = 1; j <= sdata->atom.ntypes; ++j) {
        coeff2[i * cuda_ntypes + j] = (F_CFLOAT) sdata->pair.coeff2[i][j];
      }
    }

    if(use_global_params) {
      hipMemcpyToSymbol(MY_AP(coeff2_gm)  , &sdata->pair.coeff2_gm.dev_data   , sizeof(F_CFLOAT*));
      hipMemcpy(sdata->pair.coeff2_gm.dev_data, coeff2, n, hipMemcpyHostToDevice);

      _coeff2_gm_tex.normalized = false;                      // access with normalized texture coordinates
      _coeff2_gm_tex.filterMode = hipFilterModePoint;        // Point mode, so no
      _coeff2_gm_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
      const textureReference* coeff2_gm_texture_ptr = &MY_AP(coeff2_gm_tex);

#if F_PRECISION == 1
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<float>();
      hipBindTexture(0, coeff2_gm_texture_ptr, sdata->pair.coeff2_gm.dev_data, &channelDescXType, sdata->atom.nmax * sizeof(F_CFLOAT));
#else
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<int2>();
      hipBindTexture(0, coeff2_gm_texture_ptr, sdata->pair.coeff2_gm.dev_data, &channelDescXType, sdata->atom.nmax * 2 * sizeof(int2));
#endif

    } else
      hipMemcpyToSymbol(MY_AP(coeff2), coeff2 , n);
  }

  CUT_CHECK_ERROR("Cuda_Pair: init Coeff1 failed");

  if(ncoeff > 2) {
    F_CFLOAT coeff3[cuda_ntypes2];

    for(int i = 1; i <= sdata->atom.ntypes; ++i) {
      for(int j = 1; j <= sdata->atom.ntypes; ++j) {
        coeff3[i * cuda_ntypes + j] = (F_CFLOAT) sdata->pair.coeff3[i][j];
      }
    }

    if(use_global_params) {
      hipMemcpyToSymbol(MY_AP(coeff3_gm)  , &sdata->pair.coeff3_gm.dev_data   , sizeof(F_CFLOAT*));
      hipMemcpy(sdata->pair.coeff3_gm.dev_data, coeff3, n, hipMemcpyHostToDevice);
      _coeff3_gm_tex.normalized = false;                      // access with normalized texture coordinates
      _coeff3_gm_tex.filterMode = hipFilterModePoint;        // Point mode, so no
      _coeff3_gm_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
      const textureReference* coeff3_gm_texture_ptr = &MY_AP(coeff3_gm_tex);

#if F_PRECISION == 1
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<float>();
      hipBindTexture(0, coeff3_gm_texture_ptr, sdata->pair.coeff3_gm.dev_data, &channelDescXType, sdata->atom.nmax * sizeof(F_CFLOAT));
#else
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<int2>();
      hipBindTexture(0, coeff3_gm_texture_ptr, sdata->pair.coeff3_gm.dev_data, &channelDescXType, sdata->atom.nmax * 2 * sizeof(int2));
#endif
    } else
      hipMemcpyToSymbol(MY_AP(coeff3), coeff3 , n);
  }

  CUT_CHECK_ERROR("Cuda_Pair: init Coeff3 failed");

  if(ncoeff > 3) {
    F_CFLOAT coeff4[cuda_ntypes2];

    for(int i = 1; i <= sdata->atom.ntypes; ++i) {
      for(int j = 1; j <= sdata->atom.ntypes; ++j) {
        coeff4[i * cuda_ntypes + j] = (F_CFLOAT) sdata->pair.coeff4[i][j];
      }
    }

    if(use_global_params) {
      hipMemcpyToSymbol(MY_AP(coeff4_gm)  , &sdata->pair.coeff4_gm.dev_data   , sizeof(F_CFLOAT*));
      hipMemcpy(sdata->pair.coeff4_gm.dev_data, coeff4, n, hipMemcpyHostToDevice);
      _coeff4_gm_tex.normalized = false;                      // access with normalized texture coordinates
      _coeff4_gm_tex.filterMode = hipFilterModePoint;        // Point mode, so no
      _coeff4_gm_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
      const textureReference* coeff4_gm_texture_ptr = &MY_AP(coeff4_gm_tex);

#if F_PRECISION == 1
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<float>();
      hipBindTexture(0, coeff4_gm_texture_ptr, sdata->pair.coeff4_gm.dev_data, &channelDescXType, sdata->atom.nmax * sizeof(F_CFLOAT));
#else
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<int2>();
      hipBindTexture(0, coeff4_gm_texture_ptr, sdata->pair.coeff4_gm.dev_data, &channelDescXType, sdata->atom.nmax * 2 * sizeof(int2));
#endif
    } else
      hipMemcpyToSymbol(MY_AP(coeff4), coeff4 , n);
  }

  CUT_CHECK_ERROR("Cuda_Pair: init Coeff4 failed");

  if(ncoeff > 4) {
    F_CFLOAT coeff5[cuda_ntypes2];

    for(int i = 1; i <= sdata->atom.ntypes; ++i) {
      for(int j = 1; j <= sdata->atom.ntypes; ++j) {
        coeff5[i * cuda_ntypes + j] = (F_CFLOAT) sdata->pair.coeff5[i][j];
      }
    }

    if(use_global_params) {
      hipMemcpyToSymbol(MY_AP(coeff5_gm)  , &sdata->pair.coeff5_gm.dev_data   , sizeof(F_CFLOAT*));
      hipMemcpy(sdata->pair.coeff5_gm.dev_data, coeff5, n, hipMemcpyHostToDevice);
      _coeff5_gm_tex.normalized = false;                      // access with normalized texture coordinates
      _coeff5_gm_tex.filterMode = hipFilterModePoint;        // Point mode, so no
      _coeff5_gm_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
      const textureReference* coeff5_gm_texture_ptr = &MY_AP(coeff5_gm_tex);

#if F_PRECISION == 1
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<float>();
      hipBindTexture(0, coeff5_gm_texture_ptr, sdata->pair.coeff5_gm.dev_data, &channelDescXType, sdata->atom.nmax * sizeof(F_CFLOAT));
#else
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<int2>();
      hipBindTexture(0, coeff5_gm_texture_ptr, sdata->pair.coeff5_gm.dev_data, &channelDescXType, sdata->atom.nmax * 2 * sizeof(int2));
#endif
    } else
      hipMemcpyToSymbol(MY_AP(coeff5), coeff5 , n);
  }

  CUT_CHECK_ERROR("Cuda_Pair: init Coeff5 failed");

  if(ncoeff > 5) {
    F_CFLOAT coeff6[cuda_ntypes2];

    for(int i = 1; i <= sdata->atom.ntypes; ++i) {
      for(int j = 1; j <= sdata->atom.ntypes; ++j) {
        coeff6[i * cuda_ntypes + j] = (F_CFLOAT) sdata->pair.coeff6[i][j];
      }
    }

    if(use_global_params) {
      hipMemcpyToSymbol(MY_AP(coeff6_gm)  , &sdata->pair.coeff6_gm.dev_data   , sizeof(F_CFLOAT*));
      hipMemcpy(sdata->pair.coeff6_gm.dev_data, coeff6, n, hipMemcpyHostToDevice);
      _coeff6_gm_tex.normalized = false;                      // access with normalized texture coordinates
      _coeff6_gm_tex.filterMode = hipFilterModePoint;        // Point mode, so no
      _coeff6_gm_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
      const textureReference* coeff6_gm_texture_ptr = &MY_AP(coeff6_gm_tex);

#if F_PRECISION == 1
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<float>();
      hipBindTexture(0, coeff6_gm_texture_ptr, sdata->pair.coeff6_gm.dev_data, &channelDescXType, sdata->atom.nmax * sizeof(F_CFLOAT));
#else
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<int2>();
      hipBindTexture(0, coeff6_gm_texture_ptr, sdata->pair.coeff6_gm.dev_data, &channelDescXType, sdata->atom.nmax * 2 * sizeof(int2));
#endif
    }
  }

  CUT_CHECK_ERROR("Cuda_Pair: init Coeff6 failed");

  if(ncoeff > 6) {
    F_CFLOAT coeff7[cuda_ntypes2];

    for(int i = 1; i <= sdata->atom.ntypes; ++i) {
      for(int j = 1; j <= sdata->atom.ntypes; ++j) {
        coeff7[i * cuda_ntypes + j] = (F_CFLOAT) sdata->pair.coeff7[i][j];
      }
    }

    if(use_global_params) {
      hipMemcpyToSymbol(MY_AP(coeff7_gm)  , &sdata->pair.coeff7_gm.dev_data   , sizeof(F_CFLOAT*));
      hipMemcpy(sdata->pair.coeff7_gm.dev_data, coeff7, n, hipMemcpyHostToDevice);
      _coeff7_gm_tex.normalized = false;                      // access with normalized texture coordinates
      _coeff7_gm_tex.filterMode = hipFilterModePoint;        // Point mode, so no
      _coeff7_gm_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
      const textureReference* coeff7_gm_texture_ptr = &MY_AP(coeff7_gm_tex);

#if F_PRECISION == 1
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<float>();
      hipBindTexture(0, coeff7_gm_texture_ptr, sdata->pair.coeff7_gm.dev_data, &channelDescXType, sdata->atom.nmax * sizeof(F_CFLOAT));
#else
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<int2>();
      hipBindTexture(0, coeff7_gm_texture_ptr, sdata->pair.coeff7_gm.dev_data, &channelDescXType, sdata->atom.nmax * 2 * sizeof(int2));
#endif
    }
  }

  CUT_CHECK_ERROR("Cuda_Pair: init Coeff7 failed");

  if(ncoeff > 7) {
    F_CFLOAT coeff8[cuda_ntypes2];

    for(int i = 1; i <= sdata->atom.ntypes; ++i) {
      for(int j = 1; j <= sdata->atom.ntypes; ++j) {
        coeff8[i * cuda_ntypes + j] = (F_CFLOAT) sdata->pair.coeff8[i][j];
      }
    }

    if(use_global_params) {
      hipMemcpyToSymbol(MY_AP(coeff8_gm)  , &sdata->pair.coeff8_gm.dev_data   , sizeof(F_CFLOAT*));
      hipMemcpy(sdata->pair.coeff8_gm.dev_data, coeff8, n, hipMemcpyHostToDevice);
      _coeff8_gm_tex.normalized = false;                      // access with normalized texture coordinates
      _coeff8_gm_tex.filterMode = hipFilterModePoint;        // Point mode, so no
      _coeff8_gm_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
      const textureReference* coeff8_gm_texture_ptr = &MY_AP(coeff8_gm_tex);

#if F_PRECISION == 1
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<float>();
      hipBindTexture(0, coeff8_gm_texture_ptr, sdata->pair.coeff8_gm.dev_data, &channelDescXType, sdata->atom.nmax * sizeof(F_CFLOAT));
#else
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<int2>();
      hipBindTexture(0, coeff8_gm_texture_ptr, sdata->pair.coeff8_gm.dev_data, &channelDescXType, sdata->atom.nmax * 2 * sizeof(int2));
#endif
    }
  }

  CUT_CHECK_ERROR("Cuda_Pair: init Coeff8 failed");

  if(ncoeff > 8) {
    F_CFLOAT coeff9[cuda_ntypes2];

    for(int i = 1; i <= sdata->atom.ntypes; ++i) {
      for(int j = 1; j <= sdata->atom.ntypes; ++j) {
        coeff9[i * cuda_ntypes + j] = (F_CFLOAT) sdata->pair.coeff9[i][j];
      }
    }

    if(use_global_params) {
      hipMemcpyToSymbol(MY_AP(coeff9_gm)  , &sdata->pair.coeff9_gm.dev_data   , sizeof(F_CFLOAT*));
      hipMemcpy(sdata->pair.coeff9_gm.dev_data, coeff9, n, hipMemcpyHostToDevice);
      _coeff9_gm_tex.normalized = false;                      // access with normalized texture coordinates
      _coeff9_gm_tex.filterMode = hipFilterModePoint;        // Point mode, so no
      _coeff9_gm_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
      const textureReference* coeff9_gm_texture_ptr = &MY_AP(coeff9_gm_tex);

#if F_PRECISION == 1
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<float>();
      hipBindTexture(0, coeff9_gm_texture_ptr, sdata->pair.coeff9_gm.dev_data, &channelDescXType, sdata->atom.nmax * sizeof(F_CFLOAT));
#else
      hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<int2>();
      hipBindTexture(0, coeff9_gm_texture_ptr, sdata->pair.coeff9_gm.dev_data, &channelDescXType, sdata->atom.nmax * 2 * sizeof(int2));
#endif
    }
  }

  CUT_CHECK_ERROR("Cuda_Pair: init Coeff9 failed");

  F_CFLOAT special_lj[4];
  special_lj[0] = sdata->pair.special_lj[0];
  special_lj[1] = sdata->pair.special_lj[1];
  special_lj[2] = sdata->pair.special_lj[2];
  special_lj[3] = sdata->pair.special_lj[3];


  X_CFLOAT box_size[3] = {
    sdata->domain.subhi[0] - sdata->domain.sublo[0],
    sdata->domain.subhi[1] - sdata->domain.sublo[1],
    sdata->domain.subhi[2] - sdata->domain.sublo[2]
  };

  hipMemcpyToSymbol(MY_AP(box_size)   	, box_size                 		, sizeof(X_CFLOAT) * 3);
  hipMemcpyToSymbol(MY_AP(cuda_ntypes)	, &cuda_ntypes            		, sizeof(unsigned));
  hipMemcpyToSymbol(MY_AP(special_lj) 	, special_lj               		, sizeof(F_CFLOAT) * 4);
  hipMemcpyToSymbol(MY_AP(virial)     	, &sdata->pair.virial.dev_data   , sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(eng_vdwl)     	, &sdata->pair.eng_vdwl.dev_data , sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(periodicity)	, sdata->domain.periodicity		, sizeof(int) * 3);
  hipMemcpyToSymbol(MY_AP(collect_forces_later), &sdata->pair.collect_forces_later  , sizeof(int));

  if(need_q) {
    F_CFLOAT qqrd2e_tmp = sdata->pppm.qqrd2e;
    F_CFLOAT special_coul[4];
    special_coul[0] = sdata->pair.special_coul[0];
    special_coul[1] = sdata->pair.special_coul[1];
    special_coul[2] = sdata->pair.special_coul[2];
    special_coul[3] = sdata->pair.special_coul[3];

    hipMemcpyToSymbol(MY_AP(special_coul)	, special_coul             		, sizeof(F_CFLOAT) * 4);
    hipMemcpyToSymbol(MY_AP(g_ewald)    	, &sdata->pair.g_ewald	   		, sizeof(F_CFLOAT));
    hipMemcpyToSymbol(MY_AP(qqrd2e)     	, &qqrd2e_tmp	   				, sizeof(F_CFLOAT));
    hipMemcpyToSymbol(MY_AP(kappa)     	, &sdata->pair.kappa				, sizeof(F_CFLOAT));
    hipMemcpyToSymbol(MY_AP(eng_coul)     , &sdata->pair.eng_coul.dev_data , sizeof(ENERGY_CFLOAT*));
  }

  CUT_CHECK_ERROR("Cuda_Pair: init failed");
}
my_times startpairtime, endpairtime;
//Function which is called prior to kernel invocation, determins grid, Binds Textures, updates constant memory if necessary
void Cuda_Pair_PreKernel_AllStyles(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist, int eflag, int vflag, dim3 &grid, dim3 &threads, int &sharedperproc, bool need_q = false, int maxthreads = 256)
{
  if(sdata->atom.nlocal == 0) return;

  if(sdata->atom.update_neigh)
    Cuda_Pair_UpdateNeighbor_AllStyles(sdata, sneighlist);

  if(sdata->atom.update_nmax)
    Cuda_Pair_UpdateNmax_AllStyles(sdata, sneighlist);

  if(sdata->atom.update_nlocal) {
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
    hipMemcpyToSymbol(MY_AP(nall)    , & sdata->atom.nall          , sizeof(int));
  }



  BindXTypeTexture(sdata);

  if(need_q) BindQTexture(sdata);


  sharedperproc = 0;

  if(sdata->pair.use_block_per_atom) sharedperproc += 3;

  if(eflag) sharedperproc += 1;

  if(need_q && eflag) sharedperproc += 1;

  if(vflag) sharedperproc += 6;

  int threadnum = sneighlist->inum;

  if(sdata->comm.comm_phase == 2)threadnum = sneighlist->inum_border2;

  if(sdata->pair.use_block_per_atom) {
    threadnum *= 64;
    maxthreads = 64;
  }

  int3 layout = getgrid(threadnum, sharedperproc * sizeof(ENERGY_CFLOAT), maxthreads, true); //need to limit to 192 threads due to register limit
  threads.x = layout.z;
  threads.y = 1;
  threads.z = 1;
  grid.x = layout.x;
  grid.y = layout.y;
  grid.z = 1;

  int size = (unsigned)(layout.y * layout.x) * sharedperproc * sizeof(ENERGY_CFLOAT);

  if(sdata->pair.collect_forces_later) size += (unsigned)(sdata->atom.nmax * 3 * sizeof(F_CFLOAT));

  Cuda_UpdateBuffer(sdata, size);

  if(sdata->pair.use_block_per_atom)
    hipMemset(sdata->buffer, 0, size);

  sdata->pair.lastgridsize = grid.x * grid.y;
  sdata->pair.n_energy_virial = sharedperproc;

  if(sdata->pair.use_block_per_atom) sdata->pair.n_energy_virial -= 3;

  my_gettime(CLOCK_REALTIME, &startpairtime);

  MYDBG(printf("# CUDA: Cuda_Pair: kernel start eflag: %i vflag: %i config: %i %i %i %i\n", eflag, vflag, grid.x, grid.y, threads.x, sharedperproc * sizeof(ENERGY_CFLOAT)*threads.x);)
}

//Function which is called after the kernel invocation, collects energy and virial
void Cuda_Pair_PostKernel_AllStyles(cuda_shared_data* sdata, dim3 &grid, int &sharedperproc, int eflag, int vflag)
{
  if((not sdata->pair.collect_forces_later) && (eflag || vflag)) { //not sdata->comm.comm_phase==2))
    hipDeviceSynchronize();
    my_gettime(CLOCK_REALTIME, &endpairtime);
    sdata->cuda_timings.pair_kernel +=
      endpairtime.tv_sec - startpairtime.tv_sec + 1.0 * (endpairtime.tv_nsec - startpairtime.tv_nsec) / 1000000000;
    CUT_CHECK_ERROR("Cuda_Pair: Kernel execution failed");

    if(eflag || vflag) {
      int n = grid.x * grid.y;

      if(sdata->pair.use_block_per_atom)
        grid.x = sharedperproc - 3;
      else
        grid.x = sharedperproc;

      grid.y = 1;
      dim3 threads(128, 1, 1);
      MYDBG(printf("# CUDA: Cuda_Pair: virial compute kernel start eflag: %i vflag: %i config: %i %i %i %i\n", eflag, vflag, grid.x, grid.y, threads.x, sharedperproc * sizeof(ENERGY_CFLOAT)*threads.x);)
      MY_AP(PairVirialCompute_reduce) <<< grid, threads, threads.x* sizeof(ENERGY_CFLOAT)>>>(n);
      hipDeviceSynchronize();
      CUT_CHECK_ERROR("Cuda_Pair: virial compute Kernel execution failed");
    }

    MYDBG(printf("# CUDA: Cuda_Pair: kernel done\n");)
  }
}


#include "pair_born_coul_long_cuda.cu"
#include "pair_buck_coul_cut_cuda.cu"
#include "pair_buck_coul_long_cuda.cu"
#include "pair_buck_cuda.cu"
#include "pair_lj_sdk_cuda.cu"
#include "pair_lj_sdk_coul_cut_cuda.cu"
#include "pair_lj_sdk_coul_debye_cuda.cu"
#include "pair_lj_sdk_coul_long_cuda.cu"
#include "pair_gran_hooke_cuda.cu"
#include "pair_lj_charmm_coul_charmm_implicit_cuda.cu"
#include "pair_lj_charmm_coul_charmm_cuda.cu"
#include "pair_lj_charmm_coul_long_cuda.cu"
#include "pair_lj_class2_coul_cut_cuda.cu"
#include "pair_lj_class2_coul_long_cuda.cu"
#include "pair_lj_class2_cuda.cu"
#include "pair_lj_cut_coul_cut_cuda.cu"
#include "pair_lj_cut_coul_debye_cuda.cu"
#include "pair_lj_cut_coul_long_cuda.cu"
#include "pair_lj_cut_cuda.cu"
#include "pair_lj_cut_experimental_cuda.cu"
#include "pair_lj_expand_cuda.cu"
#include "pair_lj_gromacs_cuda.cu"
#include "pair_lj_gromacs_coul_gromacs_cuda.cu"
#include "pair_lj_smooth_cuda.cu"
#include "pair_lj96_cut_cuda.cu"
#include "pair_morse_coul_long_cuda.cu"
#include "pair_morse_cuda.cu"
#include "pair_eam_cuda.cu"

#include "cuda_pair_kernel.cu"

#include "pair_manybody_const.h"
#include "pair_tersoff_cuda.cu"
#include "pair_sw_cuda.cu"

void Cuda_Pair_UpdateNmax(cuda_shared_data* sdata)
{
  CUT_CHECK_ERROR("Cuda_Pair: before updateNmax failed");
  hipMemcpyToSymbol(MY_AP(nlocal)    , & sdata->atom.nlocal             , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nall)      , & sdata->atom.nall               , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)      , & sdata->atom.nmax               , sizeof(int));
  hipMemcpyToSymbol(MY_AP(type)      , & sdata->atom.type       .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(x)         , & sdata->atom.x          .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(x_type)    , & sdata->atom.x_type     .dev_data, sizeof(X_CFLOAT4*));
  hipMemcpyToSymbol(MY_AP(xhold)     , & sdata->atom.xhold      .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(v)         , & sdata->atom.v          .dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(radius)    , & sdata->atom.radius     .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(v_radius)  , & sdata->atom.v_radius   .dev_data, sizeof(V_CFLOAT4*));
  hipMemcpyToSymbol(MY_AP(omega)     , & sdata->atom.omega      .dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(rmass)     , & sdata->atom.rmass      .dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(omega_rmass), & sdata->atom.omega_rmass.dev_data, sizeof(V_CFLOAT4*));
  hipMemcpyToSymbol(MY_AP(map_array), & sdata->atom.map_array .dev_data, sizeof(int*));
  CUT_CHECK_ERROR("Cuda_Pair: updateNmax failed");
}


void Cuda_Pair_GenerateXType(cuda_shared_data* sdata)
{
  MYDBG(printf(" # CUDA: GenerateXType ... start %i %i %i %p %p %p %p\n", sdata->atom.nlocal, sdata->atom.nall, sdata->atom.nmax, sdata->atom.x.dev_data, sdata->atom.x_type.dev_data, sdata->atom.xhold.dev_data, sdata->atom.type.dev_data);)

  if(sdata->atom.update_nmax)
    Cuda_Pair_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal) {
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
    hipMemcpyToSymbol(MY_AP(nall)    , & sdata->atom.nall          , sizeof(int));
  }

  MYDBG(printf(" # CUDA: GenerateXType ... getgrid\n"); fflush(stdout);)

  int3 layout = getgrid(sdata->atom.nall);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  MYDBG(printf(" # CUDA: GenerateXType ... kernel start test\n");  fflush(stdout);)
  Pair_GenerateXType_Kernel <<< grid, threads, 0>>>();
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Pair GenerateXType: Kernel failed");
  MYDBG(printf(" # CUDA: GenerateXType ... end\n");  fflush(stdout);)
}

void Cuda_Pair_RevertXType(cuda_shared_data* sdata)
{
  MYDBG(printf(" # CUDA: RevertXType ... start\n");)

  if(sdata->atom.update_nmax)
    Cuda_Pair_UpdateNmax(sdata);

  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nall)    , & sdata->atom.nall          , sizeof(int));

  int3 layout = getgrid(sdata->atom.nall);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  Pair_RevertXType_Kernel <<< grid, threads, 0>>>();
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Pair GenerateXType: Kernel failed");
  MYDBG(printf(" # CUDA: RevertXType ... end\n");)
}

void Cuda_Pair_GenerateVRadius(cuda_shared_data* sdata)
{
  MYDBG(printf(" # CUDA: GenerateVRadius ... start %i %i %i %p %p %p %p\n", sdata->atom.nlocal, sdata->atom.nall, sdata->atom.nmax, sdata->atom.x.dev_data, sdata->atom.x_type.dev_data, sdata->atom.xhold.dev_data, sdata->atom.type.dev_data);)

  if(sdata->atom.update_nmax)
    Cuda_Pair_UpdateNmax(sdata);

  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nall)    , & sdata->atom.nall          , sizeof(int));
  MYDBG(printf(" # CUDA: GenerateVRadius ... getgrid\n"); fflush(stdout);)

  int3 layout = getgrid(sdata->atom.nall);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  MYDBG(printf(" # CUDA: GenerateVRadius ... kernel start test\n");  fflush(stdout);)
  Pair_GenerateVRadius_Kernel <<< grid, threads, 0>>>();
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Pair GenerateVRadius: Kernel failed");
  MYDBG(printf(" # CUDA: GenerateVRadius ... end\n");  fflush(stdout);)
}

void Cuda_Pair_GenerateOmegaRmass(cuda_shared_data* sdata)
{
  MYDBG(printf(" # CUDA: GenerateOmegaRmass ... start %i %i %i %p %p %p %p\n", sdata->atom.nlocal, sdata->atom.nall, sdata->atom.nmax, sdata->atom.x.dev_data, sdata->atom.x_type.dev_data, sdata->atom.xhold.dev_data, sdata->atom.type.dev_data);)

  if(sdata->atom.update_nmax)
    Cuda_Pair_UpdateNmax(sdata);

  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nall)    , & sdata->atom.nall          , sizeof(int));
  MYDBG(printf(" # CUDA: GenerateOmegaRmass ... getgrid\n"); fflush(stdout);)

  int3 layout = getgrid(sdata->atom.nall);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  MYDBG(printf(" # CUDA: GenerateOmegaRmass ... kernel start test\n");  fflush(stdout);)
  Pair_GenerateOmegaRmass_Kernel <<< grid, threads, 0>>>();
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Pair GenerateOmegaRmass: Kernel failed");
  MYDBG(printf(" # CUDA: GenerateOmegaRmass ... end\n");  fflush(stdout);)
}

void Cuda_Pair_BuildXHold(cuda_shared_data* sdata)
{
  if(sdata->atom.update_nmax)
    Cuda_Pair_UpdateNmax(sdata);

  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nall)    , & sdata->atom.nall          , sizeof(int));

  int3 layout = getgrid(sdata->atom.nall);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  Pair_BuildXHold_Kernel <<< grid, threads, 0>>>();
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Pair GenerateXType: Kernel failed");
}

void Cuda_Pair_CollectForces(cuda_shared_data* sdata, int eflag, int vflag)
{
  hipDeviceSynchronize();
  my_gettime(CLOCK_REALTIME, &endpairtime);
  sdata->cuda_timings.pair_kernel +=
    endpairtime.tv_sec - startpairtime.tv_sec + 1.0 * (endpairtime.tv_nsec - startpairtime.tv_nsec) / 1000000000;
  CUT_CHECK_ERROR("Cuda_Pair: Kernel execution failed");
  dim3 threads;
  dim3 grid;

  if(eflag || vflag) {
    int n = sdata->pair.lastgridsize;
    grid.x = sdata->pair.n_energy_virial;
    grid.y = 1;
    threads.x = 128;
    //printf("A grid.x: %i\n",grid.x);
    MY_AP(PairVirialCompute_reduce) <<< grid, threads, threads.x* sizeof(ENERGY_CFLOAT)>>>(n);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_Pair_CollectForces: virial compute Kernel execution failed");
  }

  int3 layout = getgrid(sdata->atom.nlocal);
  threads.x = layout.z;
  grid.x = layout.x;
  grid.y = layout.y;
  Pair_CollectForces_Kernel <<< grid, threads, 0>>>(sdata->pair.n_energy_virial, sdata->pair.lastgridsize);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Pair_CollectForces: Force Summation Kernel execution failed");

}
