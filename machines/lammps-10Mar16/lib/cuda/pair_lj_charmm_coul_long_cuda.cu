/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>

#define _lj1 MY_AP(coeff1_gm)
#define _lj2 MY_AP(coeff2_gm)
#define _lj3 MY_AP(coeff3_gm)
#define _lj4 MY_AP(coeff4_gm)

#include "pair_lj_charmm_coul_long_cuda_cu.h"

#include <time.h>

void Cuda_PairLJCharmmCoulLongCuda_Init(cuda_shared_data* sdata, F_CFLOAT denom_lj_inv)
{
  Cuda_Pair_Init_AllStyles(sdata, 4, true, true, true);
  hipMemcpyToSymbol(MY_AP(denom_lj_inv) , &denom_lj_inv  , sizeof(F_CFLOAT));

  return;
}



void Cuda_PairLJCharmmCoulLongCuda(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist, int eflag, int vflag,
                                   int eflag_atom, int vflag_atom, F_CFLOAT denom_lj)
{

  static  short init = 0;

  if(! init) {
    init = 1;
    Cuda_PairLJCharmmCoulLongCuda_Init(sdata, 1.0 / denom_lj);
  }

  dim3 grid, threads;
  int sharedperproc;

  Cuda_Pair_PreKernel_AllStyles(sdata, sneighlist, eflag, vflag, grid, threads, sharedperproc, true, 192);

  hipStream_t* streams = (hipStream_t*) CudaWrapper_returnStreams();

  if(sdata->pair.use_block_per_atom)
    Pair_Kernel_BpA<PAIR_LJ_CHARMM, COUL_LONG, DATA_NONE>
    <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x, streams[1]>>> (eflag, vflag, eflag_atom, vflag_atom);
  else
    Pair_Kernel_TpA<PAIR_LJ_CHARMM, COUL_LONG, DATA_NONE>
    <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x, streams[1]>>> (eflag, vflag, eflag_atom, vflag_atom);

  Cuda_Pair_PostKernel_AllStyles(sdata, grid, sharedperproc, eflag, vflag);
}

#undef _lj1
#undef _lj2
#undef _lj3
#undef _lj4
