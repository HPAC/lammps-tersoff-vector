/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>


#include "pair_tersoff_cuda_cu.h"
__device__ __constant__ Param_Float params[MANYBODY_NPAIR* MANYBODY_NPAIR* MANYBODY_NPAIR];
__device__ __constant__ F_CFLOAT* _glob_zeta_ij; //zeta_ij
__device__ __constant__ F_CFLOAT4* _glob_r_ij; //r_ij (x,y,z,r^2) for pairs within force cutoff
__device__ __constant__ bool _zbl; //is tersoff zbl?


#include "pair_tersoff_cuda_kernel_nc.cu"

#include <time.h>


void Cuda_PairTersoffCuda_Init(cuda_shared_data* sdata, Param_Float* params_host, void* map_host, void* elem2param_host, int nelements_h, bool zbl)
{
  unsigned cuda_ntypes = sdata->atom.ntypes + 1;
  X_CFLOAT box_size[3] = {
    sdata->domain.subhi[0] - sdata->domain.sublo[0],
    sdata->domain.subhi[1] - sdata->domain.sublo[1],
    sdata->domain.subhi[2] - sdata->domain.sublo[2]
  };

  hipMemcpyToSymbol(MY_AP(box_size)     , box_size                      , sizeof(X_CFLOAT) * 3);
  hipMemcpyToSymbol(MY_AP(cuda_ntypes)  , &cuda_ntypes                   , sizeof(unsigned));
  hipMemcpyToSymbol(MY_AP(virial)       , &sdata->pair.virial.dev_data   , sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(eng_vdwl)     , &sdata->pair.eng_vdwl.dev_data , sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(periodicity)  , sdata->domain.periodicity     , sizeof(int) * 3);
  hipMemcpyToSymbol(MY_AP(collect_forces_later), &sdata->pair.collect_forces_later  , sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(params), params_host  , sizeof(Param_Float)*nelements_h * nelements_h * nelements_h);
  hipMemcpyToSymbol(HIP_SYMBOL(elem2param), elem2param_host  , sizeof(int)*nelements_h * nelements_h * nelements_h);
  hipMemcpyToSymbol(HIP_SYMBOL(map), map_host  , sizeof(int)*cuda_ntypes);
  hipMemcpyToSymbol(HIP_SYMBOL(nelements), &nelements_h, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(_zbl), &zbl, sizeof(bool));

}

void Cuda_PairTersoffCuda(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist, int eflag, int vflag, int eflag_atom, int vflag_atom)
{
  static F_CFLOAT* glob_zeta_ij = NULL;
  static int glob_zeta_ij_size = 0;
  static F_CFLOAT4* glob_r_ij = NULL;
  static int* glob_numneigh_red = NULL;
  static int* glob_neighbors_red = NULL;
  static int* glob_neightype_red = NULL;

  if(glob_zeta_ij_size < sdata->atom.nall * sneighlist->maxneighbors * sizeof(F_CFLOAT)) {
    glob_zeta_ij_size = sdata->atom.nall * sneighlist->maxneighbors * sizeof(F_CFLOAT);
    hipFree(glob_zeta_ij);
    hipFree(glob_r_ij);
    hipFree(glob_numneigh_red);
    hipFree(glob_neighbors_red);
    hipFree(glob_neightype_red);
    hipMalloc(&glob_zeta_ij, glob_zeta_ij_size);
    hipMalloc(&glob_r_ij, glob_zeta_ij_size * 4);
    hipMalloc(&glob_numneigh_red, sdata->atom.nall * sizeof(int));
    hipMalloc(&glob_neighbors_red, sdata->atom.nall * sneighlist->maxneighbors * sizeof(int));
    hipMalloc(&glob_neightype_red, sdata->atom.nall * sneighlist->maxneighbors * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(_glob_numneigh_red), &glob_numneigh_red  , sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(_glob_neighbors_red), &glob_neighbors_red  , sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(_glob_neightype_red), &glob_neightype_red  , sizeof(int*));
    hipMemcpyToSymbol(HIP_SYMBOL(_glob_r_ij), &glob_r_ij  , sizeof(F_CFLOAT4*));
    hipMemcpyToSymbol(HIP_SYMBOL(_glob_zeta_ij), &glob_zeta_ij  , sizeof(F_CFLOAT*));
  }

  dim3 grid, threads;
  int sharedperproc;

  Cuda_Pair_PreKernel_AllStyles(sdata, sneighlist, eflag, vflag, grid, threads, sharedperproc, false, 64);
  hipStream_t* streams = (hipStream_t*) CudaWrapper_returnStreams();



  dim3 grid2;

  if(sdata->atom.nall <= 256 * 64000) {
    grid2.x = (sdata->atom.nall + 255) / 256;
    grid2.y = 1;
  } else {
    grid2.x = (sdata->atom.nall + 256 * 128 - 1) / (256 * 128);
    grid2.y = 128;
  }

  grid2.z = 1;
  dim3 threads2;
  threads2.x = 256;
  threads2.y = 1;
  threads2.z = 1;

  my_times time1, time2;

  //pre-calculate all neighbordistances and zeta_ij
  my_gettime(CLOCK_REALTIME, &time1);
  Pair_Tersoff_Kernel_TpA_RIJ <<< grid2, threads2, 0, streams[1]>>>
  ();
  hipDeviceSynchronize();
  Pair_Tersoff_Kernel_TpA_ZetaIJ <<< grid2, threads2, 0, streams[1]>>>
  ();
  hipDeviceSynchronize();
  my_gettime(CLOCK_REALTIME, &time2);
  sdata->cuda_timings.test1 +=
    time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;
  my_gettime(CLOCK_REALTIME, &time1);

  //actual force calculation
  unsigned int sharedsize = (sharedperproc * sizeof(ENERGY_CFLOAT) + 4 * sizeof(F_CFLOAT)) * threads.x; //extra 4 floats per thread used to reduce register pressure

  if(eflag) {
    if(vflag)
      Pair_Tersoff_Kernel_TpA<1, 1> <<< grid, threads, sharedsize, streams[1]>>>
      (eflag_atom, vflag_atom);
    else
      Pair_Tersoff_Kernel_TpA<1, 0> <<< grid, threads, sharedsize, streams[1]>>>
      (eflag_atom, vflag_atom);
  } else {
    if(vflag)
      Pair_Tersoff_Kernel_TpA<0, 1> <<< grid, threads, sharedsize, streams[1]>>>
      (eflag_atom, vflag_atom);
    else
      Pair_Tersoff_Kernel_TpA<0, 0> <<< grid, threads, sharedsize, streams[1]>>>
      (eflag_atom, vflag_atom);
  }
  hipDeviceSynchronize();
  my_gettime(CLOCK_REALTIME, &time2);
  sdata->cuda_timings.test2 +=
    time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

  Cuda_Pair_PostKernel_AllStyles(sdata, grid, sharedperproc, eflag, vflag);
}

