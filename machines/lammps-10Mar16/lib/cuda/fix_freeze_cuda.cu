#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX fix_freeze_cuda
#include "cuda_shared.h"
#include "cuda_common.h"
#include "crm_cuda_utils.cu"

#include "fix_freeze_cuda_cu.h"
#include "fix_freeze_cuda_kernel.cu"

void Cuda_FixFreezeCuda_UpdateBuffer(cuda_shared_data* sdata)
{
  int3 layout = getgrid(sdata->atom.nlocal, 3 * sizeof(F_CFLOAT));
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  int size = (unsigned)(layout.z * layout.y * layout.x) * 3 * sizeof(F_CFLOAT);

  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_FixFreezeCuda Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
    CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)

  }

  hipMemcpyToSymbol(MY_AP(buffer) , & sdata->buffer, sizeof(int*));
}

void Cuda_FixFreezeCuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(x)       , & sdata->atom.x    .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(f)       , & sdata->atom.f    .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(torque)  , & sdata->atom.torque .dev_data, sizeof(F_CFLOAT*));
}


void Cuda_FixFreezeCuda_Init(cuda_shared_data* sdata)
{
  Cuda_FixFreezeCuda_UpdateNmax(sdata);

}


void Cuda_FixFreezeCuda_PostForce(cuda_shared_data* sdata, int groupbit, F_CFLOAT* foriginal)
{
  if(sdata->atom.update_nmax)
    Cuda_FixFreezeCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  if(sdata->buffer_new)
    Cuda_FixFreezeCuda_UpdateBuffer(sdata);


  int3 layout = getgrid(sdata->atom.nlocal, 3 * sizeof(F_CFLOAT));
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  Cuda_FixFreezeCuda_PostForce_Kernel <<< grid, threads, threads.x* 3* sizeof(F_CFLOAT)>>> (groupbit);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Cuda_FixFreezeCuda_PostForce: fix add_force post_force compute Kernel execution failed");

  int oldgrid = grid.x;
  grid.x = 3;
  threads.x = 512;
  Cuda_FixFreezeCuda_Reduce_FOriginal <<< grid, threads, threads.x* sizeof(F_CFLOAT)>>> (oldgrid, foriginal);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Cuda_FixFreezeCuda_PostForce: fix add_force post_force reduce Kernel execution failed");

}
