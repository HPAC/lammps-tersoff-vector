#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

static inline __device__ void check_distance(X_CFLOAT &xtmp, X_CFLOAT &ytmp, X_CFLOAT &ztmp, int &i, int groupbit)
{
  if(_dist_check) {
    X_CFLOAT tmp = xtmp - _xhold[i];
    X_CFLOAT d = tmp * tmp;
    tmp = ytmp - _xhold[i + _maxhold];
    d += tmp * tmp;
    tmp = ztmp - _xhold[i + 2 * _maxhold];
    d += tmp * tmp;

    d = ((i < _nlocal) && (_mask[i] & groupbit)) ? d : X_F(0.0);

    if(not __all(d <= _triggerneighsq))
      _reneigh_flag[0] = 1;
  }
}


__global__ void FixNVECuda_InitialIntegrate_Kernel(int groupbit)
{
  X_CFLOAT xtmp, ytmp, ztmp;
#ifdef CUDA_USE_BINNING

  const unsigned bin = gridDim.y * blockIdx.x + blockIdx.y;

  if(threadIdx.x < _bin_count_local[bin]) {
    const int i = 3 * blockDim.x * bin + threadIdx.x;

    if(_mask[i] & groupbit) {
      F_CFLOAT* my_f = _binned_f + i;
      V_CFLOAT* my_v = _binned_v + i;
      X_CFLOAT* my_x = _binned_x + i;

      V_CFLOAT 		dtfm = _dtf

                         if(_rmass_flag) dtfm *= V_F(1.0) / _binned_rmass[i];
      else 			dtfm *= V_F(1.0) / _mass[_binned_type[blockDim.x * bin + threadIdx.x]];

      V_CFLOAT v_mem;
      v_mem = *my_v += dtfm * (*my_f);
      xtmp = *my_x += _dtv * v_mem;
      my_f += blockDim.x;
      my_v += blockDim.x;
      my_x += blockDim.x;
      v_mem = *my_v += dtfm * (*my_f);
      ytmp = *my_x += _dtv * v_mem;
      my_f += blockDim.x;
      my_v += blockDim.x;
      my_x += blockDim.x;
      v_mem = *my_v += dtfm * (*my_f);
      ztmp = *my_x += _dtv * v_mem;
    }
  }

#else

  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal && _mask[i] & groupbit) {
    F_CFLOAT* my_f = _f + i;
    V_CFLOAT* my_v = _v + i;
    X_CFLOAT* my_x = _x + i;

    V_CFLOAT 		dtfm = _dtf;

    if(_rmass_flag) dtfm *= V_F(1.0) / _rmass[i];
    else 			dtfm *= V_F(1.0) / _mass[_type[i]];

    V_CFLOAT v_mem;
    v_mem = *my_v += dtfm * (*my_f);
    xtmp = *my_x += _dtv * v_mem;
    my_f += _nmax;
    my_v += _nmax;
    my_x += _nmax;
    v_mem = *my_v += dtfm * (*my_f);
    ytmp = *my_x += _dtv * v_mem;
    my_f += _nmax;
    my_v += _nmax;
    my_x += _nmax;
    v_mem = *my_v += dtfm * (*my_f);
    ztmp = *my_x += _dtv * v_mem;
  }

#endif

  check_distance(xtmp, ytmp, ztmp, i, groupbit);
}

__global__ void FixNVECuda_FinalIntegrate_Kernel(int groupbit)
{
#ifdef CUDA_USE_BINNING

  const unsigned bin = gridDim.y * blockIdx.x + blockIdx.y;

  if(threadIdx.x < _bin_count_local[bin]) {
    const int i = 3 * blockDim.x * bin + threadIdx.x;

    if(_mask[i] & groupbit) {
      F_CFLOAT* my_f = _binned_f + i;
      V_CFLOAT* my_v = _binned_v + i;

      V_CFLOAT 		dtfm = _dtf

                         if(_rmass_flag) dtfm *= V_F(1.0) / _binned_rmass[i];
      else 			dtfm *= V_F(1.0) / _mass[_binned_type[blockDim.x * bin + threadIdx.x]];

      *my_v += dtfm * (*my_f);
      my_f += blockDim.x;
      my_v += blockDim.x;
      *my_v += dtfm * (*my_f);
      my_f += blockDim.x;
      my_v += blockDim.x;
      *my_v += dtfm * (*my_f);
    }
  }

#else

  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal && _mask[i] & groupbit) {
    F_CFLOAT* my_f = _f + i;
    V_CFLOAT* my_v = _v + i;

    V_CFLOAT 		dtfm = _dtf;

    if(_rmass_flag) dtfm *= V_F(1.0) / _rmass[i];
    else 			dtfm *= V_F(1.0) / _mass[_type[i]];

    *my_v += dtfm * (*my_f);
    my_f += _nmax;
    my_v += _nmax;
    *my_v += dtfm * (*my_f);
    my_f += _nmax;
    my_v += _nmax;
    *my_v += dtfm * (*my_f);
  }

#endif
}



