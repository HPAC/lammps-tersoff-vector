#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */
#define Pi F_F(3.1415926535897932384626433832795)
#define PI Pi
#define PI2 F_F(0.5)*Pi
#define PI4 F_F(0.25)*Pi



__device__ void twobody(int iparam, F_CFLOAT rsq, F_CFLOAT &fforce,
                        int eflag, ENERGY_CFLOAT &eng)
{
  F_CFLOAT r, rp, rq, rainv, expsrainv;

  r = sqrt(rsq);
  rp = pow(r, -params_sw[iparam].powerp);
  rq = pow(r, -params_sw[iparam].powerq);
  rainv = 1.0 / (r - params_sw[iparam].cut);
  expsrainv = exp(params_sw[iparam].sigma * rainv);
  fforce = (params_sw[iparam].c1 * rp - params_sw[iparam].c2 * rq +
            (params_sw[iparam].c3 * rp - params_sw[iparam].c4 * rq) * rainv * rainv * r) * expsrainv / rsq;

  if(eflag) eng += (params_sw[iparam].c5 * rp - params_sw[iparam].c6 * rq) * expsrainv;
}

__device__ void threebody(int paramij, int paramik, int paramijk,
                          F_CFLOAT4 &delr1,
                          F_CFLOAT4 &delr2,
                          F_CFLOAT3 &fj, F_CFLOAT3 &fk, int eflag, ENERGY_CFLOAT &eng)
{
  F_CFLOAT r1, rinvsq1, rainv1, gsrainv1, gsrainvsq1, expgsrainv1;
  F_CFLOAT r2, rinvsq2, rainv2, gsrainv2, gsrainvsq2, expgsrainv2;
  F_CFLOAT rinv12, cs, delcs, delcssq, facexp, facrad, frad1, frad2;
  F_CFLOAT facang, facang12, csfacang, csfac1, csfac2;

  r1 = sqrt(delr1.w);
  rinvsq1 = F_F(1.0) / delr1.w;
  rainv1 = F_F(1.0) / (r1 - params_sw[paramij].cut);
  gsrainv1 = params_sw[paramij].sigma_gamma * rainv1;
  gsrainvsq1 = gsrainv1 * rainv1 / r1;
  expgsrainv1 = exp(gsrainv1);

  r2 = sqrt(delr2.w);
  rinvsq2 = F_F(1.0) / delr2.w;
  rainv2 = F_F(1.0) / (r2 - params_sw[paramik].cut);
  gsrainv2 = params_sw[paramik].sigma_gamma * rainv2;
  gsrainvsq2 = gsrainv2 * rainv2 / r2;
  expgsrainv2 = exp(gsrainv2);

  rinv12 = F_F(1.0) / (r1 * r2);
  cs = (delr1.x * delr2.x + delr1.y * delr2.y + delr1.z * delr2.z) * rinv12;
  delcs = cs - params_sw[paramijk].costheta;
  delcssq = delcs * delcs;

  facexp = expgsrainv1 * expgsrainv2;

  // facrad = sqrt(paramij->lambda_epsilon*paramik->lambda_epsilon) *
  //          facexp*delcssq;

  facrad = params_sw[paramijk].lambda_epsilon * facexp * delcssq;
  frad1 = facrad * gsrainvsq1;
  frad2 = facrad * gsrainvsq2;
  facang = params_sw[paramijk].lambda_epsilon2 * facexp * delcs;
  facang12 = rinv12 * facang;
  csfacang = cs * facang;
  csfac1 = rinvsq1 * csfacang;

  fj.x = delr1.x * (frad1 + csfac1) - delr2.x * facang12;
  fj.y = delr1.y * (frad1 + csfac1) - delr2.y * facang12;
  fj.z = delr1.z * (frad1 + csfac1) - delr2.z * facang12;

  csfac2 = rinvsq2 * csfacang;

  fk.x = delr2.x * (frad2 + csfac2) - delr1.x * facang12;
  fk.y = delr2.y * (frad2 + csfac2) - delr1.y * facang12;
  fk.z = delr2.z * (frad2 + csfac2) - delr1.z * facang12;

  if(eflag) eng += F_F(2.0) * facrad;
}

__device__ void threebody_fj(int paramij, int paramik, int paramijk,
                             F_CFLOAT4 &delr1,
                             F_CFLOAT4 &delr2,
                             F_CFLOAT3 &fj)
{
  F_CFLOAT r1, rinvsq1, rainv1, gsrainv1, gsrainvsq1, expgsrainv1;
  F_CFLOAT r2, rainv2, gsrainv2, expgsrainv2;
  F_CFLOAT rinv12, cs, delcs, delcssq, facexp, facrad, frad1;
  F_CFLOAT facang, facang12, csfacang, csfac1;

  r1 = sqrt(delr1.w);
  rinvsq1 = F_F(1.0) / delr1.w;
  rainv1 = F_F(1.0) / (r1 - params_sw[paramij].cut);
  gsrainv1 = params_sw[paramij].sigma_gamma * rainv1;
  gsrainvsq1 = gsrainv1 * rainv1 / r1;
  expgsrainv1 = exp(gsrainv1);

  r2 = sqrt(delr2.w);
  rainv2 = F_F(1.0) / (r2 - params_sw[paramik].cut);
  gsrainv2 = params_sw[paramik].sigma_gamma * rainv2;
  expgsrainv2 = exp(gsrainv2);

  rinv12 = F_F(1.0) / (r1 * r2);
  cs = (delr1.x * delr2.x + delr1.y * delr2.y + delr1.z * delr2.z) * rinv12;
  delcs = cs - params_sw[paramijk].costheta;
  delcssq = delcs * delcs;

  facexp = expgsrainv1 * expgsrainv2;

  // facrad = sqrt(paramij->lambda_epsilon*paramik->lambda_epsilon) *
  //          facexp*delcssq;

  facrad = params_sw[paramijk].lambda_epsilon * facexp * delcssq;
  frad1 = facrad * gsrainvsq1;
  facang = params_sw[paramijk].lambda_epsilon2 * facexp * delcs;
  facang12 = rinv12 * facang;
  csfacang = cs * facang;
  csfac1 = rinvsq1 * csfacang;

  fj.x = delr1.x * (frad1 + csfac1) - delr2.x * facang12;
  fj.y = delr1.y * (frad1 + csfac1) - delr2.y * facang12;
  fj.z = delr1.z * (frad1 + csfac1) - delr2.z * facang12;
}


__global__ void Pair_SW_Kernel_TpA_RIJ()//F_CFLOAT4* _glob_r_ij,int* _glob_numneigh_red,int* _glob_neighbors_red,int* _glob_neightype_red)
{
  int ii = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(ii >= _nall) return;

  X_CFLOAT4 myxtype;
  F_CFLOAT4 delij;
  F_CFLOAT xtmp, ytmp, ztmp;
  int itype, jnum, i, j;
  int* jlist;
  int neigh_red = 0;
  i = ii;//_ilist[ii];
  myxtype = fetchXType(i);

  xtmp = myxtype.x;
  ytmp = myxtype.y;
  ztmp = myxtype.z;
  itype = map[(static_cast <int>(myxtype.w))];

  jnum = _numneigh[i];
  jlist = &_neighbors[i];

  __syncthreads();

  for(int jj = 0; jj < jnum; jj++) {
    if(jj < jnum) {

      j = jlist[jj * _nall];
      j &= NEIGHMASK;
      myxtype = fetchXType(j);
      delij.x = xtmp - myxtype.x;
      delij.y = ytmp - myxtype.y;
      delij.z = ztmp - myxtype.z;
      int jtype = map[(static_cast <int>(myxtype.w))];
      int iparam_ij = elem2param[(itype * nelements + jtype) * nelements + jtype];
      delij.w = vec3_dot(delij, delij);

      if(delij.w < params_sw[iparam_ij].cutsq) {
        _glob_neighbors_red[i + neigh_red * _nall] = j;
        _glob_neightype_red[i + neigh_red * _nall] = jtype;
        _glob_r_ij[i + neigh_red * _nall] = delij;
        neigh_red++;
      }
    }
  }

  _glob_numneigh_red[i] = neigh_red;
}


template <int eflag, int vflagm>
__global__ void Pair_SW_Kernel_TpA(int eflag_atom, int vflag_atom) //,F_CFLOAT* _glob_zeta_ij,F_CFLOAT4* _glob_r_ij,int* _glob_numneigh_red,int* _glob_neighbors_red,int* _glob_neightype_red)
{
  ENERGY_CFLOAT evdwl = ENERGY_F(0.0);

  ENERGY_CFLOAT* sharedE = &sharedmem[threadIdx.x];
  ENERGY_CFLOAT* sharedV = &sharedmem[threadIdx.x];

  F_CFLOAT* shared_F_F = (F_CFLOAT*) sharedmem;

  if((eflag || eflag_atom) && (vflagm || vflag_atom)) shared_F_F = (F_CFLOAT*) &sharedmem[7 * blockDim.x];
  else if(eflag) shared_F_F = (F_CFLOAT*) &sharedmem[blockDim.x];
  else if(vflagm) shared_F_F = (F_CFLOAT*) &sharedmem[6 * blockDim.x];

  shared_F_F += threadIdx.x;

  if(eflag_atom || eflag) {
    sharedE[0] = ENERGY_F(0.0);
    sharedV += blockDim.x;
  }

  if(vflagm || vflag_atom) {
    sharedV[0 * blockDim.x] = ENERGY_F(0.0);
    sharedV[1 * blockDim.x] = ENERGY_F(0.0);
    sharedV[2 * blockDim.x] = ENERGY_F(0.0);
    sharedV[3 * blockDim.x] = ENERGY_F(0.0);
    sharedV[4 * blockDim.x] = ENERGY_F(0.0);
    sharedV[5 * blockDim.x] = ENERGY_F(0.0);
  }

  int jnum_red = 0;
#define fxtmp shared_F_F[0]
#define fytmp shared_F_F[blockDim.x]
#define fztmp shared_F_F[2*blockDim.x]
  //#define jnum_red (static_cast <int> (shared_F_F[3*blockDim.x]))

  int ii = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  X_CFLOAT4 myxtype_i, myxtype_j, myxtype_k;
  F_CFLOAT4 delij, delik, deljk;
  F_CFLOAT fpair;

  int itype, i, j;
  int* jlist_red;

  if(ii < _inum) {
    i = _ilist[ii];

    if(vflagm)
      myxtype_i = fetchXType(i);

    //itype=map[(static_cast <int> (myxtype_i.w))];
    itype = map[_type[i]];


    fxtmp = F_F(0.0);
    fytmp = F_F(0.0);
    fztmp = F_F(0.0);


    //shared_F_F[3*blockDim.x] = _glob_numneigh_red[i];
    jnum_red = _glob_numneigh_red[i];
    jlist_red = &_glob_neighbors_red[i];
  }

  __syncthreads();
#pragma unroll 1

  for(int jj = 0; jj < jnum_red; jj++) {
    if(i < _nlocal) {
      fpair = F_F(0.0);
      j = jlist_red[jj * _nall];
      j &= NEIGHMASK;

      if(vflagm)
        myxtype_j = fetchXType(j);

      int jtype = _glob_neightype_red[i + jj * _nall];
      delij = _glob_r_ij[i + jj * _nall];

      volatile int iparam_ij = elem2param[(itype * nelements + jtype) * nelements + jtype];
      volatile int iparam_ji = elem2param[(jtype * nelements + itype) * nelements + itype];

      if(delij.w < params_sw[iparam_ij].cutsq) {
        F_CFLOAT dxfp, dyfp, dzfp;
        twobody(iparam_ij, delij.w, fpair, eflag, evdwl);
        fxtmp += dxfp = delij.x * fpair;
        fytmp += dyfp = delij.y * fpair;
        fztmp += dzfp = delij.z * fpair;

        if(vflagm) {
          sharedV[0 * blockDim.x] += delij.x * dxfp;
          sharedV[1 * blockDim.x] += delij.y * dyfp;
          sharedV[2 * blockDim.x] += delij.z * dzfp;
          sharedV[3 * blockDim.x] += delij.x * dyfp;
          sharedV[4 * blockDim.x] += delij.x * dzfp;
          sharedV[5 * blockDim.x] += delij.y * dzfp;
        }






        vec3_scale(F_F(-1.0), delij, delij);

#pragma unroll 1

        for(int kk = jj + 1; kk < jnum_red; kk++) {
          int k = jlist_red[kk * _nall];
          k &= NEIGHMASK;

          if(vflagm)
            myxtype_k = fetchXType(k);

          delik = _glob_r_ij[i + kk * _nall];

          int ktype = _glob_neightype_red[i + kk * _nall];
          int iparam_ik = elem2param[(itype * nelements + ktype) * nelements + ktype];
          int iparam_ijk = elem2param[(itype * nelements + jtype) * nelements + ktype];
          vec3_scale(F_F(-1.0), delik, delik);

          if(delik.w <= params_sw[iparam_ijk].cutsq) {
            F_CFLOAT3 fj, fk;
            threebody(iparam_ij, iparam_ik, iparam_ijk,
                      delij, delik, fj, fk, eflag, evdwl);
            fxtmp -= fj.x + fk.x;
            fytmp -= fj.y + fk.y;
            fztmp -= fj.z + fk.z;

            if(vflagm) {
              sharedV[0 * blockDim.x] -= ENERGY_F(2.0) * myxtype_i.x * (fj.x + fk.x);
              sharedV[1 * blockDim.x] -= ENERGY_F(2.0) * myxtype_i.y * (fj.y + fk.y);
              sharedV[2 * blockDim.x] -= ENERGY_F(2.0) * myxtype_i.z * (fj.z + fk.z);
              sharedV[3 * blockDim.x] -= ENERGY_F(2.0) * myxtype_i.x * (fj.y + fk.y);
              sharedV[4 * blockDim.x] -= ENERGY_F(2.0) * myxtype_i.x * (fj.z + fk.z);
              sharedV[5 * blockDim.x] -= ENERGY_F(2.0) * myxtype_i.y * (fj.z + fk.z);

              sharedV[0 * blockDim.x] += ENERGY_F(2.0) * myxtype_j.x * fj.x;
              sharedV[1 * blockDim.x] += ENERGY_F(2.0) * myxtype_j.y * fj.y;
              sharedV[2 * blockDim.x] += ENERGY_F(2.0) * myxtype_j.z * fj.z;
              sharedV[3 * blockDim.x] += ENERGY_F(2.0) * myxtype_j.x * fj.y;
              sharedV[4 * blockDim.x] += ENERGY_F(2.0) * myxtype_j.x * fj.z;
              sharedV[5 * blockDim.x] += ENERGY_F(2.0) * myxtype_j.y * fj.z;

              sharedV[0 * blockDim.x] += ENERGY_F(2.0) * myxtype_k.x * fk.x;
              sharedV[1 * blockDim.x] += ENERGY_F(2.0) * myxtype_k.y * fk.y;
              sharedV[2 * blockDim.x] += ENERGY_F(2.0) * myxtype_k.z * fk.z;
              sharedV[3 * blockDim.x] += ENERGY_F(2.0) * myxtype_k.x * fk.y;
              sharedV[4 * blockDim.x] += ENERGY_F(2.0) * myxtype_k.x * fk.z;
              sharedV[5 * blockDim.x] += ENERGY_F(2.0) * myxtype_k.y * fk.z;
            }
          }
        }

        int j_jnum_red = _glob_numneigh_red[j];
        int* j_jlist_red = &_glob_neighbors_red[j];

        int j_ii = 0;

        //#pragma unroll 1
        for(int j_kk = 0; j_kk < j_jnum_red; j_kk++) {
          if(j_jlist_red[j_kk * _nall] == i) j_ii = j_kk;
        }

#pragma unroll 1

        for(int kk = 0; kk < j_jnum_red; kk++) {
          if(j_ii == kk) continue;

          int k = j_jlist_red[kk * _nall];
          k &= NEIGHMASK;
          deljk = _glob_r_ij[j + kk * _nall];
          vec3_scale(F_F(-1.0), deljk, deljk);
          int ktype = _glob_neightype_red[j + kk * _nall];

          int iparam_ji = elem2param[(jtype * nelements + itype) * nelements + itype];
          int iparam_jk = elem2param[(jtype * nelements + ktype) * nelements + ktype];
          int iparam_jik = elem2param[(jtype * nelements + itype) * nelements + ktype];


          vec3_scale(F_F(-1.0), delij, delij);

          if(deljk.w <= params_sw[iparam_jik].cutsq) {
            F_CFLOAT3 fj;

            threebody_fj(iparam_ji, iparam_jk, iparam_jik,
                         delij, deljk, fj);
            fxtmp += fj.x;
            fytmp += fj.y;
            fztmp += fj.z;

          }

          vec3_scale(F_F(-1.0), delij, delij);
        }
      }
    }

  }

  __syncthreads();

  if(ii < _inum) {
    F_CFLOAT* my_f;

    if(_collect_forces_later) {
      ENERGY_CFLOAT* buffer = (ENERGY_CFLOAT*) _buffer;

      if(eflag) {
        buffer = &buffer[1 * gridDim.x * gridDim.y];
      }

      if(vflagm) {
        buffer = &buffer[6 * gridDim.x * gridDim.y];
      }

      my_f = (F_CFLOAT*) buffer;
      my_f += i;
      *my_f = fxtmp;
      my_f += _nmax;
      *my_f = fytmp;
      my_f += _nmax;
      *my_f = fztmp;
    } else {
      my_f = _f + i;
      *my_f += fxtmp;
      my_f += _nmax;
      *my_f += fytmp;
      my_f += _nmax;
      *my_f += fztmp;
    }
  }

  __syncthreads();

  if(eflag) {
    sharedE[0] = evdwl;
  }

  if(eflag_atom && i < _nlocal) {
    _eatom[i] = ENERGY_F(0.5) * evdwl;
  }

  if(vflag_atom && i < _nlocal) {
    _vatom[i]         = ENERGY_F(0.5) * sharedV[0 * blockDim.x];
    _vatom[i + _nmax]   = ENERGY_F(0.5) * sharedV[1 * blockDim.x];
    _vatom[i + 2 * _nmax] = ENERGY_F(0.5) * sharedV[2 * blockDim.x];
    _vatom[i + 3 * _nmax] = ENERGY_F(0.5) * sharedV[3 * blockDim.x];
    _vatom[i + 4 * _nmax] = ENERGY_F(0.5) * sharedV[4 * blockDim.x];
    _vatom[i + 5 * _nmax] = ENERGY_F(0.5) * sharedV[5 * blockDim.x];
  }

  if(vflagm && eflag) PairVirialCompute_A_Kernel_Template<1, 1>();
  else if(eflag) PairVirialCompute_A_Kernel_Template<1, 0>();
  else if(vflagm) PairVirialCompute_A_Kernel_Template<0, 1>();

#undef fxtmp
#undef fytmp
#undef fztmp
  //#undef jnum_red
}
