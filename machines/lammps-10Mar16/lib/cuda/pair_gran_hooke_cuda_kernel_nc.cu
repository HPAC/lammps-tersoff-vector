#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */


__global__ void PairGranHookeCuda_Kernel(int eflag, int vflag, int eflag_atom, int vflag_atom, int** firstneight, int* binned_id
    , F_CFLOAT kn, F_CFLOAT gamman, F_CFLOAT gammat, F_CFLOAT xmu)
{
  ENERGY_CFLOAT evdwl = ENERGY_F(0.0);

  ENERGY_CFLOAT* sharedE;
  ENERGY_CFLOAT* sharedV;

  if(eflag || eflag_atom) {
    sharedE = &sharedmem[threadIdx.x];
    sharedV = &sharedmem[0];
    sharedE[0] = ENERGY_F(0.0);
    sharedV += blockDim.x;
  }

  if(vflag || vflag_atom) {
    sharedV += threadIdx.x;
    sharedV[0 * blockDim.x] = ENERGY_F(0.0);
    sharedV[1 * blockDim.x] = ENERGY_F(0.0);
    sharedV[2 * blockDim.x] = ENERGY_F(0.0);
    sharedV[3 * blockDim.x] = ENERGY_F(0.0);
    sharedV[4 * blockDim.x] = ENERGY_F(0.0);
    sharedV[5 * blockDim.x] = ENERGY_F(0.0);
  }

  int ii = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  MYEMUDBG(if(ii == 0) printf("# CUDA: PairGranHookeCuda_Kernel: -- no binning --\n");)

    X_CFLOAT xtmp, ytmp, ztmp;

  X_CFLOAT4 myxtype;
  V_CFLOAT4 myvradius, ovradius;
  F_CFLOAT fxtmp, fytmp, fztmp, torquextmp, torqueytmp, torqueztmp;
  F_CFLOAT delx, dely, delz;
  F_CFLOAT radi, radj, radsum, r, rsqinv;
  F_CFLOAT vr1, vr2, vr3, vnnr, vn1, vn2, vn3, vt1, vt2, vt3;
  F_CFLOAT wr1, wr2, wr3;
  F_CFLOAT vtr1, vtr2, vtr3, vrel;
  F_CFLOAT meff, damp, ccel, tor1, tor2, tor3;
  F_CFLOAT fn, fs, ft, fs1, fs2, fs3;

  int jnum = 0;
  int i, j;
  int* jlist;

  if(ii < _inum) {
    i = _ilist[ii];

    myxtype = fetchXType(i);
    myvradius = fetchVRadius(i);

    xtmp = myxtype.x;
    ytmp = myxtype.y;
    ztmp = myxtype.z;
    radi = myvradius.w;

    fxtmp = F_F(0.0);
    fytmp = F_F(0.0);
    fztmp = F_F(0.0);
    torquextmp = F_F(0.0);
    torqueytmp = F_F(0.0);
    torqueztmp = F_F(0.0);

    jnum = _numneigh[i];

    jlist = &_neighbors[i];
  }

  __syncthreads();

  for(int jj = 0; jj < jnum; jj++) {
    if(ii < _inum)
      if(jj < jnum) {
        j = jlist[jj * _nlocal];

        myxtype = fetchXType(j);
        ovradius = fetchVRadius(j);

        delx = xtmp - myxtype.x;
        dely = ytmp - myxtype.y;
        delz = ztmp - myxtype.z;

        radj = ovradius.w;
        radsum = radi + radj;

        const F_CFLOAT rsq = delx * delx + dely * dely + delz * delz;

        if(rsq < radsum * radsum) {
          const F_CFLOAT rinv = _RSQRT_(rsq);
          r = F_F(1.0) / rinv;
          rsqinv = F_F(1.0) / rsq;

          // relative translational velocity

          vr1 = myvradius.x - ovradius.x;
          vr2 = myvradius.y - ovradius.y;
          vr3 = myvradius.z - ovradius.z;

          // normal component

          vnnr = vr1 * delx + vr2 * dely + vr3 * delz;
          vn1 = delx * vnnr * rsqinv;
          vn2 = dely * vnnr * rsqinv;
          vn3 = delz * vnnr * rsqinv;

          // tangential component

          vt1 = vr1 - vn1;
          vt2 = vr2 - vn2;
          vt3 = vr3 - vn3;

          // relative rotational velocity
          V_CFLOAT4 omegarmass_i = fetchOmegaRmass(i);
          V_CFLOAT4 omegarmass_j = fetchOmegaRmass(j);

          wr1 = (radi * omegarmass_i.x + radj * omegarmass_j.x) * rinv;
          wr2 = (radi * omegarmass_i.y + radj * omegarmass_j.y) * rinv;
          wr3 = (radi * omegarmass_i.z + radj * omegarmass_j.z) * rinv;

          meff = omegarmass_i.w * omegarmass_j.w / (omegarmass_i.w + omegarmass_j.w);

          if(_mask[i] & _freeze_group_bit) meff = omegarmass_j.w;

          if(_mask[j] & _freeze_group_bit) meff = omegarmass_i.w;

          damp = meff * gamman * vnnr * rsqinv;
          ccel = kn * (radsum - r) * rinv - damp;

          vtr1 = vt1 - (delz * wr2 - dely * wr3);
          vtr2 = vt2 - (delx * wr3 - delz * wr1);
          vtr3 = vt3 - (dely * wr1 - delx * wr2);
          vrel = vtr1 * vtr1 + vtr2 * vtr2 + vtr3 * vtr3;
          vrel = _SQRT_(vrel);

          fn = xmu * fabs(ccel * r);
          fs = meff * gammat * vrel;
          ft = (vrel != F_F(0.0)) ? MIN(fn, fs) / vrel : F_F(0.0);

          fs1 = -ft * vtr1;
          fs2 = -ft * vtr2;
          fs3 = -ft * vtr3;

          F_CFLOAT dxfp, dyfp, dzfp;
          fxtmp += dxfp = delx * ccel + fs1;
          fytmp += dyfp = dely * ccel + fs2;
          fztmp += dzfp = delz * ccel + fs3;

          tor1 = rinv * (dely * fs3 - delz * fs2);
          tor2 = rinv * (delz * fs1 - delx * fs3);
          tor3 = rinv * (delx * fs2 - dely * fs1);

          torquextmp -= radi * tor1;
          torqueytmp -= radi * tor2;
          torqueztmp -= radi * tor3;

          if(vflag) {
            sharedV[0 * blockDim.x] += delx * dxfp;
            sharedV[1 * blockDim.x] += dely * dyfp;
            sharedV[2 * blockDim.x] += delz * dzfp;
            sharedV[3 * blockDim.x] += delx * dyfp;
            sharedV[4 * blockDim.x] += delx * dzfp;
            sharedV[5 * blockDim.x] += dely * dzfp;
          }

        }
      }
  }

  __syncthreads();

  if(ii < _inum) {
    F_CFLOAT* my_f = _f + i;
    *my_f += fxtmp;
    my_f += _nmax;
    *my_f += fytmp;
    my_f += _nmax;
    *my_f += fztmp;
    F_CFLOAT* my_torque = _torque + i;
    *my_torque += torquextmp;
    my_torque += _nmax;
    *my_torque += torqueytmp;
    my_torque += _nmax;
    *my_torque += torqueztmp;
  }

  __syncthreads();

  if(eflag) sharedE[0] = evdwl;

  if(eflag_atom && i < _nlocal) _eatom[i] += evdwl;

  if(vflag_atom && i < _nlocal) {
    _vatom[i]         += ENERGY_F(0.5) * sharedV[0 * blockDim.x];
    _vatom[i + _nmax]   += ENERGY_F(0.5) * sharedV[1 * blockDim.x];
    _vatom[i + 2 * _nmax] += ENERGY_F(0.5) * sharedV[2 * blockDim.x];
    _vatom[i + 3 * _nmax] += ENERGY_F(0.5) * sharedV[3 * blockDim.x];
    _vatom[i + 4 * _nmax] += ENERGY_F(0.5) * sharedV[4 * blockDim.x];
    _vatom[i + 5 * _nmax] += ENERGY_F(0.5) * sharedV[5 * blockDim.x];
  }

  if(vflag || eflag) PairVirialCompute_A_Kernel(eflag, vflag, 0);
}
