/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>

#define _lj1 MY_AP(coeff1)
#define _lj2 MY_AP(coeff2)
#define _lj3 MY_AP(coeff3)
#define _lj4 MY_AP(coeff4)
#define _lj_type MY_AP(coeff5)


#include "pair_lj_sdk_coul_debye_cuda_cu.h"
#include <time.h>




void Cuda_PairLJSDKCoulDebyeCuda_Init(cuda_shared_data* sdata)
{
  Cuda_Pair_Init_AllStyles(sdata, 5, true, false);

}




void Cuda_PairLJSDKCoulDebyeCuda(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist, int eflag, int vflag, int eflag_atom, int vflag_atom)
{

  // initialize only on first call
  static  short init = 0;

  if(! init) {
    init = 1;
    Cuda_PairLJSDKCoulDebyeCuda_Init(sdata);
  }

  dim3 grid, threads;
  int sharedperproc;

  Cuda_Pair_PreKernel_AllStyles(sdata, sneighlist, eflag, vflag, grid, threads, sharedperproc, true, 128);

  hipStream_t* streams = (hipStream_t*) CudaWrapper_returnStreams();

  if(sdata->pair.use_block_per_atom)
    Pair_Kernel_BpA<PAIR_CG_CMM, COUL_DEBYE, DATA_NONE>
    <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x, streams[1]>>> (eflag, vflag, eflag_atom, vflag_atom);
  else
    Pair_Kernel_TpA<PAIR_CG_CMM, COUL_DEBYE, DATA_NONE>
    <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x, streams[1]>>> (eflag, vflag, eflag_atom, vflag_atom);

  Cuda_Pair_PostKernel_AllStyles(sdata, grid, sharedperproc, eflag, vflag);
}

#undef _lj1
#undef _lj2
#undef _lj3
#undef _lj4
#undef _lj_type

