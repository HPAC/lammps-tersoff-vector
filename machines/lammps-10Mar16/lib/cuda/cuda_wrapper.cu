/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#include "cuda_shared.h"
#include "cuda_common.h"
#include "cuda_wrapper_cu.h"
#include "cuda_wrapper_kernel.cu"

static int CudaWrapper_total_gpu_mem = 0;
static double CudaWrapper_total_upload_time = 0;
static double CudaWrapper_total_download_time = 0;
static double CudaWrapper_cpubuffer_upload_time = 0;
static double CudaWrapper_cpubuffer_download_time = 0;
static hipStream_t* streams;
static int nstreams = 0;

void CudaWrapper_Init(int argc, char** argv, int me, int ppn, int* devicelist)
{
  MYDBG(printf("# CUDA: debug mode on\n");)

#if __DEVICE_EMULATION__

  printf("# CUDA: emulation mode on\n");

#else

  // modified from cutil.h
  static int deviceCount = 0;
  static bool sharedmode = false;

  if(deviceCount && !sharedmode) return;

  if(deviceCount && sharedmode) hipDeviceReset();

  CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceCount(&deviceCount));

  if(deviceCount == 0) {
    fprintf(stderr, "cutil error: no devices supporting CUDA.\n");
    exit(EXIT_FAILURE);
  }

  MYDBG(printf("# CUDA There are %i devices supporting CUDA in this system.\n", deviceCount);)

  hipDeviceProp_t deviceProp[deviceCount];

  for(int i = 0; i < deviceCount; i++)
    CUDA_SAFE_CALL_NO_SYNC(hipGetDeviceProperties(&(deviceProp[i]), i));


  int dev_list[deviceCount];

  for(int i = 0; i < deviceCount; i++) dev_list[i] = i;

  for(int i = 0; i < deviceCount; i++) {
    for(int j = 0; j < deviceCount - 1 - i; j++)
      if(deviceProp[dev_list[j]].multiProcessorCount < deviceProp[dev_list[j + 1]].multiProcessorCount) {
        int k = dev_list[j];
        dev_list[j] = dev_list[j + 1];
        dev_list[j + 1] = k;
      }
  }

  for(int i = 0; i < deviceCount; i++) {
    if((deviceProp[dev_list[i]].computeMode == 0)) sharedmode = true;

    hipSetDevice(i);
    hipSetDeviceFlags(hipDeviceMapHost);
  }

  if(sharedmode) {
    if(ppn && (me % ppn + 1) > deviceCount) {
      printf("Asking for more GPUs per node when there are. Reduce gpu/node setting.\n");
      exit(0);
    }

    int devicea = me % ppn;

    if(devicelist) devicea = devicelist[devicea];
    else
      devicea = dev_list[devicea];

    if(devicea >= deviceCount)  {
      printf("Asking for non existent GPU %i. Found only %i GPUs.\n", devicea, deviceCount);
      exit(0);
    }

    MYDBG(
      printf(" # CUDA  myid: %i take device: %i\n", me, devicea);
    )
    CUDA_SAFE_CALL(hipSetDevice(devicea));
  } else {
    CUDA_SAFE_CALL(hipSetValidDevices(dev_list, deviceCount));
  }

  hipDeviceSynchronize();

  int dev;
  CUDA_SAFE_CALL(hipGetDevice(&dev));

  if(deviceProp[dev].major < 1) {
    fprintf(stderr, "CUDA error: device does not support CUDA.\n");
    exit(EXIT_FAILURE);
  } else if((deviceProp[dev].major == 1) && (deviceProp[dev].minor != 3)) {
    fprintf(stderr, "CUDA error: You need a device with compute capability 1.3 or higher (Device %i is a %s with CC %i.%i)\n", dev, deviceProp[dev].name, deviceProp[dev].major, deviceProp[dev].minor);
    exit(EXIT_FAILURE);
  }

  if((deviceProp[dev].major == 2) && (CUDA_ARCH < 20)) {
    fprintf(stderr, "CUDA warning: You are using a compute %i.%i or higher GPU while LAMMPScuda has been compiled for architecture 1.3\n", deviceProp[dev].major, deviceProp[dev].minor);
  }

  if((deviceProp[dev].major == 1) && (CUDA_ARCH >= 20)) {
    fprintf(stderr, "CUDA error: You are using a compute 1.3 GPU while LAMMPScuda has been compiled for architecture %i\n", CUDA_ARCH);
    exit(EXIT_FAILURE);
  }


  fprintf(stderr, "# Using device %d: %s\n", dev, deviceProp[dev].name);
  MYDBG(fprintf(stderr, "# Using device %d: %s\n", dev, deviceProp[dev].name);)

  MYDBG
  (
    printf("name = %s\n", deviceProp[dev].name);
    printf("totalGlobalMem = %u\n", deviceProp[dev].totalGlobalMem);
    printf("sharedMemPerBlock = %i\n", deviceProp[dev].sharedMemPerBlock);
    printf("regsPerBlock = %i\n", deviceProp[dev].regsPerBlock);
    printf("warpSize = %i\n", deviceProp[dev].warpSize);
    printf("memPitch = %i\n", deviceProp[dev].memPitch);
    printf("maxThreadsPerBlock = %i\n", deviceProp[dev].maxThreadsPerBlock);
    printf("maxThreadsDim = [%i, %i, %i]\n", deviceProp[dev].maxThreadsDim[0], deviceProp[dev].maxThreadsDim[1], deviceProp[dev].maxThreadsDim[2]);
    printf("maxGridSize = [%i, %i, %i]\n", deviceProp[dev].maxGridSize[0], deviceProp[dev].maxGridSize[1], deviceProp[dev].maxGridSize[2]);
    printf("totalConstMem = %i\n", deviceProp[dev].totalConstMem);
    printf("major . minor = %i . %i\n", deviceProp[dev].major, deviceProp[dev].minor);
    printf("clockRate = %i\n", deviceProp[dev].clockRate);
    printf("textureAlignment = %i\n", deviceProp[dev].textureAlignment);
    printf("deviceOverlap = %i\n", deviceProp[dev].deviceOverlap);
    printf("multiProcessorCount = %i\n", deviceProp[dev].multiProcessorCount);
    printf("computeMode = %i\n", deviceProp[dev].computeMode);
  )

#endif
}

void* CudaWrapper_AllocCudaData(unsigned nbytes)
{
  void* dev_data;
  CUDA_SAFE_CALL(hipMalloc((void**)&dev_data, nbytes));
  MYDBG(printf("# CUDA: allocated %u bytes on device at dev%p\n", nbytes, dev_data);)
  CudaWrapper_total_gpu_mem += nbytes;
  return dev_data;
}

void CudaWrapper_UploadCudaData(void* host_data, void* dev_data, unsigned nbytes)
{
  MYDBG(printf("# CUDA: uploading %u bytes to device at dev%p from %p\n", nbytes, dev_data, host_data);)
  hipDeviceSynchronize();
  my_times time1, time2;
  my_gettime(CLOCK_REALTIME, &time1);
  CUDA_SAFE_CALL(hipMemcpy(dev_data, host_data, nbytes, hipMemcpyHostToDevice));
  my_gettime(CLOCK_REALTIME, &time2);
  CudaWrapper_total_upload_time +=
    time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;
}

void CudaWrapper_UploadCudaDataAsync(void* host_data, void* dev_data, unsigned nbytes, int stream)
{
  MYDBG(printf("# CUDA: downloading %u bytes from device at dev%p\n", nbytes, dev_data);)
  hipMemcpyAsync(dev_data, host_data, nbytes, hipMemcpyHostToDevice, streams[stream]);
}

void CudaWrapper_DownloadCudaData(void* host_data, void* dev_data, unsigned nbytes)
{
  MYDBG(printf("# CUDA: downloading %u bytes from device at dev%p\n", nbytes, dev_data);)
  hipDeviceSynchronize();
  my_times time1, time2;
  my_gettime(CLOCK_REALTIME, &time1);
  CUDA_SAFE_CALL(hipMemcpy(host_data, dev_data, nbytes, hipMemcpyDeviceToHost));
  my_gettime(CLOCK_REALTIME, &time2);
  CudaWrapper_total_download_time +=
    time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;
}

void CudaWrapper_DownloadCudaDataAsync(void* host_data, void* dev_data, unsigned nbytes, int stream)
{
  MYDBG(printf("# CUDA: downloading %u bytes from device at dev%p\n", nbytes, dev_data);)
  hipMemcpyAsync(host_data, dev_data, nbytes, hipMemcpyDeviceToHost, streams[stream]);
}

void CudaWrapper_FreeCudaData(void* dev_data, unsigned nbytes)
{
  MYDBG(printf("# CUDA: freeing memory at dev%p with %i bytes (last adress: %p)\n", dev_data, nbytes, (char*)dev_data + nbytes);)
  CUDA_SAFE_CALL(hipFree(dev_data));
  CudaWrapper_total_gpu_mem -= nbytes;
}

void CudaWrapper_Memset(void* dev_data, int value, unsigned nbytes)
{
  MYDBG(printf("# CUDA: setting %u bytes to %i at dev%p\n", nbytes, value, dev_data);)
  CUDA_SAFE_CALL(hipMemset(dev_data, value, nbytes));
}

void CudaWrapper_CopyData(void* dev_dest, void* dev_source, unsigned nbytes)
{
  MYDBG(printf("# CUDA: copy %u bytes from dev%p to dev%p\n", nbytes, dev_source, dev_dest);)
  CUDA_SAFE_CALL(hipMemcpy(dev_dest, dev_source, nbytes, hipMemcpyDeviceToDevice));
}

void* CudaWrapper_AllocPinnedHostData(unsigned nbytes, bool mapped, bool writeCombined)
{
  void* host_data;
  int flags = 0;

  if(mapped) flags = flags | hipHostMallocMapped;

  if(writeCombined) flags = flags | hipHostMallocWriteCombined;

  CUDA_SAFE_CALL(hipHostAlloc((void**)&host_data, nbytes, flags));
  //	CUDA_SAFE_CALL( hipHostMalloc((void**)&host_data, nbytes) );
  MYDBG(printf("# CUDA: allocated %u bytes pinned memory on host at %p\n", nbytes, host_data);)
  return host_data;
}

void CudaWrapper_FreePinnedHostData(void* host_data)
{
  MYDBG(printf("# CUDA: freeing pinned host memory at %p \n", host_data);)

  if(host_data)
    CUDA_SAFE_CALL(hipHostFree(host_data));
}

void cuda_check_error(char* comment)
{
  printf("ERROR-CUDA %s %s\n", comment, hipGetErrorString(hipGetLastError()));
}

int CudaWrapper_CheckMemUsage()
{
  size_t free, total;
  hipMemGetInfo(&free, &total);
  return total - free; //possible with cuda 3.0 ???
  //return CudaWrapper_total_gpu_mem;
}

double CudaWrapper_CheckUploadTime(bool reset)
{
  if(reset) CudaWrapper_total_upload_time = 0.0;

  return CudaWrapper_total_upload_time;
}

double CudaWrapper_CheckDownloadTime(bool reset)
{
  if(reset) CudaWrapper_total_download_time = 0.0;

  return CudaWrapper_total_download_time;
}

double CudaWrapper_CheckCPUBufUploadTime(bool reset)
{
  if(reset) CudaWrapper_cpubuffer_upload_time = 0.0;

  return CudaWrapper_cpubuffer_upload_time;
}

double CudaWrapper_CheckCPUBufDownloadTime(bool reset)
{
  if(reset) CudaWrapper_cpubuffer_download_time = 0.0;

  return CudaWrapper_cpubuffer_download_time;
}

void CudaWrapper_AddCPUBufUploadTime(double dt)
{
  CudaWrapper_cpubuffer_upload_time += dt;
}

void CudaWrapper_AddCPUBufDownloadTime(double dt)
{
  CudaWrapper_cpubuffer_download_time += dt;
}

void CudaWrapper_Sync()
{
  hipDeviceSynchronize();
}

void CudaWrapper_SyncStream(int stream)
{
  hipStreamSynchronize(streams[stream]);
}

void CudaWrapper_AddStreams(int n)
{
  hipStream_t* new_streams = new hipStream_t[nstreams + n];

  for(int i = 0; i < nstreams; i++) new_streams[i] = streams[i];

  for(int i = nstreams; i < nstreams + n; i++) hipStreamCreate(&new_streams[i]);

  if(nstreams > 0)
    delete [] streams;

  streams = new_streams;
  nstreams += n;
}

void* CudaWrapper_returnStreams()
{
  return (void*) streams;
}

int CudaWrapper_returnNStreams()
{
  return nstreams;
}

