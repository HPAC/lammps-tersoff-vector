#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

extern __shared__ F_CFLOAT sharedmem[];


__global__ void Cuda_FixSetForceCuda_PostForce_Kernel(int groupbit, F_CFLOAT xvalue, F_CFLOAT yvalue, F_CFLOAT zvalue, int flagx, int flagy, int flagz)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  sharedmem[threadIdx.x] = 0;
  sharedmem[threadIdx.x + blockDim.x] = 0;
  sharedmem[threadIdx.x + 2 * blockDim.x] = 0;

  if(i < _nlocal)
    if(_mask[i] & groupbit) {
      sharedmem[threadIdx.x] = _f[i];
      sharedmem[threadIdx.x + blockDim.x] = _f[i + 1 * _nmax];
      sharedmem[threadIdx.x + 2 * blockDim.x] = _f[i + 2 * _nmax];

      if(flagx) _f[i] = xvalue;

      if(flagy) _f[i + 1 * _nmax] = yvalue;

      if(flagz) _f[i + 2 * _nmax] = zvalue;
    }


  reduceBlock(sharedmem);
  reduceBlock(&sharedmem[blockDim.x]);
  reduceBlock(&sharedmem[2 * blockDim.x]);
  F_CFLOAT* buffer = (F_CFLOAT*)_buffer;

  if(threadIdx.x == 0) {
    buffer[blockIdx.x * gridDim.y + blockIdx.y] = sharedmem[0];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + gridDim.x * gridDim.y] = sharedmem[blockDim.x];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + 2 * gridDim.x * gridDim.y] = sharedmem[2 * blockDim.x];
  }
}


__global__ void Cuda_FixSetForceCuda_Reduce_FOriginal(int n, F_CFLOAT* foriginal)
{
  int i = 0;
  sharedmem[threadIdx.x] = 0;
  F_CFLOAT myforig = 0.0;
  F_CFLOAT* buf = (F_CFLOAT*)_buffer;
  buf = &buf[blockIdx.x * n];

  while(i < n) {
    sharedmem[threadIdx.x] = 0;

    if(i + threadIdx.x < n)
      sharedmem[threadIdx.x] = buf[i + threadIdx.x];

    __syncthreads();
    reduceBlock(sharedmem);
    i += blockDim.x;

    if(threadIdx.x == 0)
      myforig += sharedmem[0];
  }

  if(threadIdx.x == 0)
    foriginal[blockIdx.x] = myforig;
}

