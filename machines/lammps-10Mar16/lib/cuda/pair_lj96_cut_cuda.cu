/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>

#define _lj1 MY_AP(coeff1)
#define _lj2 MY_AP(coeff2)
#define _lj3 MY_AP(coeff3)
#define _lj4 MY_AP(coeff4)

#include "pair_lj96_cut_cuda_cu.h"
#include "pair_lj96_cut_cuda_kernel_nc.cu"
#include <time.h>




void Cuda_PairLJ96CutCuda_Init(cuda_shared_data* sdata)
{
  Cuda_Pair_Init_AllStyles(sdata, 4, false, false);
}




void Cuda_PairLJ96CutCuda(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist, int eflag, int vflag, int eflag_atom, int vflag_atom)
{

  // initialize only on first call
  static  short init = 0;

  if(! init) {
    init = 1;
    Cuda_PairLJ96CutCuda_Init(sdata);
  }

  dim3 grid, threads;
  int sharedperproc;

  Cuda_Pair_PreKernel_AllStyles(sdata, sneighlist, eflag, vflag, grid, threads, sharedperproc, false, 256);

  hipStream_t* streams = (hipStream_t*) CudaWrapper_returnStreams();

  if(sdata->pair.use_block_per_atom)
    Pair_Kernel_BpA<PAIR_LJ96_CUT, COUL_NONE, DATA_NONE>
    <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x, streams[1]>>> (eflag, vflag, eflag_atom, vflag_atom);
  else
    Pair_Kernel_TpA<PAIR_LJ96_CUT, COUL_NONE, DATA_NONE>
    <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x, streams[1]>>> (eflag, vflag, eflag_atom, vflag_atom);

  Cuda_Pair_PostKernel_AllStyles(sdata, grid, sharedperproc, eflag, vflag);
}

#undef _lj1
#undef _lj2
#undef _lj3
#undef _lj4


