#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX compute_temp_partial_cuda
#include "cuda_shared.h"
#include "cuda_common.h"

#include "crm_cuda_utils.cu"

#include "compute_temp_partial_cuda_cu.h"
#include "compute_temp_partial_cuda_kernel.cu"

void Cuda_ComputeTempPartialCuda_UpdateBuffer(cuda_shared_data* sdata)
{
  int size = (unsigned)((sdata->atom.nlocal + 63) / 64.0) * 6 * sizeof(ENERGY_CFLOAT);

  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_ComputeTempPartialCuda Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
    CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)
  }

  hipMemcpyToSymbol(MY_AP(buffer), & sdata->buffer, sizeof(int*));
}

void Cuda_ComputeTempPartialCuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(mass)    , & sdata->atom.mass .dev_data, sizeof(V_CFLOAT*));

  if(sdata->atom.rmass_flag)
    hipMemcpyToSymbol(MY_AP(rmass)   , & sdata->atom.rmass.dev_data, sizeof(V_CFLOAT*));

  hipMemcpyToSymbol(MY_AP(rmass_flag)   , & sdata->atom.rmass_flag, sizeof(int));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(v)       , & sdata->atom.v    .dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(type)       , & sdata->atom.type    .dev_data, sizeof(int*));
}

void Cuda_ComputeTempPartialCuda_Init(cuda_shared_data* sdata)
{
  Cuda_ComputeTempPartialCuda_UpdateNmax(sdata);
}


void Cuda_ComputeTempPartialCuda_Vector(cuda_shared_data* sdata, int groupbit, ENERGY_CFLOAT* t, int xflag, int yflag, int zflag)
{
  //if(sdata->atom.update_nmax) //is most likely not called every timestep, therefore update of constants is necessary
  Cuda_ComputeTempPartialCuda_UpdateNmax(sdata);
  //if(sdata->atom.update_nlocal)
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  //if(sdata->buffer_new)
  Cuda_ComputeTempPartialCuda_UpdateBuffer(sdata);

  int3 layout = getgrid(sdata->atom.nlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    Cuda_ComputeTempPartialCuda_Vector_Kernel <<< grid, threads, threads.x* 6* sizeof(ENERGY_CFLOAT)>>> (groupbit, xflag, yflag, zflag);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_ComputeTempPartialCuda_Vector: compute_vector Kernel execution failed");

    int oldgrid = grid.x * grid.y;
    grid.x = 6;
    threads.x = 512;
    Cuda_ComputeTempPartialCuda_Reduce_Kernel <<< grid, threads, threads.x* sizeof(ENERGY_CFLOAT)>>> (oldgrid, t);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_ComputeTempPartialCuda_Vector: reduce_vector Kernel execution failed");
  }
}

void Cuda_ComputeTempPartialCuda_Scalar(cuda_shared_data* sdata, int groupbit, ENERGY_CFLOAT* t, int xflag, int yflag, int zflag)
{
  //if(sdata->atom.update_nmax) //is most likely not called every timestep, therefore update of constants is necessary
  Cuda_ComputeTempPartialCuda_UpdateNmax(sdata);
  //if(sdata->atom.update_nlocal)
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  //if(sdata->buffer_new)
  Cuda_ComputeTempPartialCuda_UpdateBuffer(sdata);
  MYDBG(printf("#CUDA ComputeTempPartialCuda_Scalar: %i\n", sdata->atom.nlocal);)
  int3 layout = getgrid(sdata->atom.nlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    CUT_CHECK_ERROR("Cuda_ComputeTempPartialCuda_Scalar: pre compute_scalar Kernel");
    Cuda_ComputeTempPartialCuda_Scalar_Kernel <<< grid, threads, threads.x* sizeof(ENERGY_CFLOAT)>>> (groupbit, xflag, yflag, zflag);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_ComputeTempPartialCuda_Scalar: compute_scalar Kernel execution failed");

    int oldgrid = grid.x * grid.y;
    grid.x = 1;
    threads.x = 512;
    Cuda_ComputeTempPartialCuda_Reduce_Kernel <<< grid, threads, threads.x* sizeof(ENERGY_CFLOAT)>>> (oldgrid, t);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_ComputeTempPartialCuda_Scalar: reduce_scalar Kernel execution failed");
  }
}

void Cuda_ComputeTempPartialCuda_RemoveBiasAll(cuda_shared_data* sdata, int groupbit, int xflag, int yflag, int zflag, void* vbiasall)
{
  //if(sdata->atom.update_nmax) //is most likely not called every timestep, therefore update of constants is necessary
  Cuda_ComputeTempPartialCuda_UpdateNmax(sdata);
  //if(sdata->atom.update_nlocal)
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  //if(sdata->buffer_new)
  Cuda_ComputeTempPartialCuda_UpdateBuffer(sdata);

  int3 layout = getgrid(sdata->atom.nlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    Cuda_ComputeTempPartialCuda_RemoveBiasAll_Kernel <<< grid, threads, 0>>> (groupbit, xflag, yflag, zflag, (V_CFLOAT*) vbiasall);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_ComputeTempPartialCuda_RemoveBiasAll: compute_vector Kernel execution failed");
  }
}

void Cuda_ComputeTempPartialCuda_RestoreBiasAll(cuda_shared_data* sdata, int groupbit, int xflag, int yflag, int zflag, void* vbiasall)
{
  //if(sdata->atom.update_nmax) //is most likely not called every timestep, therefore update of constants is necessary
  Cuda_ComputeTempPartialCuda_UpdateNmax(sdata);
  //if(sdata->atom.update_nlocal)
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  //if(sdata->buffer_new)
  Cuda_ComputeTempPartialCuda_UpdateBuffer(sdata);

  int3 layout = getgrid(sdata->atom.nlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    Cuda_ComputeTempPartialCuda_RestoreBiasAll_Kernel <<< grid, threads, 0>>> (groupbit, xflag, yflag, zflag, (V_CFLOAT*) vbiasall);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_ComputeTempPartialCuda_RemoveBiasAll: compute_vector Kernel execution failed");
  }
}
