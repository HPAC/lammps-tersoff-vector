/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>

#define _lj1 MY_AP(coeff1)
#define _lj2 MY_AP(coeff2)
#define _lj3 MY_AP(coeff3)
#define _lj4 MY_AP(coeff4)

#include "pair_lj_cut_experimental_cuda_cu.h"

#include <time.h>

void Cuda_PairLJCutExperimentalCuda_Init(cuda_shared_data* sdata)
{
  Cuda_Pair_Init_AllStyles(sdata, 4);
}

void Cuda_PairLJCutExperimentalCuda(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist, int eflag, int vflag, int eflag_atom, int vflag_atom)
{


  static  short init = 0;

  if(! init) {
    init = 1;
    Cuda_PairLJCutExperimentalCuda_Init(sdata);
  }

  dim3 grid, threads;
  int sharedperproc;

  //int maxthreads=192*sizeof(double)/sizeof(F_CFLOAT);
  //if(CUDA_ARCH==20) maxthreads*=2;
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(Pair_Kernel_TpA_opt<PAIR_LJ_CUT),COUL_NONE,DATA_NONE>,hipFuncCachePreferL1);
  Cuda_Pair_PreKernel_AllStyles(sdata, sneighlist, eflag, vflag, grid, threads, sharedperproc, false, 192);

  if(sharedperproc == 0) sharedperproc++;

  //printf("comm_phase: %i\n",sdata->comm.comm_phase);

  hipStream_t* streams = (hipStream_t*) CudaWrapper_returnStreams();

  if(sdata->pair.use_block_per_atom)
    Pair_Kernel_BpA<PAIR_LJ_CUT, COUL_NONE, DATA_NONE>
    <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x, streams[1]>>> (eflag, vflag, eflag_atom, vflag_atom);
  else
    Pair_Kernel_TpA_opt<PAIR_LJ_CUT, COUL_NONE, DATA_NONE>
    <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x, streams[1]>>> (eflag, vflag, eflag_atom, vflag_atom, sdata->comm.comm_phase);
  Cuda_Pair_PostKernel_AllStyles(sdata, grid, sharedperproc, eflag, vflag);
}


#undef _lj1
#undef _lj2
#undef _lj3
#undef _lj4
