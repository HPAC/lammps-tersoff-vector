#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX comm_cuda
#include "cuda_shared.h"
#include "cuda_common.h"

#include "crm_cuda_utils.cu"

#include "comm_cuda_cu.h"
#include "comm_cuda_kernel.cu"
#include <ctime>

void Cuda_CommCuda_UpdateBuffer(cuda_shared_data* sdata, int n)
{
  int size = n * 3 * sizeof(X_CFLOAT);

  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_ComputeTempCuda Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
    CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)
  }

  hipMemcpyToSymbol(MY_AP(buffer), & sdata->buffer, sizeof(int*));
}


void Cuda_CommCuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(x)       , & sdata->atom.x    .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(v)       , & sdata->atom.v    .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(f)       , & sdata->atom.f    .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(type)    , & sdata->atom.type .dev_data, sizeof(int*));
}


void Cuda_CommCuda_Init(cuda_shared_data* sdata)
{
  Cuda_CommCuda_UpdateNmax(sdata);
  int ntypesp = sdata->atom.ntypes + 1;
  hipMemcpyToSymbol(MY_AP(cuda_ntypes)   , &ntypesp, sizeof(int));
  hipMemcpyToSymbol(MY_AP(prd)   , sdata->domain.prd, 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(flag)  , &sdata->flag, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(debugdata)  , &sdata->debugdata, sizeof(int*));
}

int Cuda_CommCuda_PackComm(cuda_shared_data* sdata, int n, int iswap, void* buf_send, int* pbc, int pbc_flag)
{

  my_times time1, time2;

  if(sdata->atom.update_nmax)
    Cuda_CommCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int size = n * 3 * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_CommCuda_UpdateBuffer(sdata, n);

  X_CFLOAT dx = 0.0;
  X_CFLOAT dy = 0.0;
  X_CFLOAT dz = 0.0;

  if(pbc_flag != 0) {
    if(sdata->domain.triclinic == 0) {
      dx = pbc[0] * sdata->domain.prd[0];
      dy = pbc[1] * sdata->domain.prd[1];
      dz = pbc[2] * sdata->domain.prd[2];
    } else {
      dx = pbc[0] * sdata->domain.prd[0] + pbc[5] * sdata->domain.xy + pbc[4] * sdata->domain.xz;
      dy = pbc[1] * sdata->domain.prd[1] + pbc[3] * sdata->domain.yz;
      dz = pbc[2] * sdata->domain.prd[2];
    }
  }

  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    hipMemset(sdata->flag, 0, sizeof(int));

    my_gettime(CLOCK_REALTIME, &time1);

    void* buf = sdata->overlap_comm ? sdata->comm.buf_send_dev[iswap] : sdata->buffer;
    Cuda_CommCuda_PackComm_Kernel <<< grid, threads, 0>>>((int*) sdata->comm.sendlist.dev_data, n
        , sdata->comm.maxlistlength, iswap, dx, dy, dz, buf);
    hipDeviceSynchronize();

    my_gettime(CLOCK_REALTIME, &time2);
    sdata->cuda_timings.comm_forward_kernel_pack +=
      time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

    CUT_CHECK_ERROR("Cuda_CommCuda_PackComm: Kernel execution failed");

    if(not sdata->overlap_comm)
      hipMemcpy(buf_send, sdata->buffer, n * 3 * sizeof(X_CFLOAT), hipMemcpyDeviceToHost);
    //hipMemcpy(buf_send, sdata->comm.buf_send_dev[iswap], n*3*sizeof(X_CFLOAT), hipMemcpyDeviceToHost);

    my_gettime(CLOCK_REALTIME, &time1);
    sdata->cuda_timings.comm_forward_download +=
      time1.tv_sec - time2.tv_sec + 1.0 * (time1.tv_nsec - time2.tv_nsec) / 1000000000;

    int aflag;
    hipMemcpy(&aflag, sdata->flag, sizeof(int), hipMemcpyDeviceToHost);
    if(aflag != 0) printf("aflag PackComm: %i\n", aflag);
    CUT_CHECK_ERROR("Cuda_CommCuda_PackComm: Kernel execution failed");

  }

  return 3 * n;
}

int Cuda_CommCuda_PackCommVel(cuda_shared_data* sdata, int n, int iswap, void* buf_send, int* pbc, int pbc_flag)
{

  my_times time1, time2;

  if(sdata->atom.update_nmax)
    Cuda_CommCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int size = n * 6 * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_CommCuda_UpdateBuffer(sdata, n);

  X_CFLOAT dx = 0.0;
  X_CFLOAT dy = 0.0;
  X_CFLOAT dz = 0.0;

  if(pbc_flag != 0) {
    if(sdata->domain.triclinic == 0) {
      dx = pbc[0] * sdata->domain.prd[0];
      dy = pbc[1] * sdata->domain.prd[1];
      dz = pbc[2] * sdata->domain.prd[2];
    } else {
      dx = pbc[0] * sdata->domain.prd[0] + pbc[5] * sdata->domain.xy + pbc[4] * sdata->domain.xz;
      dy = pbc[1] * sdata->domain.prd[1] + pbc[3] * sdata->domain.yz;
      dz = pbc[2] * sdata->domain.prd[2];
    }
  }

  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    hipMemset(sdata->flag, 0, sizeof(int));

    my_gettime(CLOCK_REALTIME, &time1);

    void* buf = sdata->overlap_comm ? sdata->comm.buf_send_dev[iswap] : sdata->buffer;
    Cuda_CommCuda_PackComm_Kernel <<< grid, threads, 0>>>((int*) sdata->comm.sendlist.dev_data, n
        , sdata->comm.maxlistlength, iswap, dx, dy, dz, buf);
    hipDeviceSynchronize();

    my_gettime(CLOCK_REALTIME, &time2);
    sdata->cuda_timings.comm_forward_kernel_pack +=
      time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

    CUT_CHECK_ERROR("Cuda_CommCuda_PackComm: Kernel execution failed");

    if(not sdata->overlap_comm)
      hipMemcpy(buf_send, sdata->buffer, n * 6 * sizeof(X_CFLOAT), hipMemcpyDeviceToHost);
    //hipMemcpy(buf_send, sdata->comm.buf_send_dev[iswap], n*3*sizeof(X_CFLOAT), hipMemcpyDeviceToHost);

    my_gettime(CLOCK_REALTIME, &time1);
    sdata->cuda_timings.comm_forward_download +=
      time1.tv_sec - time2.tv_sec + 1.0 * (time1.tv_nsec - time2.tv_nsec) / 1000000000;

    int aflag;
    hipMemcpy(&aflag, sdata->flag, sizeof(int), hipMemcpyDeviceToHost);
    if(aflag != 0) printf("aflag PackComm: %i\n", aflag);
    CUT_CHECK_ERROR("Cuda_CommCuda_PackComm: Kernel execution failed");

  }

  return 6 * n;
}

int Cuda_CommCuda_PackComm_Self(cuda_shared_data* sdata, int n, int iswap, int first, int* pbc, int pbc_flag)
{
  MYDBG(printf(" # CUDA: CommCuda_PackComm_Self\n");)
  my_times time1, time2;

  if(sdata->atom.update_nmax)
    Cuda_CommCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int size = n * 3 * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_CommCuda_UpdateBuffer(sdata, n);

  static int count = -1;
  count++;
  X_CFLOAT dx = 0.0;
  X_CFLOAT dy = 0.0;
  X_CFLOAT dz = 0.0;

  if(pbc_flag != 0) {
    if(sdata->domain.triclinic == 0) {
      dx = pbc[0] * sdata->domain.prd[0];
      dy = pbc[1] * sdata->domain.prd[1];
      dz = pbc[2] * sdata->domain.prd[2];
    } else {
      dx = pbc[0] * sdata->domain.prd[0] + pbc[5] * sdata->domain.xy + pbc[4] * sdata->domain.xz;
      dy = pbc[1] * sdata->domain.prd[1] + pbc[3] * sdata->domain.yz;
      dz = pbc[2] * sdata->domain.prd[2];
    }
  }



  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {

    my_gettime(CLOCK_REALTIME, &time1);

    Cuda_CommCuda_PackComm_Self_Kernel <<< grid, threads, 0>>>((int*) sdata->comm.sendlist.dev_data, n, sdata->comm.maxlistlength, iswap, dx, dy, dz, first);
    hipDeviceSynchronize();

    my_gettime(CLOCK_REALTIME, &time2);
    sdata->cuda_timings.comm_forward_kernel_self +=
      time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

    CUT_CHECK_ERROR("Cuda_CommCuda_PackComm_Self: Kernel execution failed");
  }

  return 3 * n;
}

int Cuda_CommCuda_PackCommVel_Self(cuda_shared_data* sdata, int n, int iswap, int first, int* pbc, int pbc_flag)
{
  MYDBG(printf(" # CUDA: CommCuda_PackComm_Self\n");)
  my_times time1, time2;

  if(sdata->atom.update_nmax)
    Cuda_CommCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int size = n * 6 * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_CommCuda_UpdateBuffer(sdata, n);

  static int count = -1;
  count++;
  X_CFLOAT dx = 0.0;
  X_CFLOAT dy = 0.0;
  X_CFLOAT dz = 0.0;

  if(pbc_flag != 0) {
    if(sdata->domain.triclinic == 0) {
      dx = pbc[0] * sdata->domain.prd[0];
      dy = pbc[1] * sdata->domain.prd[1];
      dz = pbc[2] * sdata->domain.prd[2];
    } else {
      dx = pbc[0] * sdata->domain.prd[0] + pbc[5] * sdata->domain.xy + pbc[4] * sdata->domain.xz;
      dy = pbc[1] * sdata->domain.prd[1] + pbc[3] * sdata->domain.yz;
      dz = pbc[2] * sdata->domain.prd[2];
    }
  }



  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {

    my_gettime(CLOCK_REALTIME, &time1);

    Cuda_CommCuda_PackComm_Self_Kernel <<< grid, threads, 0>>>((int*) sdata->comm.sendlist.dev_data, n, sdata->comm.maxlistlength, iswap, dx, dy, dz, first);
    hipDeviceSynchronize();

    my_gettime(CLOCK_REALTIME, &time2);
    sdata->cuda_timings.comm_forward_kernel_self +=
      time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

    CUT_CHECK_ERROR("Cuda_CommCuda_PackComm_Self: Kernel execution failed");
  }

  return 6 * n;
}

void Cuda_CommCuda_UnpackComm(cuda_shared_data* sdata, int n, int first, void* buf_recv, int iswap)
{
  my_times time1, time2;

  if(sdata->atom.update_nmax)
    Cuda_CommCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int size = n * 3 * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_CommCuda_UpdateBuffer(sdata, n);

  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    my_gettime(CLOCK_REALTIME, &time1);

    if(not sdata->overlap_comm || iswap < 0)
      hipMemcpy(sdata->buffer, (void*)buf_recv, n * 3 * sizeof(X_CFLOAT), hipMemcpyHostToDevice);

    my_gettime(CLOCK_REALTIME, &time2);
    sdata->cuda_timings.comm_forward_upload +=
      time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;
    void* buf = (sdata->overlap_comm && iswap >= 0) ? sdata->comm.buf_recv_dev[iswap] : sdata->buffer;
    Cuda_CommCuda_UnpackComm_Kernel <<< grid, threads, 0>>>(n, first, buf);
    hipDeviceSynchronize();

    my_gettime(CLOCK_REALTIME, &time1);
    sdata->cuda_timings.comm_forward_kernel_unpack +=
      time1.tv_sec - time2.tv_sec + 1.0 * (time1.tv_nsec - time2.tv_nsec) / 1000000000;

    CUT_CHECK_ERROR("Cuda_CommCuda_UnpackComm: Kernel execution failed");

  }
}

void Cuda_CommCuda_UnpackCommVel(cuda_shared_data* sdata, int n, int first, void* buf_recv, int iswap)
{
  my_times time1, time2;

  if(sdata->atom.update_nmax)
    Cuda_CommCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int size = n * 6 * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_CommCuda_UpdateBuffer(sdata, n);

  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    my_gettime(CLOCK_REALTIME, &time1);

    if(not sdata->overlap_comm || iswap < 0)
      hipMemcpy(sdata->buffer, (void*)buf_recv, n * 6 * sizeof(X_CFLOAT), hipMemcpyHostToDevice);

    my_gettime(CLOCK_REALTIME, &time2);
    sdata->cuda_timings.comm_forward_upload +=
      time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;
    void* buf = (sdata->overlap_comm && iswap >= 0) ? sdata->comm.buf_recv_dev[iswap] : sdata->buffer;
    Cuda_CommCuda_UnpackComm_Kernel <<< grid, threads, 0>>>(n, first, buf);
    hipDeviceSynchronize();

    my_gettime(CLOCK_REALTIME, &time1);
    sdata->cuda_timings.comm_forward_kernel_unpack +=
      time1.tv_sec - time2.tv_sec + 1.0 * (time1.tv_nsec - time2.tv_nsec) / 1000000000;

    CUT_CHECK_ERROR("Cuda_CommCuda_UnpackComm: Kernel execution failed");

  }
}

int Cuda_CommCuda_PackReverse(cuda_shared_data* sdata, int n, int first, void* buf_send)
{
  if(sdata->atom.update_nmax)
    Cuda_CommCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int size = n * 3 * sizeof(F_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_CommCuda_UpdateBuffer(sdata, n);


  F_CFLOAT* buf = (F_CFLOAT*)buf_send;
  F_CFLOAT* f_dev = (F_CFLOAT*)sdata->atom.f.dev_data;
  f_dev += first;
  hipMemcpy(buf, f_dev, n * sizeof(F_CFLOAT), hipMemcpyDeviceToHost);
  buf += n;
  f_dev += sdata->atom.nmax;
  hipMemcpy(buf, f_dev, n * sizeof(F_CFLOAT), hipMemcpyDeviceToHost);
  buf += n;
  f_dev += sdata->atom.nmax;
  hipMemcpy(buf, f_dev, n * sizeof(F_CFLOAT), hipMemcpyDeviceToHost);
  return 	n * 3;
}


void Cuda_CommCuda_UnpackReverse(cuda_shared_data* sdata, int n, int iswap, void* buf_recv)
{
  if(sdata->atom.update_nmax)
    Cuda_CommCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int size = n * 3 * sizeof(F_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_CommCuda_UpdateBuffer(sdata, n);


  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    hipMemcpy(sdata->buffer, buf_recv, size, hipMemcpyHostToDevice);
    Cuda_CommCuda_UnpackReverse_Kernel <<< grid, threads, 0>>>((int*) sdata->comm.sendlist.dev_data, n, sdata->comm.maxlistlength, iswap);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_CommCuda_UnpackReverse: Kernel execution failed");
  }
}

void Cuda_CommCuda_UnpackReverse_Self(cuda_shared_data* sdata, int n, int iswap, int first)
{
  if(sdata->atom.update_nmax)
    Cuda_CommCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int size = n * 3 * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_CommCuda_UpdateBuffer(sdata, n);

  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    Cuda_CommCuda_UnpackReverse_Self_Kernel <<< grid, threads, 0>>>((int*) sdata->comm.sendlist.dev_data, n, sdata->comm.maxlistlength, iswap, first);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_CommCuda_PackReverse_Self: Kernel execution failed");

  }
}


int Cuda_CommCuda_BuildSendlist(cuda_shared_data* sdata, int bordergroup, int ineed, int style, int atom_nfirst, int nfirst, int nlast, int dim, int iswap)
{
  MYDBG(printf(" # CUDA: CommCuda_BuildSendlist\n");)
  my_times time1, time2;

  if(sdata->atom.update_nmax)
    Cuda_CommCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  if(sdata->buffer_new or (80 > sdata->buffersize))
    Cuda_CommCuda_UpdateBuffer(sdata, 10);

  int n;

  if(!bordergroup || ineed >= 2)
    n = nlast - nfirst + 1;
  else {
    n = atom_nfirst;

    if(nlast - sdata->atom.nlocal + 1 > n) n = nlast - sdata->atom.nlocal + 1;
  }

  int3 layout = getgrid(n, 0, 512, true);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x + 1, layout.y, 1);


  hipMemset((int*)(sdata->buffer), 0, sizeof(int));

  my_gettime(CLOCK_REALTIME, &time1);

  if(style == 1)
    Cuda_CommCuda_BuildSendlist_Single <<< grid, threads, (threads.x + 1)*sizeof(int) >>> (bordergroup, ineed, atom_nfirst, nfirst, nlast, dim, iswap, (X_CFLOAT*) sdata->comm.slablo.dev_data, (X_CFLOAT*) sdata->comm.slabhi.dev_data, (int*) sdata->comm.sendlist.dev_data, sdata->comm.maxlistlength);
  else
    Cuda_CommCuda_BuildSendlist_Multi <<< grid, threads, (threads.x + 1)*sizeof(int) >>> (bordergroup, ineed, atom_nfirst, nfirst, nlast, dim, iswap, (X_CFLOAT*) sdata->comm.multilo.dev_data, (X_CFLOAT*) sdata->comm.multihi.dev_data, (int*) sdata->comm.sendlist.dev_data, sdata->comm.maxlistlength);

  hipDeviceSynchronize();
  my_gettime(CLOCK_REALTIME, &time2);
  sdata->cuda_timings.comm_border_kernel_buildlist +=
    time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

  CUT_CHECK_ERROR("Cuda_CommCuda_BuildSendlist: Kernel execution failed");
  int nsend;
  hipMemcpy(&nsend, sdata->buffer, sizeof(int), hipMemcpyDeviceToHost);
  return nsend;


}

