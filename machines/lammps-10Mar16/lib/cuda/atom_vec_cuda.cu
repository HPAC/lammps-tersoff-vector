#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX atom_vec_cuda
#include "cuda_shared.h"
#include "cuda_common.h"
#include "cuda_wrapper_cu.h"
#include "crm_cuda_utils.cu"

#include "atom_vec_cuda_kernel.cu"

int AtomVecCuda_CountDataItems(unsigned int data_mask)
{
  int n = 0;

  if(data_mask & X_MASK) n += 3;

  if(data_mask & V_MASK) n += 3;

  if(data_mask & F_MASK) n += 3;

  if(data_mask & TAG_MASK) n++;

  if(data_mask & TYPE_MASK) n++;

  if(data_mask & MASK_MASK) n++;

  if(data_mask & IMAGE_MASK) n++;

  if(data_mask & Q_MASK) n++;

  if(data_mask & MOLECULE_MASK) n++;

  if(data_mask & RMASS_MASK) n++;

  if(data_mask & RADIUS_MASK) n++;

  if(data_mask & DENSITY_MASK) n++;

  if(data_mask & OMEGA_MASK) n += 3;

  if(data_mask & TORQUE_MASK) n++;

  //if(data_mask & NSPECIAL_MASK) n+=3;
  return n;
}

void Cuda_AtomVecCuda_UpdateBuffer(cuda_shared_data* sdata, int size)
{
  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_AtomVecCuda Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
    CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)
  }

  hipMemcpyToSymbol(MY_AP(buffer), & sdata->buffer, sizeof(int*));
}

template <const unsigned int data_mask>
void Cuda_AtomVecCuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(x)       , & sdata->atom.x    .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(v)       , & sdata->atom.v    .dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(f)       , & sdata->atom.f    .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(tag)     , & sdata->atom.tag  .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(type)    , & sdata->atom.type .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(image)   , & sdata->atom.image.dev_data, sizeof(int*));

  if(data_mask & Q_MASK) hipMemcpyToSymbol(MY_AP(q)       , & sdata->atom.q    .dev_data, sizeof(F_CFLOAT*));

  if(data_mask & MOLECULE_MASK) hipMemcpyToSymbol(MY_AP(molecule)   , & sdata->atom.molecule.dev_data, sizeof(int*));

  if(data_mask & RADIUS_MASK) hipMemcpyToSymbol(MY_AP(radius)   , & sdata->atom.radius.dev_data, sizeof(int*));

  if(data_mask & DENSITY_MASK) hipMemcpyToSymbol(MY_AP(density)   , & sdata->atom.density.dev_data, sizeof(int*));

  if(data_mask & RMASS_MASK) hipMemcpyToSymbol(MY_AP(rmass)   , & sdata->atom.rmass.dev_data, sizeof(int*));

  if(data_mask & OMEGA_MASK) hipMemcpyToSymbol(MY_AP(omega)   , & sdata->atom.omega.dev_data, sizeof(int*));

  //if(data_mask & NSPECIAL_MASK) hipMemcpyToSymbol(MY_AP(nspecial)   , & sdata->atom.nspecial.dev_data, sizeof(int*) );
  hipMemcpyToSymbol(MY_AP(flag)    , & sdata->flag, sizeof(int*));
}

template <const unsigned int data_mask>
void Cuda_AtomVecCuda_Init(cuda_shared_data* sdata)
{
  MYDBG(printf("# CUDA: Cuda_AtomVecCuda_Init ... start\n");)

  if(sdata->atom.update_nmax)
    Cuda_AtomVecCuda_UpdateNmax<data_mask>(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  MYDBG(printf("# CUDA: Cuda_AtomVecCuda_Init ... post Nmax\n");)
  hipMemcpyToSymbol(MY_AP(prd)   , sdata->domain.prd, 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(sublo)   , & sdata->domain.sublo, 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(subhi)   , & sdata->domain.subhi, 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(flag)   , & sdata->flag, sizeof(int*));
  hipDeviceSynchronize();
  MYDBG(printf("# CUDA: Cuda_AtomVecCuda_Init ... end\n");)
}


template <const unsigned int data_mask>
int Cuda_AtomVecCuda_PackComm(cuda_shared_data* sdata, int n, int iswap, void* buf_send, int* pbc, int pbc_flag)
{

  my_times time1, time2;

  if(sdata->atom.update_nmax)
    Cuda_AtomVecCuda_UpdateNmax<data_mask>(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int n_data_items = AtomVecCuda_CountDataItems(data_mask);
  int size = (n * n_data_items) * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_AtomVecCuda_UpdateBuffer(sdata, size);

  X_CFLOAT dx = 0.0;
  X_CFLOAT dy = 0.0;
  X_CFLOAT dz = 0.0;

  if(pbc_flag != 0) {
    if(sdata->domain.triclinic == 0) {
      dx = pbc[0] * sdata->domain.prd[0];
      dy = pbc[1] * sdata->domain.prd[1];
      dz = pbc[2] * sdata->domain.prd[2];
    } else {
      dx = pbc[0] * sdata->domain.prd[0] + pbc[5] * sdata->domain.xy + pbc[4] * sdata->domain.xz;
      dy = pbc[1] * sdata->domain.prd[1] + pbc[3] * sdata->domain.yz;
      dz = pbc[2] * sdata->domain.prd[2];
    }
  }

  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    hipMemset(sdata->flag, 0, sizeof(int));

    my_gettime(CLOCK_REALTIME, &time1);

    void* buf = sdata->overlap_comm ? sdata->comm.buf_send_dev[iswap] : sdata->buffer;
    Cuda_AtomVecCuda_PackComm_Kernel<data_mask> <<< grid, threads, 0>>>((int*) sdata->comm.sendlist.dev_data, n
        , sdata->comm.maxlistlength, iswap, dx, dy, dz, buf);
    hipDeviceSynchronize();

    my_gettime(CLOCK_REALTIME, &time2);
    sdata->cuda_timings.comm_forward_kernel_pack +=
      time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

    CUT_CHECK_ERROR("Cuda_AtomVecCuda_PackComm: Kernel execution failed");

    if(not sdata->overlap_comm)
      hipMemcpy(buf_send, sdata->buffer, n* n_data_items* sizeof(X_CFLOAT), hipMemcpyDeviceToHost);
    //hipMemcpy(buf_send, sdata->comm.buf_send_dev[iswap], n*3*sizeof(X_CFLOAT), hipMemcpyDeviceToHost);

    my_gettime(CLOCK_REALTIME, &time1);
    sdata->cuda_timings.comm_forward_download +=
      time1.tv_sec - time2.tv_sec + 1.0 * (time1.tv_nsec - time2.tv_nsec) / 1000000000;

    int aflag;
    hipMemcpy(&aflag, sdata->flag, sizeof(int), hipMemcpyDeviceToHost);
    if(aflag != 0) printf("aflag PackComm: %i\n", aflag);
    CUT_CHECK_ERROR("Cuda_AtomVecCuda_PackComm: Kernel execution failed");

  }

  return n_data_items * n;
}


template <const unsigned int data_mask>
int Cuda_AtomVecCuda_PackComm_Self(cuda_shared_data* sdata, int n, int iswap, int first, int* pbc, int pbc_flag)
{
  MYDBG(printf(" # CUDA: AtomVecCuda_PackComm_Self\n");)
  my_times time1, time2;

  if(sdata->atom.update_nmax)
    Cuda_AtomVecCuda_UpdateNmax<data_mask>(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int n_data_items = AtomVecCuda_CountDataItems(data_mask);
  int size = (n * n_data_items) * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_AtomVecCuda_UpdateBuffer(sdata, size);

  static int count = -1;
  count++;
  X_CFLOAT dx = 0.0;
  X_CFLOAT dy = 0.0;
  X_CFLOAT dz = 0.0;

  if(pbc_flag != 0) {
    if(sdata->domain.triclinic == 0) {
      dx = pbc[0] * sdata->domain.prd[0];
      dy = pbc[1] * sdata->domain.prd[1];
      dz = pbc[2] * sdata->domain.prd[2];
    } else {
      dx = pbc[0] * sdata->domain.prd[0] + pbc[5] * sdata->domain.xy + pbc[4] * sdata->domain.xz;
      dy = pbc[1] * sdata->domain.prd[1] + pbc[3] * sdata->domain.yz;
      dz = pbc[2] * sdata->domain.prd[2];
    }
  }



  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {

    my_gettime(CLOCK_REALTIME, &time1);
    CUT_CHECK_ERROR("Cuda_AtomVecCuda_PackComm_Self:Pre Kernel execution failed");

    Cuda_AtomVecCuda_PackComm_Self_Kernel<data_mask> <<< grid, threads, 0>>>((int*) sdata->comm.sendlist.dev_data, n, sdata->comm.maxlistlength, iswap, dx, dy, dz, first);
    hipDeviceSynchronize();

    my_gettime(CLOCK_REALTIME, &time2);
    sdata->cuda_timings.comm_forward_kernel_self +=
      time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

    CUT_CHECK_ERROR("Cuda_AtomVecCuda_PackComm_Self: Kernel execution failed");
  }

  return n_data_items * n;
}


template <const unsigned int data_mask>
void Cuda_AtomVecCuda_UnpackComm(cuda_shared_data* sdata, int n, int first, void* buf_recv, int iswap)
{
  my_times time1, time2;

  if(sdata->atom.update_nmax)
    Cuda_AtomVecCuda_UpdateNmax<data_mask>(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int n_data_items = AtomVecCuda_CountDataItems(data_mask);
  int size = (n * n_data_items) * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_AtomVecCuda_UpdateBuffer(sdata, size);

  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    my_gettime(CLOCK_REALTIME, &time1);

    if(not sdata->overlap_comm || iswap < 0)
      hipMemcpy(sdata->buffer, (void*)buf_recv, n_data_items * n * sizeof(X_CFLOAT), hipMemcpyHostToDevice);

    my_gettime(CLOCK_REALTIME, &time2);
    sdata->cuda_timings.comm_forward_upload +=
      time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;
    void* buf = (sdata->overlap_comm && iswap >= 0) ? sdata->comm.buf_recv_dev[iswap] : sdata->buffer;
    Cuda_AtomVecCuda_UnpackComm_Kernel<data_mask> <<< grid, threads, 0>>>(n, first, buf);
    hipDeviceSynchronize();

    my_gettime(CLOCK_REALTIME, &time1);
    sdata->cuda_timings.comm_forward_kernel_unpack +=
      time1.tv_sec - time2.tv_sec + 1.0 * (time1.tv_nsec - time2.tv_nsec) / 1000000000;

    CUT_CHECK_ERROR("Cuda_AtomVecCuda_UnpackComm: Kernel execution failed");

  }
}

template <const unsigned int data_mask>
int Cuda_AtomVecCuda_PackExchangeList(cuda_shared_data* sdata, int n, int dim, void* buf_send)
{
  MYDBG(printf("# CUDA: Cuda_AtomVecCuda_PackExchangeList ... start dim %i \n", dim);)
  CUT_CHECK_ERROR("Cuda_AtomVecCuda_PackExchangeList: pre Kernel execution failed");
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  Cuda_AtomVecCuda_Init<data_mask>(sdata);
  int size = n * sizeof(double);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_AtomVecCuda_UpdateBuffer(sdata, size);

  hipMemset((int*)(sdata->buffer), 0, sizeof(int));

  int3 layout = getgrid(sdata->atom.nlocal, sizeof(int), 256, true);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  my_times time1, time2;
  my_gettime(CLOCK_REALTIME, &time1);

  Cuda_AtomVecCuda_PackExchangeList_Kernel <<< grid, threads, (threads.x + 1)*sizeof(int) >>> (n - 1, dim);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_AtomVecCuda_PackExchangeList: Kernel execution failed");

  my_gettime(CLOCK_REALTIME, &time2);
  sdata->cuda_timings.comm_exchange_kernel_pack +=
    time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

  hipMemcpy(buf_send, sdata->buffer, sizeof(double), hipMemcpyDeviceToHost);
  int return_value = ((int*) buf_send)[0];

  if(n > 1 + return_value)
    hipMemcpy(buf_send, sdata->buffer, (1 + return_value)*sizeof(double), hipMemcpyDeviceToHost);

  CUT_CHECK_ERROR("Cuda_AtomVecCuda_PackExchangeList: return copy failed");

  my_gettime(CLOCK_REALTIME, &time1);
  sdata->cuda_timings.comm_exchange_download +=
    time1.tv_sec - time2.tv_sec + 1.0 * (time1.tv_nsec - time2.tv_nsec) / 1000000000;

  MYDBG(printf("# CUDA: Cuda_AtomVecCuda_PackExchangeList ... done\n");)
  return return_value;
}

template <const unsigned int data_mask>
int Cuda_AtomVecCuda_PackExchange(cuda_shared_data* sdata, int nsend, void* buf_send, void* copylist)
{
  MYDBG(printf("# CUDA: Cuda_AtomVecCuda_PackExchange ... start \n");)

  if(sdata->atom.update_nmax)
    Cuda_AtomVecCuda_UpdateNmax<data_mask>(sdata);

  //if(sdata->atom.update_nlocal)
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int n_data_items = AtomVecCuda_CountDataItems(data_mask) + 1;
  int size = (nsend * n_data_items + 1) * sizeof(double);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_AtomVecCuda_UpdateBuffer(sdata, size);

  hipMemset((int*)(sdata->buffer), 0, sizeof(int));

  int3 layout = getgrid(nsend, 0);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  my_times time1, time2;
  my_gettime(CLOCK_REALTIME, &time1);

  Cuda_AtomVecCuda_PackExchange_Kernel<data_mask> <<< grid, threads, 0>>>(nsend, (int*) copylist);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_AtomVecCuda_PackExchange: Kernel execution failed");

  my_gettime(CLOCK_REALTIME, &time2);
  sdata->cuda_timings.comm_exchange_kernel_pack +=
    time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

  hipMemcpy(buf_send, sdata->buffer, size, hipMemcpyDeviceToHost);

  my_gettime(CLOCK_REALTIME, &time1);
  sdata->cuda_timings.comm_exchange_download +=
    time1.tv_sec - time2.tv_sec + 1.0 * (time1.tv_nsec - time2.tv_nsec) / 1000000000;

  MYDBG(printf("# CUDA: Cuda_AtomVecCuda_PackExchange ... done\n");)
  return nsend * n_data_items + 1;
}


template <const unsigned int data_mask>
int Cuda_AtomVecCuda_UnpackExchange(cuda_shared_data* sdata, int nsend, void* buf_send, void* copylist)
{
  Cuda_AtomVecCuda_UpdateNmax<data_mask>(sdata);
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  int n_data_items = AtomVecCuda_CountDataItems(data_mask) + 1;

  int size = (nsend * n_data_items + 1) * sizeof(double);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_AtomVecCuda_UpdateBuffer(sdata, size);

  hipMemcpyToSymbol(MY_AP(flag)   , & sdata->flag, sizeof(int*));

  hipMemset((int*)(sdata->flag), 0, sizeof(int));

  if(nsend) {
    int3 layout = getgrid(nsend, 0);
    dim3 threads(layout.z, 1, 1);
    dim3 grid(layout.x, layout.y, 1);

    if(sdata->atom.nlocal > 0) {
      my_times time1, time2;
      my_gettime(CLOCK_REALTIME, &time1);

      hipMemcpy(sdata->buffer, buf_send , size, hipMemcpyHostToDevice);

      my_gettime(CLOCK_REALTIME, &time2);
      sdata->cuda_timings.comm_exchange_upload +=
        time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

      Cuda_AtomVecCuda_UnpackExchange_Kernel<data_mask> <<< grid, threads, 0>>>(sdata->exchange_dim, nsend, (int*) copylist);
      hipDeviceSynchronize();

      my_gettime(CLOCK_REALTIME, &time1);
      sdata->cuda_timings.comm_exchange_kernel_unpack +=
        time1.tv_sec - time2.tv_sec + 1.0 * (time1.tv_nsec - time2.tv_nsec) / 1000000000;

      CUT_CHECK_ERROR("Cuda_AtomVecCuda_UnpackExchange: Kernel execution failed");
    }
  }

  int naccept;
  hipMemcpy((void*)&naccept, sdata->flag, sizeof(int), hipMemcpyDeviceToHost);

  return naccept;
}

template <const unsigned int data_mask>
int Cuda_AtomVecCuda_PackBorder(cuda_shared_data* sdata, int nsend, int iswap, void* buf_send, int* pbc, int pbc_flag)
{
  my_times atime1, atime2;
  my_gettime(CLOCK_REALTIME, &atime1);

  if(sdata->atom.update_nmax)
    Cuda_AtomVecCuda_UpdateNmax<data_mask>(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  my_gettime(CLOCK_REALTIME, &atime2);
  sdata->cuda_timings.test1 +=
    atime2.tv_sec - atime1.tv_sec + 1.0 * (atime2.tv_nsec - atime1.tv_nsec) / 1000000000;

  int n_data_items = AtomVecCuda_CountDataItems(data_mask);

  int size = nsend * n_data_items * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_AtomVecCuda_UpdateBuffer(sdata, size);

  X_CFLOAT dx = 0.0;
  X_CFLOAT dy = 0.0;
  X_CFLOAT dz = 0.0;

  if(pbc_flag != 0) {
    if(sdata->domain.triclinic == 0) {
      dx = pbc[0] * sdata->domain.prd[0];
      dy = pbc[1] * sdata->domain.prd[1];
      dz = pbc[2] * sdata->domain.prd[2];
    } else {
      dx = pbc[0];
      dy = pbc[1];
      dz = pbc[2];
    }
  }

  int3 layout = getgrid(nsend);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    my_times time1, time2;
    my_gettime(CLOCK_REALTIME, &time1);

    Cuda_AtomVecCuda_PackBorder_Kernel<data_mask> <<< grid, threads, 0>>>((int*) sdata->comm.sendlist.dev_data, nsend, sdata->comm.maxlistlength, iswap, dx, dy, dz);
    hipDeviceSynchronize();

    my_gettime(CLOCK_REALTIME, &time2);
    sdata->cuda_timings.comm_border_kernel_pack +=
      time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

    hipMemcpy(buf_send, sdata->buffer, size, hipMemcpyDeviceToHost);
    CUT_CHECK_ERROR("Cuda_AtomVecCuda_PackBorder: Kernel execution failed");

    my_gettime(CLOCK_REALTIME, &time1);
    sdata->cuda_timings.comm_border_download +=
      time1.tv_sec - time2.tv_sec + 1.0 * (time1.tv_nsec - time2.tv_nsec) / 1000000000;

  }

  return nsend * n_data_items;
}

template <const unsigned int data_mask>
int Cuda_AtomVecCuda_PackBorder_Self(cuda_shared_data* sdata, int n, int iswap, int first, int* pbc, int pbc_flag)
{
  if(sdata->atom.update_nmax)
    Cuda_AtomVecCuda_UpdateNmax<data_mask>(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int n_data_items = AtomVecCuda_CountDataItems(data_mask);

  int size = n * n_data_items * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_AtomVecCuda_UpdateBuffer(sdata, size);

  X_CFLOAT dx = 0.0;
  X_CFLOAT dy = 0.0;
  X_CFLOAT dz = 0.0;

  if(pbc_flag != 0) {
    if(sdata->domain.triclinic == 0) {
      dx = pbc[0] * sdata->domain.prd[0];
      dy = pbc[1] * sdata->domain.prd[1];
      dz = pbc[2] * sdata->domain.prd[2];
    } else {
      dx = pbc[0];
      dy = pbc[1];
      dz = pbc[2];
    }
  }

  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    my_times time1, time2;
    my_gettime(CLOCK_REALTIME, &time1);

    Cuda_AtomVecCuda_PackBorder_Self_Kernel<data_mask> <<< grid, threads, 0>>>((int*) sdata->comm.sendlist.dev_data, n, sdata->comm.maxlistlength, iswap, dx, dy, dz, first);
    hipDeviceSynchronize();

    my_gettime(CLOCK_REALTIME, &time2);
    sdata->cuda_timings.comm_border_kernel_self +=
      time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

    CUT_CHECK_ERROR("Cuda_AtomVecCuda_PackBorder_Self: Kernel execution failed");

  }

  return n * n_data_items;
}


template <const unsigned int data_mask>
int Cuda_AtomVecCuda_UnpackBorder(cuda_shared_data* sdata, int n, int first, void* buf_recv)
{
  my_times atime1, atime2;
  my_gettime(CLOCK_REALTIME, &atime1);

  if(sdata->atom.update_nmax)
    Cuda_AtomVecCuda_UpdateNmax<data_mask>(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  my_gettime(CLOCK_REALTIME, &atime2);
  sdata->cuda_timings.test1 +=
    atime2.tv_sec - atime1.tv_sec + 1.0 * (atime2.tv_nsec - atime1.tv_nsec) / 1000000000;

  int n_data_items = AtomVecCuda_CountDataItems(data_mask);

  int size = n * n_data_items * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_AtomVecCuda_UpdateBuffer(sdata, size);

  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    my_times time1, time2;
    my_gettime(CLOCK_REALTIME, &time1);

    hipMemset((int*)(sdata->flag), 0, sizeof(int));
    hipMemcpy(sdata->buffer, (void*)buf_recv, size, hipMemcpyHostToDevice);

    my_gettime(CLOCK_REALTIME, &time2);
    sdata->cuda_timings.comm_border_upload +=
      time2.tv_sec - time1.tv_sec + 1.0 * (time2.tv_nsec - time1.tv_nsec) / 1000000000;

    Cuda_AtomVecCuda_UnpackBorder_Kernel<data_mask> <<< grid, threads, 0>>>(n, first);
    hipDeviceSynchronize();

    my_gettime(CLOCK_REALTIME, &time1);
    sdata->cuda_timings.comm_border_kernel_unpack +=
      time1.tv_sec - time2.tv_sec + 1.0 * (time1.tv_nsec - time2.tv_nsec) / 1000000000;

    hipMemcpy(&sdata->comm.grow_flag, sdata->flag, sizeof(int), hipMemcpyDeviceToHost);

    CUT_CHECK_ERROR("Cuda_AtomVecCuda_UnpackBorder: Kernel execution failed");

  }

  return sdata->comm.grow_flag;
}


#include "atom_vec_angle_cuda.cu"
#include "atom_vec_atomic_cuda.cu"
#include "atom_vec_charge_cuda.cu"
#include "atom_vec_full_cuda.cu"
//#include "atom_vec_granular_cuda.cu"
