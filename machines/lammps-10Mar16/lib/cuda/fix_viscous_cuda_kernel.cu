#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

__global__ void Cuda_FixViscousCuda_PostForce_Kernel(int groupbit, F_CFLOAT* gamma)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal)
    if(_mask[i] & groupbit) {
      F_CFLOAT drag = gamma[_type[i]];
      _f[i] -= drag * _v[i];
      _f[i + 1 * _nmax] -= drag * _v[i + 1 * _nmax];
      _f[i + 2 * _nmax] -= drag * _v[i + 2 * _nmax];
    }
}
