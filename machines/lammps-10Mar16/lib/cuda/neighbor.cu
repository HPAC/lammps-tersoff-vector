#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#include <time.h>
#define MY_PREFIX neighbor
#define IncludeCommonNeigh
#include "cuda_shared.h"
#include "cuda_common.h"
#include "crm_cuda_utils.cu"
#include "cuda_wrapper_cu.h"

#define _cutneighsq     MY_AP(cutneighsq)
#define _ex_type     	MY_AP(ex_type)
#define _nex_type     	MY_AP(nex_type)
#define _ex1_bit     	MY_AP(ex1_bit)
#define _ex2_bit     	MY_AP(ex2_bit)
#define _nex_group     	MY_AP(nex_group)
#define _ex_mol_bit     MY_AP(ex_mol_bit)
#define _nex_mol     	MY_AP(nex_mol)
__device__ __constant__ CUDA_CFLOAT* _cutneighsq;
__device__ __constant__ int* _ex_type;
__device__ __constant__ int _nex_type;
__device__ __constant__ int* _ex1_bit;
__device__ __constant__ int* _ex2_bit;
__device__ __constant__ int _nex_group;
__device__ __constant__ int* _ex_mol_bit;
__device__ __constant__ int _nex_mol;

#include "neighbor_cu.h"
#include "neighbor_kernel.cu"

void Cuda_Neighbor_UpdateBuffer(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist)
{
  CUT_CHECK_ERROR("Cuda_PairLJCutCuda: before updateBuffer failed");

  int size = (unsigned)(sizeof(int) * 20 + sneighlist->bin_dim[0] * sneighlist->bin_dim[1] * sneighlist->bin_dim[2] * (sizeof(int) + sneighlist->bin_nmax * 3 * sizeof(CUDA_CFLOAT)));

  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_Neighbor Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)

    if(sdata->buffer != NULL) CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);

    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)
  }

  hipMemcpyToSymbol(MY_AP(buffer), & sdata->buffer, sizeof(int*));
  CUT_CHECK_ERROR("Cuda_PairLJCutCuda: updateBuffer failed");
}

int Cuda_BinAtoms(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist)
{
  if(sdata->buffer_new)
    Cuda_Neighbor_UpdateBuffer(sdata, sneighlist);

  // initialize only on first call
  CUDA_CFLOAT rez_bin_size[3] = {
    (1.0 * sneighlist->bin_dim[0] - 4.0) / (sdata->domain.subhi[0] - sdata->domain.sublo[0]),
    (1.0 * sneighlist->bin_dim[1] - 4.0) / (sdata->domain.subhi[1] - sdata->domain.sublo[1]),
    (1.0 * sneighlist->bin_dim[2] - 4.0) / (sdata->domain.subhi[2] - sdata->domain.sublo[2])
  };

  short init = 0;

  if(! init) {
    init = 0;
    hipMemcpyToSymbol(MY_AP(x)              , & sdata->atom.x         .dev_data, sizeof(X_CFLOAT*));
    hipMemcpyToSymbol(MY_AP(nall)         , & sdata->atom.nall                    , sizeof(unsigned));
    hipMemcpyToSymbol(MY_AP(nmax)           , & sdata->atom.nmax                    , sizeof(unsigned));
    hipMemcpyToSymbol(MY_AP(sublo)          ,   sdata->domain.sublo                 , sizeof(X_CFLOAT) * 3);
  }


  int3 layout = getgrid(sdata->atom.nall); // sneighlist->inum
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  my_times starttime, endtime;
  my_gettime(CLOCK_REALTIME, &starttime);

  hipMemset((int*)(sdata->buffer), 0, sizeof(int) * (20 + (sneighlist->bin_dim[0]) * (sneighlist->bin_dim[1]) * (sneighlist->bin_dim[2])) + 3 * sizeof(CUDA_CFLOAT) * (sneighlist->bin_dim[0]) * (sneighlist->bin_dim[1]) * (sneighlist->bin_dim[2]) * (sneighlist->bin_nmax));

  Binning_Kernel <<< grid, threads>>> (sneighlist->binned_id, sneighlist->bin_nmax, sneighlist->bin_dim[0], sneighlist->bin_dim[1], sneighlist->bin_dim[2], rez_bin_size[0], rez_bin_size[1], rez_bin_size[2]);
  hipDeviceSynchronize();

  my_gettime(CLOCK_REALTIME, &endtime);
  sdata->cuda_timings.neigh_bin +=
    endtime.tv_sec - starttime.tv_sec + 1.0 * (endtime.tv_nsec - starttime.tv_nsec) / 1000000000;


  int binning_error;
  hipMemcpy((void*) &binning_error, (void*) sdata->buffer, 1 * sizeof(int), hipMemcpyDeviceToHost);

  if(binning_error) {
    sneighlist->bin_extraspace += 0.05;
  } else {
    MYDBG(printf("CUDA: binning successful\n");)
  }
  CUT_CHECK_ERROR("Cuda_Binning: binning Kernel execution failed");
  return binning_error;
}

int Cuda_NeighborBuildFullBin(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist)
{
  //Cuda_Neighbor_UpdateBuffer(sdata,sneighlist);
  CUDA_CFLOAT globcutoff = -1.0;

  short init = 0;

  if(! init) {
    init = 1;

    // !! LAMMPS indexes atom types starting with 1 !!

    unsigned cuda_ntypes = sdata->atom.ntypes + 1;

    unsigned nx = sizeof(CUDA_CFLOAT) * cuda_ntypes * cuda_ntypes;

    CUDA_CFLOAT* acutneighsq = (CUDA_CFLOAT*) malloc(nx);
    //printf("Allocate: %i\n",nx);
    sneighlist->cu_cutneighsq = (CUDA_CFLOAT*) CudaWrapper_AllocCudaData(nx);

    if(sneighlist->cutneighsq) {
      int cutoffsdiffer = 0;
      double cutoff0 = sneighlist->cutneighsq[1][1];

      for(int i = 1; i <= sdata->atom.ntypes; ++i) {
        for(int j = 1; j <= sdata->atom.ntypes; ++j) {
          acutneighsq[i * cuda_ntypes + j] = (CUDA_CFLOAT)(sneighlist->cutneighsq[i][j]);

          if((sneighlist->cutneighsq[i][j] - cutoff0) * (sneighlist->cutneighsq[i][j] - cutoff0) > 1e-6) cutoffsdiffer++;
        }
      }

      if(not cutoffsdiffer) globcutoff = (CUDA_CFLOAT) cutoff0;
    } else {
      MYEMUDBG(printf("# CUDA: Cuda_NeighborBuild: cutneighsq == NULL\n");)
      return 0;
    }

    int size = 100;

    if(sdata->buffersize < size) {
      MYDBG(printf("Cuda_NeighborBuild Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
      CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
      sdata->buffer = CudaWrapper_AllocCudaData(size);
      sdata->buffersize = size;
      sdata->buffer_new++;
      MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)
    }

    CudaWrapper_UploadCudaData(acutneighsq, sneighlist->cu_cutneighsq, nx);
    hipMemcpyToSymbol(MY_AP(cutneighsq)       , &sneighlist->cu_cutneighsq       , sizeof(CUDA_CFLOAT*));

    hipMemcpyToSymbol(MY_AP(cuda_ntypes)      , & cuda_ntypes                    , sizeof(unsigned));
    hipMemcpyToSymbol(MY_AP(special_flag)     , sdata->atom.special_flag         , 4 * sizeof(int));
    hipMemcpyToSymbol(MY_AP(molecular)        , & sdata->atom.molecular          , sizeof(int));
  }

  hipMemcpyToSymbol(MY_AP(neighbor_maxlocal), & sneighlist->firstneigh.dim[0]  , sizeof(unsigned));
  //hipMemcpyToSymbol(MY_AP(firstneigh)       , & sneighlist->firstneigh.dev_data, sizeof(int*)     );
  hipMemcpyToSymbol(MY_AP(ilist)            , & sneighlist->ilist     .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(inum)             , & sneighlist->inum               , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nlocal)           , & sdata->atom.nlocal             , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nall)             , & sdata->atom.nall            , sizeof(int));
  hipMemcpyToSymbol(MY_AP(numneigh)         , & sneighlist->numneigh  .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(type)             , & sdata->atom.type      .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(mask)             , & sdata->atom.mask      .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(tag)              , & sdata->atom.tag       .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(special)          , & sdata->atom.special   .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(maxspecial)       , & sdata->atom.maxspecial         , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nspecial)         , & sdata->atom.nspecial  .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(maxneighbors)     , & sneighlist->maxneighbors	 , sizeof(int));
  hipMemcpyToSymbol(MY_AP(debugdata)        , & sdata->debugdata	 , sizeof(int*));
  hipMemcpyToSymbol(MY_AP(overlap_comm)     , & sdata->overlap_comm, sizeof(int));
  hipMemcpyToSymbol(MY_AP(neighbors) 		  , & sneighlist->neighbors.dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(ex_type) 		  , & sneighlist->ex_type.dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(ex1_bit) 		  , & sneighlist->ex1_bit.dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(ex2_bit) 		  , & sneighlist->ex2_bit.dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(ex_mol_bit) 	  , & sneighlist->ex_mol_bit.dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(nex_type)     	  , & sneighlist->nex_type, sizeof(int));
  hipMemcpyToSymbol(MY_AP(nex_group)     	  , & sneighlist->nex_group, sizeof(int));
  hipMemcpyToSymbol(MY_AP(nex_mol)     	  , & sneighlist->nex_mol, sizeof(int));

  if(sdata->overlap_comm) {
    hipMemcpyToSymbol(MY_AP(numneigh_border)  , & sneighlist->numneigh_border .dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(numneigh_inner)   , & sneighlist->numneigh_inner  .dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(neighbors_border) , & sneighlist->neighbors_border.dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(neighbors_inner)  , & sneighlist->neighbors_inner .dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(ilist_border)     , & sneighlist->ilist_border    .dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(inum_border)      , & sneighlist->inum_border     .dev_data, sizeof(int*));
  }

  //dim3 threads(sneighlist->bin_nmax,1,1);
  dim3 threads(MIN(128, sneighlist->bin_nmax), 1, 1);
  dim3 grid(sneighlist->bin_dim[0]*sneighlist->bin_dim[1], sneighlist->bin_dim[2], 1);

  //printf("Configuration: %i %i %i %i %i\n",grid.x,grid.y,threads.x,(sizeof(int)+3*sizeof(X_CFLOAT))*threads.x,sneighlist->bin_nmax);
  int buffer[20];
  buffer[0] = 1;
  buffer[1] = 0;
  CudaWrapper_UploadCudaData(buffer, sdata->buffer, 2 * sizeof(int));
  CUT_CHECK_ERROR("Cuda_NeighborBuild: pre neighbor build kernel error");
  //hipMemset(sdata->debugdata,0,100*sizeof(int));
  unsigned int shared_size = (sizeof(int) + 3 * sizeof(CUDA_CFLOAT)) * threads.x;
  MYDBG(printf("Configuration: %i %i %i %u %i\n", grid.x, grid.y, threads.x, shared_size, sneighlist->bin_nmax);)
  //shared_size=2056;
  my_times starttime, endtime;
  my_gettime(CLOCK_REALTIME, &starttime);
  //for(int i=0;i<100;i++)
  {
    if(sdata->overlap_comm)
      NeighborBuildFullBin_OverlapComm_Kernel <<< grid, threads, shared_size>>>
      (sneighlist->binned_id, sneighlist->bin_nmax, sneighlist->bin_dim[0], sneighlist->bin_dim[1], globcutoff, sdata->pair.use_block_per_atom);
    else {
      int exclude = sneighlist->nex_mol | sneighlist->nex_group | sneighlist->nex_type;

      if(exclude)
        NeighborBuildFullBin_Kernel<1> <<< grid, threads, shared_size>>>
        (sneighlist->binned_id, sneighlist->bin_nmax, sneighlist->bin_dim[0], sneighlist->bin_dim[1], globcutoff, sdata->pair.use_block_per_atom, sdata->pair.neighall);
      else
        NeighborBuildFullBin_Kernel<0> <<< grid, threads, shared_size>>>
        (sneighlist->binned_id, sneighlist->bin_nmax, sneighlist->bin_dim[0], sneighlist->bin_dim[1], globcutoff, sdata->pair.use_block_per_atom, sdata->pair.neighall);
    }
    //NeighborBuildFullBin_Kernel_Restrict<<<grid,threads,(2*sizeof(int)+3*sizeof(X_CFLOAT))*threads.x+sizeof(int)>>>
    //	(sneighlist->binned_id,sneighlist->bin_nmax,sneighlist->bin_dim[0],sneighlist->bin_dim[1],globcutoff);

    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_NeighborBuild: neighbor build kernel execution failed");
    my_gettime(CLOCK_REALTIME, &endtime);
    sdata->cuda_timings.neigh_build +=
      endtime.tv_sec - starttime.tv_sec + 1.0 * (endtime.tv_nsec - starttime.tv_nsec) / 1000000000;
    //dim3 threads,grid;
    CudaWrapper_DownloadCudaData(buffer, sdata->buffer, sizeof(int));

    if(buffer[0] >= 0 && true && sdata->atom.molecular) {
      //printf("Find Special: %i %i\n",sneighlist->inum,sdata->atom.nall);
      my_gettime(CLOCK_REALTIME, &starttime);
      int3 layout = getgrid(sdata->atom.nlocal, 0, 512);
      threads.x = layout.z;
      threads.y = 1;
      threads.z = 1;
      grid.x = layout.x;
      grid.y = layout.y;
      grid.z = 1;
      FindSpecial <<< grid, threads>>>(sdata->pair.use_block_per_atom);
      hipDeviceSynchronize();
      CUT_CHECK_ERROR("Cuda_NeighborBuild: FindSpecial kernel execution failed");
      my_gettime(CLOCK_REALTIME, &endtime);
      sdata->cuda_timings.neigh_special +=
        endtime.tv_sec - starttime.tv_sec + 1.0 * (endtime.tv_nsec - starttime.tv_nsec) / 1000000000;
    }
  }
  //printf("Neightime: %lf\n",sdata->cuda_timings.test1);
  CUT_CHECK_ERROR("Cuda_NeighborBuild: neighbor build kernel execution failed");

  //CudaWrapper_DownloadCudaData(buffer, sneighlist->numneigh_border .dev_data, sizeof(int));

  MYDBG(printf("Cuda_NeighborBuildFullBin build neighbor list ... end\n");)
  return buffer[0];
}

int Cuda_NeighborBuildFullNsq(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist)
{
  MYDBG(printf("Cuda_NeighborBuildFullNsq build neighbor list ... start\n");)
  // initialize only on first call
  /*static*/ short init = 0;

  if(! init) {
    init = 1;

    // !! LAMMPS indexes atom types starting with 1 !!

    unsigned cuda_ntypes = sdata->atom.ntypes + 1;

    if(cuda_ntypes * cuda_ntypes > CUDA_MAX_TYPES2)
      printf("# CUDA: Cuda_PairLJCutCuda_Init: you need %u types. this is more than %u "
             "(assumed at compile time). re-compile with -DCUDA_MAX_TYPES_PLUS_ONE=32 "
             "or ajust this in cuda_common.h\n", cuda_ntypes, CUDA_MAX_TYPES2);

    unsigned nx = sizeof(CUDA_CFLOAT) * cuda_ntypes * cuda_ntypes;
    CUDA_CFLOAT* acutneighsq = (CUDA_CFLOAT*) malloc(nx);

    if(sneighlist->cutneighsq) {
      for(int i = 1; i <= sdata->atom.ntypes; ++i) {
        for(int j = 1; j <= sdata->atom.ntypes; ++j) {
          acutneighsq[i * cuda_ntypes + j] = (CUDA_CFLOAT)(sneighlist->cutneighsq[i][j]);
          //printf("CUTOFFS: %i %i %i %e\n",i,j,cuda_ntypes,acutneighsq[i * cuda_ntypes + j]);
        }
      }
    } else {
      MYEMUDBG(printf("# CUDA: Cuda_NeighborBuild: cutneighsq == NULL\n");)
      return 0;
    }

    int size = 100;

    if(sdata->buffersize < size) {
      MYDBG(printf("Cuda_NeighborBuild Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
      CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
      sdata->buffer = CudaWrapper_AllocCudaData(size);
      sdata->buffersize = size;
      sdata->buffer_new++;
      MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)
    }

    hipMemcpyToSymbol(MY_AP(buffer)           , & sdata->buffer                  , sizeof(int*));
    hipMemcpyToSymbol(MY_AP(cuda_ntypes)      , & cuda_ntypes                    , sizeof(unsigned));
    hipMemcpyToSymbol(MY_AP(cutneighsq)       , acutneighsq                    , nx);
    hipMemcpyToSymbol(MY_AP(neighbor_maxlocal), & sneighlist->firstneigh.dim[0]  , sizeof(unsigned));
    hipMemcpyToSymbol(MY_AP(firstneigh)       , & sneighlist->firstneigh.dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(ilist)            , & sneighlist->ilist     .dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(inum)             , & sneighlist->inum               , sizeof(int));
    hipMemcpyToSymbol(MY_AP(nlocal)           , & sdata->atom.nlocal             , sizeof(int));
    hipMemcpyToSymbol(MY_AP(nall)             , & sdata->atom.nall               , sizeof(int));
    hipMemcpyToSymbol(MY_AP(nmax)             , & sdata->atom.nmax               , sizeof(int));
    hipMemcpyToSymbol(MY_AP(numneigh)         , & sneighlist->numneigh  .dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(type)             , & sdata->atom.type      .dev_data, sizeof(int*));
    hipMemcpyToSymbol(MY_AP(x)                , & sdata->atom.x         .dev_data, sizeof(X_CFLOAT*));
    hipMemcpyToSymbol(MY_AP(maxneighbors)     , & sneighlist->maxneighbors	 , sizeof(int));

    free(acutneighsq);
  }

  int3 layout = getgrid(sdata->atom.nlocal); // sneighlist->inum
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  int return_value = 1;
  CudaWrapper_UploadCudaData(& return_value, sdata->buffer, sizeof(int));

  CUT_CHECK_ERROR("Cuda_NeighborBuild: pre neighbor build kernel execution failed");
  NeighborBuildFullNsq_Kernel <<< grid, threads>>> ();
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_NeighborBuild: neighbor build kernel execution failed");

  int buffer[20];
  CudaWrapper_DownloadCudaData(buffer, sdata->buffer, sizeof(int) * 20);
  MYDBG(printf("Cuda_NeighborBuildFullNSQ build neighbor list ... end\n");)
  return return_value = buffer[0];
}
