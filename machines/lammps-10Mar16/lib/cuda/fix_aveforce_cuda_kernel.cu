#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

extern __shared__ F_CFLOAT sharedmem[];


__global__ void Cuda_FixAveForceCuda_PostForce_FOrg_Kernel(int groupbit)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  sharedmem[threadIdx.x] = 0;
  sharedmem[threadIdx.x + blockDim.x] = 0;
  sharedmem[threadIdx.x + 2 * blockDim.x] = 0;
  sharedmem[threadIdx.x + 3 * blockDim.x] = 0;

  if(i < _nlocal)
    if(_mask[i] & groupbit) {
      sharedmem[threadIdx.x] = _f[i];
      sharedmem[threadIdx.x + blockDim.x] = _f[i + 1 * _nmax];
      sharedmem[threadIdx.x + 2 * blockDim.x] = _f[i + 2 * _nmax];
      sharedmem[threadIdx.x + 3 * blockDim.x] = 1;
    }

  reduceBlock(sharedmem);
  reduceBlock(&sharedmem[blockDim.x]);
  reduceBlock(&sharedmem[2 * blockDim.x]);
  reduceBlock(&sharedmem[3 * blockDim.x]);
  F_CFLOAT* buffer = (F_CFLOAT*) _buffer;

  if(threadIdx.x == 0) {
    buffer[blockIdx.x * gridDim.y + blockIdx.y] = sharedmem[0];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + gridDim.x * gridDim.y] = sharedmem[blockDim.x];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + 2 * gridDim.x * gridDim.y] = sharedmem[2 * blockDim.x];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + 3 * gridDim.x * gridDim.y] = sharedmem[3 * blockDim.x];
  }
}


__global__ void Cuda_FixAveForceCuda_reduce_foriginal(int n, F_CFLOAT* foriginal)
{
  int i = 0;
  sharedmem[threadIdx.x] = 0;
  F_CFLOAT myforig = 0.0;
  F_CFLOAT* buf = (F_CFLOAT*) _buffer;
  buf = &buf[blockIdx.x * n];

  while(i < n) {
    sharedmem[threadIdx.x] = 0;

    if(i + threadIdx.x < n)
      sharedmem[threadIdx.x] = buf[i + threadIdx.x];

    __syncthreads();
    reduceBlock(sharedmem);
    i += blockDim.x;

    if(threadIdx.x == 0)
      myforig += sharedmem[0];
  }

  if(threadIdx.x == 0)
    foriginal[blockIdx.x] = myforig;
}

__global__ void Cuda_FixAveForceCuda_PostForce_Set_Kernel(int groupbit, int xflag, int yflag, int zflag, F_CFLOAT xvalue, F_CFLOAT yvalue, F_CFLOAT zvalue)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal)
    if(_mask[i] & groupbit) {
      if(xflag) _f[i] = xvalue;

      if(yflag) _f[i + 1 * _nmax] = yvalue;

      if(zflag) _f[i + 2 * _nmax] = zvalue;
    }
}
