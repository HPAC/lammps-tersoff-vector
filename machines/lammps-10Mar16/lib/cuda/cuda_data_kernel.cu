#include "hip/hip_runtime.h"
__global__ void CudaData_Upload_Kernel_DoubleFloat(double* buffer, float* dev_data,
    unsigned nx, unsigned ny, unsigned nz, copy_mode mode)
{
  if(mode == x) mode = xx;

  unsigned length = nx;

  if(ny > 0) length *= ny;

  if(nz > 0) length *= nz;

  unsigned i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x, j, k, l;


  if(i >= length) return;

  switch(mode) {
    case xx: {
      dev_data[i] = buffer[i];
    }

    case xy: {
      dev_data[i] = buffer[i];
    }

    case yx: {
      j = i / ny;
      k = i % ny;
      dev_data[k * nx + j] = buffer[j * ny + k];
    }

    case xyz: {
      dev_data[i] = buffer[i];
    }

    case xzy: {
      j = i / (ny * nz);
      k = (i % (ny * nz)) / nz;
      l = i % nz;
      dev_data[j * ny * nz + l * ny + k] = buffer[j * ny * nz + k * nz + l];
    }
  }
}

__global__ void CudaData_Upload_Kernel_DoubleDouble(double* buffer, double* dev_data,
    unsigned nx, unsigned ny, unsigned nz, copy_mode mode)
{
  if(mode == x) mode = xx;

  unsigned length = nx;

  if(ny > 0) length *= ny;

  if(nz > 0) length *= nz;

  unsigned i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x, j, k, l;

  if(i >= length) return;

  switch(mode) {
    case xx:
      dev_data[i] = buffer[i];

    case xy:
      dev_data[i] = buffer[i];

    case yx:
      j = i / ny;
      k = i % ny;
      dev_data[k * nx + j] = buffer[j * ny + k];

    case xyz:
      dev_data[i] = buffer[i];

    case xzy:
      j = i / (ny * nz);
      k = (i % (ny * nz)) / nz;
      l = i % nz;
      dev_data[j * ny * nz + l * ny + k] = buffer[j * ny * nz + k * nz + l];
  }
}

__global__ void CudaData_Upload_Kernel_FloatDouble(float* buffer, double* dev_data,
    unsigned nx, unsigned ny, unsigned nz, copy_mode mode)
{
  if(mode == x) mode = xx;

  unsigned length = nx;

  if(ny > 0) length *= ny;

  if(nz > 0) length *= nz;

  unsigned i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x, j, k, l;

  if(i >= length) return;

  switch(mode) {
    case xx:
      dev_data[i] = buffer[i];

    case xy:
      dev_data[i] = buffer[i];

    case yx:
      j = i / ny;
      k = i % ny;
      dev_data[k * nx + j] = buffer[j * ny + k];

    case xyz:
      dev_data[i] = buffer[i];

    case xzy:
      j = i / (ny * nz);
      k = (i % (ny * nz)) / nz;
      l = i % nz;
      dev_data[j * ny * nz + l * ny + k] = buffer[j * ny * nz + k * nz + l];
  }
}

__global__ void CudaData_Upload_Kernel_FloatFloat(float* buffer, float* dev_data,
    unsigned nx, unsigned ny, unsigned nz, copy_mode mode)
{
  if(mode == x) mode = xx;

  unsigned length = nx;

  if(ny > 0) length *= ny;

  if(nz > 0) length *= nz;

  unsigned i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x, j, k, l;

  if(i >= length) return;

  switch(mode) {
    case xx:
      dev_data[i] = buffer[i];

    case xy:
      dev_data[i] = buffer[i];

    case yx:
      j = i / ny;
      k = i % ny;
      dev_data[k * nx + j] = buffer[j * ny + k];

    case xyz:
      dev_data[i] = buffer[i];

    case xzy:
      j = i / (ny * nz);
      k = (i % (ny * nz)) / nz;
      l = i % nz;
      dev_data[j * ny * nz + l * ny + k] = buffer[j * ny * nz + k * nz + l];
  }
}

__global__ void CudaData_Upload_Kernel_IntInt(int* buffer, int* dev_data,
    unsigned nx, unsigned ny, unsigned nz, copy_mode mode)
{
  if(mode == x) mode = xx;

  unsigned length = nx;

  if(ny > 0) length *= ny;

  if(nz > 0) length *= nz;

  unsigned i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x, j, k, l;

  if(i >= length) return;

  switch(mode) {
    case xx:
      dev_data[i] = buffer[i];

    case xy:
      dev_data[i] = buffer[i];

    case yx:
      j = i / ny;
      k = i % ny;
      dev_data[k * nx + j] = buffer[j * ny + k];

    case xyz:
      dev_data[i] = buffer[i];

    case xzy:
      j = i / (ny * nz);
      k = (i % (ny * nz)) / nz;
      l = i % nz;
      dev_data[j * ny * nz + l * ny + k] = buffer[j * ny * nz + k * nz + l];
  }
}
