#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */



__global__ void Cuda_FixTempRescaleCuda_EndOfStep_Kernel(int groupbit, V_CFLOAT factor)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal)
    if(_mask[i] & groupbit) {
      _v[i] *= factor;
      _v[i + _nmax] *= factor;
      _v[i + 2 * _nmax] *= factor;
    }
}

