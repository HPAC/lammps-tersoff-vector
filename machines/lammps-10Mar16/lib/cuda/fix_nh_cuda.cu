#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX fix_nh_cuda
#define IncludeCommonNeigh
#include "cuda_shared.h"
#include "cuda_common.h"
#include "crm_cuda_utils.cu"
#include "fix_nh_cuda_cu.h"
#include "fix_nh_cuda_kernel.cu"

void Cuda_FixNHCuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(f)       , & sdata->atom.f    .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(tag)     , & sdata->atom.tag  .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(debugdata)     , & sdata->debugdata, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(rmass)   , & sdata->atom.rmass.dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(mass)    , & sdata->atom.mass.dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(type)    , & sdata->atom.type .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(v)       , & sdata->atom.v    .dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(x)       , & sdata->atom.x    .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(xhold)   , & sdata->atom.xhold.dev_data, sizeof(X_CFLOAT*));  //might be moved to a neighbor record in sdata
  hipMemcpyToSymbol(MY_AP(maxhold)   , & sdata->atom.maxhold, sizeof(int));  //might be moved to a neighbor record in sdata
  hipMemcpyToSymbol(MY_AP(reneigh_flag), & sdata->buffer, sizeof(int*));  //might be moved to a neighbor record in sdata
  hipMemcpyToSymbol(MY_AP(triggerneighsq), & sdata->atom.triggerneighsq, sizeof(X_CFLOAT)); //might be moved to a neighbor record in sdata
}

void Cuda_FixNHCuda_UpdateBuffer(cuda_shared_data* sdata)
{
  int size = (unsigned)10 * sizeof(int);

  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_FixNHCuda Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
    CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)

  }

  hipMemcpyToSymbol(MY_AP(buffer) , & sdata->buffer, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(reneigh_flag), & sdata->buffer, sizeof(int*));  //might be moved to a neighbor record in sdata
}

void Cuda_FixNHCuda_Init(cuda_shared_data* sdata, X_CFLOAT dtv, V_CFLOAT dtf)
{
  hipMemcpyToSymbol(MY_AP(mass)    , & sdata->atom.mass.dev_data , sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(dtf)     , & dtf                       		, sizeof(V_CFLOAT));
  hipMemcpyToSymbol(MY_AP(dtv)     , & dtv                            , sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(triggerneighsq), &sdata->atom.triggerneighsq, sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(dist_check), & sdata->atom.dist_check       , sizeof(int));
  hipMemcpyToSymbol(MY_AP(rmass_flag), & sdata->atom.rmass_flag       , sizeof(int));       //
  Cuda_FixNHCuda_UpdateNmax(sdata);
}


void Cuda_FixNHCuda_nh_v_press(cuda_shared_data* sdata, int groupbit, double* factor_h, int mynlocal, int p_triclinic) //mynlocal can be nfirst if firstgroup==igroup  see cpp
{
  my_times atime1, atime2;
  my_gettime(CLOCK_REALTIME, &atime1);

  if(sdata->atom.update_nmax)
    Cuda_FixNHCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal , sizeof(int));

  my_gettime(CLOCK_REALTIME, &atime2);
  sdata->cuda_timings.test1 +=
    atime2.tv_sec - atime1.tv_sec + 1.0 * (atime2.tv_nsec - atime1.tv_nsec) / 1000000000;

  if(sdata->buffer_new)
    Cuda_FixNHCuda_UpdateBuffer(sdata);

  F_CFLOAT3 factor = {factor_h[0], factor_h[1], factor_h[2]};
  F_CFLOAT3 factor2;

  if(p_triclinic) {
    factor2.x = factor_h[3], factor2.y = factor_h[4];
    factor2.z = factor_h[5];
  }

  int3 layout = getgrid(mynlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  FixNHCuda_nh_v_press_Kernel <<< grid, threads>>> (groupbit, factor, p_triclinic, factor2);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("FixNHCuda: fix nh v_press Kernel execution failed");

}

void Cuda_FixNHCuda_nh_v_press_and_nve_v_NoBias(cuda_shared_data* sdata, int groupbit, double* factor_h, int mynlocal, int p_triclinic) //mynlocal can be nfirst if firstgroup==igroup  see cpp
{
  if(sdata->atom.update_nmax)
    Cuda_FixNHCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal , sizeof(int));

  if(sdata->buffer_new)
    Cuda_FixNHCuda_UpdateBuffer(sdata);

  F_CFLOAT3 factor = {factor_h[0], factor_h[1], factor_h[2]};
  F_CFLOAT3 factor2;

  if(p_triclinic) {
    factor2.x = factor_h[3], factor2.y = factor_h[4];
    factor2.z = factor_h[5];
  }

  int3 layout = getgrid(mynlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  CUT_CHECK_ERROR("FixNHCuda: fix nh v_press pre Kernel execution failed");
  FixNHCuda_nh_v_press_and_nve_v_NoBias_Kernel <<< grid, threads>>> (groupbit, factor, p_triclinic, factor2);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("FixNHCuda: fix nh v_press Kernel execution failed");

}

void Cuda_FixNHCuda_nh_v_temp(cuda_shared_data* sdata, int groupbit, F_CFLOAT factor_eta, int mynlocal) //mynlocal can be nfirst if firstgroup==igroup  see cpp
{
  my_times atime1, atime2;
  my_gettime(CLOCK_REALTIME, &atime1);

  if(sdata->atom.update_nmax)
    Cuda_FixNHCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal , sizeof(int));

  my_gettime(CLOCK_REALTIME, &atime2);
  sdata->cuda_timings.test1 +=
    atime2.tv_sec - atime1.tv_sec + 1.0 * (atime2.tv_nsec - atime1.tv_nsec) / 1000000000;

  if(sdata->buffer_new)
    Cuda_FixNHCuda_UpdateBuffer(sdata);

  int3 layout = getgrid(mynlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  FixNHCuda_nh_v_temp_Kernel <<< grid, threads>>> (groupbit, factor_eta);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("FixNHCuda: fix nh v_temp Kernel execution failed");

}
void Cuda_FixNHCuda_nve_v(cuda_shared_data* sdata, int groupbit, int mynlocal) //mynlocal can be nfirst if firstgroup==igroup  see cpp
{
  my_times atime1, atime2;
  my_gettime(CLOCK_REALTIME, &atime1);

  if(sdata->atom.update_nmax)
    Cuda_FixNHCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal , sizeof(int));

  my_gettime(CLOCK_REALTIME, &atime2);
  sdata->cuda_timings.test1 +=
    atime2.tv_sec - atime1.tv_sec + 1.0 * (atime2.tv_nsec - atime1.tv_nsec) / 1000000000;

  if(sdata->buffer_new)
    Cuda_FixNHCuda_UpdateBuffer(sdata);

  int3 layout = getgrid(mynlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  FixNHCuda_nve_v_Kernel <<< grid, threads>>> (groupbit);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("FixNHCuda: nve_v Kernel execution failed");
}


void Cuda_FixNHCuda_nve_x(cuda_shared_data* sdata, int groupbit, int mynlocal) //mynlocal can be nfirst if firstgroup==igroup  see cpp
{
  my_times atime1, atime2;
  my_gettime(CLOCK_REALTIME, &atime1);

  if(sdata->atom.update_nmax)
    Cuda_FixNHCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal , sizeof(int));

  my_gettime(CLOCK_REALTIME, &atime2);
  sdata->cuda_timings.test1 +=
    atime2.tv_sec - atime1.tv_sec + 1.0 * (atime2.tv_nsec - atime1.tv_nsec) / 1000000000;

  if(sdata->buffer_new)
    Cuda_FixNHCuda_UpdateBuffer(sdata);

  int3 layout = getgrid(mynlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  hipMemset(sdata->buffer, 0, sizeof(int));
  FixNHCuda_nve_x_Kernel <<< grid, threads>>> (groupbit);
  hipDeviceSynchronize();
  int reneigh_flag;
  hipMemcpy((void*)(&reneigh_flag), sdata->buffer, sizeof(int), hipMemcpyDeviceToHost);
  sdata->atom.reneigh_flag += reneigh_flag;
  CUT_CHECK_ERROR("FixNHCuda: nve_x Kernel execution failed");
}

void Cuda_FixNHCuda_nve_v_and_nh_v_press_NoBias(cuda_shared_data* sdata, int groupbit, double* factor_h, int mynlocal, int p_triclinic) //mynlocal can be nfirst if firstgroup==igroup  see cpp
{
  if(sdata->atom.update_nmax)
    Cuda_FixNHCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal , sizeof(int));

  if(sdata->buffer_new)
    Cuda_FixNHCuda_UpdateBuffer(sdata);

  F_CFLOAT3 factor = {factor_h[0], factor_h[1], factor_h[2]};
  F_CFLOAT3 factor2;

  if(p_triclinic) {
    factor2.x = factor_h[3], factor2.y = factor_h[4];
    factor2.z = factor_h[5];
  }

  int3 layout = getgrid(mynlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  FixNHCuda_nve_v_and_nh_v_press_NoBias_Kernel <<< grid, threads>>> (groupbit, factor, p_triclinic, factor2);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("FixNHCuda__nve_v_and_nh_v_press_NoBias:   Kernel execution failed");
}

