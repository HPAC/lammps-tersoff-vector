enum copy_mode {x, xx, xy, yx, xyz, xzy}; // yxz, yzx, zxy, zyx not yet implemented since they were not needed yet

#include "cuda_data_cu.h"
#include "cuda_wrapper_cu.h"
#include "cuda_data_kernel.cu"
#include <cstdio>

void CudaData_Upload_DoubleFloat(void* host_data, void* dev_data, unsigned* n, copy_mode mode, void* buffer)
{
  int size = n[0];

  if(n[1] > 0) size *= n[1];

  if(n[2] > 0) size *= n[2];

  dim3 threads;
  threads.x = 1;
  threads.y = 1;
  threads.z = 1;
  dim3 grid;
  grid.x = 1;
  grid.y = 1;
  grid.z = 1;

  if(size <= 128 * 30)
    threads.x = 32;
  else if(size <= 256 * 30)
    threads.x = 64;
  else if(size <= 512 * 30)
    threads.x = 128;
  else
    threads.x = 256;

  grid.x = ((size - 1) + threads.x) / threads.x;

  if(grid.x > 32000)
    grid.x = 32000;

  while(grid.x * grid.y * threads.x < size) grid.y++;

  float debugdata[size];
  //int* cu_debug=(int*) CudaWrapper_AllocCudaData(size*sizeof(FLOAT));
  size *= sizeof(double);
  printf("size: %i (%i %i %i) (%i %i %i) %p\n", size, grid.x, grid.y, threads.x, n[0], n[1], n[2], buffer);
  CudaWrapper_UploadCudaData(host_data, buffer, size);
  CudaData_Upload_Kernel_DoubleFloat <<< grid, threads>>>((double*)buffer, (float*)dev_data, n[0], n[1], n[2], mode);
  hipDeviceSynchronize();
  CudaWrapper_DownloadCudaData(debugdata, dev_data, size / 2);
  double sum = 0;
  printf("debugdata: ");

  for(int i = 0; i < size / sizeof(double); i++) sum += (debugdata[i] - ((double*) host_data)[i]) * (debugdata[i] - ((double*) host_data)[i]);

  printf("%lf \n", sum);

}

void CudaData_Upload_DoubleDouble(void* host_data, void* dev_data, unsigned* n, copy_mode mode, void* buffer)
{
  int size = n[0];

  if(n[1] > 0) size *= n[1];

  if(n[2] > 0) size *= n[2];

  dim3 threads;
  threads.x = 1;
  threads.y = 1;
  threads.z = 1;
  dim3 grid;
  grid.x = 1;
  grid.y = 1;
  grid.z = 1;

  if(size <= 128 * 30)
    threads.x = 32;
  else if(size <= 256 * 30)
    threads.x = 64;
  else if(size <= 512 * 30)
    threads.x = 128;
  else
    threads.x = 256;

  grid.x = ((size - 1) + threads.x) / threads.x;

  if(grid.x > 32000)
    grid.x = 32000;

  while(grid.x * grid.y * threads.x < size) grid.y++;

  size *= sizeof(double);

  CudaWrapper_UploadCudaData(host_data, buffer, size);
  CudaData_Upload_Kernel_DoubleDouble <<< grid, threads>>>((double*)buffer, (double*)dev_data, n[0], n[1], n[2], mode);
  hipDeviceSynchronize();
}

void CudaData_Upload_FloatDouble(void* host_data, void* dev_data, unsigned* n, copy_mode mode, void* buffer)
{
  int size = n[0];

  if(n[1] > 0) size *= n[1];

  if(n[2] > 0) size *= n[2];

  dim3 threads;
  threads.x = 1;
  threads.y = 1;
  threads.z = 1;
  dim3 grid;
  grid.x = 1;
  grid.y = 1;
  grid.z = 1;

  if(size <= 128 * 30)
    threads.x = 32;
  else if(size <= 256 * 30)
    threads.x = 64;
  else if(size <= 512 * 30)
    threads.x = 128;
  else
    threads.x = 256;

  grid.x = ((size - 1) + threads.x) / threads.x;

  if(grid.x > 32000)
    grid.x = 32000;

  while(grid.x * grid.y * threads.x < size) grid.y++;

  size *= sizeof(float);

  CudaWrapper_UploadCudaData(host_data, buffer, size);
  CudaData_Upload_Kernel_FloatDouble <<< grid, threads>>>((float*)buffer, (double*)dev_data, n[0], n[1], n[2], mode);
  hipDeviceSynchronize();
}

void CudaData_Upload_FloatFloat(void* host_data, void* dev_data, unsigned* n, copy_mode mode, void* buffer)
{
  int size = n[0];

  if(n[1] > 0) size *= n[1];

  if(n[2] > 0) size *= n[2];

  dim3 threads;
  threads.x = 1;
  threads.y = 1;
  threads.z = 1;
  dim3 grid;
  grid.x = 1;
  grid.y = 1;
  grid.z = 1;

  if(size <= 128 * 30)
    threads.x = 32;
  else if(size <= 256 * 30)
    threads.x = 64;
  else if(size <= 512 * 30)
    threads.x = 128;
  else
    threads.x = 256;

  grid.x = ((size - 1) + threads.x) / threads.x;

  if(grid.x > 32000)
    grid.x = 32000;

  while(grid.x * grid.y * threads.x < size) grid.y++;

  size *= sizeof(float);

  CudaWrapper_UploadCudaData(host_data, buffer, size);
  CudaData_Upload_Kernel_FloatFloat <<< grid, threads>>>((float*)buffer, (float*)dev_data, n[0], n[1], n[2], mode);
  hipDeviceSynchronize();
}

void CudaData_Upload_IntInt(void* host_data, void* dev_data, unsigned* n, copy_mode mode, void* buffer)
{
  int size = n[0];

  if(n[1] > 0) size *= n[1];

  if(n[2] > 0) size *= n[2];

  dim3 threads;
  threads.x = 1;
  threads.y = 1;
  threads.z = 1;
  dim3 grid;
  grid.x = 1;
  grid.y = 1;
  grid.z = 1;

  if(size <= 128 * 30)
    threads.x = 32;
  else if(size <= 256 * 30)
    threads.x = 64;
  else if(size <= 512 * 30)
    threads.x = 128;
  else
    threads.x = 256;

  grid.x = ((size - 1) + threads.x) / threads.x;

  if(grid.x > 32000)
    grid.x = 32000;

  while(grid.x * grid.y * threads.x < size) grid.y++;

  size *= sizeof(int);

  CudaWrapper_UploadCudaData(host_data, buffer, size);
  CudaData_Upload_Kernel_IntInt <<< grid, threads>>>((int*)buffer, (int*)dev_data, n[0], n[1], n[2], mode);
  hipDeviceSynchronize();
}

void CudaData_Download(void* host_data, void* dev_data, int host_size, int dev_size, unsigned* n, copy_mode mode, void* buffer)
{
}
