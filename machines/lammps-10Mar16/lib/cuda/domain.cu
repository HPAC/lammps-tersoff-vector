#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX domain
#include "cuda_shared.h"
#include "cuda_common.h"

#include "crm_cuda_utils.cu"

#include "domain_cu.h"
#include "domain_kernel.cu"

void Cuda_Domain_UpdateBuffer(cuda_shared_data* sdata, int size)
{
  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_Domain Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
    CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)
  }

  hipMemcpyToSymbol(MY_AP(buffer), & sdata->buffer, sizeof(int*));
}

void Cuda_Domain_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(x)       , & sdata->atom.x    .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(v)       , & sdata->atom.v    .dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(tag)    , & sdata->atom.tag .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(image)   , & sdata->atom.image.dev_data, sizeof(int*));
}

void Cuda_Domain_UpdateDomain(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(boxlo)   ,  sdata->domain.boxlo       , 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(boxhi)   ,  sdata->domain.boxhi       , 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(sublo)   ,  sdata->domain.sublo       , 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(subhi)   ,  sdata->domain.subhi       , 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(prd)     ,  sdata->domain.prd         , 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(periodicity)   ,   sdata->domain.periodicity , 3 * sizeof(int));
  hipMemcpyToSymbol(MY_AP(triclinic)     , & sdata->domain.triclinic   , sizeof(int));
  hipMemcpyToSymbol(MY_AP(boxlo_lamda)   ,   sdata->domain.boxlo_lamda , 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(boxhi_lamda)   ,   sdata->domain.boxhi_lamda , 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(prd_lamda)	   ,   sdata->domain.prd_lamda   , 3 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(h)	   	 ,   sdata->domain.h   		  , 6 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(h_inv)	 ,   sdata->domain.h_inv   	  , 6 * sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(h_rate)	 ,   sdata->domain.h_rate     , 6 * sizeof(V_CFLOAT));
  hipMemcpyToSymbol(MY_AP(flag)	 ,   &sdata->flag     , sizeof(int*));
  hipMemcpyToSymbol(MY_AP(debugdata)	 ,   &sdata->debugdata     , sizeof(int*));
}

void Cuda_Domain_Init(cuda_shared_data* sdata)
{
  Cuda_Domain_UpdateNmax(sdata);
  Cuda_Domain_UpdateDomain(sdata);
}

void Cuda_Domain_PBC(cuda_shared_data* sdata, int deform_remap, int deform_groupbit, double* extent)
{
  Cuda_Domain_UpdateNmax(sdata);
  //if(sdata->domain.update)
  Cuda_Domain_UpdateDomain(sdata);
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int box_change = 0;

  if(extent) box_change = 1;

  int sharedmem = 0;

  if(box_change) sharedmem = 6 * sizeof(X_CFLOAT);

  int3 layout = getgrid(sdata->atom.nlocal, sharedmem);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  sharedmem *= threads.x;

  if((box_change) && (sdata->buffer_new or (6 * sizeof(X_CFLOAT)*grid.x * grid.y > sdata->buffersize)))
    Cuda_Domain_UpdateBuffer(sdata, layout.x * layout.y * 6 * sizeof(X_CFLOAT));


  Domain_PBC_Kernel <<< grid, threads, sharedmem>>>(deform_remap, deform_groupbit, box_change);
  hipDeviceSynchronize();

  CUT_CHECK_ERROR("Cuda_Domain_PBC: Kernel execution failed");

  if(box_change) {
    X_CFLOAT buf2[6 * layout.x * layout.y];
    X_CFLOAT* buf = buf2;
    int flag;
    hipMemcpy(buf, sdata->buffer, 6 * layout.x * layout.y * sizeof(X_CFLOAT), hipMemcpyDeviceToHost);
    hipMemcpy(&flag, sdata->flag, sizeof(int), hipMemcpyDeviceToHost);
    //printf("Flag: %i\n",flag);
    X_CFLOAT min, max;
    min = 1.0 * BIG;
    max = -1.0 * BIG;

    for(int i = 0; i < layout.x * layout.y; i++) {
      if(buf[i] < min) min = buf[i];

      if(buf[i + layout.x * layout.y] > max) max = buf[i + layout.x * layout.y];
    }

    extent[0] = min;
    extent[1] = max;

    buf += 2 * layout.x * layout.y;
    min = 1.0 * BIG;
    max = -1.0 * BIG;

    for(int i = 0; i < layout.x * layout.y; i++) {
      if(buf[i] < min) min = buf[i];

      if(buf[i + layout.x * layout.y] > max) max = buf[i + layout.x * layout.y];
    }

    extent[2] = min;
    extent[3] = max;

    buf += 2 * layout.x * layout.y;
    min = 1.0 * BIG;
    max = -1.0 * BIG;

    for(int i = 0; i < layout.x * layout.y; i++) {
      if(buf[i] < min) min = buf[i];

      if(buf[i + layout.x * layout.y] > max) max = buf[i + layout.x * layout.y];
    }

    extent[4] = min;
    extent[5] = max;
    //printf("Extent: %lf %lf %lf %lf %lf %lf\n",extent[0],extent[1],extent[2],extent[3],extent[4],extent[5]);
    /*	   int n=grid.x*grid.y;
    	   if(n<128) threads.x=32;
    	   else if(n<256) threads.x=64;
    	   else threads.x=128;
    	   sharedmem=n*sizeof(X_CFLOAT);
    	   grid.x=6;
    	   grid.y=1;
    	   Domain_reduceBoxExtent<<<grid, threads,sharedmem>>>(extent,n);
    	   hipDeviceSynchronize();
    	   CUT_CHECK_ERROR("Cuda_Domain_reduceBoxExtent: Kernel execution failed");*/
  }
}

void Cuda_Domain_lamda2x(cuda_shared_data* sdata, int n)
{
  Cuda_Domain_UpdateNmax(sdata);
  //if(sdata->domain.update)
  Cuda_Domain_UpdateDomain(sdata);
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  Domain_lamda2x_Kernel <<< grid, threads, 0>>>(n);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Domain_lamda2x: Kernel execution failed");
}

void Cuda_Domain_x2lamda(cuda_shared_data* sdata, int n)
{
  Cuda_Domain_UpdateNmax(sdata);
  //if(sdata->domain.update)
  Cuda_Domain_UpdateDomain(sdata);
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  Domain_x2lamda_Kernel <<< grid, threads, 0>>>(n);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Domain_x2lamda: Kernel execution failed");
}
