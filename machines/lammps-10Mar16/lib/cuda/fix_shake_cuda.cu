#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX fix_shake_cuda
#include "cuda_shared.h"
#include "cuda_common.h"
#include "crm_cuda_utils.cu"
#include "fix_shake_cuda_cu.h"
#include "cuda_pair_virial_kernel_nc.cu"

#define _shake_atom           MY_AP(shake_atom)
#define _shake_type           MY_AP(shake_type)
#define _shake_flag           MY_AP(shake_flag)
#define _xshake               MY_AP(xshake)
#define _dtfsq                MY_AP(dtfsq)
#define _bond_distance        MY_AP(bond_distance)
#define _angle_distance       MY_AP(angle_distance)
#define _max_iter			  MY_AP(max_iter)
#define _tolerance			  MY_AP(tolerance)
__device__ __constant__ int* _shake_atom;
__device__ __constant__ int* _shake_type;
__device__ __constant__ int* _shake_flag;
__device__ __constant__ X_CFLOAT3* _xshake;
__device__ __constant__ F_CFLOAT _dtfsq;
__device__ __constant__ X_CFLOAT* _bond_distance;
__device__ __constant__ X_CFLOAT* _angle_distance;
__device__ __constant__ int _max_iter;
__device__ __constant__ X_CFLOAT _tolerance;

#include "fix_shake_cuda_kernel.cu"

void Cuda_FixShakeCuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(x)       , & sdata->atom.x    .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(v)       , & sdata->atom.v    .dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(f)       , & sdata->atom.f    .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(tag)     , & sdata->atom.tag  .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(rmass)   , & sdata->atom.rmass.dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(type)    , & sdata->atom.type .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(map_array), & sdata->atom.map_array .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(vatom)   , & sdata->atom.vatom.dev_data, sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(debugdata), & sdata->debugdata         , sizeof(int*));
}

void Cuda_FixShakeCuda_UpdateDomain(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(periodicity), sdata->domain.periodicity		, sizeof(int) * 3);
  hipMemcpyToSymbol(MY_AP(prd)		, sdata->domain.prd				, sizeof(X_CFLOAT) * 3);
  hipMemcpyToSymbol(MY_AP(triclinic)  , &sdata->domain.triclinic		, sizeof(int));
  hipMemcpyToSymbol(MY_AP(h)			, sdata->domain.h				, sizeof(X_CFLOAT) * 6);
}

void Cuda_FixShakeCuda_UpdateBuffer(cuda_shared_data* sdata, int size)
{
  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_FixShakeCuda Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
    CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)

  }

  hipMemcpyToSymbol(MY_AP(buffer) , & sdata->buffer, sizeof(int*));
}

void Cuda_FixShakeCuda_Init(cuda_shared_data* sdata, X_CFLOAT dtv, F_CFLOAT dtfsq,
                            void* shake_flag, void* shake_atom, void* shake_type, void* xshake,
                            void* bond_distance, void* angle_distance, void* virial,
                            int max_iter, X_CFLOAT tolerance)
{
  Cuda_FixShakeCuda_UpdateNmax(sdata);
  Cuda_FixShakeCuda_UpdateDomain(sdata);
  hipMemcpyToSymbol(MY_AP(shake_atom)        , & shake_atom 	  , sizeof(void*));
  hipMemcpyToSymbol(MY_AP(shake_type)        , & shake_type 	  , sizeof(void*));
  hipMemcpyToSymbol(MY_AP(shake_flag)        , & shake_flag 	  , sizeof(void*));
  hipMemcpyToSymbol(MY_AP(xshake)            , & xshake     	  , sizeof(void*));
  hipMemcpyToSymbol(MY_AP(dtv)               , & dtv        	  , sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(dtfsq)             , & dtfsq      	  , sizeof(F_CFLOAT));
  hipMemcpyToSymbol(MY_AP(bond_distance)     , & bond_distance  , sizeof(void*));
  hipMemcpyToSymbol(MY_AP(angle_distance)    , & angle_distance , sizeof(void*));
  hipMemcpyToSymbol(MY_AP(virial)     	   , & virial  		  , sizeof(void*));
  hipMemcpyToSymbol(MY_AP(flag)  			   , &sdata->flag	  , sizeof(int*));
  hipMemcpyToSymbol(MY_AP(max_iter)  		   , &max_iter  	  , sizeof(int));
  hipMemcpyToSymbol(MY_AP(tolerance)  	   , &tolerance  	  , sizeof(X_CFLOAT));

  if(sdata->atom.mass_host)
    hipMemcpyToSymbol(MY_AP(mass), & sdata->atom.mass.dev_data , sizeof(V_CFLOAT*));

  hipMemcpyToSymbol(MY_AP(rmass_flag), & sdata->atom.rmass_flag       , sizeof(int));       //

  hipMemcpyToSymbol(MY_AP(flag)  , &sdata->flag, sizeof(int*));

}

void Cuda_FixShakeCuda_UnconstrainedUpdate(cuda_shared_data* sdata)
{
  if(sdata->atom.update_nmax)
    Cuda_FixShakeCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal , sizeof(int));

  if(sdata->buffer_new)
    Cuda_FixShakeCuda_UpdateBuffer(sdata, 10 * sizeof(double));

  int3 layout = getgrid(sdata->atom.nlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  FixShakeCuda_UnconstrainedUpdate_Kernel <<< grid, threads>>> ();
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("FixShakeCuda_UnconstrainedUpdate: Kernel execution failed");
}

void Cuda_FixShakeCuda_Shake(cuda_shared_data* sdata, int vflag, int vflag_atom, int* list, int nlist)
{
  if(sdata->atom.update_nmax)
    Cuda_FixShakeCuda_UpdateNmax(sdata);

  if(sdata->domain.update)
    Cuda_FixShakeCuda_UpdateDomain(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal , sizeof(int));

  int3 layout = getgrid(sdata->atom.nlocal, 6 * sizeof(ENERGY_CFLOAT), 64);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->buffer_new)
    Cuda_FixShakeCuda_UpdateBuffer(sdata, grid.x * grid.y * 6 * sizeof(ENERGY_CFLOAT));

  BindXTypeTexture(sdata);

  FixShakeCuda_Shake_Kernel <<< grid, threads, 6* threads.x* sizeof(ENERGY_CFLOAT)>>> (vflag, vflag_atom, list, nlist);
  hipDeviceSynchronize();

  CUT_CHECK_ERROR("FixShakeCuda_Shake: Kernel execution failed");

  if(vflag) {
    int n = grid.x * grid.y;
    grid.x = 6;
    grid.y = 1;
    threads.x = 256;
    MY_AP(PairVirialCompute_reduce) <<< grid, threads, threads.x* sizeof(ENERGY_CFLOAT)>>>(n);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_FixShakeCuda: (no binning) virial compute Kernel execution failed");
  }

}

int Cuda_FixShakeCuda_PackComm(cuda_shared_data* sdata, int n, int iswap, void* buf_send, int* pbc, int pbc_flag)
{
  if(sdata->atom.update_nmax)
    Cuda_FixShakeCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int size = n * 3 * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_FixShakeCuda_UpdateBuffer(sdata, size);

  X_CFLOAT dx = 0.0;
  X_CFLOAT dy = 0.0;
  X_CFLOAT dz = 0.0;

  if(pbc_flag != 0) {
    if(sdata->domain.triclinic == 0) {
      dx = pbc[0] * sdata->domain.prd[0];
      dy = pbc[1] * sdata->domain.prd[1];
      dz = pbc[2] * sdata->domain.prd[2];
    } else {
      dx = pbc[0] * sdata->domain.prd[0] + pbc[5] * sdata->domain.xy + pbc[4] * sdata->domain.xz;
      dy = pbc[1] * sdata->domain.prd[1] + pbc[3] * sdata->domain.yz;
      dz = pbc[2] * sdata->domain.prd[2];
    }
  }

  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    hipMemset(sdata->flag, 0, sizeof(int));
    FixShakeCuda_PackComm_Kernel <<< grid, threads, 0>>>((int*) sdata->comm.sendlist.dev_data, n, sdata->comm.maxlistlength, iswap, dx, dy, dz);
    hipDeviceSynchronize();
    hipMemcpy(buf_send, sdata->buffer, n * 3 * sizeof(X_CFLOAT), hipMemcpyDeviceToHost);
    int aflag;
    hipMemcpy(&aflag, sdata->flag, sizeof(int), hipMemcpyDeviceToHost);

    if(aflag != 0) printf("aflag PackComm: %i\n", aflag);
    CUT_CHECK_ERROR("Cuda_FixShakeCuda_PackComm: Kernel execution failed");

  }

  return 3 * n;
}

int Cuda_FixShakeCuda_PackComm_Self(cuda_shared_data* sdata, int n, int iswap, int first, int* pbc, int pbc_flag)
{
  if(sdata->atom.update_nmax)
    Cuda_FixShakeCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int size = n * 3 * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_FixShakeCuda_UpdateBuffer(sdata, size);

  static int count = -1;
  count++;
  X_CFLOAT dx = 0.0;
  X_CFLOAT dy = 0.0;
  X_CFLOAT dz = 0.0;

  if(pbc_flag != 0) {
    if(sdata->domain.triclinic == 0) {
      dx = pbc[0] * sdata->domain.prd[0];
      dy = pbc[1] * sdata->domain.prd[1];
      dz = pbc[2] * sdata->domain.prd[2];
    } else {
      dx = pbc[0] * sdata->domain.prd[0] + pbc[5] * sdata->domain.xy + pbc[4] * sdata->domain.xz;
      dy = pbc[1] * sdata->domain.prd[1] + pbc[3] * sdata->domain.yz;
      dz = pbc[2] * sdata->domain.prd[2];
    }
  }



  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    FixShakeCuda_PackComm_Self_Kernel <<< grid, threads, 0>>>((int*) sdata->comm.sendlist.dev_data, n, sdata->comm.maxlistlength, iswap, dx, dy, dz, first);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_CommCuda_PackComm_Self: Kernel execution failed");
  }

  return 3 * n;
}

void Cuda_FixShakeCuda_UnpackComm(cuda_shared_data* sdata, int n, int first, void* buf_recv)
{
  if(sdata->atom.update_nmax)
    Cuda_FixShakeCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int size = n * 3 * sizeof(X_CFLOAT);

  if(sdata->buffer_new or (size > sdata->buffersize))
    Cuda_FixShakeCuda_UpdateBuffer(sdata, size);

  int3 layout = getgrid(n);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  if(sdata->atom.nlocal > 0) {
    hipMemcpy(sdata->buffer, (void*)buf_recv, n * 3 * sizeof(X_CFLOAT), hipMemcpyHostToDevice);
    FixShakeCuda_UnpackComm_Kernel <<< grid, threads, 0>>>(n, first);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_FixShakeCuda_UnpackComm: Kernel execution failed");

  }
}
