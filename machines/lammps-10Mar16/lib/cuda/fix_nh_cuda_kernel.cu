#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

static inline __device__ void check_distance(X_CFLOAT &xtmp, X_CFLOAT &ytmp, X_CFLOAT &ztmp, int &i, int groupbit)
{
  if(_dist_check) {

    X_CFLOAT d = X_F(0.0);

    if(i < _nlocal) {
      X_CFLOAT tmp = xtmp - _xhold[i];
      d = tmp * tmp;
      tmp = ytmp - _xhold[i + _maxhold];
      d += tmp * tmp;
      tmp = ztmp - _xhold[i + 2 * _maxhold];
      d += tmp * tmp;

      d = ((_mask[i] & groupbit)) ? d : X_F(0.0);
    }

    if(not __all(d <= _triggerneighsq))
      _reneigh_flag[0] = 1;
  }
}

__global__ void FixNHCuda_nh_v_press_Kernel(int groupbit, F_CFLOAT3 factor, int p_triclinic, F_CFLOAT3 factor2)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal && _mask[i] & groupbit) {
    V_CFLOAT* my_v = _v + i;
    V_CFLOAT vx = my_v[0];
    V_CFLOAT vy = my_v[_nmax];
    V_CFLOAT vz = my_v[2 * _nmax];
    vx *= factor.x;
    vy *= factor.y;
    vz *= factor.z;

    if(p_triclinic) {
      vx += vy * factor2.z + vz * factor2.y;
      vy += vz * factor2.x;
    }

    vx *= factor.x;
    vy *= factor.y;
    vz *= factor.z;
    my_v[0]       = vx;
    my_v[_nmax]   = vy;
    my_v[2 * _nmax] = vz;
  }

}

__global__ void FixNHCuda_nh_v_temp_Kernel(int groupbit, F_CFLOAT factor_eta)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal && _mask[i] & groupbit) {
    V_CFLOAT* my_v = _v + i;
    my_v[0] *= factor_eta;
    my_v[_nmax] *= factor_eta;
    my_v[2 * _nmax] *= factor_eta;
  }

}

__global__ void FixNHCuda_nh_v_press_and_nve_v_NoBias_Kernel(int groupbit, F_CFLOAT3 factor, int p_triclinic, F_CFLOAT3 factor2)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal && _mask[i] & groupbit) {
    F_CFLOAT* my_f = _f + i;
    V_CFLOAT* my_v = _v + i;

    V_CFLOAT 		dtfm = _dtf;

    if(_rmass_flag) dtfm *= V_F(1.0) / _rmass[i];
    else 			dtfm *= V_F(1.0) / _mass[_type[i]];

    V_CFLOAT vx = my_v[0];
    V_CFLOAT vy = my_v[_nmax];
    V_CFLOAT vz = my_v[2 * _nmax];
    vx *= factor.x;
    vy *= factor.y;
    vz *= factor.z;

    if(p_triclinic) {
      vx += vy * factor2.z + vz * factor2.y;
      vy += vz * factor2.x;
    }

    vx *= factor.x;
    vy *= factor.y;
    vz *= factor.z;
    my_v[0]       = vx + dtfm * my_f[0];
    my_v[_nmax]   = vy + dtfm * my_f[_nmax];
    my_v[2 * _nmax] = vz + dtfm * my_f[_nmax * 2];
  }

}

__global__ void FixNHCuda_nve_v_Kernel(int groupbit)
{

  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal && _mask[i] & groupbit) {
    F_CFLOAT* my_f = _f + i;
    V_CFLOAT* my_v = _v + i;

    V_CFLOAT 		dtfm = _dtf;

    if(_rmass_flag) dtfm *= V_F(1.0) / _rmass[i];
    else 			dtfm *= V_F(1.0) / _mass[_type[i]];

    *my_v = (*my_v + dtfm * (*my_f));
    my_f += _nmax;
    my_v += _nmax;
    *my_v = (*my_v + dtfm * (*my_f));
    my_f += _nmax;
    my_v += _nmax;
    *my_v = (*my_v + dtfm * (*my_f));
  }
}

__global__ void FixNHCuda_nve_x_Kernel(int groupbit)
{
  X_CFLOAT xtmp, ytmp, ztmp;

  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal && _mask[i] & groupbit) {
    V_CFLOAT* my_v = _v + i;
    X_CFLOAT* my_x = _x + i;

    xtmp = *my_x += _dtv * *my_v;
    my_v += _nmax;
    my_x += _nmax;
    ytmp = *my_x += _dtv * *my_v;
    my_v += _nmax;
    my_x += _nmax;
    ztmp = *my_x += _dtv * *my_v;
  }

  check_distance(xtmp, ytmp, ztmp, i, groupbit);
}


__global__ void FixNHCuda_nve_v_and_nh_v_press_NoBias_Kernel(int groupbit, F_CFLOAT3 factor, int p_triclinic, F_CFLOAT3 factor2)
{

  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal && _mask[i] & groupbit) {
    F_CFLOAT* my_f = _f + i;
    V_CFLOAT* my_v = _v + i;

    V_CFLOAT 		dtfm = _dtf;

    if(_rmass_flag) dtfm *= V_F(1.0) / _rmass[i];
    else 			dtfm *= V_F(1.0) / _mass[_type[i]];

    V_CFLOAT vx = my_v[0] + dtfm * my_f[0];
    V_CFLOAT vy = my_v[_nmax] + dtfm * my_f[_nmax];
    V_CFLOAT vz = my_v[2 * _nmax] + dtfm * my_f[2 * _nmax];

    vx *= factor.x;
    vy *= factor.y;
    vz *= factor.z;

    if(p_triclinic) {
      vx += vy * factor2.z + vz * factor2.y;
      vy += vz * factor2.x;
    }

    vx *= factor.x;
    vy *= factor.y;
    vz *= factor.z;
    my_v[0]       = vx;
    my_v[_nmax]   = vy;
    my_v[2 * _nmax] = vz;

  }
}

