#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

extern __shared__ F_CFLOAT sharedmem[];


__global__ void Cuda_FixAddForceCuda_PostForce_Kernel(int groupbit, F_CFLOAT xvalue, F_CFLOAT yvalue, F_CFLOAT zvalue)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  sharedmem[threadIdx.x] = 0;
  sharedmem[threadIdx.x + blockDim.x] = 0;
  sharedmem[threadIdx.x + 2 * blockDim.x] = 0;
  sharedmem[threadIdx.x + 3 * blockDim.x] = 0;

  if(i < _nlocal)
    if(_mask[i] & groupbit)
      //if (iregion >= 0 &&
      //match(x[i][0],x[i][1],x[i][2],iregion)) //currently not supported
    {
      sharedmem[threadIdx.x] = -xvalue * _x[i] - yvalue * _x[i + 1 * _nmax] - zvalue * _x[i + 2 * _nmax];
      sharedmem[threadIdx.x + blockDim.x] = _f[i];
      sharedmem[threadIdx.x + 2 * blockDim.x] = _f[i + 1 * _nmax];
      sharedmem[threadIdx.x + 3 * blockDim.x] = _f[i + 2 * _nmax];
      _f[i] += xvalue;
      _f[i + 1 * _nmax] += yvalue;
      _f[i + 2 * _nmax] += zvalue;
    }

  reduceBlock(sharedmem);
  reduceBlock(&sharedmem[blockDim.x]);
  reduceBlock(&sharedmem[2 * blockDim.x]);
  reduceBlock(&sharedmem[3 * blockDim.x]);
  F_CFLOAT* buffer = (F_CFLOAT*) _buffer;

  if(threadIdx.x == 0) {
    buffer[blockIdx.x * gridDim.y + blockIdx.y] = sharedmem[0];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + gridDim.x * gridDim.y] = sharedmem[blockDim.x];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + 2 * gridDim.x * gridDim.y] = sharedmem[2 * blockDim.x];
    buffer[blockIdx.x * gridDim.y + blockIdx.y + 3 * gridDim.x * gridDim.y] = sharedmem[3 * blockDim.x];
  }

}


__global__ void reduce_foriginal(int n, F_CFLOAT* foriginal)
{
  int i = 0;
  sharedmem[threadIdx.x] = 0;
  F_CFLOAT myforig = 0.0;
  F_CFLOAT* buf = (F_CFLOAT*) _buffer;
  buf = &buf[blockIdx.x * n];

  while(i < n) {
    sharedmem[threadIdx.x] = 0;

    if(i + threadIdx.x < n)
      sharedmem[threadIdx.x] = buf[i + threadIdx.x];

    __syncthreads();
    reduceBlock(sharedmem);
    i += blockDim.x;

    if(threadIdx.x == 0)
      myforig += sharedmem[0];
  }

  if(threadIdx.x == 0)
    foriginal[blockIdx.x] = myforig;
}
