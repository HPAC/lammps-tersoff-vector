#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#define OFFSET 4096
__device__ int negativCUDA(float f)
{
  return ((unsigned int)1 << 31 & (__float_as_int(f))) >> 31;
}

__device__ void reduceBlock(float* data)
{
  int p2 = 1;

  while(p2 * 2 < blockDim.x) p2 *= 2;

  if(threadIdx.x < blockDim.x - p2)
    data[threadIdx.x] += data[threadIdx.x + p2];

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    if(threadIdx.x < p2 / i)
      data[threadIdx.x] += data[threadIdx.x + p2 / i];

    __syncthreads();
  }
}

__device__ void reduceBlock(double* data)
{
  int p2 = 1;

  while(p2 * 2 < blockDim.x) p2 *= 2;

  if(threadIdx.x < blockDim.x - p2)
    data[threadIdx.x] += data[threadIdx.x + p2];

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    if(threadIdx.x < p2 / i)
      data[threadIdx.x] += data[threadIdx.x + p2 / i];

    __syncthreads();
  }
}

extern __shared__ PPPM_CFLOAT sharedmem[];

__global__ void setup_fkxyz_vg(PPPM_CFLOAT unitkx, PPPM_CFLOAT unitky, PPPM_CFLOAT unitkz, PPPM_CFLOAT g_ewald)
{
  PPPM_CFLOAT my_fkx = unitkx * (int(threadIdx.x) - nx_pppm * (2 * int(threadIdx.x) / nx_pppm));
  PPPM_CFLOAT my_fky = unitky * (int(blockIdx.y) - ny_pppm * (2 * int(blockIdx.y) / ny_pppm));
  PPPM_CFLOAT my_fkz = unitkz * (int(blockIdx.x) - nz_pppm * (2 * int(blockIdx.x) / nz_pppm));

  if((blockIdx.x == 0) && (blockIdx.y == 0)) fkx[threadIdx.x] = my_fkx;

  if((blockIdx.x == 0) && (threadIdx.x == 0)) fky[blockIdx.y] = my_fky;

  if((threadIdx.x == 0) && (blockIdx.y == 0)) fkz[blockIdx.x] = my_fkz;

  __syncthreads();

  if((blockIdx.x >= nzlo_fft) && (blockIdx.x <= nzhi_fft) &&
      (blockIdx.y >= nylo_fft) && (blockIdx.y <= nyhi_fft) &&
      (threadIdx.x >= nxlo_fft) && (threadIdx.x <= nxhi_fft)) {
    int n = ((int(blockIdx.x) - nzlo_fft) * (nyhi_fft - nylo_fft + 1) + int(blockIdx.y) - nylo_fft) * (nxhi_fft - nxlo_fft + 1) + int(threadIdx.x) - nxlo_fft;
    PPPM_CFLOAT sqk = my_fkx * my_fkx + my_fky * my_fky + my_fkz * my_fkz;
    PPPM_CFLOAT vterm = (sqk == PPPM_F(0.0)) ? PPPM_F(0.0) : PPPM_F(-2.0) * (PPPM_F(1.0) / sqk + PPPM_F(0.25) / (g_ewald * g_ewald));
    vg[6 * n + 0] = (sqk == PPPM_F(0.0)) ? PPPM_F(0.0) : PPPM_F(1.0) + vterm * my_fkx * my_fkx;
    vg[6 * n + 1] = (sqk == PPPM_F(0.0)) ? PPPM_F(0.0) : PPPM_F(1.0) + vterm * my_fky * my_fky;
    vg[6 * n + 2] = (sqk == PPPM_F(0.0)) ? PPPM_F(0.0) : PPPM_F(1.0) + vterm * my_fkz * my_fkz;
    vg[6 * n + 3] = (sqk == PPPM_F(0.0)) ? PPPM_F(0.0) : vterm * my_fkx * my_fky;
    vg[6 * n + 4] = (sqk == PPPM_F(0.0)) ? PPPM_F(0.0) : vterm * my_fkx * my_fkz;
    vg[6 * n + 5] = (sqk == PPPM_F(0.0)) ? PPPM_F(0.0) : vterm * my_fky * my_fkz;

  }
}

__device__ PPPM_CFLOAT gf_denom(PPPM_CFLOAT x, PPPM_CFLOAT y, PPPM_CFLOAT z)
{
  PPPM_CFLOAT sx, sy, sz;
  sz = sy = sx = PPPM_F(0.0);

  for(int l = order - 1; l >= 0; l--) {
    sx = gf_b[l] + sx * x;
    sy = gf_b[l] + sy * y;
    sz = gf_b[l] + sz * z;
  }

  PPPM_CFLOAT s = sx * sy * sz;
  return s * s;
}

__global__ void setup_greensfn(PPPM_CFLOAT unitkx, PPPM_CFLOAT unitky, PPPM_CFLOAT unitkz, PPPM_CFLOAT g_ewald,
                               int nbx, int nby, int nbz,
                               PPPM_CFLOAT xprd, PPPM_CFLOAT yprd, PPPM_CFLOAT zprd_slab)
{
  PPPM_CFLOAT sqk;
  int nx, ny, nz, kper, lper, mper, k, l, m;
  PPPM_CFLOAT snx, sny, snz, snx2, sny2, snz2;
  PPPM_CFLOAT argx, argy, argz, wx, wy, wz, sx, sy, sz, qx, qy, qz;
  PPPM_CFLOAT sum1, dot1, dot2;
  PPPM_CFLOAT numerator, denominator;

  PPPM_CFLOAT form = PPPM_F(1.0);
  int n = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  m = blockIdx.x;
  l = blockIdx.y;
  k = threadIdx.x;

  mper = m - nz_pppm * (2 * m / nz_pppm);
  snz = sin(PPPM_F(0.5) * unitkz * mper * zprd_slab / nz_pppm);
  snz2 = snz * snz;


  lper = l - ny_pppm * (2 * l / ny_pppm);
  sny = sin(PPPM_F(0.5) * unitky * lper * yprd / ny_pppm);
  sny2 = sny * sny;

  kper = k - nx_pppm * (2 * k / nx_pppm);
  snx = sin(PPPM_F(0.5) * unitkx * kper * xprd / nx_pppm);
  snx2 = snx * snx;

  sqk = pow(unitkx * kper, PPPM_F(2.0)) + pow(unitky * lper, PPPM_F(2.0)) +
        pow(unitkz * mper, PPPM_F(2.0));

  if(sqk != PPPM_F(0.0)) {
    numerator = form * PPPM_F(12.5663706) / sqk;
    denominator = gf_denom(snx2, sny2, snz2);
    sum1 = PPPM_F(0.0);

    for(nx = -nbx; nx <= nbx; nx++) {
      qx = unitkx * (kper + nx_pppm * nx);
      sx = exp(PPPM_F(-.25) * pow(qx / g_ewald, PPPM_F(2.0)));
      wx = PPPM_F(1.0);
      argx = PPPM_F(0.5) * qx * xprd / nx_pppm;

      if(argx != PPPM_F(0.0)) wx = pow(sin(argx) / argx, order);

      for(ny = -nby; ny <= nby; ny++) {
        qy = unitky * (lper + ny_pppm * ny);
        sy = exp(PPPM_F(-.25) * pow(qy / g_ewald, PPPM_F(2.0)));
        wy = PPPM_F(1.0);
        argy = PPPM_F(0.5) * qy * yprd / ny_pppm;

        if(argy != PPPM_F(0.0)) wy = pow(sin(argy) / argy, order);

        for(nz = -nbz; nz <= nbz; nz++) {
          qz = unitkz * (mper + nz_pppm * nz);
          sz = exp(PPPM_F(-.25) * pow(qz / g_ewald, PPPM_F(2.0)));
          wz = PPPM_F(1.0);
          argz = PPPM_F(0.5) * qz * zprd_slab / nz_pppm;

          if(argz != PPPM_F(0.0)) wz = pow(sin(argz) / argz, order);

          dot1 = unitkx * kper * qx + unitky * lper * qy + unitkz * mper * qz;
          dot2 = qx * qx + qy * qy + qz * qz;
          sum1 += (dot1 / dot2) * sx * sy * sz * pow(wx * wy * wz, PPPM_F(2.0));
        }
      }
    }

    greensfn[n] = numerator * sum1 / denominator;
  } else greensfn[n] = PPPM_F(0.0);
}

__global__ void poisson_scale_kernel()
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  FFT_CFLOAT scaleinv = FFT_F(1.0) / (gridDim.x * gridDim.y * blockDim.x);
  work1[2 * i] *= scaleinv * greensfn[i];
  work1[2 * i + 1] *= scaleinv * greensfn[i];
}

__global__ void poisson_xgrad_kernel()
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  work2[2 * i] = fkx[threadIdx.x] * work1[2 * i + 1];
  work2[2 * i + 1] = -fkx[threadIdx.x] * work1[2 * i];
}

__global__ void poisson_ygrad_kernel()
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  work2[2 * i] = fky[blockIdx.y] * work1[2 * i + 1];
  work2[2 * i + 1] = -fky[blockIdx.y] * work1[2 * i];
}

__global__ void poisson_zgrad_kernel()
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  work2[2 * i] = fkz[blockIdx.x] * work1[2 * i + 1];
  work2[2 * i + 1] = -fkz[blockIdx.x] * work1[2 * i];
}

__global__ void poisson_vdx_brick_kernel(int ilo, int jlo, int klo)
{
  int k = blockIdx.x + klo;
  k += nz_pppm * negativCUDA(CUDA_F(1.0) * k) - nz_pppm * negativCUDA(CUDA_F(1.0) * (nz_pppm - k - 1));
  int j = blockIdx.y + jlo;
  j += ny_pppm * negativCUDA(CUDA_F(1.0) * j) - ny_pppm * negativCUDA(CUDA_F(1.0) * (ny_pppm - j - 1));
  int i = threadIdx.x + ilo;
  i += nx_pppm * negativCUDA(CUDA_F(1.0) * i) - nx_pppm * negativCUDA(CUDA_F(1.0) * (nx_pppm - i - 1));
  vdx_brick[((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1) + threadIdx.x] = work3[2 * (((k) * ny_pppm + (j)) * nx_pppm + i)];
}

__global__ void poisson_vdy_brick_kernel(int ilo, int jlo, int klo)
{
  int k = blockIdx.x + klo;
  k += nz_pppm * negativCUDA(CUDA_F(1.0) * k) - nz_pppm * negativCUDA(CUDA_F(1.0) * (nz_pppm - k - 1));
  int j = blockIdx.y + jlo;
  j += ny_pppm * negativCUDA(CUDA_F(1.0) * j) - ny_pppm * negativCUDA(CUDA_F(1.0) * (ny_pppm - j - 1));
  int i = threadIdx.x + ilo;
  i += nx_pppm * negativCUDA(CUDA_F(1.0) * i) - nx_pppm * negativCUDA(CUDA_F(1.0) * (nx_pppm - i - 1));
  vdy_brick[((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1) + threadIdx.x] = work3[2 * (((k) * ny_pppm + (j)) * nx_pppm + i)];
}

__global__ void poisson_vdz_brick_kernel(int ilo, int jlo, int klo)
{
  int k = blockIdx.x + klo;
  k += nz_pppm * negativCUDA(CUDA_F(1.0) * k) - nz_pppm * negativCUDA(CUDA_F(1.0) * (nz_pppm - k - 1));
  int j = blockIdx.y + jlo;
  j += ny_pppm * negativCUDA(CUDA_F(1.0) * j) - ny_pppm * negativCUDA(CUDA_F(1.0) * (ny_pppm - j - 1));
  int i = threadIdx.x + ilo;
  i += nx_pppm * negativCUDA(CUDA_F(1.0) * i) - nx_pppm * negativCUDA(CUDA_F(1.0) * (nx_pppm - i - 1));
  vdz_brick[((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1) + threadIdx.x] = work3[2 * (((k) * ny_pppm + (j)) * nx_pppm + i)];
}

__global__ void poisson_energy_kernel(int nxlo_fft, int nylo_fft, int nzlo_fft, int vflag)
{
  ENERGY_CFLOAT scaleinv = FFT_F(1.0) / (nx_pppm * ny_pppm * nz_pppm);
  int i = (blockIdx.x + nzlo_fft) * ny_pppm * nx_pppm + (blockIdx.y + nylo_fft) * nx_pppm + threadIdx.x + nxlo_fft;
  ENERGY_CFLOAT* s_energy = (ENERGY_CFLOAT*) sharedmem;
  ENERGY_CFLOAT myenergy = scaleinv * scaleinv * greensfn[i] * (work1[2 * i] * work1[2 * i] + work1[2 * i + 1] * work1[2 * i + 1]);
  s_energy[threadIdx.x] = myenergy;

  __syncthreads();
  reduceBlock(s_energy);

  if(threadIdx.x == 0)
    energy[blockIdx.x * ny_pppm + blockIdx.y] = s_energy[0];

  if(vflag) {
    __syncthreads();

    for(int j = 0; j < 6; j++) {
      s_energy[threadIdx.x] = myenergy * vg[((blockIdx.x * gridDim.y + blockIdx.y) * (blockDim.x) + threadIdx.x) * 6 + j];
      __syncthreads();
      reduceBlock(s_energy);

      if(threadIdx.x == 0)
        virial[blockIdx.x * ny_pppm + blockIdx.y + j * nz_pppm * ny_pppm] = s_energy[0];
    }
  }
}


__global__ void sum_energy_kernel1(int vflag)
{
  ENERGY_CFLOAT myenergy = energy[(blockIdx.x * ny_pppm + threadIdx.x)];
  ENERGY_CFLOAT* s_energy = (ENERGY_CFLOAT*) sharedmem;
  s_energy[threadIdx.x] = myenergy;
  __syncthreads();
  reduceBlock(s_energy);

  if(threadIdx.x == 0)
    energy[blockIdx.x * ny_pppm] = s_energy[0];

  if(vflag) {
    __syncthreads();

    for(int j = 0; j < 6; j++) {
      myenergy = virial[blockIdx.x * ny_pppm + threadIdx.x + j * ny_pppm * nz_pppm];
      s_energy[threadIdx.x] = myenergy;
      __syncthreads();
      reduceBlock(s_energy);

      if(threadIdx.x == 0)
        virial[blockIdx.x * ny_pppm + j * ny_pppm * nz_pppm] = s_energy[0];
    }
  }

}

__global__ void sum_energy_kernel2(int vflag)
{
  ENERGY_CFLOAT myenergy = energy[threadIdx.x * ny_pppm];
  ENERGY_CFLOAT* s_energy = (ENERGY_CFLOAT*) sharedmem;
  s_energy[threadIdx.x] = myenergy;
  __syncthreads();
  reduceBlock(s_energy);

  if(threadIdx.x == 0)
    energy[0] = s_energy[0];

  if(vflag) {
    __syncthreads();

    for(int j = 0; j < 6; j++) {
      myenergy = virial[threadIdx.x * ny_pppm + j * ny_pppm * nz_pppm];
      s_energy[threadIdx.x] = myenergy;
      __syncthreads();
      reduceBlock(s_energy);

      if(threadIdx.x == 0)
        virial[j] = s_energy[0];
    }
  }
}

__device__ PPPM_CFLOAT rho1d(int k, PPPM_CFLOAT d, PPPM_CFLOAT* srho_coeff)
{
  PPPM_CFLOAT rho1d_tmp = PPPM_F(0.0);

  for(int l = order - 1; l >= 0; l--)
    rho1d_tmp = srho_coeff[l * order + k - (1 - order) / 2] + rho1d_tmp * d;

  return rho1d_tmp;
}

__global__ void particle_map_kernel(int* flag)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < nlocal) {
    int nx, ny, nz;
    PPPM_CFLOAT shift = PPPM_F(0.5) - shiftone; //+OFFSET;
    nx = (int)((_x[i] - _boxlo[0]) * delxinv + shift); // - OFFSET;
    ny = (int)((_x[i + nmax] - _boxlo[1]) * delyinv + shift); // - OFFSET;
    nz = (int)((_x[i + 2 * nmax] - _boxlo[2]) * delzinv + shift); // - OFFSET;

    part2grid[i] = nx;
    part2grid[i + nmax] = ny;
    part2grid[i + 2 * nmax] = nz;

    // check that entire stencil around nx,ny,nz will fit in my 3d brick
    if(nx + nlower < nxlo_out || nx + nupper > nxhi_out ||
        ny + nlower < nylo_out || ny + nupper > nyhi_out ||
        nz + nlower < nzlo_out || nz + nupper > nzhi_out) {
      flag[0]++;
      debugdata[0] = i;
      debugdata[1] = _boxlo[0];
      debugdata[2] = _boxlo[1];
      debugdata[3] = _boxlo[2];
      debugdata[4] = nx;
      debugdata[5] = ny;
      debugdata[6] = nz;
      debugdata[7] = _x[i];
      debugdata[8] = _x[i + _nmax];
      debugdata[9] = _x[i + 2 * _nmax];
      debugdata[10] = nlocal;

    }
  }
}

__global__ void make_rho_kernelA()
{
  int i, l, m, n, nx, ny, nz, mx, my, mz;

  // clear 3d density array


  // loop over my charges, add their contribution to nearby grid points
  // (nx,ny,nz) = global coords of grid pt to "lower left" of charge
  // (dx,dy,dz) = distance to "lower left" grid pt
  // (mx,my,mz) = global coords of moving stencil pt

  i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < nlocal) {

    PPPM_CFLOAT dx, dy, dz, x0, y0, z0;
    nx = part2grid[i];
    ny = part2grid[i + nmax];
    nz = part2grid[i + 2 * nmax];
    dx = nx + shiftone - (_x[i] - _boxlo[0]) * delxinv;
    dy = ny + shiftone - (_x[i + nmax] - _boxlo[1]) * delyinv;
    dz = nz + shiftone - (_x[i + 2 * nmax] - _boxlo[2]) * delzinv;

    z0 = delxinv * delyinv * delzinv * _q[i];

    for(n = nlower; n <= nupper; n++) {
      mz = n + nz;
      y0 = z0 * rho1d(n, dz, rho_coeff);

      for(m = nlower; m <= nupper; m++) {
        my = m + ny;
        x0 = y0 * rho1d(m, dy, rho_coeff);

        for(l = nlower; l <= nupper; l++) {
          mx = l + nx;
          int mzyx = ((mz - nzlo_out) * (nyhi_out - nylo_out + 1) + my - nylo_out) * (nxhi_out - nxlo_out + 1) + mx - nxlo_out;

          while(atomicAdd(&density_brick_int[mzyx], 1) != 0) atomicAdd(&density_brick_int[mzyx], -1);

          density_brick[mzyx] += x0 * rho1d(l, dx, rho_coeff);
          __threadfence();
          atomicAdd(&density_brick_int[mzyx], -1);
          __syncthreads();

        }
      }
    }
  }
}

__global__ void make_rho_kernel(int* flag, int read_threads_at_same_time)
{
  int i, l, m, n, nx, ny, nz, mx, my, mz, a, b;

  // clear 3d density array


  // loop over my charges, add their contribution to nearby grid points
  // (nx,ny,nz) = global coords of grid pt to "lower left" of charge
  // (dx,dy,dz) = distance to "lower left" grid pt
  // (mx,my,mz) = global coords of moving stencil pt
  // int nzxy=blockIdx.x*gridDim.y+blockIdx.y;

  int nelements = nupper - nlower + 1;
  int* idx = (int*) sharedmem;
  int* sdensity_brick_int = &idx[blockDim.x];
  PPPM_CFLOAT* srho_coeff = (PPPM_CFLOAT*) &sdensity_brick_int[nelements * blockDim.x];

  if(threadIdx.x < order * (order / 2 - (1 - order) / 2 + 1))
    srho_coeff[threadIdx.x] = rho_coeff[threadIdx.x];

  __syncthreads();

  i = blockIdx.x * blockDim.x + threadIdx.x;

  if(false) {
    if(i < nlocal) {

      PPPM_CFLOAT dx, dy, dz, x0, y0, z0;
      nx = part2grid[i];
      ny = part2grid[i + nmax];
      nz = part2grid[i + 2 * nmax];
      dx = nx + shiftone - (_x[i] - _boxlo[0]) * delxinv;
      dy = ny + shiftone - (_x[i + nmax] - _boxlo[1]) * delyinv;
      dz = nz + shiftone - (_x[i + 2 * nmax] - _boxlo[2]) * delzinv;

      z0 = delxinv * delyinv * delzinv * _q[i];

      for(n = nlower; n <= nupper; n++) {
        mz = n + nz;
        y0 = z0 * rho1d(n, dz, srho_coeff);

        for(m = nlower; m <= nupper; m++) {
          my = m + ny;
          x0 = y0 * rho1d(m, dy, srho_coeff);

          for(l = nlower; l <= nupper; l++) {
            mx = l + nx;
            int mzyx = ((mz - nzlo_out) * (nyhi_out - nylo_out + 1) + my - nylo_out) * (nxhi_out - nxlo_out + 1) + mx - nxlo_out;

            a = int(x0 * rho1d(l, dx, srho_coeff) * density_intScale);
            b = (atomicAdd(&density_brick_int[mzyx], a) | a);

            if(((b) & (0x7c000000)) && (not((b) & (0x80000000)))) {
              flag[1]++;

              if((b) & (0x60000000)) flag[0]++;
            }

            __syncthreads();
          }
        }
      }
    }

    return;
  }

  i = blockIdx.x * blockDim.x + threadIdx.x;
  {

    PPPM_CFLOAT dx, dy, dz, x0, y0, z0, qtmp;

    if(i < nlocal) {
      qtmp = _q[i];
      nx = part2grid[i];
      ny = part2grid[i + nmax];
      nz = part2grid[i + 2 * nmax];
      dx = nx + shiftone - (_x[i] - _boxlo[0]) * delxinv;
      dy = ny + shiftone - (_x[i + nmax] - _boxlo[1]) * delyinv;
      dz = nz + shiftone - (_x[i + 2 * nmax] - _boxlo[2]) * delzinv;
      z0 = delxinv * delyinv * delzinv * qtmp;
    } else {
      nx = ny = nz = 1;
      dx = dy = dz = PPPM_F(0.1);
    }

    __syncthreads();

    for(n = nlower; n <= nupper; n++) {
      mz = n + nz;
      y0 = z0 * rho1d(n, dz, srho_coeff);

      for(m = nlower; m <= nupper; m++) {
        my = m + ny;
        x0 = y0 * rho1d(m, dy, srho_coeff);

        if(i < nlocal) {
          idx[threadIdx.x] = ((mz - nzlo_out) * (nyhi_out - nylo_out + 1) + my - nylo_out) * (nxhi_out - nxlo_out + 1) + nx + nlower - nxlo_out;

          for(l = nlower; l <= nupper; l++) {
            sdensity_brick_int[threadIdx.x * nelements + l - nlower] = int(x0 * rho1d(l, dx, srho_coeff) * density_intScale);
          }
        } else idx[threadIdx.x] = -1;

        __syncthreads();

        for(int ii = 0; ii < blockDim.x; ii += read_threads_at_same_time) {
          int kk = threadIdx.x / nelements;

          if((threadIdx.x < nelements * read_threads_at_same_time) && (kk + ii < blockDim.x) && (idx[ii + kk] > -1)) {
            a = sdensity_brick_int[ii * nelements + threadIdx.x];
            //if(a*a>1e-100)
            b = (atomicAdd(&density_brick_int[idx[ii + kk] + threadIdx.x - kk * nelements], a) | a);

            //else
            //b=(density_brick_int[idx[ii+kk]+threadIdx.x-kk*nelements]|a);
            if(((b) & (0x7c000000)) && (not((b) & (0x80000000)))) {
              flag[1]++;

              if((b) & (0x60000000)) flag[0]++;
            }
          }
        }

        __syncthreads();	   //*/
      }
    }

  }
}

__global__ void scale_rho_kernel()
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  density_brick[i] = (1.0 / density_intScale) * density_brick_int[i];
}

__global__ void fieldforce_kernel(int elements_per_thread, int read_threads_at_same_time, int* flag) //20*x64 0.36
{
  int i;

  // loop over my charges, interpolate electric field from nearby grid points
  // (nx,ny,nz) = global coords of grid pt to "lower left" of charge
  // (dx,dy,dz) = distance to "lower left" grid pt
  // (mx,my,mz) = global coords of moving stencil pt
  // ek = 3 components of E-field on particle
  i = blockIdx.x * blockDim.x + threadIdx.x;
  int* idx = (int*) sharedmem;
  PPPM_CFLOAT* tmp_brick = (PPPM_CFLOAT*) &idx[blockDim.x];
  PPPM_CFLOAT* srho_coeff = (PPPM_CFLOAT*) &tmp_brick[3 * blockDim.x * elements_per_thread];

  if(threadIdx.x < order * (order / 2 - (1 - order) / 2 + 1))
    srho_coeff[threadIdx.x] = rho_coeff[threadIdx.x];

  __syncthreads();
  {
    int l, m, n, nx, ny, nz, my, mz;
    PPPM_CFLOAT dx, dy, dz, x0, y0, z0;
    PPPM_CFLOAT ek[3];

    if(i < nlocal) {
      nx = part2grid[i];
      ny = part2grid[i + nmax];
      nz = part2grid[i + 2 * nmax];
      dx = nx + shiftone - (_x[i] - _boxlo[0]) * delxinv;
      dy = ny + shiftone - (_x[i + nmax] - _boxlo[1]) * delyinv;
      dz = nz + shiftone - (_x[i + 2 * nmax] - _boxlo[2]) * delzinv;

      ek[0] = ek[1] = ek[2] = PPPM_F(0.0);
    } else {
      nx = ny = nz = 1;
      dx = dy = dz = PPPM_F(0.1);
    }

    __syncthreads();

    for(n = nlower; n <= nupper; n++) {
      mz = n + nz;
      z0 = rho1d(n, dz, srho_coeff);

      for(m = nlower; m <= nupper; m++) {
        my = m + ny;
        y0 = z0 * rho1d(m, dy, srho_coeff);


        if(i < nlocal)
          idx[threadIdx.x] = ((mz - nzlo_out) * (nyhi_out - nylo_out + 1) + my - nylo_out) * (nxhi_out - nxlo_out + 1) + nx + nlower - nxlo_out;
        else idx[threadIdx.x] = -1;

        __syncthreads();

        for(int ii = 0; ii < blockDim.x; ii += read_threads_at_same_time) {
          int kk = threadIdx.x / elements_per_thread;

          if((threadIdx.x < elements_per_thread * read_threads_at_same_time) && (kk + ii < blockDim.x) && (idx[ii + kk] > -1)) {
            tmp_brick[ii * elements_per_thread + threadIdx.x] = vdx_brick[idx[ii + kk] + threadIdx.x - kk * elements_per_thread];
            tmp_brick[(ii + blockDim.x)*elements_per_thread + threadIdx.x] = vdy_brick[idx[ii + kk] + threadIdx.x - kk * elements_per_thread];
            tmp_brick[(ii + 2 * blockDim.x)*elements_per_thread + threadIdx.x] = vdz_brick[idx[ii + kk] + threadIdx.x - kk * elements_per_thread];
          }
        }

        __syncthreads();

        if(i < nlocal)
          for(l = nlower; l <= nupper; l++) {
            x0 = y0 * rho1d(l, dx, srho_coeff);

            ek[0] -= x0 * tmp_brick[threadIdx.x * elements_per_thread + l - nlower];
            ek[1] -= x0 * tmp_brick[threadIdx.x * elements_per_thread + l - nlower + blockDim.x * elements_per_thread];
            ek[2] -= x0 * tmp_brick[threadIdx.x * elements_per_thread + l - nlower + 2 * blockDim.x * elements_per_thread];
          }

        __syncthreads();
      }
    }

    // convert E-field to force


    _f[i] += qqrd2e * _q[i] * ek[0];
    _f[i + nmax] += qqrd2e * _q[i] * ek[1];
    _f[i + 2 * nmax] += qqrd2e * _q[i] * ek[2];
  }
}

__global__ void slabcorr_energy_kernel(ENERGY_CFLOAT* buf)
{
  ENERGY_CFLOAT* dipole = (ENERGY_CFLOAT*) sharedmem;
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < nlocal)
    dipole[threadIdx.x] = _q[i] * _x[i + 2 * nmax];
  else
    dipole[threadIdx.x] = ENERGY_F(0.0);

  __syncthreads();
  reduceBlock(dipole);

  if(threadIdx.x == 0) buf[blockIdx.x] = dipole[0];
}

__global__ void slabcorr_force_kernel(F_CFLOAT ffact)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(i < nlocal)
    _f[i + 2 * nmax] += qqrd2e * _q[i] * ffact;
}


__global__ void initfftdata_core_kernel(PPPM_CFLOAT* in, FFT_CFLOAT* out)
{
  out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] = in[(((blockIdx.x + nzlo_in - nzlo_out) * (nyhi_out - nylo_out + 1) + blockIdx.y + nylo_in - nylo_out) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxlo_in - nxlo_out];
  out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x) + 1] = 0;
}

__global__ void initfftdata_z_kernel(PPPM_CFLOAT* in, FFT_CFLOAT* out)
{
  if(slabflag) {
    if(blockIdx.x < nzlo_in - nzlo_out)
      out[2 * (((nzhi_in - nzlo_in + 2 - nupper - slabflag + blockIdx.x) * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1) + threadIdx.x)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y + nylo_in - nylo_out) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxlo_in - nxlo_out];
  } else {
    if(blockIdx.x < nzlo_in - nzlo_out)
      out[2 * (((blockIdx.x + 2 * (nzhi_in + 1) - nzlo_in - nzhi_out) * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1) + threadIdx.x)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y + nylo_in - nylo_out) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxlo_in - nxlo_out];
  }

  if(blockIdx.x < nzhi_out - nzhi_in)
    out[2 * ((((blockIdx.x) * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x + (nzhi_out - nzlo_in)) * (nyhi_out - nylo_out + 1) + blockIdx.y + nylo_in - nylo_out) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxlo_in - nxlo_out];
}

__global__ void initfftdata_y_kernel(PPPM_CFLOAT* in, FFT_CFLOAT* out)
{
  if(blockIdx.y < nylo_in - nylo_out)
    out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + (2 * (nyhi_in + 1) - nylo_in - nyhi_out) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x + nzlo_in - nzlo_out) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxlo_in - nxlo_out];

  if(blockIdx.y < nyhi_out - nyhi_in)
    out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x + nzlo_in - nzlo_out) * (nyhi_out - nylo_out + 1) + blockIdx.y + (nyhi_out - nylo_in)) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxlo_in - nxlo_out];
}

__global__ void initfftdata_x_kernel(PPPM_CFLOAT* in, FFT_CFLOAT* out)
{
  if(threadIdx.x < nxlo_in - nxlo_out)
    out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x + 2 * (nxhi_in + 1) - nxlo_in - nxhi_out)] += in[(((blockIdx.x + nzlo_in - nzlo_out) * (nyhi_out - nylo_out + 1) + blockIdx.y + nylo_in - nylo_out) * (nxhi_out - nxlo_out + 1)) + threadIdx.x];

  if(threadIdx.x < nxhi_out - nxhi_in)
    out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x + nzlo_in - nzlo_out) * (nyhi_out - nylo_out + 1) + blockIdx.y + nylo_in - nylo_out) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxhi_in - nxlo_out + 1];
}

__global__ void initfftdata_yz_kernel(PPPM_CFLOAT* in, FFT_CFLOAT* out)
{
  if(slabflag) {
    if(blockIdx.x < nzlo_in - nzlo_out)
      if(blockIdx.y < nyhi_out - nyhi_in)
        out[2 * ((((nzhi_in - nzlo_in + 2 - nupper - slabflag + blockIdx.x) * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y + nyhi_in - nylo_out + 1) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxlo_in - nxlo_out];

    if(blockIdx.x < nzlo_in - nzlo_out)
      if(blockIdx.y < nylo_in - nylo_out)
        out[2 * ((((nzhi_in - nzlo_in + 2 - nupper - slabflag + blockIdx.x) * (nyhi_in - nylo_in + 1) + blockIdx.y + 2 * (nyhi_in + 1) - nylo_in - nyhi_out) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxlo_in - nxlo_out];
  } else {
    if(blockIdx.x < nzlo_in - nzlo_out)
      if(blockIdx.y < nyhi_out - nyhi_in)
        out[2 * ((((blockIdx.x + 2 * (nzhi_in + 1) - nzlo_in - nzhi_out) * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y + nyhi_in - nylo_out + 1) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxlo_in - nxlo_out];

    if(blockIdx.x < nzlo_in - nzlo_out)
      if(blockIdx.y < nylo_in - nylo_out)
        out[2 * ((((blockIdx.x + 2 * (nzhi_in + 1) - nzlo_in - nzhi_out) * (nyhi_in - nylo_in + 1) + blockIdx.y + 2 * (nyhi_in + 1) - nylo_in - nyhi_out) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxlo_in - nxlo_out];
  }

  if(blockIdx.x < nzhi_out - nzhi_in)
    if(blockIdx.y < nyhi_out - nyhi_in)
      out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x + nzhi_in - nzlo_out + 1) * (nyhi_out - nylo_out + 1) + blockIdx.y + nyhi_in - nylo_out + 1) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxlo_in - nxlo_out];

  if(blockIdx.x < nzhi_out - nzhi_in)
    if(blockIdx.y < nylo_in - nylo_out)
      out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y + 2 * (nyhi_in + 1) - nylo_in - nyhi_out) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x + nzhi_in - nzlo_out + 1) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxlo_in - nxlo_out];
}

__global__ void initfftdata_xz_kernel(PPPM_CFLOAT* in, FFT_CFLOAT* out)
{
  if(blockIdx.x < nzhi_out - nzhi_in)
    if(threadIdx.x < nxlo_in - nxlo_out)
      out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x + 2 * (nxhi_in + 1) - nxlo_in - nxhi_out)] += in[(((blockIdx.x + nzhi_in - nzlo_out + 1) * (nyhi_out - nylo_out + 1) + blockIdx.y + nylo_in - nylo_out) * (nxhi_out - nxlo_out + 1)) + threadIdx.x];

  if(blockIdx.x < nzhi_out - nzhi_in)
    if(threadIdx.x < nxhi_out - nxhi_in)
      out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x + nzhi_in - nzlo_out + 1) * (nyhi_out - nylo_out + 1) + blockIdx.y + nylo_in - nylo_out) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxhi_in - nxlo_out + 1];

  if(slabflag) {
    if(blockIdx.x < nzlo_in - nzlo_out)
      if(threadIdx.x < nxlo_in - nxlo_out)
        out[2 * ((((nzhi_in - nzlo_in + 2 - nupper - slabflag + blockIdx.x) * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x + 2 * (nxhi_in + 1) - nxlo_in - nxhi_out)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y + nylo_in - nylo_out) * (nxhi_out - nxlo_out + 1)) + threadIdx.x];

    if(blockIdx.x < nzlo_in - nzlo_out)
      if(threadIdx.x < nxhi_out - nxhi_in)
        out[2 * ((((nzhi_in - nzlo_in + 2 - nupper - slabflag + blockIdx.x) * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y + nylo_in - nylo_out) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxhi_in - nxlo_out + 1];
  } else {
    if(blockIdx.x < nzlo_in - nzlo_out)
      if(threadIdx.x < nxlo_in - nxlo_out)
        out[2 * ((((blockIdx.x + 2 * (nzhi_in + 1) - nzlo_in - nzhi_out) * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x + 2 * (nxhi_in + 1) - nxlo_in - nxhi_out)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y + nylo_in - nylo_out) * (nxhi_out - nxlo_out + 1)) + threadIdx.x];

    if(blockIdx.x < nzlo_in - nzlo_out)
      if(threadIdx.x < nxhi_out - nxhi_in)
        out[2 * ((((blockIdx.x + 2 * (nzhi_in + 1) - nzlo_in - nzhi_out) * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y + nylo_in - nylo_out) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxhi_in - nxlo_out + 1];
  }
}

__global__ void initfftdata_xy_kernel(PPPM_CFLOAT* in, FFT_CFLOAT* out)
{
  if(blockIdx.y < nyhi_out - nyhi_in)
    if(threadIdx.x < nxlo_in - nxlo_out)
      out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x + 2 * (nxhi_in + 1) - nxlo_in - nxhi_out)] += in[(((blockIdx.x + nzlo_in - nzlo_out) * (nyhi_out - nylo_out + 1) + blockIdx.y + nyhi_in - nylo_out + 1) * (nxhi_out - nxlo_out + 1)) + threadIdx.x];

  if(blockIdx.y < nyhi_out - nyhi_in)
    if(threadIdx.x < nxhi_out - nxhi_in)
      out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x + nzlo_in - nzlo_out) * (nyhi_out - nylo_out + 1) + blockIdx.y + nyhi_in - nylo_out + 1) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxhi_in - nxlo_out + 1];

  if(blockIdx.y < nylo_in - nylo_out)
    if(threadIdx.x < nxlo_in - nxlo_out)
      out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y + 2 * (nyhi_in + 1) - nylo_in - nyhi_out) * (nxhi_in - nxlo_in + 1)) + threadIdx.x + 2 * (nxhi_in + 1) - nxlo_in - nxhi_out)] += in[(((blockIdx.x + nzlo_in - nzlo_out) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1)) + threadIdx.x];

  if(blockIdx.y < nylo_in - nylo_out)
    if(threadIdx.x < nxhi_out - nxhi_in)
      out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y + 2 * (nyhi_in + 1) - nylo_in - nyhi_out) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x + nzlo_in - nzlo_out) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxhi_in - nxlo_out + 1];
}

__global__ void initfftdata_xyz_kernel(PPPM_CFLOAT* in, FFT_CFLOAT* out)
{
  if(blockIdx.x < nzhi_out - nzhi_in)
    if(blockIdx.y < nyhi_out - nyhi_in)
      if(threadIdx.x < nxlo_in - nxlo_out)
        out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x + 2 * (nxhi_in + 1) - nxlo_in - nxhi_out)] += in[(((blockIdx.x + nzhi_in - nzlo_out + 1) * (nyhi_out - nylo_out + 1) + blockIdx.y + nyhi_in - nylo_out + 1) * (nxhi_out - nxlo_out + 1)) + threadIdx.x];

  if(blockIdx.x < nzhi_out - nzhi_in)
    if(blockIdx.y < nyhi_out - nyhi_in)
      if(threadIdx.x < nxhi_out - nxhi_in)
        out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x + nzhi_in - nzlo_out + 1) * (nyhi_out - nylo_out + 1) + blockIdx.y + nyhi_in - nylo_out + 1) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxhi_in - nxlo_out + 1];

  if(blockIdx.x < nzhi_out - nzhi_in)
    if(blockIdx.y < nylo_in - nylo_out)
      if(threadIdx.x < nxlo_in - nxlo_out)
        out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y + 2 * (nyhi_in + 1) - nylo_in - nyhi_out) * (nxhi_in - nxlo_in + 1)) + threadIdx.x + 2 * (nxhi_in + 1) - nxlo_in - nxhi_out)] += in[(((blockIdx.x + nzhi_in - nzlo_out + 1) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1)) + threadIdx.x];

  if(blockIdx.x < nzhi_out - nzhi_in)
    if(blockIdx.y < nylo_in - nylo_out)
      if(threadIdx.x < nxhi_out - nxhi_in)
        out[2 * (((blockIdx.x * (nyhi_in - nylo_in + 1) + blockIdx.y + 2 * (nyhi_in + 1) - nylo_in - nyhi_out) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x + nzhi_in - nzlo_out + 1) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxhi_in - nxlo_out + 1];

  if(slabflag) {
    if(blockIdx.x < nzlo_in - nzlo_out)
      if(blockIdx.y < nyhi_out - nyhi_in)
        if(threadIdx.x < nxlo_in - nxlo_out)
          out[2 * ((((nzhi_in - nzlo_in + 2 - nupper - slabflag + blockIdx.x) * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x + 2 * (nxhi_in + 1) - nxlo_in - nxhi_out)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y + nyhi_in - nylo_out + 1) * (nxhi_out - nxlo_out + 1)) + threadIdx.x];

    if(blockIdx.x < nzlo_in - nzlo_out)
      if(blockIdx.y < nyhi_out - nyhi_in)
        if(threadIdx.x < nxhi_out - nxhi_in)
          out[2 * ((((nzhi_in - nzlo_in + 2 - nupper - slabflag + blockIdx.x) * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y + nyhi_in - nylo_out + 1) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxhi_in - nxlo_out + 1];

    if(blockIdx.x < nzlo_in - nzlo_out)
      if(blockIdx.y < nylo_in - nylo_out)
        if(threadIdx.x < nxlo_in - nxlo_out)
          out[2 * ((((nzhi_in - nzlo_in + 2 - nupper - slabflag + blockIdx.x) * (nyhi_in - nylo_in + 1) + blockIdx.y + 2 * (nyhi_in + 1) - nylo_in - nyhi_out) * (nxhi_in - nxlo_in + 1)) + threadIdx.x + 2 * (nxhi_in + 1) - nxlo_in - nxhi_out)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1)) + threadIdx.x];

    if(blockIdx.x < nzlo_in - nzlo_out)
      if(blockIdx.y < nylo_in - nylo_out)
        if(threadIdx.x < nxhi_out - nxhi_in)
          out[2 * ((((nzhi_in - nzlo_in + 2 - nupper - slabflag + blockIdx.x) * (nyhi_in - nylo_in + 1) + blockIdx.y + 2 * (nyhi_in + 1) - nylo_in - nyhi_out) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxhi_in - nxlo_out + 1];
  } else {
    if(blockIdx.x < nzlo_in - nzlo_out)
      if(blockIdx.y < nyhi_out - nyhi_in)
        if(threadIdx.x < nxlo_in - nxlo_out)
          out[2 * ((((blockIdx.x + 2 * (nzhi_in + 1) - nzlo_in - nzhi_out) * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x + 2 * (nxhi_in + 1) - nxlo_in - nxhi_out)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y + nyhi_in - nylo_out + 1) * (nxhi_out - nxlo_out + 1)) + threadIdx.x];

    if(blockIdx.x < nzlo_in - nzlo_out)
      if(blockIdx.y < nyhi_out - nyhi_in)
        if(threadIdx.x < nxhi_out - nxhi_in)
          out[2 * ((((blockIdx.x + 2 * (nzhi_in + 1) - nzlo_in - nzhi_out) * (nyhi_in - nylo_in + 1) + blockIdx.y) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y + nyhi_in - nylo_out + 1) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxhi_in - nxlo_out + 1];

    if(blockIdx.x < nzlo_in - nzlo_out)
      if(blockIdx.y < nylo_in - nylo_out)
        if(threadIdx.x < nxlo_in - nxlo_out)
          out[2 * ((((blockIdx.x + 2 * (nzhi_in + 1) - nzlo_in - nzhi_out) * (nyhi_in - nylo_in + 1) + blockIdx.y + 2 * (nyhi_in + 1) - nylo_in - nyhi_out) * (nxhi_in - nxlo_in + 1)) + threadIdx.x + 2 * (nxhi_in + 1) - nxlo_in - nxhi_out)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1)) + threadIdx.x];

    if(blockIdx.x < nzlo_in - nzlo_out)
      if(blockIdx.y < nylo_in - nylo_out)
        if(threadIdx.x < nxhi_out - nxhi_in)
          out[2 * ((((blockIdx.x + 2 * (nzhi_in + 1) - nzlo_in - nzhi_out) * (nyhi_in - nylo_in + 1) + blockIdx.y + 2 * (nyhi_in + 1) - nylo_in - nyhi_out) * (nxhi_in - nxlo_in + 1)) + threadIdx.x)] += in[(((blockIdx.x) * (nyhi_out - nylo_out + 1) + blockIdx.y) * (nxhi_out - nxlo_out + 1)) + threadIdx.x + nxhi_in - nxlo_out + 1];
  }
}
