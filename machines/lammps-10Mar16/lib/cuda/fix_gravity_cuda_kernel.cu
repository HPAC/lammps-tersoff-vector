#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

__global__ void Cuda_FixGravityCuda_PostForce_Kernel(int groupbit, F_CFLOAT xacc, F_CFLOAT yacc, F_CFLOAT zacc)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nlocal)
    if(_mask[i] & groupbit) {
      F_CFLOAT mass = _rmass_flag ? _rmass[i] : _mass[_type[i]];
      _f[i] += mass * xacc;
      _f[i + 1 * _nmax] += mass * yacc;
      _f[i + 2 * _nmax] += mass * zacc;
    }
}

