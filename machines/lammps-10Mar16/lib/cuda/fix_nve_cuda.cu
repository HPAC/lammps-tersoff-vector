#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX fix_nve_cuda
#define IncludeCommonNeigh
#include "cuda_shared.h"
#include "cuda_common.h"
#include "crm_cuda_utils.cu"
#include "fix_nve_cuda_cu.h"
#include "fix_nve_cuda_kernel.cu"

void Cuda_FixNVECuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(f)       , & sdata->atom.f    .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(rmass)   , & sdata->atom.rmass.dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(mass)    , & sdata->atom.mass.dev_data , sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(type)    , & sdata->atom.type .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(v)       , & sdata->atom.v    .dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(x)       , & sdata->atom.x    .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(xhold)   , & sdata->atom.xhold.dev_data, sizeof(X_CFLOAT*));  //might be moved to a neighbor record in sdata
  hipMemcpyToSymbol(MY_AP(maxhold)   , & sdata->atom.maxhold, sizeof(int));  //might be moved to a neighbor record in sdata
  hipMemcpyToSymbol(MY_AP(reneigh_flag), & sdata->buffer, sizeof(int*));  //might be moved to a neighbor record in sdata
  hipMemcpyToSymbol(MY_AP(triggerneighsq), & sdata->atom.triggerneighsq, sizeof(X_CFLOAT)); //might be moved to a neighbor record in sdata
}

void Cuda_FixNVECuda_UpdateBuffer(cuda_shared_data* sdata)
{
  int size = (unsigned)10 * sizeof(int);

  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_FixNVECuda Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
    CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)

  }

  hipMemcpyToSymbol(MY_AP(buffer) , & sdata->buffer, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(reneigh_flag), & sdata->buffer, sizeof(int*));  //might be moved to a neighbor record in sdata
}

void Cuda_FixNVECuda_Init(cuda_shared_data* sdata, X_CFLOAT dtv, V_CFLOAT dtf)
{
  hipMemcpyToSymbol(MY_AP(mass)    , & sdata->atom.mass.dev_data , sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(dtf)     , & dtf                       		, sizeof(V_CFLOAT));
  hipMemcpyToSymbol(MY_AP(dtv)     , & dtv                            , sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(triggerneighsq), &sdata->atom.triggerneighsq, sizeof(X_CFLOAT));
  hipMemcpyToSymbol(MY_AP(dist_check), & sdata->atom.dist_check       , sizeof(int));
  hipMemcpyToSymbol(MY_AP(rmass_flag), & sdata->atom.rmass_flag       , sizeof(int));       //
  Cuda_FixNVECuda_UpdateNmax(sdata);
}


void Cuda_FixNVECuda_InitialIntegrate(cuda_shared_data* sdata, int groupbit, int mynlocal)//mynlocal can be nfirst if firstgroup==igroup  see cpp
{
  if(sdata->atom.update_nmax)
    Cuda_FixNVECuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal , sizeof(int));

  if(sdata->buffer_new)
    Cuda_FixNVECuda_UpdateBuffer(sdata);

  int3 layout = getgrid(mynlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  hipMemset(sdata->buffer, 0, sizeof(int));
  FixNVECuda_InitialIntegrate_Kernel <<< grid, threads>>> (groupbit);
  hipDeviceSynchronize();
  int reneigh_flag;
  hipMemcpy((void*)(&reneigh_flag), sdata->buffer, sizeof(int), hipMemcpyDeviceToHost);
  sdata->atom.reneigh_flag += reneigh_flag;
  CUT_CHECK_ERROR("Cuda_FixNVECuda_InitialIntegrate_N: fix nve initial integrate Kernel execution failed");

}

void Cuda_FixNVECuda_FinalIntegrate(cuda_shared_data* sdata, int groupbit, int mynlocal)//mynlocal can be nfirst if firstgroup==igroup  see cpp
{
  if(sdata->atom.update_nmax)
    Cuda_FixNVECuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal) , & sdata->atom.nlocal , sizeof(int));

  if(sdata->buffer_new)
    Cuda_FixNVECuda_UpdateBuffer(sdata);

#ifdef CUDA_USE_BINNING

  dim3 grid(sdata->domain.bin_dim[0], sdata->domain.bin_dim[1] * sdata->domain.bin_neighbors[2], 1);
  dim3 threads(sdata->domain.bin_nmax, 1, 1);
  FixNVECuda_FinalIntegrate_Kernel <<< grid, threads>>> (groupbit);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_FixNVECuda_FinalIntegrate: fix nve final integrate (binning) Kernel execution failed");

#else

  int3 layout = getgrid(mynlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  FixNVECuda_FinalIntegrate_Kernel <<< grid, threads>>> (groupbit);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_FixNVECuda_FinalIntegrate: fix nve final integrate Kernel execution failed");

#endif
}

