#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

__global__ void Cuda_CommCuda_PackComm_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_CFLOAT dx, X_CFLOAT dy, X_CFLOAT dz, void* buffer)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];

    if(j > _nmax) _flag[0] = 1;

    ((X_CFLOAT*) buffer)[i] = _x[j] + dx;
    ((X_CFLOAT*) buffer)[i + 1 * n] = _x[j + _nmax] + dy;
    ((X_CFLOAT*) buffer)[i + 2 * n] = _x[j + 2 * _nmax] + dz;
  }
}

__global__ void Cuda_CommCuda_PackCommVel_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_CFLOAT dx, X_CFLOAT dy, X_CFLOAT dz, void* buffer)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];

    if(j > _nmax) _flag[0] = 1;

    ((X_CFLOAT*) buffer)[i] = _x[j] + dx;
    ((X_CFLOAT*) buffer)[i + 1 * n] = _x[j + _nmax] + dy;
    ((X_CFLOAT*) buffer)[i + 2 * n] = _x[j + 2 * _nmax] + dz;
    ((X_CFLOAT*) buffer)[i + 3 * n] = _v[j];
    ((X_CFLOAT*) buffer)[i + 4 * n] = _v[j + _nmax];
    ((X_CFLOAT*) buffer)[i + 5 * n] = _v[j + 2 * _nmax];
  }
}

__global__ void Cuda_CommCuda_PackComm_Self_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_CFLOAT dx, X_CFLOAT dy, X_CFLOAT dz, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = i;
    j = list[i];

    _x[i + first] = _x[j] + dx;
    _x[i + first + _nmax] = _x[j + _nmax] + dy;
    _x[i + first + 2 * _nmax] = _x[j + 2 * _nmax] + dz;
  }
}

__global__ void Cuda_CommCuda_PackCommVel_Self_Kernel(int* sendlist, int n, int maxlistlength, int iswap, X_CFLOAT dx, X_CFLOAT dy, X_CFLOAT dz, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = i;
    j = list[i];

    _x[i + first] = _x[j] + dx;
    _x[i + first + _nmax] = _x[j + _nmax] + dy;
    _x[i + first + 2 * _nmax] = _x[j + 2 * _nmax] + dz;
    _v[i + first] = _v[j];
    _v[i + first + _nmax] = _v[j + _nmax];
    _v[i + first + 2 * _nmax] = _v[j + 2 * _nmax];
  }
}

__global__ void Cuda_CommCuda_UnpackComm_Kernel(int n, int first, void* buffer)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < n) {
    _x[i + first] = ((X_CFLOAT*) buffer)[i];
    _x[i + first + _nmax] = ((X_CFLOAT*) buffer)[i + 1 * n];
    _x[i + first + 2 * _nmax] = ((X_CFLOAT*) buffer)[i + 2 * n];
  }
}


__global__ void Cuda_CommCuda_UnpackCommVel_Kernel(int n, int first, void* buffer)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < n) {
    _x[i + first] = ((X_CFLOAT*) buffer)[i];
    _x[i + first + _nmax] = ((X_CFLOAT*) buffer)[i + 1 * n];
    _x[i + first + 2 * _nmax] = ((X_CFLOAT*) buffer)[i + 2 * n];
    _v[i + first] = ((X_CFLOAT*) buffer)[i + 3 * n];
    _v[i + first + _nmax] = ((X_CFLOAT*) buffer)[i + 4 * n];
    _v[i + first + 2 * _nmax] = ((X_CFLOAT*) buffer)[i + 5 * n];
  }
}

__global__ void Cuda_CommCuda_PackReverse_Kernel(int n, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < n) {
    ((F_CFLOAT*) _buffer)[i] = _f[i + first];
    ((F_CFLOAT*) _buffer)[i + n] = _f[i + first + _nmax];
    ((F_CFLOAT*) _buffer)[i + 2 * n] = _f[i + first + 2 * _nmax];
  }

}

__global__ void Cuda_CommCuda_UnpackReverse_Kernel(int* sendlist, int n, int maxlistlength, int iswap)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];
    _f[j] += ((F_CFLOAT*)_buffer)[i];
    _f[j + _nmax] += ((F_CFLOAT*) _buffer)[i + n];
    _f[j + 2 * _nmax] += ((F_CFLOAT*) _buffer)[i + 2 * n];
  }

}

__global__ void Cuda_CommCuda_UnpackReverse_Self_Kernel(int* sendlist, int n, int maxlistlength, int iswap, int first)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];

    _f[j] += _f[i + first];
    _f[j + _nmax] += _f[i + first + _nmax];
    _f[j + 2 * _nmax] += _f[i + first + 2 * _nmax];
  }

}

extern __shared__ int shared[];

__global__ void Cuda_CommCuda_BuildSendlist_Single(int bordergroup, int ineed, int atom_nfirst,
    int nfirst, int nlast, int dim, int iswap, X_CFLOAT* slablo, X_CFLOAT* slabhi, int* sendlist, int maxlistlength)
{
  int* list = sendlist + iswap * maxlistlength;
  X_CFLOAT lo = slablo[iswap];
  X_CFLOAT hi = slabhi[iswap];
  bool add = false;

  if(!bordergroup || ineed >= 2) {
    int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x + nfirst;

    if(i < nlast)
      if(_x[i + dim * _nmax] >= lo && _x[i + dim * _nmax] <= hi) {
        add = true;
      }

    shared[threadIdx.x] = add ? 1 : 0;

    __syncthreads();

    int nsend = 0;

    if(threadIdx.x == 0) {
      for(int k = 0; k < blockDim.x; k++) {
        if(shared[k]) {
          nsend++;
          shared[k] = nsend;
        }
      }

      shared[blockDim.x] = atomicAdd((int*) _buffer, nsend);
    }

    __syncthreads();

    nsend = shared[blockDim.x] + shared[threadIdx.x] - 1;

    if(add && nsend < maxlistlength)
      list[nsend] = i;


  } else {

    int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

    if(i < atom_nfirst)
      if(_x[i + dim * _nmax] >= lo && _x[i + dim * _nmax] <= hi) {
        add = true;
      }

    shared[threadIdx.x] = add ? 1 : 0;

    __syncthreads();

    int nsend = 0;

    if(threadIdx.x == 0) {
      for(int k = 0; k < blockDim.x; k++) {
        if(shared[k]) {
          nsend++;
          shared[k] = nsend;
        }
      }

      shared[blockDim.x] = atomicAdd((int*) _buffer, nsend);
    }

    __syncthreads();

    nsend = shared[blockDim.x] + shared[threadIdx.x] - 1;

    if(add && nsend < maxlistlength)
      list[nsend] = i;

    __syncthreads();

    add = false;
    i += _nlocal;

    if(i < nlast)
      if(_x[i + dim * _nmax] >= lo && _x[i + dim * _nmax] <= hi) {
        add = true;
      }

    shared[threadIdx.x] = add ? 1 : 0;

    __syncthreads();

    nsend = 0;

    if(threadIdx.x == 0) {
      for(int k = 0; k < blockDim.x; k++) {
        if(shared[k]) {
          nsend++;
          shared[k] = nsend;
        }
      }

      shared[blockDim.x] = atomicAdd((int*) _buffer, nsend);
    }

    __syncthreads();

    nsend = shared[blockDim.x] + shared[threadIdx.x] - 1;

    if(add && nsend < maxlistlength)
      list[nsend] = i;

  }
}


__global__ void Cuda_CommCuda_BuildSendlist_Multi(int bordergroup, int ineed, int atom_nfirst
    , int nfirst, int nlast, int dim, int iswap, X_CFLOAT* multilo, X_CFLOAT* multihi, int* sendlist, int maxlistlength)
{
  int* list = sendlist + iswap * maxlistlength;
  X_CFLOAT* mlo = &multilo[iswap * _cuda_ntypes];
  X_CFLOAT* mhi = &multihi[iswap * _cuda_ntypes];
  int itype = 0;
  bool add = false;

  if(!bordergroup || ineed >= 2) {
    int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x + nfirst;

    if(i < nlast) {
      itype = _type[i];

      if(_x[i + dim * _nmax] >= mlo[itype] && _x[i + dim * _nmax] <= mhi[itype]) {
        add = true;
      }
    }

    shared[threadIdx.x] = add ? 1 : 0;

    __syncthreads();

    int nsend = 0;

    if(threadIdx.x == 0) {
      for(int k = 0; k < blockDim.x; k++) {
        if(shared[k]) {
          nsend++;
          shared[k] = nsend;
        }
      }

      shared[blockDim.x] = atomicAdd((int*) _buffer, nsend);
    }

    __syncthreads();

    nsend = shared[blockDim.x] + shared[threadIdx.x] - 1;

    if(add && nsend < maxlistlength)
      list[nsend] = i;


  } else {

    int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

    if(i < atom_nfirst) {
      itype = _type[i];

      if(_x[i + dim * _nmax] >= mlo[itype] && _x[i + dim * _nmax] <= mhi[itype]) {
        add = true;
      }
    }

    shared[threadIdx.x] = add ? 1 : 0;

    __syncthreads();

    int nsend = 0;

    if(threadIdx.x == 0) {
      for(int k = 0; k < blockDim.x; k++) {
        if(shared[k]) {
          nsend++;
          shared[k] = nsend;
        }
      }

      shared[blockDim.x] = atomicAdd((int*) _buffer, nsend);
    }

    __syncthreads();

    nsend = shared[blockDim.x] + shared[threadIdx.x] - 1;

    if(add && nsend < maxlistlength)
      list[nsend] = i;

    __syncthreads();

    add = false;
    i += _nlocal;

    if(i < nlast) {
      itype = _type[i];

      if(_x[i + dim * _nmax] >= mlo[itype] && _x[i + dim * _nmax] <= mhi[itype]) {
        add = true;
      }
    }

    shared[threadIdx.x] = add ? 1 : 0;

    __syncthreads();

    nsend = 0;

    if(threadIdx.x == 0) {
      for(int k = 0; k < blockDim.x; k++) {
        if(shared[k]) {
          nsend++;
          shared[k] = nsend;
        }
      }

      shared[blockDim.x] = atomicAdd((int*) _buffer, nsend);
    }

    __syncthreads();

    nsend = shared[blockDim.x] + shared[threadIdx.x] - 1;

    if(add && nsend < maxlistlength)
      list[nsend] = i;

  }
}
