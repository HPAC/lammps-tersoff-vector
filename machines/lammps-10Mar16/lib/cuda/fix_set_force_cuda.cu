#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX fix_set_force_cuda
#include "cuda_shared.h"
#include "cuda_common.h"
#include "crm_cuda_utils.cu"

#include "fix_set_force_cuda_cu.h"
#include "fix_set_force_cuda_kernel.cu"

void Cuda_FixSetForceCuda_UpdateBuffer(cuda_shared_data* sdata)
{
  int3 layout = getgrid(sdata->atom.nlocal, 3 * sizeof(F_CFLOAT));
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  int size = (unsigned)(layout.z * layout.y * layout.x) * 3 * sizeof(F_CFLOAT);

  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_FixSetForceCuda Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
    CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)

  }

  hipMemcpyToSymbol(MY_AP(buffer) , & sdata->buffer, sizeof(int*));
}

void Cuda_FixSetForceCuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(x)       , & sdata->atom.x    .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(f)       , & sdata->atom.f    .dev_data, sizeof(F_CFLOAT*));
}

void Cuda_FixSetForceCuda_Init(cuda_shared_data* sdata)
{
  Cuda_FixSetForceCuda_UpdateNmax(sdata);

}


void Cuda_FixSetForceCuda_PostForce(cuda_shared_data* sdata, int groupbit, F_CFLOAT xvalue, F_CFLOAT yvalue, F_CFLOAT zvalue, F_CFLOAT* foriginal, int flagx, int flagy, int flagz)
{
  if(sdata->atom.update_nmax)
    Cuda_FixSetForceCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  if(sdata->buffer_new)
    Cuda_FixSetForceCuda_UpdateBuffer(sdata);


  int3 layout = getgrid(sdata->atom.nlocal, 3 * sizeof(F_CFLOAT));
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  Cuda_FixSetForceCuda_PostForce_Kernel <<< grid, threads, threads.x* 3* sizeof(F_CFLOAT)>>> (groupbit, xvalue, yvalue, zvalue, flagx, flagy, flagz);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Cuda_FixSetForceCuda_PostForce: fix add_force post_force compute Kernel execution failed");

  int oldgrid = grid.x;
  grid.x = 3;
  threads.x = 512;
  Cuda_FixSetForceCuda_Reduce_FOriginal <<< grid, threads, threads.x* sizeof(F_CFLOAT)>>> (oldgrid, foriginal);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Cuda_FixSetForceCuda_PostForce: fix add_force post_force reduce Kernel execution failed");

}
