#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX fix_temp_berendsen_cuda
#include "cuda_shared.h"
#include "cuda_common.h"
#include "crm_cuda_utils.cu"

#include "fix_temp_berendsen_cuda_cu.h"
#include "fix_temp_berendsen_cuda_kernel.cu"


void Cuda_FixTempBerendsenCuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(v)       , & sdata->atom.v    .dev_data, sizeof(X_CFLOAT*));
}

void Cuda_FixTempBerendsenCuda_Init(cuda_shared_data* sdata)
{
  Cuda_FixTempBerendsenCuda_UpdateNmax(sdata);

}


void Cuda_FixTempBerendsenCuda_EndOfStep(cuda_shared_data* sdata, int groupbit, double afactor)
{
  V_CFLOAT factor = afactor;

  if(sdata->atom.update_nmax)
    Cuda_FixTempBerendsenCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int3 layout = getgrid(sdata->atom.nlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  Cuda_FixTempBerendsenCuda_EndOfStep_Kernel <<< grid, threads, 0>>> (groupbit, factor);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Cuda_FixTempBerendsenCuda_PostForce: fix add_force post_force compute Kernel execution failed");
}
