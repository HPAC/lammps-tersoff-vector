#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

extern __shared__ X_CFLOAT sharedmem[];

#define BIG 1e10
__global__ void Domain_PBC_Kernel(int deform_remap, int deform_groupbit, int box_change)
{
  int idim, otherdims;
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  X_CFLOAT lo[3];
  X_CFLOAT hi[3];
  X_CFLOAT* period;

  if(_triclinic == 0) {
    lo[0] = _boxlo[0];
    lo[1] = _boxlo[1];
    lo[2] = _boxlo[2];

    hi[0] = _boxhi[0];
    hi[1] = _boxhi[1];
    hi[2] = _boxhi[2];
    period = _prd;
  } else {
    lo[0] = _boxlo_lamda[0];
    lo[1] = _boxlo_lamda[1];
    lo[2] = _boxlo_lamda[2];

    hi[0] = _boxhi_lamda[0];
    hi[1] = _boxhi_lamda[1];
    hi[2] = _boxhi_lamda[2];
    period = _prd_lamda;
  }


  X_CFLOAT tmpx = X_F(0.5) * (hi[0] + lo[0]);
  X_CFLOAT tmpy = X_F(0.5) * (hi[1] + lo[1]);
  X_CFLOAT tmpz = X_F(0.5) * (hi[2] + lo[2]);

  X_CFLOAT* buf = (X_CFLOAT*) _buffer;
  buf += blockIdx.x * gridDim.y + blockIdx.y;
  buf[0] = tmpx;
  buf += gridDim.x * gridDim.y;
  buf[0] = tmpx;
  buf += gridDim.x * gridDim.y;
  buf[0] = tmpy;
  buf += gridDim.x * gridDim.y;
  buf[0] = tmpy;
  buf += gridDim.x * gridDim.y;
  buf[0] = tmpz;
  buf += gridDim.x * gridDim.y;
  buf[0] = tmpz;

  if(i < _nlocal) {

    if(_periodicity[0]) {
      if(_x[i] < lo[0]) {
        _x[i] += period[0];

        if(deform_remap && _mask[i] & deform_groupbit) _v[i] += _h_rate[0];

        idim = _image[i] & 1023;
        otherdims = _image[i] ^ idim;
        idim--;
        idim &= 1023;
        _image[i] = otherdims | idim;
      }

      if(_x[i] >= hi[0]) {
        _x[i] -= period[0];
        _x[i] = MAX(_x[i], lo[0]);

        if(deform_remap && _mask[i] & deform_groupbit) _v[i] -= _h_rate[0];

        idim = _image[i] & 1023;
        otherdims = _image[i] ^ idim;
        idim++;
        idim &= 1023;
        _image[i] = otherdims | idim;
      }
    }

    if(_periodicity[1]) {
      if(_x[i + _nmax] < lo[1]) {
        _x[i + _nmax] += period[1];

        if(deform_remap && _mask[i] & deform_groupbit) {
          _v[i] += _h_rate[5];
          _v[i + _nmax] += _h_rate[1];
        }

        idim = (_image[i] >> 10) & 1023;
        otherdims = _image[i] ^ (idim << 10);
        idim--;
        idim &= 1023;
        _image[i] = otherdims | (idim << 10);
      }

      if(_x[i + _nmax] >= hi[1]) {
        _x[i + _nmax] -= period[1];
        _x[i + _nmax] = MAX(_x[i + _nmax], lo[1]);

        if(deform_remap && _mask[i] & deform_groupbit) {
          _v[i] -= _h_rate[5];
          _v[i + _nmax] -= _h_rate[1];
        }

        idim = (_image[i] >> 10) & 1023;
        otherdims = _image[i] ^ (idim << 10);
        idim++;
        idim &= 1023;
        _image[i] = otherdims | (idim << 10);
      }
    }

    if(_periodicity[2]) {
      if(_x[i + 2 * _nmax] < lo[2]) {
        _x[i + 2 * _nmax] += period[2];

        if(deform_remap && _mask[i] & deform_groupbit) {
          _v[i] += _h_rate[4];
          _v[i + _nmax] += _h_rate[3];
          _v[i + 2 * _nmax] += _h_rate[2];
        }

        idim = _image[i] >> 20;
        otherdims = _image[i] ^ (idim << 20);
        idim--;
        idim &= 1023;
        _image[i] = otherdims | (idim << 20);
      }

      if(_x[i + 2 * _nmax] >= hi[2]) {
        _x[i + 2 * _nmax] -= period[2];
        _x[i + 2 * _nmax] = MAX(_x[i + 2 * _nmax], lo[2]);

        if(deform_remap && _mask[i] & deform_groupbit) {
          _v[i] -= _h_rate[4];
          _v[i + _nmax] -= _h_rate[3];
          _v[i + 2 * _nmax] -= _h_rate[2];
        }

        idim = _image[i] >> 20;
        otherdims = _image[i] ^ (idim << 20);
        idim++;
        idim &= 1023;
        _image[i] = otherdims | (idim << 20);
      }
    }

    if(box_change) {
      tmpx = _x[i];
      tmpy = _x[i + _nmax];
      tmpz = _x[i + 2 * _nmax];


    }
  }

  __syncthreads();

  if(box_change) {
    X_CFLOAT minx = BIG;
    X_CFLOAT maxx = -BIG;
    X_CFLOAT miny = BIG;
    X_CFLOAT maxy = -BIG;
    X_CFLOAT minz = BIG;
    X_CFLOAT maxz = -BIG;

    if(not _periodicity[0]) {
      sharedmem[threadIdx.x] = tmpx;
      minOfBlock(sharedmem);
      minx = sharedmem[0];
      __syncthreads();
      sharedmem[threadIdx.x] = tmpx;
      maxOfBlock(sharedmem);
      maxx = sharedmem[0];
      __syncthreads();
    } else {
      minx = lo[0];
      maxx = hi[0];
    }

    if(not _periodicity[1]) {
      sharedmem[threadIdx.x] = tmpy;
      minOfBlock(sharedmem);
      miny = sharedmem[0];
      __syncthreads();
      sharedmem[threadIdx.x] = tmpy;
      maxOfBlock(sharedmem);
      maxy = sharedmem[0];
      __syncthreads();
    } else {
      minx = lo[1];
      maxx = hi[1];
    }

    if(not _periodicity[2]) {
      sharedmem[threadIdx.x] = tmpz;
      minOfBlock(sharedmem);
      minz = sharedmem[0];
      __syncthreads();
      sharedmem[threadIdx.x] = tmpz;
      maxOfBlock(sharedmem);
      maxz = sharedmem[0];
      __syncthreads();
    } else {
      minz = lo[2];
      maxz = hi[2];
    }

    if(threadIdx.x == 0) {
      buf = (X_CFLOAT*) _buffer;
      buf += blockIdx.x * gridDim.y + blockIdx.y;
      buf[0] = minx;
      buf += gridDim.x * gridDim.y;
      buf[0] = maxx;
      buf += gridDim.x * gridDim.y;
      buf[0] = miny;
      buf += gridDim.x * gridDim.y;
      buf[0] = maxy;
      buf += gridDim.x * gridDim.y;
      buf[0] = minz;
      buf += gridDim.x * gridDim.y;
      buf[0] = maxz;
    }
  }
}

__global__ void Domain_reduceBoxExtent(double* extent, int n)
{
  X_CFLOAT* buf = (X_CFLOAT*) _buffer;
  buf += blockIdx.x * n;
  copyGlobToShared(buf, sharedmem, n);

  if(blockIdx.x % 2 == 0)
    minOfData(sharedmem, n);
  else
    maxOfData(sharedmem, n);

  extent[blockIdx.x] = sharedmem[0];
}

__global__ void Domain_lamda2x_Kernel(int n)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < n) {
    X_CFLOAT ytmp = _x[i + _nmax];
    X_CFLOAT ztmp = _x[i + 2 * _nmax];
    _x[i] = _h[0] * _x[i] + _h[5] * ytmp + _h[4] * ztmp + _boxlo[0];
    _x[i + _nmax] = _h[1] * ytmp + _h[3] * ztmp + _boxlo[1];
    _x[i + 2 * _nmax] = _h[2] * ztmp + _boxlo[2];
  }
}

__global__ void Domain_x2lamda_Kernel(int n)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  X_CFLOAT delta[3];

  if(i < n) {
    delta[0] = _x[i] - _boxlo[0];
    delta[1] = _x[i + _nmax] - _boxlo[1];
    delta[2] = _x[i + 2 * _nmax] - _boxlo[2];

    _x[i] = _h_inv[0] * delta[0] + _h_inv[5] * delta[1] + _h_inv[4] * delta[2];
    _x[i + _nmax] = _h_inv[1] * delta[1] + _h_inv[3] * delta[2];
    _x[i + 2 * _nmax] = _h_inv[2] * delta[2];
  }
}
