#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */




static __device__ inline F_CFLOAT4 fetchRhor(int i)
{
#ifdef CUDA_USE_TEXTURE
#if F_PRECISION == 1
  return tex1Dfetch(_rhor_spline_tex, i);
#else
  return tex1Dfetch_double_f(_rhor_spline_tex, i);
#endif
#else
  return _rhor_spline[i];
#endif
}

static __device__ inline F_CFLOAT4 fetchZ2r(int i)
{
#ifdef CUDA_USE_TEXTURE
#if F_PRECISION == 1
  return tex1Dfetch(_z2r_spline_tex, i);
#else
  return tex1Dfetch_double_f(_z2r_spline_tex, i);
#endif
#else
  return _z2r_spline[i];
#endif
}

__global__ void PairEAMCuda_Kernel1(int eflag, int vflag, int eflag_atom, int vflag_atom)
{
  ENERGY_CFLOAT* sharedE;
  ENERGY_CFLOAT* sharedV = &sharedmem[threadIdx.x];


  if(eflag || eflag_atom) {
    sharedE = &sharedmem[threadIdx.x];
    sharedE[0] = ENERGY_F(0.0);
    sharedV += blockDim.x;
  }

  if(vflag || vflag_atom) {
    sharedV[0 * blockDim.x] = ENERGY_F(0.0);
    sharedV[1 * blockDim.x] = ENERGY_F(0.0);
    sharedV[2 * blockDim.x] = ENERGY_F(0.0);
    sharedV[3 * blockDim.x] = ENERGY_F(0.0);
    sharedV[4 * blockDim.x] = ENERGY_F(0.0);
    sharedV[5 * blockDim.x] = ENERGY_F(0.0);
  }

  int ii = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  X_CFLOAT xtmp, ytmp, ztmp;
  X_CFLOAT4 myxtype;
  F_CFLOAT delx, dely, delz;
  int itype;
  int i = _nlocal;
  int jnum = 0;
  int* jlist;

  if(ii < _inum) {
    i = _ilist[ii];

    myxtype = fetchXType(i);
    xtmp = myxtype.x;
    ytmp = myxtype.y;
    ztmp = myxtype.z;
    itype = static_cast <int>(myxtype.w);

    jnum = _numneigh[i];

    jlist = &_neighbors[i];

    if(i < _nlocal)
      _rho[i] = F_F(0.0);
  }

  __syncthreads();

  for(int jj = 0; jj < jnum; jj++) {
    if(ii < _inum)
      if(jj < jnum) {
        const int j = jlist[jj * _nlocal];
        myxtype = fetchXType(j);
        delx = xtmp - myxtype.x;
        dely = ytmp - myxtype.y;
        delz = ztmp - myxtype.z;
        int jtype = static_cast <int>(myxtype.w);
        const F_CFLOAT rsq = delx * delx + dely * dely + delz * delz;

        if(rsq < _cutsq_global) {
          F_CFLOAT p = sqrt(rsq) * _rdr + F_F(1.0);
          int m = static_cast<int>(p);
          m = MIN(m, _nr - 1);
          p -= m;
          p = MIN(p, F_F(1.0));

          int k = (static_cast <int>(_type2rhor[jtype * _cuda_ntypes + itype]) * (_nr + 1) + m) * 2;
          F_CFLOAT4 c = fetchRhor(k + 1);
          _rho[i] += ((c.w * p + c.x) * p + c.y) * p + c.z;
        }
      }
  }

  if(ii < _inum) {

    F_CFLOAT p = _rho[i] * _rdrho + F_F(1.0);
    int m = static_cast<int>(p);
    m = MAX(1, MIN(m, _nrho - 1));
    p -= m;
    p = MIN(p, F_F(1.0));
    F_CFLOAT* coeff = &_frho_spline[(static_cast <int>(_type2frho[itype]) * (_nrho + 1) + m) * EAM_COEFF_LENGTH];
    _fp[i] = (coeff[0] * p + coeff[1]) * p + coeff[2];

    if(eflag || eflag_atom) {
      sharedmem[threadIdx.x] += ((coeff[3] * p + coeff[4]) * p + coeff[5]) * p + coeff[6];
    }

  }

  __syncthreads();

  if(eflag || eflag_atom) {
    if(i < _nlocal && eflag_atom)
      _eatom[i] += sharedmem[threadIdx.x];

    reduceBlock(sharedmem);
    ENERGY_CFLOAT* buffer = (ENERGY_CFLOAT*) _buffer;
    buffer[blockIdx.x * gridDim.y + blockIdx.y] = ENERGY_F(2.0) * sharedmem[0];
  }
}

__global__ void PairEAMCuda_Kernel2(int eflag, int vflag, int eflag_atom, int vflag_atom)
{
  ENERGY_CFLOAT evdwl = ENERGY_F(0.0);

  ENERGY_CFLOAT* sharedE;
  ENERGY_CFLOAT* sharedV = &sharedmem[threadIdx.x];


  if(eflag || eflag_atom) {
    sharedE = &sharedmem[threadIdx.x];
    sharedE[0] = ENERGY_F(0.0);
    sharedV += blockDim.x;
  }

  if(vflag || vflag_atom) {
    sharedV[0 * blockDim.x] = ENERGY_F(0.0);
    sharedV[1 * blockDim.x] = ENERGY_F(0.0);
    sharedV[2 * blockDim.x] = ENERGY_F(0.0);
    sharedV[3 * blockDim.x] = ENERGY_F(0.0);
    sharedV[4 * blockDim.x] = ENERGY_F(0.0);
    sharedV[5 * blockDim.x] = ENERGY_F(0.0);
  }

  int ii = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  X_CFLOAT xtmp, ytmp, ztmp;
  X_CFLOAT4 myxtype;
  F_CFLOAT fxtmp, fytmp, fztmp, fpair;
  F_CFLOAT delx, dely, delz;
  int itype, i;
  int jnum = 0;
  int* jlist;

  if(ii < _inum) {
    i = _ilist[ii];

    myxtype = fetchXType(i);
    xtmp = myxtype.x;
    ytmp = myxtype.y;
    ztmp = myxtype.z;
    itype = static_cast <int>(myxtype.w);
    fxtmp = F_F(0.0);
    fytmp = F_F(0.0);
    fztmp = F_F(0.0);

    jnum = _numneigh[i];

    jlist = &_neighbors[i];

    if(i < _nlocal)
      _rho[i] = F_F(0.0);
  }

  if(ii < gridDim.x * gridDim.y) evdwl = ((ENERGY_CFLOAT*) _buffer)[ii];

  __syncthreads();

  for(int jj = 0; jj < jnum; jj++) {
    if(ii < _inum)
      if(jj < jnum) {
        const int j = jlist[jj * _nlocal];
        myxtype = fetchXType(j);
        delx = xtmp - myxtype.x;
        dely = ytmp - myxtype.y;
        delz = ztmp - myxtype.z;
        int jtype = static_cast <int>(myxtype.w);
        const F_CFLOAT rsq = delx * delx + dely * dely + delz * delz;

        if(rsq < _cutsq_global) {
          F_CFLOAT r = _SQRT_(rsq);
          F_CFLOAT p = r * _rdr + F_F(1.0);
          int m = static_cast<int>(p);
          m = MIN(m, _nr - 1);
          p -= m;
          p = MIN(p, F_F(1.0));

          int k = (static_cast <int>(_type2rhor[itype * _cuda_ntypes + jtype]) * (_nr + 1) + m) * 2;
          F_CFLOAT4 c = fetchRhor(k);
          F_CFLOAT rhoip = (c.x * p + c.y) * p + c.z;
          k = (static_cast <int>(_type2rhor[jtype * _cuda_ntypes + itype]) * (_nr + 1) + m) * 2;
          c = fetchRhor(k);
          F_CFLOAT rhojp = (c.x * p + c.y) * p + c.z;
          k = (static_cast <int>(_type2z2r[itype * _cuda_ntypes + jtype]) * (_nr + 1) + m) * 2;
          c = fetchZ2r(k);
          F_CFLOAT z2p = (c.x * p + c.y) * p + c.z;
          c = fetchZ2r(k + 1);
          F_CFLOAT z2 = ((c.w * p + c.x) * p + c.y) * p + c.z;

          F_CFLOAT recip = F_F(1.0) / r;
          F_CFLOAT phi = z2 * recip;
          F_CFLOAT phip = z2p * recip - phi * recip;
          F_CFLOAT psip = _fp[i] * rhojp + _fp[j] * rhoip + phip;
          fpair = -psip * recip;

          F_CFLOAT dxfp, dyfp, dzfp;
          fxtmp += dxfp = delx * fpair;
          fytmp += dyfp = dely * fpair;
          fztmp += dzfp = delz * fpair;
          evdwl += phi;

          if(vflag || vflag_atom) {
            sharedV[0 * blockDim.x] += delx * dxfp;
            sharedV[1 * blockDim.x] += dely * dyfp;
            sharedV[2 * blockDim.x] += delz * dzfp;
            sharedV[3 * blockDim.x] += delx * dyfp;
            sharedV[4 * blockDim.x] += delx * dzfp;
            sharedV[5 * blockDim.x] += dely * dzfp;
          }
        }
      }
  }

  __syncthreads();

  if(ii < _inum) {
    F_CFLOAT* my_f;

    if(_collect_forces_later) {
      ENERGY_CFLOAT* buffer = (ENERGY_CFLOAT*) _buffer;

      if(eflag) {
        buffer = &buffer[1 * gridDim.x * gridDim.y];
      }

      if(vflag) {
        buffer = &buffer[6 * gridDim.x * gridDim.y];
      }

      my_f = (F_CFLOAT*) buffer;
      my_f += i;
      *my_f = fxtmp;
      my_f += _nmax;
      *my_f = fytmp;
      my_f += _nmax;
      *my_f = fztmp;
    } else {
      my_f = _f + i;
      *my_f += fxtmp;
      my_f += _nmax;
      *my_f += fytmp;
      my_f += _nmax;
      *my_f += fztmp;
    }
  }

  __syncthreads();

  if(eflag) {
    sharedE[0] = evdwl;
  }

  if(eflag_atom && i < _nlocal) {
    _eatom[i] += evdwl;
  }

  if(vflag_atom && i < _nlocal) {
    _vatom[i]         += ENERGY_F(0.5) * sharedV[0 * blockDim.x];
    _vatom[i + _nmax]   += ENERGY_F(0.5) * sharedV[1 * blockDim.x];
    _vatom[i + 2 * _nmax] += ENERGY_F(0.5) * sharedV[2 * blockDim.x];
    _vatom[i + 3 * _nmax] += ENERGY_F(0.5) * sharedV[3 * blockDim.x];
    _vatom[i + 4 * _nmax] += ENERGY_F(0.5) * sharedV[4 * blockDim.x];
    _vatom[i + 5 * _nmax] += ENERGY_F(0.5) * sharedV[5 * blockDim.x];
  }

  if(vflag || eflag) PairVirialCompute_A_Kernel(eflag, vflag, 0);
}

__global__ void PairEAMCuda_PackComm_Kernel(int* sendlist, int n, int maxlistlength, int iswap, F_CFLOAT* buffer)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* list = sendlist + iswap * maxlistlength;

  if(i < n) {
    int j = list[i];
    buffer[i] = _fp[j];
  }
}

__global__ void PairEAMCuda_UnpackComm_Kernel(int n, int first, F_CFLOAT* buffer)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < n) {
    _fp[i + first] = buffer[i];
  }
}
