#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

extern __shared__ ENERGY_CFLOAT sharedmem[];

static inline __device__ void PairVirialCompute_A_Kernel(int eflag, int vflag, int coulflag = 0)
{
  __syncthreads();
  ENERGY_CFLOAT* shared = sharedmem;

  if(eflag) {
    reduceBlock(shared);
    shared += blockDim.x;

    if(coulflag) {
      reduceBlock(shared);
      shared += blockDim.x;
    }
  }

  if(vflag) {
    reduceBlock(shared + 0 * blockDim.x);
    reduceBlock(shared + 1 * blockDim.x);
    reduceBlock(shared + 2 * blockDim.x);
    reduceBlock(shared + 3 * blockDim.x);
    reduceBlock(shared + 4 * blockDim.x);
    reduceBlock(shared + 5 * blockDim.x);
  }

  if(threadIdx.x == 0) {
    shared = sharedmem;
    ENERGY_CFLOAT* buffer = (ENERGY_CFLOAT*) _buffer;

    if(eflag) {
      buffer[blockIdx.x * gridDim.y + blockIdx.y] = ENERGY_F(0.5) * shared[0];
      shared += blockDim.x;
      buffer += gridDim.x * gridDim.y;

      if(coulflag) {
        buffer[blockIdx.x * gridDim.y + blockIdx.y] = ENERGY_F(0.5) * shared[0];
        shared += blockDim.x;
        buffer += gridDim.x * gridDim.y;
      }
    }

    if(vflag) {
      buffer[blockIdx.x * gridDim.y + blockIdx.y + 0 * gridDim.x * gridDim.y] = ENERGY_F(0.5) * shared[0 * blockDim.x];
      buffer[blockIdx.x * gridDim.y + blockIdx.y + 1 * gridDim.x * gridDim.y] = ENERGY_F(0.5) * shared[1 * blockDim.x];
      buffer[blockIdx.x * gridDim.y + blockIdx.y + 2 * gridDim.x * gridDim.y] = ENERGY_F(0.5) * shared[2 * blockDim.x];
      buffer[blockIdx.x * gridDim.y + blockIdx.y + 3 * gridDim.x * gridDim.y] = ENERGY_F(0.5) * shared[3 * blockDim.x];
      buffer[blockIdx.x * gridDim.y + blockIdx.y + 4 * gridDim.x * gridDim.y] = ENERGY_F(0.5) * shared[4 * blockDim.x];
      buffer[blockIdx.x * gridDim.y + blockIdx.y + 5 * gridDim.x * gridDim.y] = ENERGY_F(0.5) * shared[5 * blockDim.x];
    }
  }

  __syncthreads();
}

__global__ void MY_AP(PairVirialCompute_reduce)(int n)
{
  sharedmem[threadIdx.x] = ENERGY_F(0.0);
  ENERGY_CFLOAT sum = ENERGY_F(0.0);
  ENERGY_CFLOAT* buf = (ENERGY_CFLOAT*) _buffer;
  buf = &buf[blockIdx.x * n];
  //if(blockIdx.x==2) buf=&buf[n];

  for(int i = 0; i < n; i += blockDim.x) {
    sharedmem[threadIdx.x] = (i + threadIdx.x < n) ? buf[i + threadIdx.x] : ENERGY_F(0.0);
    __syncthreads();
    reduceBlock(sharedmem);

    if(threadIdx.x == 0) sum += sharedmem[0];
  }

  if(threadIdx.x == 0) {
    if(gridDim.x == 1) { //evdwl
      _eng_vdwl[0] += sum;
    }

    if(gridDim.x == 2) { //evdwl + ecoul only
      if(blockIdx.x == 0)
        _eng_vdwl[0] += sum;
      else
        _eng_coul[0] += sum;
    }

    if(gridDim.x == 6) { //virial
      _virial[blockIdx.x] += sum;
    }

    if(gridDim.x == 7) { //evdwl+virial
      if(blockIdx.x == 0)
        _eng_vdwl[0] += sum;
      else _virial[blockIdx.x - 1] += sum;
    }

    if(gridDim.x == 8) { //evdwl+ecoul+virial
      if(blockIdx.x == 0)
        _eng_vdwl[0] += sum;
      else if(blockIdx.x == 1)
        _eng_coul[0] += sum;
      else
        _virial[blockIdx.x - 2] += sum;
    }
  }
}
