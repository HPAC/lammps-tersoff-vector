#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX fix_ave_force_cuda
#include "cuda_shared.h"
#include "cuda_common.h"

#include "crm_cuda_utils.cu"

#include "fix_aveforce_cuda_cu.h"
#include "fix_aveforce_cuda_kernel.cu"

void Cuda_FixAveForceCuda_UpdateBuffer(cuda_shared_data* sdata)
{
  int3 layout = getgrid(sdata->atom.nlocal, 4 * sizeof(F_CFLOAT));
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  int size = (unsigned)(layout.z * layout.y * layout.x) * 4 * sizeof(F_CFLOAT);

  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_FixAveForceCuda Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
    CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)
  }

  hipMemcpyToSymbol(MY_AP(buffer), & sdata->buffer, sizeof(int*));
}

void Cuda_FixAveForceCuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(x)       , & sdata->atom.x    .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(f)       , & sdata->atom.f    .dev_data, sizeof(F_CFLOAT*));
}

void Cuda_FixAveForceCuda_Init(cuda_shared_data* sdata)
{
  Cuda_FixAveForceCuda_UpdateNmax(sdata);
}

void Cuda_FixAveForceCuda_PostForce_FOrg(cuda_shared_data* sdata, int groupbit, F_CFLOAT* aforiginal)
{
  if(sdata->atom.update_nmax)
    Cuda_FixAveForceCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  if(sdata->buffer_new)
    Cuda_FixAveForceCuda_UpdateBuffer(sdata);

  int3 layout = getgrid(sdata->atom.nlocal, 4 * sizeof(F_CFLOAT));
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);


  Cuda_FixAveForceCuda_PostForce_FOrg_Kernel <<< grid, threads, threads.x* 4* sizeof(F_CFLOAT)>>> (groupbit);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_FixAveForceCuda_PostForce: fix ave_force post_force Compute Kernel execution failed");

  int oldgrid = grid.x;
  grid.x = 4;
  threads.x = 512;
  Cuda_FixAveForceCuda_reduce_foriginal <<< grid, threads, threads.x* sizeof(F_CFLOAT)>>> (oldgrid, aforiginal);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_FixAveForceCuda_PostForce: fix ave_force post_force Reduce Kernel execution failed");

}

void Cuda_FixAveForceCuda_PostForce_Set(cuda_shared_data* sdata, int groupbit, int xflag, int yflag, int zflag, F_CFLOAT axvalue, F_CFLOAT ayvalue, F_CFLOAT azvalue)
{
  int3 layout = getgrid(sdata->atom.nlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);


  Cuda_FixAveForceCuda_PostForce_Set_Kernel <<< grid, threads, 0>>> (groupbit, xflag, yflag, zflag, axvalue, ayvalue, azvalue);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_FixAveForceCuda_PostForce_Set: fix ave_force post_force Compute Kernel execution failed");

}
