/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>

#define _lj1 MY_AP(coeff1)
#define _lj2 MY_AP(coeff2)
#define _lj3 MY_AP(coeff3)
#define _lj4 MY_AP(coeff4)

#include "pair_lj_class2_cuda_cu.h"
#include "pair_lj_class2_cuda_kernel_nc.cu"

#include <time.h>

void Cuda_PairLJClass2Cuda_Init(cuda_shared_data* sdata)
{
  Cuda_Pair_Init_AllStyles(sdata, 4);
}

void Cuda_PairLJClass2Cuda(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist, int eflag, int vflag, int eflag_atom, int vflag_atom)
{


  static  short init = 0;

  if(! init) {
    init = 1;
    Cuda_PairLJClass2Cuda_Init(sdata);
  }

  dim3 grid, threads;
  int sharedperproc;

  //int maxthreads=192*sizeof(double)/sizeof(F_CFLOAT);
  //if(CUDA_ARCH==20) maxthreads*=2;
  //hipFuncSetCacheConfig(reinterpret_cast<const void*>(Pair_Kernel_TpA_opt<PAIR_LJ_CUT),COUL_NONE,DATA_NONE>,hipFuncCachePreferL1);
  Cuda_Pair_PreKernel_AllStyles(sdata, sneighlist, eflag, vflag, grid, threads, sharedperproc, false, 192);
  hipStream_t* streams = (hipStream_t*) CudaWrapper_returnStreams();

  if(sdata->pair.use_block_per_atom)
    Pair_Kernel_BpA<PAIR_LJ_CLASS2, COUL_NONE, DATA_NONE>
    <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x, streams[1]>>> (eflag, vflag, eflag_atom, vflag_atom);
  else
    Pair_Kernel_TpA<PAIR_LJ_CLASS2, COUL_NONE, DATA_NONE>
    <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x, streams[1]>>> (eflag, vflag, eflag_atom, vflag_atom);

  Cuda_Pair_PostKernel_AllStyles(sdata, grid, sharedperproc, eflag, vflag);
}


#undef _lj1
#undef _lj2
#undef _lj3
#undef _lj4
