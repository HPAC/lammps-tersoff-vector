#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

extern __shared__ ENERGY_CFLOAT sharedmem[];


__global__ void Cuda_ComputeTempCuda_Scalar_Kernel(int groupbit)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  sharedmem[threadIdx.x] = 0;

  if(i < _nlocal) {
    if(_rmass_flag) {
      if(_mask[i] & groupbit)
        sharedmem[threadIdx.x] = (_v[i] * _v[i] + _v[i + _nmax] * _v[i + _nmax] + _v[i + 2 * _nmax] * _v[i + 2 * _nmax]) * _rmass[i];
    } else {
      if(_mask[i] & groupbit)
        sharedmem[threadIdx.x] = (_v[i] * _v[i] + _v[i + _nmax] * _v[i + _nmax] + _v[i + 2 * _nmax] * _v[i + 2 * _nmax]) * (_mass[_type[i]]);
    }
  }

  reduceBlock(sharedmem);
  ENERGY_CFLOAT* buffer = (ENERGY_CFLOAT*) _buffer;

  if(threadIdx.x == 0) {
    buffer[(blockIdx.x * gridDim.y + blockIdx.y)] = sharedmem[0];
  }
}

__global__ void Cuda_ComputeTempCuda_Vector_Kernel(int groupbit)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  sharedmem[threadIdx.x] = 0;
  sharedmem[threadIdx.x + blockDim.x] = 0;
  sharedmem[threadIdx.x + 2 * blockDim.x] = 0;
  sharedmem[threadIdx.x + 3 * blockDim.x] = 0;
  sharedmem[threadIdx.x + 4 * blockDim.x] = 0;
  sharedmem[threadIdx.x + 5 * blockDim.x] = 0;

  if(i < _nlocal)
    if(_mask[i] & groupbit) {
      V_CFLOAT massone;

      if(_rmass_flag) massone = _rmass[i];
      else massone = _mass[_type[i]];

      sharedmem[threadIdx.x] = massone * _v[i] * _v[i];
      sharedmem[threadIdx.x + blockDim.x] = massone * _v[i + _nmax] * _v[i + _nmax];
      sharedmem[threadIdx.x + 2 * blockDim.x] = massone * _v[i + 2 * _nmax] * _v[i + 2 * _nmax];
      sharedmem[threadIdx.x + 3 * blockDim.x] = massone * _v[i] * _v[i + _nmax];
      sharedmem[threadIdx.x + 4 * blockDim.x] = massone * _v[i] * _v[i + 2 * _nmax];
      sharedmem[threadIdx.x + 5 * blockDim.x] = massone * _v[i + _nmax] * _v[i + 2 * _nmax];
    }

  reduceBlock(sharedmem);
  reduceBlock(&sharedmem[blockDim.x]);
  reduceBlock(&sharedmem[2 * blockDim.x]);
  reduceBlock(&sharedmem[3 * blockDim.x]);
  reduceBlock(&sharedmem[4 * blockDim.x]);
  reduceBlock(&sharedmem[5 * blockDim.x]);
  ENERGY_CFLOAT* buffer = (ENERGY_CFLOAT*) _buffer;

  if(threadIdx.x == 0) {
    buffer[(blockIdx.x * gridDim.y + blockIdx.y)] = sharedmem[0];
    buffer[(blockIdx.x * gridDim.y + blockIdx.y) + gridDim.x * gridDim.y] = sharedmem[blockDim.x];
    buffer[(blockIdx.x * gridDim.y + blockIdx.y) + 2 * gridDim.x * gridDim.y] = sharedmem[2 * blockDim.x];
    buffer[(blockIdx.x * gridDim.y + blockIdx.y) + 3 * gridDim.x * gridDim.y] = sharedmem[3 * blockDim.x];
    buffer[(blockIdx.x * gridDim.y + blockIdx.y) + 4 * gridDim.x * gridDim.y] = sharedmem[4 * blockDim.x];
    buffer[(blockIdx.x * gridDim.y + blockIdx.y) + 5 * gridDim.x * gridDim.y] = sharedmem[5 * blockDim.x];
  }
}


__global__ void Cuda_ComputeTempCuda_Reduce_Kernel(int n, ENERGY_CFLOAT* t)
{
  int i = 0;
  sharedmem[threadIdx.x] = 0;
  ENERGY_CFLOAT myforig = 0.0;
  ENERGY_CFLOAT* buf = (ENERGY_CFLOAT*) _buffer;
  buf = &buf[blockIdx.x * n];

  while(i < n) {
    sharedmem[threadIdx.x] = 0;

    if(i + threadIdx.x < n)
      sharedmem[threadIdx.x] = buf[i + threadIdx.x];

    __syncthreads();
    reduceBlock(sharedmem);
    i += blockDim.x;

    if(threadIdx.x == 0)
      myforig += sharedmem[0];
  }

  if(threadIdx.x == 0)
    t[blockIdx.x] = myforig;
}
