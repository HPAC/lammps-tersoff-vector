#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#ifndef CRM_CUDA_UTILS
#define CRM_CUDA_UTILS

//split n threads into 2 dimensional grid + threads, return values are grid.x grid.y and threads.x
#define MIN(a,b) ((a) < (b) ? (a) : (b))
#define MAX(a,b) ((a) > (b) ? (a) : (b))

inline int3 getgrid(int n, int shared_per_thread = 0, int threadsmax = 256, bool p2 = false)
{
  int3 gridparams;
  int sharedsize = 16000;

  if(shared_per_thread > 0) threadsmax = sharedsize / shared_per_thread < threadsmax ? sharedsize / shared_per_thread : threadsmax;

  if((n < 60 * 32) || (threadsmax < 64))
    gridparams.z = 32;
  else if((n < 60 * 64) || (threadsmax < 128))
    gridparams.z = 64;
  else if((n < 60 * 128) || (threadsmax < 256))
    gridparams.z = 128;
  else if((n < 60 * 256) || (threadsmax < 512))
    gridparams.z = 256;
  else gridparams.z = 512;

  if(p2) {
    gridparams.z = 16;

    while(gridparams.z * 2 <= threadsmax) gridparams.z *= 2;
  }


  int blocks = (n + gridparams.z - 1) / gridparams.z;

  if(blocks > 10000)
    gridparams.x = gridparams.y = int(sqrt(blocks));
  else {
    gridparams.x = blocks;
    gridparams.y = 1;
  }

  while(gridparams.x * gridparams.y * gridparams.z < n) gridparams.x++;

  if(gridparams.x == 0) gridparams.x = 1;

  return gridparams;
}

//return value: 1 if f<0; else: 0
//take care if working with values as "blockId.x-n" for f: it might be interpreted as a unsigned int
static inline __device__ int negativCUDA(float f)
{
  return ((unsigned int)1 << 31 & (__float_as_int(f))) >> 31;
}

//return value: -1 if f<0; else +1
static inline __device__ float fsignCUDA(float f)
{
  return f < 0.0f ? -1.0f : 1.0f;
}

//functions to copy data between global and shared memory (indeed you can copy data between two arbitrary memory regims on device - as long as you have read respectively write rights)
//blockDim.y and blockDim.z are assumed to be 1
static inline __device__ void copySharedToGlob(int* shared, int* glob, const int &n)
{
  int i, k;
  k = n - blockDim.x;

  for(i = 0; i < k; i += blockDim.x) {
    glob[i + threadIdx.x] = shared[i + threadIdx.x];
  }

  if(threadIdx.x < n - i) {
    glob[i + threadIdx.x] = shared[i + threadIdx.x];
  }

  __syncthreads();
}

static inline __device__ void copySharedToGlob(float* shared, float* glob, const int &n)
{
  int i, k;
  k = n - blockDim.x;

  for(i = 0; i < k; i += blockDim.x) {
    glob[i + threadIdx.x] = shared[i + threadIdx.x];
  }

  if(threadIdx.x < n - i) {
    glob[i + threadIdx.x] = shared[i + threadIdx.x];
  }

  __syncthreads();
}

static inline __device__ void copySharedToGlob(double* shared, double* glob, const int &n)
{
  int i, k;
  k = n - blockDim.x;

  for(i = 0; i < k; i += blockDim.x) {
    glob[i + threadIdx.x] = shared[i + threadIdx.x];
  }

  if(threadIdx.x < n - i) {
    glob[i + threadIdx.x] = shared[i + threadIdx.x];
  }

  __syncthreads();
}

static inline __device__ void copyGlobToShared(int* glob, int* shared, const int &n)
{
  int i, k;
  k = n - blockDim.x;

  for(i = 0; i < k; i += blockDim.x) {
    shared[i + threadIdx.x] = glob[i + threadIdx.x];
  }

  if(threadIdx.x < n - i) {
    shared[i + threadIdx.x] = glob[i + threadIdx.x];
  }

  __syncthreads();
}

static __device__ inline void copyGlobToShared(float* glob, float* shared, const int &n)
{
  int i, k;
  k = n - blockDim.x;

  for(i = 0; i < k; i += blockDim.x) {
    shared[i + threadIdx.x] = glob[i + threadIdx.x];
  }

  if(threadIdx.x < n - i) {
    shared[i + threadIdx.x] = glob[i + threadIdx.x];
  }

  __syncthreads();
}

static __device__ inline void copyGlobToShared(double* glob, double* shared, const int &n)
{
  int i;

  for(i = 0; i < n - blockDim.x; i += blockDim.x) {
    shared[i + threadIdx.x] = glob[i + threadIdx.x];
  }

  if(threadIdx.x < n - i) {
    shared[i + threadIdx.x] = glob[i + threadIdx.x];
  }

  __syncthreads();
}

//copy data between two memory areas on device, 3d BlockDims are allowed
static __device__ inline void copyData(double* source, double* target, const int &n)
{
  int i;
  int offset = threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z;

  for(i = 0; i < n - blockDim.x * blockDim.y * blockDim.z; i += blockDim.x * blockDim.y * blockDim.z) {
    target[i + offset] = source[i + offset];
  }

  if(offset < n - i) {
    target[i + offset] = source[i + offset];
  }

  __syncthreads();
}

static __device__ inline void copyData(float* source, float* target, const int &n)
{
  int i;
  int offset = threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z;

  for(i = 0; i < n - blockDim.x * blockDim.y * blockDim.z; i += blockDim.x * blockDim.y * blockDim.z) {
    target[i + offset] = source[i + offset];
  }

  if(offset < n - i) {
    target[i + offset] = source[i + offset];
  }

  __syncthreads();
}

static __device__ inline void copyData(int* source, int* target, const int &n)
{
  int i;
  int offset = threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z;

  for(i = 0; i < n - blockDim.x * blockDim.y * blockDim.z; i += blockDim.x * blockDim.y * blockDim.z) {
    target[i + offset] = source[i + offset];
  }

  if(offset < n - i) {
    target[i + offset] = source[i + offset];
  }

  __syncthreads();
}

static __device__ inline void copyData(unsigned int* source, unsigned int* target, const int &n)
{
  int i;
  int offset = threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z;

  for(i = 0; i < n - blockDim.x * blockDim.y * blockDim.z; i += blockDim.x * blockDim.y * blockDim.z) {
    target[i + offset] = source[i + offset];
  }

  if(offset < n - i) {
    target[i + offset] = source[i + offset];
  }

  __syncthreads();
}

//functions in order to sum over values of one block. P2 means blockdim MUST be a power of 2 otherwise the behaviour is not well defined
//in the end in data[0]=sum_i=0^blockDim.x data[i]
//for reduceBlockP2 and reduceBlock blockDim.y=1 and blockDim.z=1
static __device__ inline void reduceBlockP2(int* data)
{
  __syncthreads();

  for(int i = 2; i <= blockDim.x; i *= 2) {
    if(threadIdx.x < blockDim.x / i)
      data[threadIdx.x] += data[threadIdx.x + blockDim.x / i];

    __syncthreads();
  }
}

static __device__ inline void reduceBlockP2(unsigned int* data)
{
  __syncthreads();

  for(int i = 2; i <= blockDim.x; i *= 2) {
    if(threadIdx.x < blockDim.x / i)
      data[threadIdx.x] += data[threadIdx.x + blockDim.x / i];

    __syncthreads();
  }
}

static __device__ inline void reduceBlockP2(float* data)
{
  __syncthreads();

  for(int i = 2; i <= blockDim.x; i *= 2) {
    if(threadIdx.x < blockDim.x / i)
      data[threadIdx.x] += data[threadIdx.x + blockDim.x / i];

    __syncthreads();
  }
}

static __device__ inline void reduceBlockP2(double* data)
{
  __syncthreads();

  for(int i = 2; i <= blockDim.x; i *= 2) {
    if(threadIdx.x < blockDim.x / i)
      data[threadIdx.x] += data[threadIdx.x + blockDim.x / i];

    __syncthreads();
  }
}

static __device__ inline void reduceBlock(float* data)
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < blockDim.x) p2 *= 2;

  if(threadIdx.x < blockDim.x - p2)
    data[threadIdx.x] += data[threadIdx.x + p2];

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    if(threadIdx.x < p2 / i)
      data[threadIdx.x] += data[threadIdx.x + p2 / i];

    __syncthreads();
  }
}

static __device__ inline void reduceBlock(int* data)
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < blockDim.x) p2 *= 2;

  if(threadIdx.x < blockDim.x - p2)
    data[threadIdx.x] += data[threadIdx.x + p2];

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    if(threadIdx.x < p2 / i)
      data[threadIdx.x] += data[threadIdx.x + p2 / i];

    __syncthreads();
  }
}

static __device__ inline void reduceBlock(unsigned int* data)
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < blockDim.x) p2 *= 2;

  if(threadIdx.x < blockDim.x - p2)
    data[threadIdx.x] += data[threadIdx.x + p2];

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    if(threadIdx.x < p2 / i)
      data[threadIdx.x] += data[threadIdx.x + p2 / i];

    __syncthreads();
  }
}

static __device__ inline void reduceBlock(double* data)
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < blockDim.x) p2 *= 2;

  if(threadIdx.x < blockDim.x - p2)
    data[threadIdx.x] += data[threadIdx.x + p2];

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    if(threadIdx.x < p2 / i)
      data[threadIdx.x] += data[threadIdx.x + p2 / i];

    __syncthreads();
  }
}

static __device__ inline void cudaFillBlockData_int(int* data, const int &n, const int &value)
{
  int i;

  for(i = 0; i < n - blockDim.x; i += blockDim.x) {
    data[i + threadIdx.x] = value;
  }

  if(threadIdx.x < n - i) data[i + threadIdx.x] = value;
}

static __device__ inline void cudaFillBlockData_float(float* data, const int &n, const float &value)
{
  int i;

  for(i = 0; i < n - blockDim.x; i += blockDim.x) {
    data[i + threadIdx.x] = value;
  }

  if(threadIdx.x < n - i) data[i + threadIdx.x] = value;
}

static __device__ inline void reduce(float* data, int n) //cautious not sure if working
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < n) p2 *= 2;

  int j = 0;

  while((threadIdx.x + blockDim.x * j) * 2 < n - p2) {
    data[threadIdx.x + blockDim.x * j] += data[(threadIdx.x + blockDim.x * j) + p2];
    j++;
  }

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    while((threadIdx.x + blockDim.x * j) < p2 / i) {
      data[threadIdx.x + blockDim.x * j] += data[(threadIdx.x + blockDim.x * j) + p2 / i];
      j++;
    }

    __syncthreads();
  }
}

static __device__ inline void reduce(double* data, int n) //cautious not sure if working
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < n) p2 *= 2;

  int j = 0;

  while((threadIdx.x + blockDim.x * j) * 2 < n - p2) {
    data[threadIdx.x + blockDim.x * j] += data[(threadIdx.x + blockDim.x * j) + p2];
    j++;
  }

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    while((threadIdx.x + blockDim.x * j) < p2 / i) {
      data[threadIdx.x + blockDim.x * j] += data[(threadIdx.x + blockDim.x * j) + p2 / i];
      j++;
    }

    __syncthreads();
  }
}

static __device__ inline void minOfBlock(float* data)
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < blockDim.x) p2 *= 2;

  if(threadIdx.x < blockDim.x - p2)
    data[threadIdx.x] = MIN(data[threadIdx.x + p2], data[threadIdx.x]);

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    if(threadIdx.x < p2 / i)
      data[threadIdx.x] = MIN(data[threadIdx.x + p2 / i], data[threadIdx.x]);

    __syncthreads();
  }
}

static __device__ inline void maxOfBlock(float* data)
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < blockDim.x) p2 *= 2;

  if(threadIdx.x < blockDim.x - p2)
    data[threadIdx.x] = MAX(data[threadIdx.x + p2], data[threadIdx.x]);

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    if(threadIdx.x < p2 / i)
      data[threadIdx.x] = MAX(data[threadIdx.x + p2 / i], data[threadIdx.x]);

    __syncthreads();
  }
}

static __device__ inline void minOfBlock(double* data)
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < blockDim.x) p2 *= 2;

  if(threadIdx.x < blockDim.x - p2)
    data[threadIdx.x] = MIN(data[threadIdx.x + p2], data[threadIdx.x]);

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    if(threadIdx.x < p2 / i)
      data[threadIdx.x] = MIN(data[threadIdx.x + p2 / i], data[threadIdx.x]);

    __syncthreads();
  }
}

static __device__ inline void maxOfBlock(double* data)
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < blockDim.x) p2 *= 2;

  if(threadIdx.x < blockDim.x - p2)
    data[threadIdx.x] = MAX(data[threadIdx.x + p2], data[threadIdx.x]);

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    if(threadIdx.x < p2 / i)
      data[threadIdx.x] = MAX(data[threadIdx.x + p2 / i], data[threadIdx.x]);

    __syncthreads();
  }
}


static __device__ inline void minOfData(double* data, int n) //cautious not sure if working
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < n) p2 *= 2;

  int j = 0;

  while((threadIdx.x + blockDim.x * j) < n - p2) {
    data[threadIdx.x + blockDim.x * j] = MIN(data[threadIdx.x + blockDim.x * j], data[(threadIdx.x + blockDim.x * j) + p2]);
    j++;
  }

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    while((threadIdx.x + blockDim.x * j) < p2 / i) {
      data[threadIdx.x + blockDim.x * j] = MIN(data[threadIdx.x + blockDim.x * j], data[(threadIdx.x + blockDim.x * j) + p2 / i]);
      j++;
    }

    __syncthreads();
  }
}

static __device__ inline void maxOfData(double* data, int n) //cautious not sure if working
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < n) p2 *= 2;

  int j = 0;

  while((threadIdx.x + blockDim.x * j) < n - p2) {
    data[threadIdx.x + blockDim.x * j] = MAX(data[threadIdx.x + blockDim.x * j], data[(threadIdx.x + blockDim.x * j) + p2]);
    j++;
  }

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    while((threadIdx.x + blockDim.x * j) < p2 / i) {
      data[threadIdx.x + blockDim.x * j] = MAX(data[threadIdx.x + blockDim.x * j], data[(threadIdx.x + blockDim.x * j) + p2 / i]);
      j++;
    }

    __syncthreads();
  }
}

static __device__ inline void minOfData(float* data, int n) //cautious not sure if working
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < n) p2 *= 2;

  int j = 0;

  while((threadIdx.x + blockDim.x * j) < n - p2) {
    data[threadIdx.x + blockDim.x * j] = MIN(data[threadIdx.x + blockDim.x * j], data[(threadIdx.x + blockDim.x * j) + p2]);
    j++;
  }

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    while((threadIdx.x + blockDim.x * j) < p2 / i) {
      data[threadIdx.x + blockDim.x * j] = MIN(data[threadIdx.x + blockDim.x * j], data[(threadIdx.x + blockDim.x * j) + p2 / i]);
      j++;
    }

    __syncthreads();
  }
}

static __device__ inline void maxOfData(float* data, int n) //cautious not sure if working
{
  __syncthreads();
  int p2 = 1;

  while(p2 * 2 < n) p2 *= 2;

  int j = 0;

  while((threadIdx.x + blockDim.x * j) < n - p2) {
    data[threadIdx.x + blockDim.x * j] = MAX(data[threadIdx.x + blockDim.x * j], data[(threadIdx.x + blockDim.x * j) + p2]);
    j++;
  }

  __syncthreads();

  for(int i = 2; i <= p2; i *= 2) {
    while((threadIdx.x + blockDim.x * j) < p2 / i) {
      data[threadIdx.x + blockDim.x * j] = MAX(data[threadIdx.x + blockDim.x * j], data[(threadIdx.x + blockDim.x * j) + p2 / i]);
      j++;
    }

    __syncthreads();
  }
}

#if X_PRECISION == 2
static __device__ inline double tex1Dfetch_double(texture<int2, 1> t, int i)
{
  int2 v = tex1Dfetch(t, i);
  return __hiloint2double(v.y, v.x);
}

static __device__ inline X_CFLOAT4 tex1Dfetch_double(texture<int4, 1> t, int i)
{
  int4 v = tex1Dfetch(t, 2 * i);
  int4 u = tex1Dfetch(t, 2 * i + 1);
  X_CFLOAT4 w;

  w.x = __hiloint2double(v.y, v.x);
  w.y = __hiloint2double(v.w, v.z);
  w.z = __hiloint2double(u.y, u.x);
  w.w = __hiloint2double(u.w, u.z);
  return w;
}
#endif

inline void BindXTypeTexture(cuda_shared_data* sdata)
{
#ifdef CUDA_USE_TEXTURE
  _x_type_tex.normalized = false;                      // access with normalized texture coordinates
  _x_type_tex.filterMode = hipFilterModePoint;        // Point mode, so no
  _x_type_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
  const textureReference* x_type_texture_ptr = &MY_AP(x_type_tex);

#if X_PRECISION == 1
  hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<float4>();
  hipBindTexture(0, x_type_texture_ptr, sdata->atom.x_type.dev_data, &channelDescXType, sdata->atom.nmax * sizeof(X_CFLOAT4));
#else
  hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<int4>();
  hipBindTexture(0, x_type_texture_ptr, sdata->atom.x_type.dev_data, &channelDescXType, sdata->atom.nmax * 2 * sizeof(int4));
#endif
#endif
}

static __device__ inline X_CFLOAT4 fetchXType(int i)
{
#ifdef CUDA_USE_TEXTURE
#if X_PRECISION == 1
  return tex1Dfetch(_x_type_tex, i);
#else
  return tex1Dfetch_double(_x_type_tex, i);
#endif
#else
  return _x_type[i];
#endif
}

#if V_PRECISION == 2
static __device__ inline double tex1Dfetch_double_v(texture<int2, 1> t, int i)
{
  int2 v = tex1Dfetch(t, i);
  return __hiloint2double(v.y, v.x);
}

static __device__ inline V_CFLOAT4 tex1Dfetch_double_v(texture<int4, 1> t, int i)
{
  int4 v = tex1Dfetch(t, 2 * i);
  int4 u = tex1Dfetch(t, 2 * i + 1);
  V_CFLOAT4 w;

  w.x = __hiloint2double(v.y, v.x);
  w.y = __hiloint2double(v.w, v.z);
  w.z = __hiloint2double(u.y, u.x);
  w.w = __hiloint2double(u.w, u.z);
  return w;
}
#endif

inline void BindVRadiusTexture(cuda_shared_data* sdata)
{
#ifdef CUDA_USE_TEXTURE
  _v_radius_tex.normalized = false;                      // access with normalized texture coordinates
  _v_radius_tex.filterMode = hipFilterModePoint;        // Point mode, so no
  _v_radius_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
  const textureReference* v_radius_texture_ptr = &MY_AP(v_radius_tex);

#if V_PRECISION == 1
  hipChannelFormatDesc channelDescVRadius = hipCreateChannelDesc<float4>();
  hipBindTexture(0, v_radius_texture_ptr, sdata->atom.v_radius.dev_data, &channelDescVRadius, sdata->atom.nmax * sizeof(X_CFLOAT4));
#else
  hipChannelFormatDesc channelDescVRadius = hipCreateChannelDesc<int4>();
  hipBindTexture(0, v_radius_texture_ptr, sdata->atom.v_radius.dev_data, &channelDescVRadius, sdata->atom.nmax * 2 * sizeof(int4));
#endif
#endif
}

static __device__ inline V_CFLOAT4 fetchVRadius(int i)
{
#ifdef CUDA_USE_TEXTURE
#if V_PRECISION == 1
  return tex1Dfetch(_v_radius_tex, i);
#else
  return tex1Dfetch_double_v(_v_radius_tex, i);
#endif
#else
  return _v_radius[i];
#endif
}

inline void BindOmegaRmassTexture(cuda_shared_data* sdata)
{
#ifdef CUDA_USE_TEXTURE
  _omega_rmass_tex.normalized = false;                      // access with normalized texture coordinates
  _omega_rmass_tex.filterMode = hipFilterModePoint;        // Point mode, so no
  _omega_rmass_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
  const textureReference* omega_rmass_texture_ptr = &MY_AP(omega_rmass_tex);

#if V_PRECISION == 1
  hipChannelFormatDesc channelDescOmegaRmass = hipCreateChannelDesc<float4>();
  hipBindTexture(0, omega_rmass_texture_ptr, sdata->atom.omega_rmass.dev_data, &channelDescOmegaRmass, sdata->atom.nmax * sizeof(X_CFLOAT4));
#else
  hipChannelFormatDesc channelDescOmegaRmass = hipCreateChannelDesc<int4>();
  hipBindTexture(0, omega_rmass_texture_ptr, sdata->atom.omega_rmass.dev_data, &channelDescOmegaRmass, sdata->atom.nmax * 2 * sizeof(int4));
#endif
#endif
}

static __device__ inline V_CFLOAT4 fetchOmegaRmass(int i)
{
#ifdef CUDA_USE_TEXTURE
#if V_PRECISION == 1
  return tex1Dfetch(_omega_rmass_tex, i);
#else
  return tex1Dfetch_double_v(_omega_rmass_tex, i);
#endif
#else
  return _omega_rmass[i];
#endif
}

#if F_PRECISION == 2
static __device__ inline double tex1Dfetch_double_f(texture<int2, 1> t, int i)
{
  int2 v = tex1Dfetch(t, i);
  return __hiloint2double(v.y, v.x);
}

static __device__ inline F_CFLOAT4 tex1Dfetch_double_f(texture<int4, 1> t, int i)
{
  int4 v = tex1Dfetch(t, 2 * i);
  int4 u = tex1Dfetch(t, 2 * i + 1);
  F_CFLOAT4 w;

  w.x = __hiloint2double(v.y, v.x);
  w.y = __hiloint2double(v.w, v.z);
  w.z = __hiloint2double(u.y, u.x);
  w.w = __hiloint2double(u.w, u.z);
  return w;
}
#endif

inline void BindQTexture(cuda_shared_data* sdata)
{
#ifdef CUDA_USE_TEXTURE
  _q_tex.normalized = false;                      // access with normalized texture coordinates
  _q_tex.filterMode = hipFilterModePoint;        // Point mode, so no
  _q_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
  const textureReference* q_texture_ptr = &MY_AP(q_tex);

#if F_PRECISION == 1
  hipChannelFormatDesc channelDescQ = hipCreateChannelDesc<float>();
  hipBindTexture(0, q_texture_ptr, sdata->atom.q.dev_data, &channelDescQ, sdata->atom.nmax * sizeof(F_CFLOAT));
#else
  hipChannelFormatDesc channelDescQ = hipCreateChannelDesc<int2>();
  hipBindTexture(0, q_texture_ptr, sdata->atom.q.dev_data, &channelDescQ, sdata->atom.nmax * sizeof(int2));
#endif
#endif
}

static __device__ inline F_CFLOAT fetchQ(int i)
{
#ifdef CUDA_USE_TEXTURE
#if F_PRECISION == 1
  return tex1Dfetch(_q_tex, i);
#else
  return tex1Dfetch_double_f(_q_tex, i);
#endif
#else
  return _q[i];
#endif
}

#endif

/*

inline void BindPairCoeffTypeTexture(cuda_shared_data* sdata,coeff_tex)
{
	#ifdef CUDA_USE_TEXTURE
		_coeff_tex.normalized = false;                      // access with normalized texture coordinates
		_coeff_tex.filterMode = hipFilterModePoint;        // Point mode, so no
		_coeff_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates
		const textureReference* coeff_texture_ptr;
		hipGetTextureReference(&coeff_texture_ptr, HIP_SYMBOL(&MY_AP(coeff_tex)));

		#if F_PRECISION == 1
		hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<float4>();
		hipBindTexture(0,x_type_texture_ptr, sdata->atom.x_type.dev_data, &channelDescXType, sdata->atom.nmax*sizeof(X_CFLOAT4));
		#else
		hipChannelFormatDesc channelDescXType = hipCreateChannelDesc<int4>();
		hipBindTexture(0,x_type_texture_ptr, sdata->atom.x_type.dev_data, &channelDescXType, sdata->atom.nmax*2*sizeof(int4));
		#endif
	#endif
}

static __device__ inline X_CFLOAT4 fetchXType(int i)
{
		#ifdef CUDA_USE_TEXTURE
		  #if X_PRECISION == 1
		     return tex1Dfetch(_x_type_tex,i);
		  #else
		     return tex1Dfetch_double(_x_type_tex,i);
		  #endif
		#else
		  return _x_type[i];
		#endif
}
*/
#define SBBITS 30

static inline __device__ int sbmask(int j)
{
  return j >> SBBITS & 3;
}

static inline __device__ void minimum_image(X_CFLOAT4 &delta)
{
  if(_triclinic == 0) {
    if(_periodicity[0]) {
      delta.x += delta.x < -X_F(0.5) * _prd[0] ? _prd[0] :
                 (delta.x >  X_F(0.5) * _prd[0] ? -_prd[0] : X_F(0.0));
    }

    if(_periodicity[1]) {
      delta.y += delta.y < -X_F(0.5) * _prd[1] ? _prd[1] :
                 (delta.y >  X_F(0.5) * _prd[1] ? -_prd[1] : X_F(0.0));
    }

    if(_periodicity[2]) {
      delta.z += delta.z < -X_F(0.5) * _prd[2] ? _prd[2] :
                 (delta.z >  X_F(0.5) * _prd[2] ? -_prd[2] : X_F(0.0));
    }

  } else {
    if(_periodicity[1]) {
      delta.z += delta.z < -X_F(0.5) * _prd[2] ? _prd[2] :
                 (delta.z >  X_F(0.5) * _prd[2] ? -_prd[2] : X_F(0.0));
      delta.y += delta.z < -X_F(0.5) * _prd[2] ? _h[3] :
                 (delta.z >  X_F(0.5) * _prd[2] ? -_h[3] : X_F(0.0));
      delta.x += delta.z < -X_F(0.5) * _prd[2] ? _h[4] :
                 (delta.z >  X_F(0.5) * _prd[2] ? -_h[4] : X_F(0.0));

    }

    if(_periodicity[1]) {
      delta.y += delta.y < -X_F(0.5) * _prd[1] ? _prd[1] :
                 (delta.y >  X_F(0.5) * _prd[1] ? -_prd[1] : X_F(0.0));
      delta.x += delta.y < -X_F(0.5) * _prd[1] ? _h[5] :
                 (delta.y >  X_F(0.5) * _prd[1] ? -_h[5] : X_F(0.0));

    }

    if(_periodicity[0]) {
      delta.x += delta.x < -X_F(0.5) * _prd[0] ? _prd[0] :
                 (delta.x >  X_F(0.5) * _prd[0] ? -_prd[0] : X_F(0.0));
    }
  }
}

static inline __device__ void closest_image(X_CFLOAT4 &x1, X_CFLOAT4 &x2, X_CFLOAT4 &ci)
{
  ci.x = x2.x - x1.x;
  ci.y = x2.y - x1.y;
  ci.z = x2.z - x1.z;
  minimum_image(ci);
  ci.x += x1.x;
  ci.y += x1.y;
  ci.z += x1.z;
}
