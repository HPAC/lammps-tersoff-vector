#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>

#define _kn MY_AP(coeff1)  //[0]
#define _kt MY_AP(coeff1)  //[1]
#define _gamman MY_AP(coeff1) //[2]
#define _gammat MY_AP(coeff3) //[0]
#define _xmu MY_AP(coeff2) //[0]
#define _dampflag MY_AP(coeff2) //[1]

#include "pair_gran_hooke_cuda_cu.h"
#include "pair_gran_hooke_cuda_kernel_nc.cu"
#include <time.h>

void Cuda_PairGranHookeCuda_UpdateBuffer(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist)
{
  CUT_CHECK_ERROR("Cuda_PairGranHookeCuda: before updateBuffer failed");
  int3 layout = getgrid(sneighlist->inum, 7 * sizeof(ENERGY_CFLOAT));
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  int size = (unsigned)(layout.y * layout.x) * 7 * sizeof(ENERGY_CFLOAT);

  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_PairGranHookeCuda Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)

    if(sdata->buffer != NULL) hipFree(sdata->buffer);

    hipMalloc((void**)&sdata->buffer, size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)
  }

  hipMemcpyToSymbol(MY_AP(buffer), & sdata->buffer, sizeof(int*));
  CUT_CHECK_ERROR("Cuda_PairGranHookeCuda: updateBuffer failed");
}

void Cuda_PairGranHookeCuda_UpdateNmax(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist)
{
  CUT_CHECK_ERROR("Cuda_PairGranHookeCuda: before updateNmax failed");
  hipMemcpyToSymbol(MY_AP(neighbor_maxlocal) , & sneighlist->firstneigh.dim[0]  , sizeof(unsigned));
  //hipMemcpyToSymbol(MY_AP(firstneigh), & sneighlist->firstneigh.dev_data, sizeof(int*) );
  hipMemcpyToSymbol(MY_AP(ilist)     , & sneighlist->ilist     .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(inum)      , & sneighlist->inum               , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nlocal)    , & sdata->atom.nlocal             , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nall)      , & sdata->atom.nall               , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)      , & sdata->atom.nmax               , sizeof(int));
  hipMemcpyToSymbol(MY_AP(numneigh)  , & sneighlist->numneigh  .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(neighbors) , & sneighlist->neighbors  .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(type)      , & sdata->atom.type      .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(tag)       , & sdata->atom.tag       .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(mask)      , & sdata->atom.mask      .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(f)         , & sdata->atom.f         .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(x)         , & sdata->atom.x         .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(x_type)    , & sdata->atom.x_type    .dev_data, sizeof(X_CFLOAT4*));
  hipMemcpyToSymbol(MY_AP(v_radius)  , & sdata->atom.v_radius  .dev_data, sizeof(V_CFLOAT4*));
  hipMemcpyToSymbol(MY_AP(omega_rmass), & sdata->atom.omega_rmass.dev_data, sizeof(V_CFLOAT4*));
  hipMemcpyToSymbol(MY_AP(torque)    , & sdata->atom.torque    .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(maxneighbors), &sneighlist->maxneighbors	 	  , sizeof(int));
  hipMemcpyToSymbol(MY_AP(eatom)     , & sdata->atom.eatom     .dev_data, sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(vatom)     , & sdata->atom.vatom     .dev_data, sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(debugdata) , & sdata->debugdata      		  , sizeof(int*));
  hipMemcpyToSymbol(MY_AP(freeze_group_bit) , & sdata->pair.freeze_group_bit, sizeof(int));


  CUT_CHECK_ERROR("Cuda_PairGranHookeCuda: updateNmax failed");
}


void Cuda_PairGranHookeCuda_Init(cuda_shared_data* sdata)
{
  // !! LAMMPS indexes atom types starting with 1 !!

  unsigned cuda_ntypes = sdata->atom.ntypes + 2;

  if(cuda_ntypes * cuda_ntypes > CUDA_MAX_TYPES2)
    printf("# CUDA: Cuda_PairGranHookeCuda_Init: you need %u types. this is more than %u "
           "(assumed at compile time). re-compile with -DCUDA_MAX_TYPES_PLUS_ONE=32 "
           "or ajust this in cuda_common.h\n", cuda_ntypes, CUDA_MAX_TYPES_PLUS_ONE - 1);

  unsigned cuda_ntypes2 = cuda_ntypes * cuda_ntypes;
  unsigned n = sizeof(F_CFLOAT) * cuda_ntypes2;

  F_CFLOAT coeffs1[cuda_ntypes2];
  coeffs1[0] = (F_CFLOAT) sdata->pair.coeff1[0][0];
  coeffs1[1] = (F_CFLOAT) sdata->pair.coeff1[0][1];
  coeffs1[2] = (F_CFLOAT) sdata->pair.coeff1[1][0];
  F_CFLOAT coeffs3[cuda_ntypes2];
  coeffs3[0] = (F_CFLOAT) sdata->pair.coeff1[1][1];
  F_CFLOAT coeffs2[cuda_ntypes2];
  coeffs2[0] = (F_CFLOAT) sdata->pair.coeff2[0][0];
  coeffs2[1] = (F_CFLOAT) sdata->pair.coeff2[0][1];


  X_CFLOAT box_size[3] = {
    sdata->domain.subhi[0] - sdata->domain.sublo[0],
    sdata->domain.subhi[1] - sdata->domain.sublo[1],
    sdata->domain.subhi[2] - sdata->domain.sublo[2]
  };
  //printf("n: %i %i\n",n,CUDA_MAX_TYPES2);
  hipMemcpyToSymbol(MY_AP(box_size)   , box_size                 , sizeof(X_CFLOAT) * 3);
  hipMemcpyToSymbol(MY_AP(cuda_ntypes), & cuda_ntypes            , sizeof(unsigned));
  hipMemcpyToSymbol(MY_AP(coeff1)        , coeffs1                   , n);
  hipMemcpyToSymbol(MY_AP(coeff2)        , coeffs2                   , n);
  hipMemcpyToSymbol(MY_AP(coeff3)        , coeffs3                   , n);
  hipMemcpyToSymbol(MY_AP(virial)     , &sdata->pair.virial.dev_data   , sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(eng_vdwl)     , &sdata->pair.eng_vdwl.dev_data   , sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(periodicity), sdata->domain.periodicity, sizeof(int) * 3);
  CUT_CHECK_ERROR("Cuda_PairGranHookeCuda: init failed");
}



void Cuda_PairGranHookeCuda(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist, int eflag, int vflag, int eflag_atom, int vflag_atom)
{

  //if(sdata->atom.update_nmax)
  Cuda_PairGranHookeCuda_UpdateNmax(sdata, sneighlist);
  //if(sdata->atom.update_nlocal)
  {
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
    hipMemcpyToSymbol(MY_AP(nall)    , & sdata->atom.nall          , sizeof(int));
  }
  //if(sdata->buffer_new)
  Cuda_PairGranHookeCuda_UpdateBuffer(sdata, sneighlist);

  BindXTypeTexture(sdata);
  BindVRadiusTexture(sdata);
  BindOmegaRmassTexture(sdata);

  int sharedperproc = 0;

  if(eflag) sharedperproc += 1;

  if(vflag) sharedperproc += 6;

  int3 layout = getgrid(sneighlist->inum, sharedperproc * sizeof(ENERGY_CFLOAT), 128);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  // initialize only on first call
  static  short init = 0;

  if(! init) {
    init = 1;
    Cuda_PairGranHookeCuda_Init(sdata);
  }

  MYDBG(printf("# CUDA: Cuda_PairGranHookeCuda: kernel start eflag: %i vflag: %i config: %i %i %i %i\n", eflag, vflag, grid.x, grid.y, threads.x, sharedperproc * sizeof(ENERGY_CFLOAT)*threads.x);)

  CUT_CHECK_ERROR("Cuda_PairGranHookeCuda: (no binning) pre pair lj cut Kernel problems before kernel invocation");
  PairGranHookeCuda_Kernel <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x>>> (eflag, vflag, eflag_atom, vflag_atom, (int**)sneighlist->firstneigh.dev_data, sneighlist->binned_id
      , (F_CFLOAT) sdata->pair.coeff1[0][0], (F_CFLOAT) sdata->pair.coeff1[1][0], (F_CFLOAT) sdata->pair.coeff1[1][1], (F_CFLOAT) sdata->pair.coeff2[0][0]);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_PairGranHookeCuda: (no binning) pair lj cut Kernel execution failed");

  if(eflag || vflag) {
    int n = grid.x * grid.y;
    grid.x = sharedperproc;
    grid.y = 1;
    threads.x = 256;
    MY_AP(PairVirialCompute_reduce) <<< grid, threads, threads.x* sizeof(ENERGY_CFLOAT)>>>(n);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_PairGranHookeCuda: (no binning) virial compute Kernel execution failed");
  }

  MYDBG(printf("# CUDA: Cuda_PairGranHookeCoulLongCuda: kernel done\n");)

}


#undef _kn
#undef _kt
#undef _gamman
#undef _gammat
#undef _xmu
#undef _dampflag


