#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */
#define EWALD_F   1.12837917
#define EWALD_P   0.3275911
#define A1        0.254829592
#define A2       -0.284496736
#define A3        1.421413741
#define A4       -1.453152027
#define A5        1.061405429


template <const PAIR_FORCES pair_type, const COUL_FORCES coul_type, const unsigned int extended_data>
__global__ void Pair_Kernel_TpA(int eflag, int vflag, int eflag_atom, int vflag_atom)
{
  ENERGY_CFLOAT evdwl = ENERGY_F(0.0);
  ENERGY_CFLOAT ecoul = ENERGY_F(0.0);

  ENERGY_CFLOAT* sharedE;
  ENERGY_CFLOAT* sharedECoul;
  ENERGY_CFLOAT* sharedV = &sharedmem[threadIdx.x];

  if(eflag || eflag_atom) {
    sharedE = &sharedmem[threadIdx.x];
    sharedE[0] = ENERGY_F(0.0);
    sharedV += blockDim.x;

    if(coul_type != COUL_NONE) {
      sharedECoul = sharedE + blockDim.x;
      sharedECoul[0] = ENERGY_F(0.0);
      sharedV += blockDim.x;
    }
  }

  if(vflag || vflag_atom) {
    sharedV[0 * blockDim.x] = ENERGY_F(0.0);
    sharedV[1 * blockDim.x] = ENERGY_F(0.0);
    sharedV[2 * blockDim.x] = ENERGY_F(0.0);
    sharedV[3 * blockDim.x] = ENERGY_F(0.0);
    sharedV[4 * blockDim.x] = ENERGY_F(0.0);
    sharedV[5 * blockDim.x] = ENERGY_F(0.0);
  }

  int ii = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  X_CFLOAT xtmp, ytmp, ztmp;
  X_CFLOAT4 myxtype;
  F_CFLOAT fxtmp, fytmp, fztmp, fpair;
  F_CFLOAT delx, dely, delz;
  F_CFLOAT factor_lj, factor_coul;
  F_CFLOAT qtmp;
  int itype, i, j;
  int jnum = 0;
  int* jlist;

  if(ii < _inum) {
    i = _ilist[ii];

    myxtype = fetchXType(i);
    xtmp = myxtype.x;
    ytmp = myxtype.y;
    ztmp = myxtype.z;
    itype = static_cast <int>(myxtype.w);


    fxtmp = F_F(0.0);
    fytmp = F_F(0.0);
    fztmp = F_F(0.0);

    if(coul_type != COUL_NONE)
      qtmp = fetchQ(i);

    jnum = _numneigh[i];
    jlist = &_neighbors[i];
  }

  __syncthreads();

  for(int jj = 0; jj < jnum; jj++) {
    if(ii < _inum)
      if(jj < jnum) {
        fpair = F_F(0.0);
        j = jlist[jj * _nlocal];
        factor_lj =  _special_lj[sbmask(j)];

        if(coul_type != COUL_NONE)
          factor_coul = _special_coul[sbmask(j)];

        j &= NEIGHMASK;

        myxtype = fetchXType(j);
        delx = xtmp - myxtype.x;
        dely = ytmp - myxtype.y;
        delz = ztmp - myxtype.z;
        int jtype = static_cast <int>(myxtype.w);


        const F_CFLOAT rsq = delx * delx + dely * dely + delz * delz;

        bool in_cutoff = rsq < (_cutsq_global > X_F(0.0) ? _cutsq_global : _cutsq[itype * _cuda_ntypes + jtype]);

        if(in_cutoff) {
          switch(pair_type) {
            case PAIR_BORN:
              fpair += PairBornCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_BUCK:
              fpair += PairBuckCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_CG_CMM:
              fpair += PairLJSDKCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ_CHARMM:
              fpair += PairLJCharmmCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ_CLASS2:
              fpair += PairLJClass2Cuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ_CUT:
              fpair += PairLJCutCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ_EXPAND:
              fpair += PairLJExpandCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ_GROMACS:
              fpair += PairLJGromacsCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ_SMOOTH:
              fpair += PairLJSmoothCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ96_CUT:
              fpair += PairLJ96CutCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_MORSE_R6:
              fpair += PairMorseR6Cuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_MORSE:
              fpair += PairMorseCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;
          }
        }

        if(coul_type != COUL_NONE) {
          const F_CFLOAT qiqj = qtmp * fetchQ(j);

          if(qiqj * qiqj > 1e-8) {
            const bool in_coul_cutoff =
              rsq < (_cut_coulsq_global > X_F(0.0) ? _cut_coulsq_global : _cut_coulsq[itype * _cuda_ntypes + jtype]);

            if(in_coul_cutoff) {
              switch(coul_type) {
                case COUL_CHARMM:
                  fpair += CoulCharmmCuda_Eval(rsq, factor_coul, eflag, ecoul, qiqj);
                  break;

                case COUL_CHARMM_IMPLICIT:
                  fpair += CoulCharmmImplicitCuda_Eval(rsq, factor_coul, eflag, ecoul, qiqj);
                  break;

                case COUL_CUT: {
                  const F_CFLOAT forcecoul = factor_coul * _qqrd2e * qiqj * _RSQRT_(rsq);

                  if(eflag) {
                    ecoul += forcecoul;
                  }

                  fpair += forcecoul * (F_F(1.0) / rsq);
                }
                break;

                case COUL_DEBYE: {
                  const F_CFLOAT r2inv = F_F(1.0) / rsq;
                  const X_CFLOAT r = _RSQRT_(r2inv);
                  const X_CFLOAT rinv = F_F(1.0) / r;
                  const F_CFLOAT screening = _EXP_(-_kappa * r);
                  F_CFLOAT forcecoul = factor_coul * _qqrd2e * qiqj * screening ;

                  if(eflag) {
                    ecoul += forcecoul * rinv;
                  }

                  forcecoul *= (_kappa + rinv);
                  fpair += forcecoul * r2inv;
                }
                break;

                case COUL_GROMACS:
                  fpair += CoulGromacsCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_coul, eflag, ecoul, qiqj);
                  break;

                case COUL_LONG: {
                  const F_CFLOAT r2inv = F_F(1.0) / rsq;
                  const F_CFLOAT r = _RSQRT_(r2inv);
                  const F_CFLOAT grij = _g_ewald * r;
                  const F_CFLOAT expm2 = _EXP_(-grij * grij);
                  const F_CFLOAT t = F_F(1.0) / (F_F(1.0) + EWALD_P * grij);
                  const F_CFLOAT erfc = t * (A1 + t * (A2 + t * (A3 + t * (A4 + t * A5)))) * expm2;
                  const F_CFLOAT prefactor = _qqrd2e * qiqj * (F_F(1.0) / r);
                  F_CFLOAT forcecoul = prefactor * (erfc + EWALD_F * grij * expm2);

                  if(factor_coul < 1.0) forcecoul -= (1.0 - factor_coul) * prefactor;

                  if(eflag) {
                    ecoul += prefactor * erfc;

                    if(factor_coul < 1.0) ecoul -= (1.0 - factor_coul) * prefactor;
                  }

                  fpair += forcecoul * r2inv;
                }
                break;
              }
            }

            in_cutoff = in_cutoff || in_coul_cutoff;
          }
        }


        if(in_cutoff) {
          F_CFLOAT dxfp, dyfp, dzfp;
          fxtmp += dxfp = delx * fpair;
          fytmp += dyfp = dely * fpair;
          fztmp += dzfp = delz * fpair;

          if(vflag) {
            sharedV[0 * blockDim.x] += delx * dxfp;
            sharedV[1 * blockDim.x] += dely * dyfp;
            sharedV[2 * blockDim.x] += delz * dzfp;
            sharedV[3 * blockDim.x] += delx * dyfp;
            sharedV[4 * blockDim.x] += delx * dzfp;
            sharedV[5 * blockDim.x] += dely * dzfp;
          }
        }
      }
  }

  __syncthreads();

  if(ii < _inum) {
    F_CFLOAT* my_f;

    if(_collect_forces_later) {
      ENERGY_CFLOAT* buffer = (ENERGY_CFLOAT*) _buffer;

      if(eflag) {
        buffer = &buffer[1 * gridDim.x * gridDim.y];

        if(coul_type != COUL_NONE)
          buffer = &buffer[1 * gridDim.x * gridDim.y];
      }

      if(vflag) {
        buffer = &buffer[6 * gridDim.x * gridDim.y];
      }

      my_f = (F_CFLOAT*) buffer;
      my_f += i;
      *my_f = fxtmp;
      my_f += _nmax;
      *my_f = fytmp;
      my_f += _nmax;
      *my_f = fztmp;
    } else {
      my_f = _f + i;
      *my_f += fxtmp;
      my_f += _nmax;
      *my_f += fytmp;
      my_f += _nmax;
      *my_f += fztmp;
    }
  }

  __syncthreads();

  if(eflag) {
    sharedE[0] = evdwl;

    if(coul_type != COUL_NONE)
      sharedECoul[0] = ecoul;
  }

  if(eflag_atom && i < _nlocal) {
    if(coul_type != COUL_NONE)
      _eatom[i] += evdwl + ecoul;
    else
      _eatom[i] += evdwl;
  }

  if(vflag_atom && i < _nlocal) {
    _vatom[i]         += ENERGY_F(0.5) * sharedV[0 * blockDim.x];
    _vatom[i + _nmax]   += ENERGY_F(0.5) * sharedV[1 * blockDim.x];
    _vatom[i + 2 * _nmax] += ENERGY_F(0.5) * sharedV[2 * blockDim.x];
    _vatom[i + 3 * _nmax] += ENERGY_F(0.5) * sharedV[3 * blockDim.x];
    _vatom[i + 4 * _nmax] += ENERGY_F(0.5) * sharedV[4 * blockDim.x];
    _vatom[i + 5 * _nmax] += ENERGY_F(0.5) * sharedV[5 * blockDim.x];
  }

  if(vflag || eflag) PairVirialCompute_A_Kernel(eflag, vflag, coul_type != COUL_NONE ? 1 : 0);
}

template <const PAIR_FORCES pair_type, const COUL_FORCES coul_type, const unsigned int extended_data>
__global__ void Pair_Kernel_BpA(int eflag, int vflag, int eflag_atom, int vflag_atom)
{
  int ii = (blockIdx.x * gridDim.y + blockIdx.y);

  if(ii >= _inum)
    return;

  ENERGY_CFLOAT evdwl = ENERGY_F(0.0);
  ENERGY_CFLOAT ecoul = ENERGY_F(0.0);
  F_CFLOAT3* sharedVirial1;
  F_CFLOAT3* sharedVirial2;
  F_CFLOAT* sharedEnergy;
  F_CFLOAT* sharedEnergyCoul;

  F_CFLOAT3* sharedForce = (F_CFLOAT3*) &sharedmem[0];

  if(vflag) {
    sharedVirial1 = &sharedForce[64];
    sharedVirial2 = &sharedVirial1[64];
  } else {
    sharedVirial1 = &sharedForce[0];
    sharedVirial2 = &sharedVirial1[0];
  }

  if(eflag) {
    if(vflag || vflag_atom)
      sharedEnergy = (F_CFLOAT*) &sharedVirial2[64];
    else
      sharedEnergy = (F_CFLOAT*) &sharedForce[64];

    if(coul_type != COUL_NONE)
      sharedEnergyCoul = (F_CFLOAT*) &sharedEnergy[64];

  }

  F_CFLOAT3 partialForce = { F_F(0.0),  F_F(0.0),  F_F(0.0) };
  F_CFLOAT3 partialVirial1 = {  F_F(0.0),  F_F(0.0),  F_F(0.0) };
  F_CFLOAT3 partialVirial2 = {  F_F(0.0),  F_F(0.0),  F_F(0.0) };

  X_CFLOAT xtmp, ytmp, ztmp;
  X_CFLOAT4 myxtype;
  F_CFLOAT delx, dely, delz;
  F_CFLOAT factor_lj, factor_coul;
  F_CFLOAT fpair;
  F_CFLOAT qtmp;
  int itype, jnum, i, j;
  int* jlist;

  i = _ilist[ii];

  myxtype = fetchXType(i);

  xtmp = myxtype.x;
  ytmp = myxtype.y;
  ztmp = myxtype.z;
  itype = static_cast <int>(myxtype.w);

  if(coul_type != COUL_NONE)
    qtmp = fetchQ(i);

  jnum = _numneigh[i];

  jlist = &_neighbors[i * _maxneighbors];
  __syncthreads();

  for(int jj = threadIdx.x; jj < jnum + blockDim.x; jj += blockDim.x) {
    if(jj < jnum) {
      fpair = F_F(0.0);
      j = jlist[jj];
      factor_lj =  _special_lj[sbmask(j)];

      if(coul_type != COUL_NONE)
        factor_coul = _special_coul[sbmask(j)];

      j &= NEIGHMASK;

      myxtype = fetchXType(j);

      delx = xtmp - myxtype.x;
      dely = ytmp - myxtype.y;
      delz = ztmp - myxtype.z;
      int jtype = static_cast <int>(myxtype.w);

      const F_CFLOAT rsq = delx * delx + dely * dely + delz * delz;

      bool in_cutoff = rsq < (_cutsq_global > X_F(0.0) ? _cutsq_global : _cutsq[itype * _cuda_ntypes + jtype]);
      bool in_coul_cutoff;

      if(in_cutoff) {
        switch(pair_type) {
          case PAIR_BORN:
            fpair += PairBornCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_BUCK:
            fpair += PairBuckCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_CG_CMM:
            fpair += PairLJSDKCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ_CHARMM:
            fpair += PairLJCharmmCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ_CLASS2:
            fpair += PairLJClass2Cuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ_CUT:
            fpair += PairLJCutCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ_EXPAND:
            fpair += PairLJExpandCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ_GROMACS:
            fpair += PairLJGromacsCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ_SMOOTH:
            fpair += PairLJSmoothCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ96_CUT:
            fpair += PairLJ96CutCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_MORSE_R6:
            fpair += PairMorseR6Cuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_MORSE:
            fpair += PairMorseCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;
        }
      }

      if(coul_type != COUL_NONE) {
        const F_CFLOAT qiqj = qtmp * fetchQ(j);

        if(qiqj * qiqj > (1e-8f)) {
          in_coul_cutoff =
            rsq < (_cut_coulsq_global > X_F(0.0) ? _cut_coulsq_global : _cut_coulsq[itype * _cuda_ntypes + jtype]);

          if(in_coul_cutoff) {
            switch(coul_type) {
              case COUL_CHARMM:
                fpair += CoulCharmmCuda_Eval(rsq, factor_coul, eflag, ecoul, qiqj);
                break;

              case COUL_CHARMM_IMPLICIT:
                fpair += CoulCharmmImplicitCuda_Eval(rsq, factor_coul, eflag, ecoul, qiqj);
                break;

              case COUL_GROMACS:
                fpair += CoulGromacsCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_coul, eflag, ecoul, qiqj);
                break;

              case COUL_LONG: {
                const F_CFLOAT r2inv = F_F(1.0) / rsq;
                const F_CFLOAT r = _RSQRT_(r2inv);
                const F_CFLOAT grij = _g_ewald * r;
                const F_CFLOAT expm2 = _EXP_(-grij * grij);
                const F_CFLOAT t = F_F(1.0) / (F_F(1.0) + EWALD_P * grij);
                const F_CFLOAT erfc = t * (A1 + t * (A2 + t * (A3 + t * (A4 + t * A5)))) * expm2;
                const F_CFLOAT prefactor = _qqrd2e * qiqj * (F_F(1.0) / r);
                F_CFLOAT forcecoul = prefactor * (erfc + EWALD_F * grij * expm2);

                if(factor_coul < 1.0) forcecoul -= (1.0 - factor_coul) * prefactor;

                if(eflag) {
                  ecoul += prefactor * erfc;

                  if(factor_coul < 1.0) ecoul -= (1.0 - factor_coul) * prefactor;
                }

                fpair += forcecoul * r2inv;
              }
              break;

              case COUL_DEBYE: {
                const F_CFLOAT r2inv = F_F(1.0) / rsq;
                const X_CFLOAT r = _RSQRT_(r2inv);
                const X_CFLOAT rinv = F_F(1.0) / r;
                const F_CFLOAT screening = _EXP_(-_kappa * r);
                F_CFLOAT forcecoul = factor_coul * _qqrd2e * qiqj * screening ;

                if(eflag) {
                  ecoul += forcecoul * rinv;
                }

                forcecoul *= (_kappa + rinv);
                fpair += forcecoul * r2inv;
              }
              break;

              case COUL_CUT: {
                const F_CFLOAT forcecoul = factor_coul * _qqrd2e * qiqj * _RSQRT_(rsq);

                if(eflag) {
                  ecoul += forcecoul;
                }

                fpair += forcecoul * (F_F(1.0) / rsq);
              }
              break;


            }
          }
        }
      }



      if(in_cutoff || in_coul_cutoff) {
        F_CFLOAT dxfp, dyfp, dzfp;
        partialForce.x += dxfp = delx * fpair;
        partialForce.y += dyfp = dely * fpair;
        partialForce.z += dzfp = delz * fpair;

        if(vflag) {
          partialVirial1.x += delx * dxfp;
          partialVirial1.y += dely * dyfp;
          partialVirial1.z += delz * dzfp;
          partialVirial2.x += delx * dyfp;
          partialVirial2.y += delx * dzfp;
          partialVirial2.z += dely * dzfp;
        }
      }
    }
  }

  if(eflag) {
    sharedEnergy[threadIdx.x] = evdwl;

    if(coul_type != COUL_NONE)
      sharedEnergyCoul[threadIdx.x] = ecoul;
  }

  sharedForce[threadIdx.x] = partialForce;

  if(vflag) {
    sharedVirial1[threadIdx.x] = partialVirial1;
    sharedVirial2[threadIdx.x] = partialVirial2;
  }

  __syncthreads();


  for(unsigned int s = blockDim.x >> 1; s > 0; s >>= 1) {

    if(threadIdx.x < s) {
      sharedForce[ threadIdx.x ].x += sharedForce[ threadIdx.x + s ].x;
      sharedForce[ threadIdx.x ].y += sharedForce[ threadIdx.x + s ].y;
      sharedForce[ threadIdx.x ].z += sharedForce[ threadIdx.x + s ].z;

      if(vflag) {
        sharedVirial1[ threadIdx.x ].x += sharedVirial1[ threadIdx.x + s ].x;
        sharedVirial1[ threadIdx.x ].y += sharedVirial1[ threadIdx.x + s ].y;
        sharedVirial1[ threadIdx.x ].z += sharedVirial1[ threadIdx.x + s ].z;

        sharedVirial2[ threadIdx.x ].x += sharedVirial2[ threadIdx.x + s ].x;
        sharedVirial2[ threadIdx.x ].y += sharedVirial2[ threadIdx.x + s ].y;
        sharedVirial2[ threadIdx.x ].z += sharedVirial2[ threadIdx.x + s ].z;
      }

      if(eflag) {
        sharedEnergy[ threadIdx.x ] += sharedEnergy[ threadIdx.x + s ];

        if(coul_type != COUL_NONE)
          sharedEnergyCoul[ threadIdx.x ] += sharedEnergyCoul[ threadIdx.x + s ];
      }
    }

    __syncthreads();
  }

  if(threadIdx.x == 0) {

    ENERGY_CFLOAT* buffer = (ENERGY_CFLOAT*) _buffer;

    if(eflag) {
      ENERGY_CFLOAT tmp_evdwl;
      buffer[blockIdx.x * gridDim.y + blockIdx.y + 0 * gridDim.x * gridDim.y] = tmp_evdwl = ENERGY_F(0.5) * sharedEnergy[0];

      if(eflag_atom)
        _eatom[i] = tmp_evdwl;

      buffer = &buffer[gridDim.x * gridDim.y];

      if(coul_type != COUL_NONE) {
        buffer[blockIdx.x * gridDim.y + blockIdx.y + 0 * gridDim.x * gridDim.y] = tmp_evdwl = ENERGY_F(0.5) * sharedEnergyCoul[0];

        if(eflag_atom)
          _eatom[i] += tmp_evdwl;

        buffer = &buffer[gridDim.x * gridDim.y];
      }
    }

    if(vflag) {
      ENERGY_CFLOAT tmp;
      buffer[blockIdx.x * gridDim.y + blockIdx.y + 0 * gridDim.x * gridDim.y] = tmp = ENERGY_F(0.5) * sharedVirial1[0].x;

      if(vflag_atom) _vatom[i + 0 * _nmax] = tmp;

      buffer[blockIdx.x * gridDim.y + blockIdx.y + 1 * gridDim.x * gridDim.y] = tmp = ENERGY_F(0.5) * sharedVirial1[0].y;

      if(vflag_atom) _vatom[i + 1 * _nmax] = tmp;

      buffer[blockIdx.x * gridDim.y + blockIdx.y + 2 * gridDim.x * gridDim.y] = tmp = ENERGY_F(0.5) * sharedVirial1[0].z;

      if(vflag_atom) _vatom[i + 2 * _nmax] = tmp;

      buffer[blockIdx.x * gridDim.y + blockIdx.y + 3 * gridDim.x * gridDim.y] = tmp = ENERGY_F(0.5) * sharedVirial2[0].x;

      if(vflag_atom) _vatom[i + 3 * _nmax] = tmp;

      buffer[blockIdx.x * gridDim.y + blockIdx.y + 4 * gridDim.x * gridDim.y] = tmp = ENERGY_F(0.5) * sharedVirial2[0].y;

      if(vflag_atom) _vatom[i + 4 * _nmax] = tmp;

      buffer[blockIdx.x * gridDim.y + blockIdx.y + 5 * gridDim.x * gridDim.y] = tmp = ENERGY_F(0.5) * sharedVirial2[0].z;

      if(vflag_atom) _vatom[i + 5 * _nmax] = tmp;

      buffer = &buffer[6 * gridDim.x * gridDim.y];
    }

    F_CFLOAT* my_f;

    if(_collect_forces_later) {
      my_f = (F_CFLOAT*) buffer;
      my_f += i;
      *my_f = sharedForce[0].x;
      my_f += _nmax;
      *my_f = sharedForce[0].y;
      my_f += _nmax;
      *my_f = sharedForce[0].z;
    } else {
      my_f = _f + i;
      *my_f += sharedForce[0].x;
      my_f += _nmax;
      *my_f += sharedForce[0].y;
      my_f += _nmax;
      *my_f += sharedForce[0].z;
    }
  }
}


template <const PAIR_FORCES pair_type, const COUL_FORCES coul_type, const unsigned int extended_data>
__global__ void Pair_Kernel_TpA_opt(int eflag, int vflag, int eflag_atom, int vflag_atom, int comm_phase)
{
  ENERGY_CFLOAT evdwl = ENERGY_F(0.0);
  ENERGY_CFLOAT ecoul = ENERGY_F(0.0);

  ENERGY_CFLOAT* sharedE;
  ENERGY_CFLOAT* sharedECoul;
  ENERGY_CFLOAT* sharedV = &sharedmem[threadIdx.x];

  if(eflag || eflag_atom) {
    sharedE = &sharedmem[threadIdx.x];
    sharedE[0] = ENERGY_F(0.0);
    sharedV += blockDim.x;

    if(coul_type != COUL_NONE) {
      sharedECoul = sharedE + blockDim.x;
      sharedECoul[0] = ENERGY_F(0.0);
      sharedV += blockDim.x;
    }
  }

  if(vflag || vflag_atom) {
    sharedV[0 * blockDim.x] = ENERGY_F(0.0);
    sharedV[1 * blockDim.x] = ENERGY_F(0.0);
    sharedV[2 * blockDim.x] = ENERGY_F(0.0);
    sharedV[3 * blockDim.x] = ENERGY_F(0.0);
    sharedV[4 * blockDim.x] = ENERGY_F(0.0);
    sharedV[5 * blockDim.x] = ENERGY_F(0.0);
  }

  int ii = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  X_CFLOAT xtmp, ytmp, ztmp;
  X_CFLOAT4 myxtype;
  F_CFLOAT fxtmp, fytmp, fztmp, fpair;
  F_CFLOAT delx, dely, delz;
  F_CFLOAT factor_lj, factor_coul;
  F_CFLOAT qtmp;
  int itype, i, j;
  int jnum = 0;
  int* jlist;

  if(ii < (comm_phase < 2 ? _inum : _inum_border[0])) {
    i = comm_phase < 2 ? _ilist[ii] : _ilist_border[ii] ;

    myxtype = fetchXType(i);
    myxtype = _x_type[i];
    xtmp = myxtype.x;
    ytmp = myxtype.y;
    ztmp = myxtype.z;
    itype = static_cast <int>(myxtype.w);


    fxtmp = F_F(0.0);
    fytmp = F_F(0.0);
    fztmp = F_F(0.0);

    if(coul_type != COUL_NONE)
      qtmp = fetchQ(i);

    jnum = comm_phase == 0 ? _numneigh[i] : (comm_phase == 1 ? _numneigh_inner[i] : _numneigh_border[ii]);


    jlist = comm_phase == 0 ? &_neighbors[i] : (comm_phase == 1 ? &_neighbors_inner[i] : &_neighbors_border[ii]);
  }

  __syncthreads();

  for(int jj = 0; jj < jnum; jj++) {
    if(ii < (comm_phase < 2 ? _inum : _inum_border[0]))
      if(jj < jnum) {
        fpair = F_F(0.0);
        j = jlist[jj * _nlocal];

        factor_lj = j < _nall ? F_F(1.0) : _special_lj[j / _nall];

        if(coul_type != COUL_NONE)
          factor_coul = j < _nall ? F_F(1.0) : _special_coul[j / _nall];

        j = j < _nall ? j : j % _nall;

        myxtype = fetchXType(j);
        delx = xtmp - myxtype.x;
        dely = ytmp - myxtype.y;
        delz = ztmp - myxtype.z;
        int jtype = static_cast <int>(myxtype.w);


        const F_CFLOAT rsq = delx * delx + dely * dely + delz * delz;

        bool in_cutoff = rsq < (_cutsq_global > X_F(0.0) ? _cutsq_global : _cutsq[itype * _cuda_ntypes + jtype]);

        if(in_cutoff) {
          switch(pair_type) {
            case PAIR_BORN:
              fpair += PairBornCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_BUCK:
              fpair += PairBuckCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_CG_CMM:
              fpair += PairLJSDKCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ_CHARMM:
              fpair += PairLJCharmmCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ_CLASS2:
              fpair += PairLJClass2Cuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ_CUT:
              fpair += PairLJCutCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ_EXPAND:
              fpair += PairLJExpandCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ_GROMACS:
              fpair += PairLJGromacsCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ_SMOOTH:
              fpair += PairLJSmoothCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_LJ96_CUT:
              fpair += PairLJ96CutCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_MORSE_R6:
              fpair += PairMorseR6Cuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;

            case PAIR_MORSE:
              fpair += PairMorseCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
              break;
          }
        }

        if(coul_type != COUL_NONE) {
          const F_CFLOAT qiqj = qtmp * fetchQ(j);

          if(qiqj * qiqj > 1e-8) {
            const bool in_coul_cutoff =
              rsq < (_cut_coulsq_global > X_F(0.0) ? _cut_coulsq_global : _cut_coulsq[itype * _cuda_ntypes + jtype]);

            if(in_coul_cutoff) {
              switch(coul_type) {
                case COUL_CHARMM:
                  fpair += CoulCharmmCuda_Eval(rsq, factor_coul, eflag, ecoul, qiqj);
                  break;

                case COUL_CHARMM_IMPLICIT:
                  fpair += CoulCharmmImplicitCuda_Eval(rsq, factor_coul, eflag, ecoul, qiqj);
                  break;

                case COUL_CUT: {
                  const F_CFLOAT forcecoul = factor_coul * _qqrd2e * qiqj * _RSQRT_(rsq);

                  if(eflag) {
                    ecoul += forcecoul;
                  }

                  fpair += forcecoul * (F_F(1.0) / rsq);
                }
                break;

                case COUL_DEBYE: {
                  const F_CFLOAT r2inv = F_F(1.0) / rsq;
                  const X_CFLOAT r = _RSQRT_(r2inv);
                  const X_CFLOAT rinv = F_F(1.0) / r;
                  const F_CFLOAT screening = _EXP_(-_kappa * r);
                  F_CFLOAT forcecoul = factor_coul * _qqrd2e * qiqj * screening ;

                  if(eflag) {
                    ecoul += forcecoul * rinv;
                  }

                  forcecoul *= (_kappa + rinv);
                  fpair += forcecoul * r2inv;
                }
                break;

                case COUL_GROMACS:
                  fpair += CoulGromacsCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_coul, eflag, ecoul, qiqj);
                  break;

                case COUL_LONG: {
                  const F_CFLOAT r2inv = F_F(1.0) / rsq;
                  const F_CFLOAT r = _RSQRT_(r2inv);
                  const F_CFLOAT grij = _g_ewald * r;
                  const F_CFLOAT expm2 = _EXP_(-grij * grij);
                  const F_CFLOAT t = F_F(1.0) / (F_F(1.0) + EWALD_P * grij);
                  const F_CFLOAT erfc = t * (A1 + t * (A2 + t * (A3 + t * (A4 + t * A5)))) * expm2;
                  const F_CFLOAT prefactor = _qqrd2e * qiqj * (F_F(1.0) / r);
                  F_CFLOAT forcecoul = prefactor * (erfc + EWALD_F * grij * expm2);

                  if(factor_coul < 1.0) forcecoul -= (1.0 - factor_coul) * prefactor;

                  if(eflag) {
                    ecoul += prefactor * erfc;

                    if(factor_coul < 1.0) ecoul -= (1.0 - factor_coul) * prefactor;
                  }

                  fpair += forcecoul * r2inv;
                }
                break;

              }
            }

            in_cutoff = in_cutoff || in_coul_cutoff;
          }
        }


        if(in_cutoff) {
          F_CFLOAT dxfp, dyfp, dzfp;
          fxtmp += dxfp = delx * fpair;
          fytmp += dyfp = dely * fpair;
          fztmp += dzfp = delz * fpair;

          if(vflag) {
            sharedV[0 * blockDim.x] += delx * dxfp;
            sharedV[1 * blockDim.x] += dely * dyfp;
            sharedV[2 * blockDim.x] += delz * dzfp;
            sharedV[3 * blockDim.x] += delx * dyfp;
            sharedV[4 * blockDim.x] += delx * dzfp;
            sharedV[5 * blockDim.x] += dely * dzfp;
          }
        }
      }
  }

  __syncthreads();

  if(ii < (comm_phase < 2 ? _inum : _inum_border[0])) {
    F_CFLOAT* my_f;

    if(_collect_forces_later) {
      ENERGY_CFLOAT* buffer = (ENERGY_CFLOAT*) _buffer;

      if(eflag) {
        buffer = &buffer[1 * gridDim.x * gridDim.y];

        if(coul_type != COUL_NONE)
          buffer = &buffer[1 * gridDim.x * gridDim.y];
      }

      if(vflag) {
        buffer = &buffer[6 * gridDim.x * gridDim.y];
      }

      my_f = (F_CFLOAT*) buffer;
      my_f += i;
      *my_f = fxtmp;
      my_f += _nmax;
      *my_f = fytmp;
      my_f += _nmax;
      *my_f = fztmp;
    } else {
      my_f = _f + i;
      *my_f += fxtmp;
      my_f += _nmax;
      *my_f += fytmp;
      my_f += _nmax;
      *my_f += fztmp;
    }
  }

  __syncthreads();

  if(eflag) {
    sharedE[0] = evdwl;

    if(coul_type != COUL_NONE)
      sharedECoul[0] = ecoul;
  }

  if(eflag_atom && i < _nlocal) {
    if(coul_type != COUL_NONE)
      _eatom[i] += evdwl + ecoul;
    else
      _eatom[i] += evdwl;
  }

  if(vflag_atom && i < _nlocal) {
    _vatom[i]         += ENERGY_F(0.5) * sharedV[0 * blockDim.x];
    _vatom[i + _nmax]   += ENERGY_F(0.5) * sharedV[1 * blockDim.x];
    _vatom[i + 2 * _nmax] += ENERGY_F(0.5) * sharedV[2 * blockDim.x];
    _vatom[i + 3 * _nmax] += ENERGY_F(0.5) * sharedV[3 * blockDim.x];
    _vatom[i + 4 * _nmax] += ENERGY_F(0.5) * sharedV[4 * blockDim.x];
    _vatom[i + 5 * _nmax] += ENERGY_F(0.5) * sharedV[5 * blockDim.x];
  }

  if(vflag || eflag) PairVirialCompute_A_Kernel(eflag, vflag, coul_type != COUL_NONE ? 1 : 0);
}

template <const PAIR_FORCES pair_type, const COUL_FORCES coul_type, const unsigned int extended_data>
__global__ void Pair_Kernel_BpA_opt(int eflag, int vflag, int eflag_atom, int vflag_atom, int comm_phase)
{
  int ii = (blockIdx.x * gridDim.y + blockIdx.y);

  if(ii >= (comm_phase < 2 ? _inum : _inum_border[0]))
    return;

  ENERGY_CFLOAT evdwl = ENERGY_F(0.0);
  ENERGY_CFLOAT ecoul = ENERGY_F(0.0);
  F_CFLOAT3* sharedVirial1;
  F_CFLOAT3* sharedVirial2;
  F_CFLOAT* sharedEnergy;
  F_CFLOAT* sharedEnergyCoul;

  F_CFLOAT3* sharedForce = (F_CFLOAT3*) &sharedmem[0];

  if(vflag) {
    sharedVirial1 = &sharedForce[64];
    sharedVirial2 = &sharedVirial1[64];
  } else {
    sharedVirial1 = &sharedForce[0];
    sharedVirial2 = &sharedVirial1[0];
  }

  if(eflag) {
    if(vflag || vflag_atom)
      sharedEnergy = (F_CFLOAT*) &sharedVirial2[64];
    else
      sharedEnergy = (F_CFLOAT*) &sharedForce[64];

    if(coul_type != COUL_NONE)
      sharedEnergyCoul = (F_CFLOAT*) &sharedEnergy[64];

  }

  F_CFLOAT3 partialForce = { F_F(0.0),  F_F(0.0),  F_F(0.0) };
  F_CFLOAT3 partialVirial1 = {  F_F(0.0),  F_F(0.0),  F_F(0.0) };
  F_CFLOAT3 partialVirial2 = {  F_F(0.0),  F_F(0.0),  F_F(0.0) };

  X_CFLOAT xtmp, ytmp, ztmp;
  X_CFLOAT4 myxtype;
  F_CFLOAT delx, dely, delz;
  F_CFLOAT factor_lj, factor_coul;
  F_CFLOAT fpair;
  F_CFLOAT qtmp;
  int itype, jnum, i, j;
  int* jlist;

  i = comm_phase < 2 ? _ilist[ii] : _ilist_border[ii];

  myxtype = fetchXType(i);

  xtmp = myxtype.x;
  ytmp = myxtype.y;
  ztmp = myxtype.z;
  itype = static_cast <int>(myxtype.w);

  if(coul_type != COUL_NONE)
    qtmp = fetchQ(i);

  jnum = comm_phase == 0 ? _numneigh[i] : (comm_phase == 1 ? _numneigh_inner[i] : _numneigh_border[ii]);

  jlist = comm_phase == 0 ? &_neighbors[i * _maxneighbors] : (comm_phase == 1 ? &_neighbors_inner[i * _maxneighbors] : &_neighbors_border[ii * _maxneighbors]);
  __syncthreads();

  for(int jj = threadIdx.x; jj < jnum + blockDim.x; jj += blockDim.x) {
    if(jj < jnum) {
      fpair = F_F(0.0);
      j = jlist[jj];
      factor_lj   = j < _nall ? F_F(1.0) : _special_lj[j / _nall];

      if(coul_type != COUL_NONE)
        factor_coul = j < _nall ? F_F(1.0) : _special_coul[j / _nall];

      j 			= j < _nall ? j : j % _nall;

      myxtype = fetchXType(j);

      delx = xtmp - myxtype.x;
      dely = ytmp - myxtype.y;
      delz = ztmp - myxtype.z;
      int jtype = static_cast <int>(myxtype.w);

      const F_CFLOAT rsq = delx * delx + dely * dely + delz * delz;

      bool in_cutoff = rsq < (_cutsq_global > X_F(0.0) ? _cutsq_global : _cutsq[itype * _cuda_ntypes + jtype]);
      bool in_coul_cutoff;

      if(in_cutoff) {
        switch(pair_type) {
          case PAIR_BORN:
            fpair += PairBornCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_BUCK:
            fpair += PairBuckCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_CG_CMM:
            fpair += PairLJSDKCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ_CHARMM:
            fpair += PairLJCharmmCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ_CLASS2:
            fpair += PairLJClass2Cuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ_CUT:
            fpair += PairLJCutCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ_EXPAND:
            fpair += PairLJExpandCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ_GROMACS:
            fpair += PairLJGromacsCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ_SMOOTH:
            fpair += PairLJSmoothCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_LJ96_CUT:
            fpair += PairLJ96CutCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_MORSE_R6:
            fpair += PairMorseR6Cuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;

          case PAIR_MORSE:
            fpair += PairMorseCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_lj, eflag, evdwl);
            break;
        }
      }

      if(coul_type != COUL_NONE) {
        const F_CFLOAT qiqj = qtmp * fetchQ(j);

        if(qiqj * qiqj > (1e-8f)) {
          in_coul_cutoff =
            rsq < (_cut_coulsq_global > X_F(0.0) ? _cut_coulsq_global : _cut_coulsq[itype * _cuda_ntypes + jtype]);

          if(in_coul_cutoff) {
            switch(coul_type) {
              case COUL_CHARMM:
                fpair += CoulCharmmCuda_Eval(rsq, factor_coul, eflag, ecoul, qiqj);
                break;

              case COUL_CHARMM_IMPLICIT:
                fpair += CoulCharmmImplicitCuda_Eval(rsq, factor_coul, eflag, ecoul, qiqj);
                break;

              case COUL_GROMACS:
                fpair += CoulGromacsCuda_Eval(rsq, itype * _cuda_ntypes + jtype, factor_coul, eflag, ecoul, qiqj);
                break;

              case COUL_LONG: {
                const F_CFLOAT r2inv = F_F(1.0) / rsq;
                const F_CFLOAT r = _RSQRT_(r2inv);
                const F_CFLOAT grij = _g_ewald * r;
                const F_CFLOAT expm2 = _EXP_(-grij * grij);
                const F_CFLOAT t = F_F(1.0) / (F_F(1.0) + EWALD_P * grij);
                const F_CFLOAT erfc = t * (A1 + t * (A2 + t * (A3 + t * (A4 + t * A5)))) * expm2;
                const F_CFLOAT prefactor = _qqrd2e * qiqj * (F_F(1.0) / r);
                F_CFLOAT forcecoul = prefactor * (erfc + EWALD_F * grij * expm2);

                if(factor_coul < 1.0) forcecoul -= (1.0 - factor_coul) * prefactor;

                if(eflag) {
                  ecoul += prefactor * erfc;

                  if(factor_coul < 1.0) ecoul -= (1.0 - factor_coul) * prefactor;
                }

                fpair += forcecoul * r2inv;
              }
              break;

              case COUL_DEBYE: {
                const F_CFLOAT r2inv = F_F(1.0) / rsq;
                const X_CFLOAT r = _RSQRT_(r2inv);
                const X_CFLOAT rinv = F_F(1.0) / r;
                const F_CFLOAT screening = _EXP_(-_kappa * r);
                F_CFLOAT forcecoul = factor_coul * _qqrd2e * qiqj * screening ;

                if(eflag) {
                  ecoul += forcecoul * rinv;
                }

                forcecoul *= (_kappa + rinv);
                fpair += forcecoul * r2inv;
              }
              break;

              case COUL_CUT: {
                const F_CFLOAT forcecoul = factor_coul * _qqrd2e * qiqj * _RSQRT_(rsq);

                if(eflag) {
                  ecoul += forcecoul;
                }

                fpair += forcecoul * (F_F(1.0) / rsq);
              }
              break;


            }
          }
        }
      }



      if(in_cutoff || in_coul_cutoff) {
        F_CFLOAT dxfp, dyfp, dzfp;
        partialForce.x += dxfp = delx * fpair;
        partialForce.y += dyfp = dely * fpair;
        partialForce.z += dzfp = delz * fpair;

        if(vflag) {
          partialVirial1.x += delx * dxfp;
          partialVirial1.y += dely * dyfp;
          partialVirial1.z += delz * dzfp;
          partialVirial2.x += delx * dyfp;
          partialVirial2.y += delx * dzfp;
          partialVirial2.z += dely * dzfp;
        }
      }
    }
  }

  if(eflag) {
    sharedEnergy[threadIdx.x] = evdwl;

    if(coul_type != COUL_NONE)
      sharedEnergyCoul[threadIdx.x] = ecoul;
  }

  sharedForce[threadIdx.x] = partialForce;

  if(vflag) {
    sharedVirial1[threadIdx.x] = partialVirial1;
    sharedVirial2[threadIdx.x] = partialVirial2;
  }

  __syncthreads();


  for(unsigned int s = blockDim.x >> 1; s > 0; s >>= 1) {

    if(threadIdx.x < s) {
      sharedForce[ threadIdx.x ].x += sharedForce[ threadIdx.x + s ].x;
      sharedForce[ threadIdx.x ].y += sharedForce[ threadIdx.x + s ].y;
      sharedForce[ threadIdx.x ].z += sharedForce[ threadIdx.x + s ].z;

      if(vflag) {
        sharedVirial1[ threadIdx.x ].x += sharedVirial1[ threadIdx.x + s ].x;
        sharedVirial1[ threadIdx.x ].y += sharedVirial1[ threadIdx.x + s ].y;
        sharedVirial1[ threadIdx.x ].z += sharedVirial1[ threadIdx.x + s ].z;

        sharedVirial2[ threadIdx.x ].x += sharedVirial2[ threadIdx.x + s ].x;
        sharedVirial2[ threadIdx.x ].y += sharedVirial2[ threadIdx.x + s ].y;
        sharedVirial2[ threadIdx.x ].z += sharedVirial2[ threadIdx.x + s ].z;
      }

      if(eflag) {
        sharedEnergy[ threadIdx.x ] += sharedEnergy[ threadIdx.x + s ];

        if(coul_type != COUL_NONE)
          sharedEnergyCoul[ threadIdx.x ] += sharedEnergyCoul[ threadIdx.x + s ];
      }
    }

    __syncthreads();
  }

  if(threadIdx.x == 0) {

    ENERGY_CFLOAT* buffer = (ENERGY_CFLOAT*) _buffer;

    if(eflag) {
      ENERGY_CFLOAT tmp_evdwl;
      buffer[blockIdx.x * gridDim.y + blockIdx.y + 0 * gridDim.x * gridDim.y] = tmp_evdwl = ENERGY_F(0.5) * sharedEnergy[0];

      if(eflag_atom)
        _eatom[i] = tmp_evdwl;

      buffer = &buffer[gridDim.x * gridDim.y];

      if(coul_type != COUL_NONE) {
        buffer[blockIdx.x * gridDim.y + blockIdx.y + 0 * gridDim.x * gridDim.y] = tmp_evdwl = ENERGY_F(0.5) * sharedEnergyCoul[0];

        if(eflag_atom)
          _eatom[i] += tmp_evdwl;

        buffer = &buffer[gridDim.x * gridDim.y];
      }
    }

    if(vflag) {
      ENERGY_CFLOAT tmp;
      buffer[blockIdx.x * gridDim.y + blockIdx.y + 0 * gridDim.x * gridDim.y] = tmp = ENERGY_F(0.5) * sharedVirial1[0].x;

      if(vflag_atom) _vatom[i + 0 * _nmax] = tmp;

      buffer[blockIdx.x * gridDim.y + blockIdx.y + 1 * gridDim.x * gridDim.y] = tmp = ENERGY_F(0.5) * sharedVirial1[0].y;

      if(vflag_atom) _vatom[i + 1 * _nmax] = tmp;

      buffer[blockIdx.x * gridDim.y + blockIdx.y + 2 * gridDim.x * gridDim.y] = tmp = ENERGY_F(0.5) * sharedVirial1[0].z;

      if(vflag_atom) _vatom[i + 2 * _nmax] = tmp;

      buffer[blockIdx.x * gridDim.y + blockIdx.y + 3 * gridDim.x * gridDim.y] = tmp = ENERGY_F(0.5) * sharedVirial2[0].x;

      if(vflag_atom) _vatom[i + 3 * _nmax] = tmp;

      buffer[blockIdx.x * gridDim.y + blockIdx.y + 4 * gridDim.x * gridDim.y] = tmp = ENERGY_F(0.5) * sharedVirial2[0].y;

      if(vflag_atom) _vatom[i + 4 * _nmax] = tmp;

      buffer[blockIdx.x * gridDim.y + blockIdx.y + 5 * gridDim.x * gridDim.y] = tmp = ENERGY_F(0.5) * sharedVirial2[0].z;

      if(vflag_atom) _vatom[i + 5 * _nmax] = tmp;

      buffer = &buffer[6 * gridDim.x * gridDim.y];
    }

    F_CFLOAT* my_f;

    if(_collect_forces_later) {
      my_f = (F_CFLOAT*) buffer;
      my_f += i;
      *my_f = sharedForce[0].x;
      my_f += _nmax;
      *my_f = sharedForce[0].y;
      my_f += _nmax;
      *my_f = sharedForce[0].z;
    } else {
      my_f = _f + i;
      *my_f += sharedForce[0].x;
      my_f += _nmax;
      *my_f += sharedForce[0].y;
      my_f += _nmax;
      *my_f += sharedForce[0].z;
    }
  }
}

__global__ void Pair_GenerateXType_Kernel()
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nall) {
    X_CFLOAT4 xtype;
    xtype.x = _x[i];
    xtype.y = _x[i + _nmax];
    xtype.z = _x[i + 2 * _nmax];
    xtype.w = _type[i];
    _x_type[i] = xtype;
  }

}

__global__ void Pair_GenerateVRadius_Kernel()
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nall) {
    V_CFLOAT4 vradius;
    vradius.x = _v[i];
    vradius.y = _v[i + _nmax];
    vradius.z = _v[i + 2 * _nmax];
    vradius.w = _radius[i];
    _v_radius[i] = vradius;
  }
}

__global__ void Pair_GenerateOmegaRmass_Kernel()
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nall) {
    V_CFLOAT4 omegarmass;
    omegarmass.x = _omega[i];
    omegarmass.y = _omega[i + _nmax];
    omegarmass.z = _omega[i + 2 * _nmax];
    omegarmass.w = _rmass[i];
    _omega_rmass[i] = omegarmass;
  }
}

__global__ void Pair_RevertXType_Kernel()
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nall) {
    X_CFLOAT4 xtype = _x_type[i];
    _x[i] = xtype.x;
    _x[i + _nmax] = xtype.y;
    _x[i + 2 * _nmax] = xtype.z;
    _type[i] = static_cast <int>(xtype.w);
  }

}

__global__ void Pair_BuildXHold_Kernel()
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i < _nall) {
    X_CFLOAT4 xtype = _x_type[i];
    _xhold[i] = xtype.x;
    _xhold[i + _nmax] = xtype.y;
    _xhold[i + 2 * _nmax] = xtype.z;
  }

}

__global__ void Pair_CollectForces_Kernel(int nperblock, int n)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  if(i >= _nlocal) return;

  ENERGY_CFLOAT* buf = (ENERGY_CFLOAT*) _buffer;

  F_CFLOAT* buf_f = (F_CFLOAT*) &buf[nperblock * n];
  F_CFLOAT* my_f = _f + i;
  buf_f += i;
  *my_f += * buf_f;
  my_f += _nmax;
  buf_f += _nmax;
  *my_f += * buf_f;
  my_f += _nmax;
  buf_f += _nmax;
  *my_f += * buf_f;
  my_f += _nmax;
}
