#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#define SBBITS 30

__global__ void Binning_Kernel(int* binned_id, int bin_nmax, int bin_dim_x, int bin_dim_y, int bin_dim_z,
                               CUDA_CFLOAT rez_bin_size_x, CUDA_CFLOAT rez_bin_size_y, CUDA_CFLOAT rez_bin_size_z)
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;

  /*int* bin_count=(int*) _buffer;
  bin_count=bin_count+20;
  CUDA_CFLOAT* binned_x=(CUDA_CFLOAT*)(bin_count+bin_dim_x*bin_dim_y*bin_dim_z);*/
  CUDA_CFLOAT* binned_x = (CUDA_CFLOAT*) _buffer;
  binned_x = &binned_x[2];
  int* bin_count = (int*) &binned_x[3 * bin_dim_x * bin_dim_y * bin_dim_z * bin_nmax];

  if(i < _nall) {
    // copy atom position from global device memory to local register
    // in this 3 steps to get as much coalesced access as possible
    X_CFLOAT* my_x = _x + i;
    CUDA_CFLOAT x_i = *my_x;
    my_x += _nmax;
    CUDA_CFLOAT y_i = *my_x;
    my_x += _nmax;
    CUDA_CFLOAT z_i = *my_x;


    // calculate flat bin index
    int bx = __float2int_rd(rez_bin_size_x * (x_i - _sublo[0])) + 2;
    int by = __float2int_rd(rez_bin_size_y * (y_i - _sublo[1])) + 2;
    int bz = __float2int_rd(rez_bin_size_z * (z_i - _sublo[2])) + 2;

    bx -= bx * negativCUDA(1.0f * bx);
    bx -= (bx - bin_dim_x + 1) * negativCUDA(1.0f * bin_dim_x - 1.0f - 1.0f * bx);
    by -= by * negativCUDA(1.0f * by);
    by -= (by - bin_dim_y + 1) * negativCUDA(1.0f * bin_dim_y - 1.0f - 1.0f * by);
    bz -= bz * negativCUDA(1.0f * bz);
    bz -= (bz - bin_dim_z + 1) * negativCUDA(1.0f * bin_dim_z - 1.0f - 1.0f * bz);


    const unsigned j = bin_dim_z * (bin_dim_y * bx + by) + bz;

    // add new atom to bin, get bin-array position
    const unsigned k = atomicAdd(& bin_count[j], 1);

    if(k < bin_nmax) {
      binned_id [bin_nmax * j + k] = i;
      binned_x [3 * bin_nmax * j + k] = x_i;
      binned_x [3 * bin_nmax * j + k + bin_nmax] = y_i;
      binned_x [3 * bin_nmax * j + k + 2 * bin_nmax] = z_i;
    } else {
      // normally, this should not happen:
      int errorn = atomicAdd((int*) _buffer, 1);
      MYEMUDBG(printf("# CUDA: Binning_Kernel: WARNING: atom %i ignored, no place left in bin %u\n", i, j);)
    }
  }
}


__device__ inline int exclusion(int &i, int &j, int &itype, int &jtype)
{
  int m;

  if(_nex_type)
    if(_ex_type[itype * _cuda_ntypes + jtype]) return 1;

  if(_nex_group) {
    for(m = 0; m < _nex_group; m++) {
      if(_mask[i] & _ex1_bit[m] && _mask[j] & _ex2_bit[m]) return 1;

      if(_mask[i] & _ex2_bit[m] && _mask[j] & _ex1_bit[m]) return 1;
    }
  }

  if(_nex_mol) {
    if(_molecule[i] == _molecule[j])
      for(m = 0; m < _nex_mol; m++)
        if(_mask[i] & _ex_mol_bit[m] && _mask[j] & _ex_mol_bit[m]) return 1;
  }

  return 0;
}

extern __shared__ CUDA_CFLOAT shared[];

__device__ inline int find_special(int3 &n, int* list, int &tag, int3 flag)
{
  int k = n.z;

  for(int l = 0; l < n.z; l++) k = ((list[l] == tag) ? l : k);

  return k < n.x ? flag.x : (k < n.y ? flag.y : (k < n.z ? flag.z : 0));
}

template <const unsigned int exclude>
__global__ void NeighborBuildFullBin_Kernel(int* binned_id, int bin_nmax, int bin_dim_x, int bin_dim_y, CUDA_CFLOAT globcutoff, int block_style, bool neighall)
{
  int natoms = neighall ? _nall : _nlocal;
  //const bool domol=false;
  int bin_dim_z = gridDim.y;
  CUDA_CFLOAT* binned_x = (CUDA_CFLOAT*) _buffer;
  binned_x = &binned_x[2];
  int* bin_count = (int*) &binned_x[3 * bin_dim_x * bin_dim_y * bin_dim_z * bin_nmax];
  int bin = __mul24(gridDim.y, blockIdx.x) + blockIdx.y;
  int bin_x = blockIdx.x / bin_dim_y;
  int bin_y = blockIdx.x - bin_x * bin_dim_y;
  int bin_z = blockIdx.y;
  int bin_c = bin_count[bin];


  CUDA_CFLOAT cut;

  if(globcutoff > 0)
    cut = globcutoff;

  int i = _nall;
  CUDA_CFLOAT* my_x;
  CUDA_CFLOAT x_i, y_i, z_i;

  for(int actOffset = 0; actOffset < bin_c; actOffset += blockDim.x) {

    int actIdx = threadIdx.x + actOffset;
    CUDA_CFLOAT* other_x = shared;
    int* other_id = (int*) &other_x[3 * blockDim.x];

    if(actIdx < bin_c) {
      i = binned_id[__mul24(bin, bin_nmax) + actIdx];
      my_x = binned_x + __mul24(__mul24(bin, 3), bin_nmax) + actIdx;
      x_i = *my_x;
      my_x += bin_nmax;
      y_i = *my_x;
      my_x += bin_nmax;
      z_i = *my_x;
    } else
      i = 2 * _nall;

    __syncthreads();

    int jnum = 0;
    int itype;

    if(i < natoms) {
      jnum = 0;
      _ilist[i] = i;
      itype = _type[i];
    }

    //__syncthreads();


    for(int otherActOffset = 0; otherActOffset < bin_c; otherActOffset += blockDim.x) {
      int otherActIdx = threadIdx.x + otherActOffset;

      if(otherActIdx < bin_c) {
        if(otherActOffset == actOffset) {
          other_id[threadIdx.x] = i;
          other_x[threadIdx.x] = x_i;
          other_x[threadIdx.x + blockDim.x] = y_i;
          other_x[threadIdx.x + 2 * blockDim.x] = z_i;
        } else {
          other_id[threadIdx.x] = binned_id[__mul24(bin, bin_nmax) + otherActIdx];
          my_x = binned_x + __mul24(__mul24(bin, 3), bin_nmax) + otherActIdx;
          other_x[threadIdx.x] = *my_x;
          my_x += bin_nmax;
          other_x[threadIdx.x + blockDim.x] = *my_x;
          my_x += bin_nmax;
          other_x[threadIdx.x + __mul24(2, blockDim.x)] = *my_x;

        }
      }

      __syncthreads();
      int kk = threadIdx.x;

      for(int k = 0; k < MIN(bin_c - otherActOffset, blockDim.x); ++k) {
        if(i < natoms) {
          kk++;
          kk = kk < MIN(bin_c - otherActOffset, blockDim.x) ? kk : 0;
          int j = other_id[kk];

          if(exclude && exclusion(i, j, itype, _type[j])) continue;

          if(globcutoff < 0) {
            int jtype = _type[j];
            cut = _cutneighsq[itype * _cuda_ntypes + jtype];
          }

          CUDA_CFLOAT delx = x_i - other_x[kk];
          CUDA_CFLOAT dely = y_i - other_x[kk + blockDim.x];
          CUDA_CFLOAT delz = z_i - other_x[kk + 2 * blockDim.x];
          CUDA_CFLOAT rsq = delx * delx + dely * dely + delz * delz;


          if(rsq <= cut && i != j) {
            if(jnum < _maxneighbors) {
              if(block_style)
                _neighbors[i * _maxneighbors + jnum] = j;
              else
                _neighbors[i + jnum * natoms] = j;
            }

            ++jnum;
          }
        }
      }

      __syncthreads();

    }

    for(int obin_x = bin_x - 1; obin_x < bin_x + 2; obin_x++)
      for(int obin_y = bin_y - 1; obin_y < bin_y + 2; obin_y++)
        for(int obin_z = bin_z - 1; obin_z < bin_z + 2; obin_z++) {
          if(obin_x < 0 || obin_y < 0 || obin_z < 0) continue;

          if(obin_x >= bin_dim_x || obin_y >= bin_dim_y || obin_z >= bin_dim_z) continue;

          int other_bin = bin_dim_z * (bin_dim_y * obin_x + obin_y) + obin_z;

          if(other_bin == bin) continue;

          int obin_c = bin_count[other_bin];

          for(int otherActOffset = 0; otherActOffset < obin_c; otherActOffset += blockDim.x) {
            int otherActIdx = otherActOffset + threadIdx.x;

            if(threadIdx.x < MIN(blockDim.x, obin_c - otherActOffset)) {
              other_id[threadIdx.x] = binned_id[__mul24(other_bin, bin_nmax) + otherActIdx];
              my_x = binned_x + __mul24(__mul24(other_bin, 3), bin_nmax) + otherActIdx;
              other_x[threadIdx.x] = *my_x;
              my_x += bin_nmax;
              other_x[threadIdx.x + blockDim.x] = *my_x;
              my_x += bin_nmax;
              other_x[threadIdx.x + 2 * blockDim.x] = *my_x;
            }

            __syncthreads();

            for(int k = 0; k < MIN(blockDim.x, obin_c - otherActOffset); ++k) {
              if(i < natoms) {
                int j = other_id[k];

                if(exclude && exclusion(i, j, itype, _type[j])) continue;

                if(globcutoff < 0) {
                  int jtype = _type[j];
                  cut = _cutneighsq[itype * _cuda_ntypes + jtype];
                }

                CUDA_CFLOAT delx = x_i - other_x[k];
                CUDA_CFLOAT dely = y_i - other_x[k + blockDim.x];
                CUDA_CFLOAT delz = z_i - other_x[k + 2 * blockDim.x];
                CUDA_CFLOAT rsq = delx * delx + dely * dely + delz * delz;

                if(rsq <= cut && i != j) {
                  if(jnum < _maxneighbors) {
                    if(block_style)
                      _neighbors[i * _maxneighbors + jnum] = j;
                    else
                      _neighbors[i + jnum * natoms] = j;
                  }

                  ++jnum;
                }
              }
            }

            __syncthreads();

          }
        }

    if(jnum > _maxneighbors)((int*)_buffer)[0] = -jnum;

    if(i < natoms)
      _numneigh[i] = jnum;
  }
}


__global__ void FindSpecial(int block_style)
{
  int ii = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int which;
  int tag_mask = 0;
  int3 spec_flag;

  int3 mynspecial = {0, 0, 1};

  if(ii >= _nlocal) return;

  int special_id[CUDA_MAX_NSPECIAL];

  int i = _ilist[ii];

  if(i >= _nlocal) return;

  int jnum = _numneigh[i];

  if(_special_flag[1] == 0) spec_flag.x = -1;
  else if(_special_flag[1] == 1) spec_flag.x = 0;
  else spec_flag.x = 1;

  if(_special_flag[2] == 0) spec_flag.y = -1;
  else if(_special_flag[2] == 1) spec_flag.y = 0;
  else spec_flag.y = 2;

  if(_special_flag[3] == 0) spec_flag.z = -1;
  else if(_special_flag[3] == 1) spec_flag.z = 0;
  else spec_flag.z = 3;

  mynspecial.x = _nspecial[i];
  mynspecial.y = _nspecial[i + _nmax];
  mynspecial.z = _nspecial[i + 2 * _nmax];

  if(i < _nlocal) {
    int* list = &_special[i];

    for(int k = 0; k < mynspecial.z; k++) {
      special_id[k] = list[k * _nmax];
      tag_mask = tag_mask | special_id[k];
    }
  }


  for(int k = 0; k < MIN(jnum, _maxneighbors); k++) {
    int j;

    if(block_style)
      j = _neighbors[i * _maxneighbors + k];
    else
      j = _neighbors[i + k * _nlocal];

    int tag_j = _tag[j];
    which = 0;

    if((tag_mask & tag_j) == tag_j) {
      which = find_special(mynspecial, special_id, tag_j, spec_flag);

      if(which > 0) {
        if(block_style)
          _neighbors[i * _maxneighbors + k] = j ^ (which << SBBITS);
        else
          _neighbors[i + k * _nlocal] = j ^ (which << SBBITS);
      } else if(which < 0) {
        if(block_style)
          _neighbors[i * _maxneighbors + k] = _neighbors[i * _maxneighbors + jnum - 1];
        else
          _neighbors[i + k * _nlocal] = _neighbors[i + (jnum - 1) * _nlocal];

        jnum--;
        k--;
      }
    }
  }

  _numneigh[i] = jnum;
}

__global__ void NeighborBuildFullBin_OverlapComm_Kernel(int* binned_id, int bin_nmax, int bin_dim_x, int bin_dim_y, CUDA_CFLOAT globcutoff, int block_style)
{
  int bin_dim_z = gridDim.y;
  CUDA_CFLOAT* binned_x = (CUDA_CFLOAT*) _buffer;
  binned_x = &binned_x[2];
  int* bin_count = (int*) &binned_x[3 * bin_dim_x * bin_dim_y * bin_dim_z * bin_nmax];
  int bin = __mul24(gridDim.y, blockIdx.x) + blockIdx.y;
  int bin_x = blockIdx.x / bin_dim_y;
  int bin_y = blockIdx.x - bin_x * bin_dim_y;
  int bin_z = blockIdx.y;
  int bin_c = bin_count[bin];


  CUDA_CFLOAT cut;

  if(globcutoff > 0)
    cut = globcutoff;

  int i = _nall;
  CUDA_CFLOAT* my_x;
  CUDA_CFLOAT x_i, y_i, z_i;

  for(int actOffset = 0; actOffset < bin_c; actOffset += blockDim.x) {

    int actIdx = threadIdx.x + actOffset;
    CUDA_CFLOAT* other_x = shared;
    int* other_id = (int*) &other_x[3 * blockDim.x];

    if(actIdx < bin_c) {
      i = binned_id[__mul24(bin, bin_nmax) + actIdx];
      my_x = binned_x + __mul24(__mul24(bin, 3), bin_nmax) + actIdx;
      x_i = *my_x;
      my_x += bin_nmax;
      y_i = *my_x;
      my_x += bin_nmax;
      z_i = *my_x;
    } else
      i = 2 * _nall;

    __syncthreads();

    int jnum = 0;
    int jnum_border = 0;
    int jnum_inner = 0;
    int i_border = -1;
    int itype;

    if(i < _nlocal) {
      jnum = 0;
      _ilist[i] = i;
      itype = _type[i];
    }

    __syncthreads();


    for(int otherActOffset = 0; otherActOffset < bin_c; otherActOffset += blockDim.x) {
      int otherActIdx = threadIdx.x + otherActOffset;

      if(otherActIdx < bin_c) {
        if(otherActOffset == actOffset) {
          other_id[threadIdx.x] = i;
          other_x[threadIdx.x] = x_i;
          other_x[threadIdx.x + blockDim.x] = y_i;
          other_x[threadIdx.x + 2 * blockDim.x] = z_i;
        } else {
          other_id[threadIdx.x] = binned_id[__mul24(bin, bin_nmax) + otherActIdx];
          my_x = binned_x + __mul24(__mul24(bin, 3), bin_nmax) + otherActIdx;
          other_x[threadIdx.x] = *my_x;
          my_x += bin_nmax;
          other_x[threadIdx.x + blockDim.x] = *my_x;
          my_x += bin_nmax;
          other_x[threadIdx.x + __mul24(2, blockDim.x)] = *my_x;

        }
      }

      __syncthreads();
      int kk = threadIdx.x;

      for(int k = 0; k < MIN(bin_c - otherActOffset, blockDim.x); ++k) {
        if(i < _nlocal) {
          kk++;
          kk = kk < MIN(bin_c - otherActOffset, blockDim.x) ? kk : 0;
          int j = other_id[kk];

          if(globcutoff < 0) {
            int jtype = _type[j];
            cut = _cutneighsq[itype * _cuda_ntypes + jtype];
          }

          CUDA_CFLOAT delx = x_i - other_x[kk];
          CUDA_CFLOAT dely = y_i - other_x[kk + blockDim.x];
          CUDA_CFLOAT delz = z_i - other_x[kk + 2 * blockDim.x];
          CUDA_CFLOAT rsq = delx * delx + dely * dely + delz * delz;


          if(rsq <= cut && i != j) {
            if((j >= _nlocal) && (i_border < 0))
              i_border = atomicAdd(_inum_border, 1);

            if(jnum < _maxneighbors) {
              if(block_style) {
                _neighbors[i * _maxneighbors + jnum] = j;

                if(j >= _nlocal) {
                  _neighbors_border[i_border * _maxneighbors + jnum_border] = j;
                } else {
                  _neighbors_inner[i * _maxneighbors + jnum_inner] = j;
                }
              } else {
                _neighbors[i + jnum * _nlocal] = j;

                if(j >= _nlocal) {
                  _neighbors_border[i_border + jnum_border * _nlocal] = j;
                } else {
                  _neighbors_inner[i + jnum_inner * _nlocal] = j;
                }
              }
            }

            ++jnum;

            if(j >= _nlocal)
              jnum_border++;
            else
              jnum_inner++;
          }
        }
      }

      __syncthreads();
    }

    for(int obin_x = bin_x - 1; obin_x < bin_x + 2; obin_x++)
      for(int obin_y = bin_y - 1; obin_y < bin_y + 2; obin_y++)
        for(int obin_z = bin_z - 1; obin_z < bin_z + 2; obin_z++) {
          if(obin_x < 0 || obin_y < 0 || obin_z < 0) continue;

          if(obin_x >= bin_dim_x || obin_y >= bin_dim_y || obin_z >= bin_dim_z) continue;

          int other_bin = bin_dim_z * (bin_dim_y * obin_x + obin_y) + obin_z;

          if(other_bin == bin) continue;

          int obin_c = bin_count[other_bin];

          for(int otherActOffset = 0; otherActOffset < obin_c; otherActOffset += blockDim.x) {
            int otherActIdx = otherActOffset + threadIdx.x;

            if(threadIdx.x < MIN(blockDim.x, obin_c - otherActOffset)) {
              other_id[threadIdx.x] = binned_id[__mul24(other_bin, bin_nmax) + otherActIdx];
              my_x = binned_x + __mul24(__mul24(other_bin, 3), bin_nmax) + otherActIdx;
              other_x[threadIdx.x] = *my_x;
              my_x += bin_nmax;
              other_x[threadIdx.x + blockDim.x] = *my_x;
              my_x += bin_nmax;
              other_x[threadIdx.x + 2 * blockDim.x] = *my_x;
            }

            __syncthreads();

            for(int k = 0; k < MIN(blockDim.x, obin_c - otherActOffset); ++k) {
              if(i < _nlocal) {
                int j = other_id[k];

                if(globcutoff < 0) {
                  int jtype = _type[j];
                  cut = _cutneighsq[itype * _cuda_ntypes + jtype];
                }

                CUDA_CFLOAT delx = x_i - other_x[k];
                CUDA_CFLOAT dely = y_i - other_x[k + blockDim.x];
                CUDA_CFLOAT delz = z_i - other_x[k + 2 * blockDim.x];
                CUDA_CFLOAT rsq = delx * delx + dely * dely + delz * delz;

                if(rsq <= cut && i != j) {
                  if((j >= _nlocal) && (i_border < 0))
                    i_border = atomicAdd(_inum_border, 1);

                  if(jnum < _maxneighbors) {
                    if(block_style) {
                      _neighbors[i * _maxneighbors + jnum] = j;

                      if(j >= _nlocal) {
                        _neighbors_border[i_border * _maxneighbors + jnum_border] = j;
                      } else {
                        _neighbors_inner[i * _maxneighbors + jnum_inner] = j;
                      }
                    } else {
                      _neighbors[i + jnum * _nlocal] = j;

                      if(j >= _nlocal) {
                        _neighbors_border[i_border + jnum_border * _nlocal] = j;
                      } else {
                        _neighbors_inner[i + jnum_inner * _nlocal] = j;
                      }
                    }
                  }

                  ++jnum;

                  if(j >= _nlocal)
                    jnum_border++;
                  else
                    jnum_inner++;
                }
              }
            }

            __syncthreads();
          }
        }

    if(jnum > _maxneighbors)((int*)_buffer)[0] = -jnum;

    if(i < _nlocal) {
      _numneigh[i] = jnum;
      _numneigh_inner[i] = jnum_inner;

      if(i_border >= 0) _numneigh_border[i_border] = jnum_border;

      if(i_border >= 0) _ilist_border[i_border] = i;

    }
  }
}

__global__ void NeighborBuildFullNsq_Kernel()
{
  int i = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
  int* buffer = (int*) _buffer;

  if(i < _nlocal) {
    X_CFLOAT* my_x = _x + i;
    CUDA_CFLOAT x_i = *my_x;
    my_x += _nmax;
    CUDA_CFLOAT y_i = *my_x;
    my_x += _nmax;
    CUDA_CFLOAT z_i = *my_x;
    int jnum = 0;
    int* jlist = _firstneigh[i];
    _ilist[i] = i;

    int itype = _type[i];
    __syncthreads();

    for(int j = 0; j < _nall; ++j) {
      my_x = _x + j;
      CUDA_CFLOAT x_j = *my_x;
      my_x += _nmax;
      CUDA_CFLOAT y_j = *my_x;
      my_x += _nmax;
      CUDA_CFLOAT z_j = *my_x;
      CUDA_CFLOAT delx = x_i - x_j;
      CUDA_CFLOAT dely = y_i - y_j;
      CUDA_CFLOAT delz = z_i - z_j;
      CUDA_CFLOAT rsq = delx * delx + dely * dely + delz * delz;
      int jtype = _type[j];

      if(rsq <= _cutneighsq[itype * _cuda_ntypes + jtype] && i != j) {
        if(jnum < _maxneighbors)
          jlist[jnum] = j;

        if(i == 151)((int*)_buffer)[jnum + 2] = j;

        ++jnum;
      }

      __syncthreads();
    }

    if(jnum > _maxneighbors) buffer[0] = 0;

    _numneigh[i] = jnum;

    if(i == 151)((int*)_buffer)[1] = jnum;
  }
}

