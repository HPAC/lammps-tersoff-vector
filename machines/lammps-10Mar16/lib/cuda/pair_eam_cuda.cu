#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>

#define _type2frho MY_AP(coeff1)
#define _type2rhor MY_AP(coeff2)
#define _type2z2r MY_AP(coeff3)
#define _rdr MY_AP(rdr)
#define _rdrho MY_AP(rdrho)
#define _nr MY_AP(nr)
#define _nrho MY_AP(nrho)
#define _nfrho MY_AP(nfrho)
#define _nrhor MY_AP(nrhor)
#define _nz2r MY_AP(nz2r)
#define _frho_spline MY_AP(frho_spline)
#define _rhor_spline MY_AP(rhor_spline)
#define _z2r_spline MY_AP(z2r_spline)
#define _rho MY_AP(rho)
#define _fp MY_AP(fp)

__device__ __constant__ F_CFLOAT MY_AP(rdr);
__device__ __constant__ F_CFLOAT MY_AP(rdrho);
__device__ __constant__ int MY_AP(nr);
__device__ __constant__ int MY_AP(nrho);
__device__ __constant__ int MY_AP(nfrho);
__device__ __constant__ int MY_AP(nrhor);
__device__ __constant__ int MY_AP(nz2r);
__device__ __constant__ F_CFLOAT* MY_AP(frho_spline);
__device__ __constant__ F_CFLOAT* MY_AP(rhor_spline);
__device__ __constant__ F_CFLOAT* MY_AP(z2r_spline);
__device__ __constant__ F_CFLOAT* MY_AP(rho);
__device__ __constant__ F_CFLOAT* MY_AP(fp);

#define _rhor_spline_tex         MY_AP(rhor_spline_tex)
#if F_PRECISION == 1
texture<float4, 1> _rhor_spline_tex;
#else
texture<int4, 1> _rhor_spline_tex;
#endif


#define _z2r_spline_tex         MY_AP(z2r_spline_tex)
#if F_PRECISION == 1
texture<float4, 1> _z2r_spline_tex;
#else
texture<int4, 1> _z2r_spline_tex;
#endif



#include "pair_eam_cuda_cu.h"
#include "pair_eam_cuda_kernel_nc.cu"
#include <time.h>

int eam_buff_offset;
int rhor_spline_size;
void* rhor_spline_pointer;
int z2r_spline_size;
void* z2r_spline_pointer;


inline void BindEAMTextures(cuda_shared_data* sdata)
{
  _rhor_spline_tex.normalized = false;                      // access with normalized texture coordinates
  _rhor_spline_tex.filterMode = hipFilterModePoint;        // Point mode, so no
  _rhor_spline_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

  const textureReference* rhor_spline_texture_ptr = &MY_AP(rhor_spline_tex);

#if F_PRECISION == 1
  hipChannelFormatDesc channelDescRhor = hipCreateChannelDesc<float4>();
  hipBindTexture(0, rhor_spline_texture_ptr, rhor_spline_pointer, &channelDescRhor, rhor_spline_size);
#else
  hipChannelFormatDesc channelDescRhor = hipCreateChannelDesc<int4>();
  hipBindTexture(0, rhor_spline_texture_ptr, rhor_spline_pointer, &channelDescRhor, rhor_spline_size);
#endif

  _z2r_spline_tex.normalized = false;                      // access with normalized texture coordinates
  _z2r_spline_tex.filterMode = hipFilterModePoint;        // Point mode, so no
  _z2r_spline_tex.addressMode[0] = hipAddressModeWrap;    // wrap texture coordinates

  const textureReference* z2r_spline_texture_ptr = &MY_AP(z2r_spline_tex);

#if F_PRECISION == 1
  hipChannelFormatDesc channelDescZ2r = hipCreateChannelDesc<float4>();
  hipBindTexture(0, z2r_spline_texture_ptr, z2r_spline_pointer, &channelDescZ2r, z2r_spline_size);
#else
  hipChannelFormatDesc channelDescZ2r = hipCreateChannelDesc<int4>();
  hipBindTexture(0, z2r_spline_texture_ptr, z2r_spline_pointer, &channelDescZ2r, z2r_spline_size);
#endif

}

void Cuda_PairEAMCuda_UpdateBuffer(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist)
{
  CUT_CHECK_ERROR("Cuda_PairEAMCuda: before updateBuffer failed");
  int3 layout = getgrid(sneighlist->inum, 7 * sizeof(F_CFLOAT));
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  int size = (unsigned)(layout.y * layout.x) * 7 * sizeof(F_CFLOAT);

  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_PairEAMCuda Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)

    if(sdata->buffer != NULL) hipFree(sdata->buffer);

    hipMalloc((void**)&sdata->buffer, size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)
  }

  hipMemcpyToSymbol(MY_AP(buffer), & sdata->buffer, sizeof(int*));
  CUT_CHECK_ERROR("Cuda_PairEAMCuda: updateBuffer failed");
}

void Cuda_PairEAMCuda_UpdateNeighbor(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist)
{
  hipMemcpyToSymbol(MY_AP(neighbor_maxlocal) , & sneighlist->firstneigh.dim[0]  , sizeof(unsigned));
  hipMemcpyToSymbol(MY_AP(firstneigh), & sneighlist->firstneigh.dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(ilist)     , & sneighlist->ilist     .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(inum)      , & sneighlist->inum               , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal               , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)      , & sdata->atom.nmax               , sizeof(int));
  hipMemcpyToSymbol(MY_AP(numneigh)  , & sneighlist->numneigh  .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(neighbors)      , & sneighlist->neighbors  .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(maxneighbors)       , & sneighlist->maxneighbors     , sizeof(int));
}

void Cuda_PairEAMCuda_UpdateNmax(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist)
{
  CUT_CHECK_ERROR("Cuda_PairEAMCuda: before updateNmax failed");
  hipMemcpyToSymbol(MY_AP(x)         , & sdata->atom.x         .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(x_type)         	, & sdata->atom.x_type    .dev_data, sizeof(X_CFLOAT4*));
  hipMemcpyToSymbol(MY_AP(f)         			, & sdata->atom.f         .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(type)      			, & sdata->atom.type      .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(tag)      			, & sdata->atom.tag       .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(eatom)     			, & sdata->atom.eatom     .dev_data, sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(vatom)     			, & sdata->atom.vatom     .dev_data, sizeof(ENERGY_CFLOAT*));
  CUT_CHECK_ERROR("Cuda_PairEAMCuda: updateNmax failed");
}


void Cuda_PairEAMCuda_Init(cuda_shared_data* sdata, double rdr, double rdrho, int nfrho, int nrhor, int nr, int nrho, int nz2r,
                           void* frho_spline, void* rhor_spline, void* z2r_spline, void* rho, void* fp,
                           int* type2frho, int** type2z2r, int** type2rhor)
{
  // !! LAMMPS indexes atom types starting with 1 !!

  unsigned cuda_ntypes = sdata->atom.ntypes + 1;

  if(cuda_ntypes * cuda_ntypes > CUDA_MAX_TYPES2)
    printf("# CUDA: Cuda_PairEAMCuda_Init: you need %u types. this is more than %u "
           "(assumed at compile time). re-compile with -DCUDA_MAX_TYPES_PLUS_ONE=99 "
           "or ajust this in cuda_common.h\n", cuda_ntypes, CUDA_MAX_TYPES2);

  unsigned nI = sizeof(F_CFLOAT) * cuda_ntypes * cuda_ntypes;

  X_CFLOAT cutsq_global;
  cutsq_global = (X_CFLOAT)(sdata->pair.cut_global);
  hipMemcpyToSymbol(MY_AP(cutsq_global)	, &cutsq_global  				, sizeof(X_CFLOAT));


  F_CFLOAT* coeff_buf = new F_CFLOAT[cuda_ntypes * cuda_ntypes];

  for(int i = 0; i < cuda_ntypes; i++) coeff_buf[i] = type2frho[i];

  hipMemcpyToSymbol(MY_AP(coeff1)        , coeff_buf             , cuda_ntypes * sizeof(F_CFLOAT));

  for(int i = 0; i < cuda_ntypes * cuda_ntypes; i++) coeff_buf[i] = (&type2rhor[0][0])[i];

  hipMemcpyToSymbol(MY_AP(coeff2)        , coeff_buf             , nI);

  for(int i = 0; i < cuda_ntypes * cuda_ntypes; i++) coeff_buf[i] = (&type2z2r[0][0])[i];

  hipMemcpyToSymbol(MY_AP(coeff3)        , coeff_buf             , nI);

  delete [] coeff_buf;
  X_CFLOAT box_size[3] = {
    sdata->domain.subhi[0] - sdata->domain.sublo[0],
    sdata->domain.subhi[1] - sdata->domain.sublo[1],
    sdata->domain.subhi[2] - sdata->domain.sublo[2]
  };
  F_CFLOAT rdr_F = rdr;
  F_CFLOAT rdrho_F = rdrho;
  hipMemcpyToSymbol(MY_AP(box_size)   , box_size                 , sizeof(X_CFLOAT) * 3);
  hipMemcpyToSymbol(MY_AP(cuda_ntypes), & cuda_ntypes            , sizeof(unsigned));
  hipMemcpyToSymbol(MY_AP(virial)     , &sdata->pair.virial.dev_data   , sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(eng_vdwl)     , &sdata->pair.eng_vdwl.dev_data   , sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(periodicity), sdata->domain.periodicity, sizeof(int) * 3);
  hipMemcpyToSymbol(MY_AP(collect_forces_later), &sdata->pair.collect_forces_later  , sizeof(int));
  hipMemcpyToSymbol(MY_AP(rdr), &rdr_F, sizeof(F_CFLOAT));
  hipMemcpyToSymbol(MY_AP(rdrho), &rdrho_F, sizeof(F_CFLOAT));
  hipMemcpyToSymbol(MY_AP(nr), &nr, sizeof(int));
  hipMemcpyToSymbol(MY_AP(nrho), &nrho, sizeof(int));
  hipMemcpyToSymbol(MY_AP(nfrho), &nfrho, sizeof(int));
  hipMemcpyToSymbol(MY_AP(nrhor), &nrhor, sizeof(int));
  hipMemcpyToSymbol(MY_AP(rho), &rho, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(fp), &fp, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(frho_spline), &frho_spline, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(rhor_spline), &rhor_spline, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(z2r_spline), &z2r_spline, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(nrhor), &nrhor, sizeof(int));

  rhor_spline_size = nrhor * (nr + 1) * EAM_COEFF_LENGTH * sizeof(F_CFLOAT);
  z2r_spline_size = nz2r * (nr + 1) * EAM_COEFF_LENGTH * sizeof(F_CFLOAT);
  rhor_spline_pointer = rhor_spline;
  z2r_spline_pointer = z2r_spline;

  CUT_CHECK_ERROR("Cuda_PairEAMCuda: init failed");

}



void Cuda_PairEAM1Cuda(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist, int eflag, int vflag, int eflag_atom, int vflag_atom)
{

  if(sdata->atom.update_nmax)
    Cuda_PairEAMCuda_UpdateNmax(sdata, sneighlist);

  if(sdata->atom.update_neigh)
    Cuda_PairEAMCuda_UpdateNeighbor(sdata, sneighlist);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  if(sdata->buffer_new)
    Cuda_PairEAMCuda_UpdateBuffer(sdata, sneighlist);

  hipMemcpyToSymbol(MY_AP(eatom)     			, & sdata->atom.eatom     .dev_data, sizeof(ENERGY_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(vatom)     			, & sdata->atom.vatom     .dev_data, sizeof(ENERGY_CFLOAT*));

  int sharedperproc = 0;

  if(eflag || eflag_atom) sharedperproc = 1;

  if(vflag || vflag_atom) sharedperproc = 7;

  int3 layout = getgrid(sneighlist->inum, sharedperproc * sizeof(ENERGY_CFLOAT));
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  eam_buff_offset = grid.x * grid.y;

  BindXTypeTexture(sdata);
  BindEAMTextures(sdata); // initialize only on first call


  MYDBG(printf("# CUDA: Cuda_PairEAMCuda: kernel start eflag: %i vflag: %i\n", eflag, vflag);)
  CUT_CHECK_ERROR("Cuda_PairEAMCuda: pre pair Kernel 1 problems before kernel invocation");
  PairEAMCuda_Kernel1 <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x>>> (eflag, vflag, eflag_atom, vflag_atom);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_PairEAMCuda: pair Kernel 1 execution failed");



  MYDBG(printf("# CUDA: Cuda_PairEAMCoulLongCuda: kernel done\n");)

}

void Cuda_PairEAM2Cuda(cuda_shared_data* sdata, cuda_shared_neighlist* sneighlist, int eflag, int vflag, int eflag_atom, int vflag_atom)
{
  int sharedperproc = 0;

  if(eflag || eflag_atom) sharedperproc = 1;

  if(vflag || vflag_atom) sharedperproc = 7;

  int3 layout = getgrid(sneighlist->inum, sharedperproc * sizeof(ENERGY_CFLOAT));
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  BindXTypeTexture(sdata);
  BindEAMTextures(sdata); // initialize only on first call
  // initialize only on first call
  sdata->pair.lastgridsize = grid.x * grid.y;
  sdata->pair.n_energy_virial = sharedperproc;

  MYDBG(printf("# CUDA: Cuda_PairEAMCuda: kernel start eflag: %i vflag: %i\n", eflag, vflag);)
  CUT_CHECK_ERROR("Cuda_PairEAMCuda: pre pair Kernel 2 problems before kernel invocation");
  PairEAMCuda_Kernel2 <<< grid, threads, sharedperproc* sizeof(ENERGY_CFLOAT)*threads.x>>> (eflag, vflag, eflag_atom, vflag_atom);
  CUT_CHECK_ERROR("Cuda_PairEAMCuda: pair Kernel 2 start failed");
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_PairEAMCuda: pair Kernel 2 execution failed");

  if(eflag || vflag) {
    int n = grid.x * grid.y;
    grid.x = sharedperproc;
    grid.y = 1;
    threads.x = 256;
    MY_AP(PairVirialCompute_reduce) <<< grid, threads, threads.x* sizeof(ENERGY_CFLOAT)*sharedperproc>>>(n);
    hipDeviceSynchronize();
    CUT_CHECK_ERROR("Cuda_PairEAMCuda: virial compute Kernel execution failed");
  }

  MYDBG(printf("# CUDA: Cuda_PairEAMCoulLongCuda: kernel done\n");)

}

void Cuda_PairEAMCuda_PackComm(cuda_shared_data* sdata, int n, int iswap, void* buf_send)
{
  int3 layout = getgrid(n, 0);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  F_CFLOAT* buf = (F_CFLOAT*)(& ((double*)sdata->buffer)[eam_buff_offset]);

  PairEAMCuda_PackComm_Kernel <<< grid, threads, 0>>> ((int*) sdata->comm.sendlist.dev_data, n
      , sdata->comm.maxlistlength, iswap, buf);
  hipDeviceSynchronize();
  hipMemcpy(buf_send, buf, n* sizeof(F_CFLOAT), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
}

void Cuda_PairEAMCuda_UnpackComm(cuda_shared_data* sdata, int n, int first, void* buf_recv, void* fp)
{
  F_CFLOAT* fp_first = &(((F_CFLOAT*) fp)[first]);
  hipMemcpy(fp_first, buf_recv, n * sizeof(F_CFLOAT), hipMemcpyHostToDevice);
}

#undef _type2frho
#undef _type2rhor
#undef _type2z2r


/* ----------------------------------------------------------------------
   tally eng_vdwl and virial into global and per-atom accumulators
   need i < nlocal test since called by bond_quartic and dihedral_charmm
------------------------------------------------------------------------- */

