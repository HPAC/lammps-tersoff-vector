#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX fix_gravity_cuda
#include "cuda_shared.h"
#include "cuda_common.h"
#include "crm_cuda_utils.cu"

#include "fix_gravity_cuda_cu.h"
#include "fix_gravity_cuda_kernel.cu"

void Cuda_FixGravityCuda_UpdateBuffer(cuda_shared_data* sdata)
{
  int3 layout = getgrid(sdata->atom.nlocal, 3 * sizeof(F_CFLOAT));
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  int size = (unsigned)(layout.z * layout.y * layout.x) * 3 * sizeof(F_CFLOAT);

  if(sdata->buffersize < size) {
    MYDBG(printf("Cuda_FixGravityCuda Resizing Buffer at %p with %i kB to\n", sdata->buffer, sdata->buffersize);)
    CudaWrapper_FreeCudaData(sdata->buffer, sdata->buffersize);
    sdata->buffer = CudaWrapper_AllocCudaData(size);
    sdata->buffersize = size;
    sdata->buffer_new++;
    MYDBG(printf("New buffer at %p with %i kB\n", sdata->buffer, sdata->buffersize);)

  }

  hipMemcpyToSymbol(MY_AP(buffer) , & sdata->buffer, sizeof(int*));
}

void Cuda_FixGravityCuda_UpdateNmax(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(x)       , & sdata->atom.x    .dev_data, sizeof(X_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(f)       , & sdata->atom.f    .dev_data, sizeof(F_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(type)       , & sdata->atom.type    .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(rmass_flag)       , & sdata->atom.rmass_flag, sizeof(int));
  hipMemcpyToSymbol(MY_AP(rmass)       , & sdata->atom.rmass    .dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(mass)       , & sdata->atom.mass    .dev_data, sizeof(V_CFLOAT*));
}

void Cuda_FixGravityCuda_Init(cuda_shared_data* sdata)
{
  Cuda_FixGravityCuda_UpdateNmax(sdata);

}


void Cuda_FixGravityCuda_PostForce(cuda_shared_data* sdata, int groupbit, F_CFLOAT xacc, F_CFLOAT yacc, F_CFLOAT zacc)
{
  if(sdata->atom.update_nmax)
    Cuda_FixGravityCuda_UpdateNmax(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  if(sdata->buffer_new)
    Cuda_FixGravityCuda_UpdateBuffer(sdata);


  int3 layout = getgrid(sdata->atom.nlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);

  Cuda_FixGravityCuda_PostForce_Kernel <<< grid, threads>>> (groupbit, xacc, yacc, zacc);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Cuda_FixGravityCuda_PostForce: fix add_force post_force compute Kernel execution failed");
}
