#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator

   Original Version:
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   See the README file in the top-level LAMMPS directory.

   -----------------------------------------------------------------------

   USER-CUDA Package and associated modifications:
   https://sourceforge.net/projects/lammpscuda/

   Christian Trott, christian.trott@tu-ilmenau.de
   Lars Winterfeld, lars.winterfeld@tu-ilmenau.de
   Theoretical Physics II, University of Technology Ilmenau, Germany

   See the README file in the USER-CUDA directory.

   This software is distributed under the GNU General Public License.
------------------------------------------------------------------------- */

#include <stdio.h>
#define MY_PREFIX fix_enforce2d_cuda
#include "cuda_shared.h"
#include "cuda_common.h"
#include "crm_cuda_utils.cu"
#include "fix_enforce2d_cuda_cu.h"
#include "fix_enforce2d_cuda_kernel.cu"

void Cuda_FixEnforce2dCuda_Init(cuda_shared_data* sdata)
{
  hipMemcpyToSymbol(MY_AP(mask)    , & sdata->atom.mask .dev_data, sizeof(int*));
  hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));
  hipMemcpyToSymbol(MY_AP(nmax)    , & sdata->atom.nmax          , sizeof(int));
  hipMemcpyToSymbol(MY_AP(v)       , & sdata->atom.v    .dev_data, sizeof(V_CFLOAT*));
  hipMemcpyToSymbol(MY_AP(f)       , & sdata->atom.f    .dev_data, sizeof(F_CFLOAT*));
}

void Cuda_FixEnforce2dCuda_PostForce(cuda_shared_data* sdata, int groupbit)
{
  if(sdata->atom.update_nmax)
    Cuda_FixEnforce2dCuda_Init(sdata);

  if(sdata->atom.update_nlocal)
    hipMemcpyToSymbol(MY_AP(nlocal)  , & sdata->atom.nlocal        , sizeof(int));

  int3 layout = getgrid(sdata->atom.nlocal);
  dim3 threads(layout.z, 1, 1);
  dim3 grid(layout.x, layout.y, 1);
  Cuda_FixEnforce2dCuda_PostForce_Kernel <<< grid, threads>>> (groupbit);
  hipDeviceSynchronize();
  CUT_CHECK_ERROR("Cuda_Cuda_FixEnforce2dCuda_PostForce: fix enforce2d post_force Kernel execution failed");
}
